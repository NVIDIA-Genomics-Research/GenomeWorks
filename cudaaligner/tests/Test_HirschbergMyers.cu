#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "../src/hirschberg_myers_gpu.cu"
#include "../src/batched_device_matrices.cuh"
#include <claragenomics/utils/device_buffer.cuh>
#include <claragenomics/utils/signed_integer_utils.hpp>
#include <vector>
#include <gtest/gtest.h>

namespace claragenomics
{

namespace cudaaligner
{

using WordType = hirschbergmyers::WordType;

namespace test
{

__global__ void myers_preprocess_kernel(batched_device_matrices<WordType>::device_interface* batched_query_pattern, char const* query, int32_t query_size)
{
    CGA_CONSTEXPR int32_t word_size            = sizeof(WordType) * CHAR_BIT;
    const int32_t n_words                      = ceiling_divide<int32_t>(query_size, word_size);
    device_matrix_view<WordType> query_pattern = batched_query_pattern->get_matrix_view(0, n_words, 8);
    hirschbergmyers::myers_preprocess(query_pattern, query, query_size);
}

__global__ void myers_get_query_pattern_test_kernel(int32_t n_words, WordType* result, batched_device_matrices<WordType>::device_interface* batched_query_pattern, int32_t idx, char x, bool reverse)
{
    int const i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 32)
    {
        device_matrix_view<WordType> patterns = batched_query_pattern->get_matrix_view(0, n_words, 8);
        result[i]                             = hirschbergmyers::get_query_pattern(patterns, idx, i, x, reverse);
    }
}

matrix<WordType> compute_myers_preprocess_matrix(std::string query_host)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    hipStream_t stream             = nullptr;
    const int32_t n_words           = ceiling_divide<int32_t>(query_host.size(), word_size);

    device_buffer<char> query(query_host.size());
    hipMemcpy(query.data(), query_host.data(), sizeof(char) * query.size(), hipMemcpyHostToDevice);

    batched_device_matrices<WordType> query_pattern(1, 8 * n_words, stream);
    myers_preprocess_kernel<<<1, 32>>>(query_pattern.get_device_interface(), query.data(), query.size());
    return query_pattern.get_matrix(0, n_words, 8, stream);
}

std::vector<WordType> myers_get_query_pattern_test(std::string query_host, int32_t idx, char x, bool reverse)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    hipStream_t stream             = nullptr;
    const int32_t n_words           = ceiling_divide<int32_t>(query_host.size(), word_size);
    device_buffer<char> query(query_host.size());
    hipMemcpy(query.data(), query_host.data(), sizeof(char) * query.size(), hipMemcpyHostToDevice);
    batched_device_matrices<WordType> query_pattern(1, 8 * n_words, stream);
    myers_preprocess_kernel<<<1, 32>>>(query_pattern.get_device_interface(), query.data(), query.size());

    device_buffer<WordType> result(32);
    myers_get_query_pattern_test_kernel<<<1, 32>>>(n_words, result.data(), query_pattern.get_device_interface(), idx, x, reverse);

    std::vector<WordType> result_host(result.size());
    hipMemcpy(result_host.data(), result.data(), sizeof(WordType) * result.size(), hipMemcpyDeviceToHost);
    return result_host;
}

} // namespace test

TEST(HirschbergMyers, myers_preprocess_test)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    static_assert(word_size == 32, "This test assumes word_size = 32bit.");
    using test::compute_myers_preprocess_matrix;
    std::string query =
        "AACCGGTTACGTACGT"
        "AAACCCGGGTTTACGT"
        "AAACCCGGGTTTACG";
    matrix<WordType> patterns = compute_myers_preprocess_matrix(query);
    ASSERT_EQ(patterns.num_rows(), 2);
    ASSERT_EQ(patterns.num_cols(), 8);
    // A=0, C=1, G=2, T=3
    EXPECT_EQ(patterns(0, 0), 0b00010000000001110001000100000011u);
    EXPECT_EQ(patterns(0, 1), 0b00100000001110000010001000001100u);
    EXPECT_EQ(patterns(0, 2), 0b01000001110000000100010000110000u);
    EXPECT_EQ(patterns(0, 3), 0b10001110000000001000100011000000u);
    EXPECT_EQ(patterns(1, 0), 0b001000000000111u);
    EXPECT_EQ(patterns(1, 1), 0b010000000111000u);
    EXPECT_EQ(patterns(1, 2), 0b100000111000000u);
    EXPECT_EQ(patterns(1, 3), 0b000111000000000u);
    // reverse: A=4, C=5, G=6, T=7
    EXPECT_EQ(patterns(0, 4), 0b01110000000001000111000000000100u);
    EXPECT_EQ(patterns(0, 6), 0b00000001110000010000000111000001u);
    EXPECT_EQ(patterns(0, 5), 0b00001110000000100000111000000010u);
    EXPECT_EQ(patterns(0, 7), 0b10000000001110001000000000111000u);
    EXPECT_EQ(patterns(1, 4), 0b110000001000100u);
    EXPECT_EQ(patterns(1, 5), 0b001100000100010u);
    EXPECT_EQ(patterns(1, 6), 0b000011000010001u);
    EXPECT_EQ(patterns(1, 7), 0b000000110001000u);

    std::reverse(query.begin(), query.end());
    matrix<WordType> patterns_reversed = compute_myers_preprocess_matrix(query);
    ASSERT_EQ(patterns.num_rows(), 2);
    ASSERT_EQ(patterns.num_cols(), 8);
    EXPECT_EQ(patterns_reversed(0, 0), patterns(0, 4));
    EXPECT_EQ(patterns_reversed(1, 0), patterns(1, 4));
    EXPECT_EQ(patterns_reversed(0, 1), patterns(0, 5));
    EXPECT_EQ(patterns_reversed(1, 1), patterns(1, 5));
    EXPECT_EQ(patterns_reversed(0, 2), patterns(0, 6));
    EXPECT_EQ(patterns_reversed(1, 2), patterns(1, 6));
    EXPECT_EQ(patterns_reversed(0, 3), patterns(0, 7));
    EXPECT_EQ(patterns_reversed(1, 3), patterns(1, 7));
    EXPECT_EQ(patterns_reversed(0, 4), patterns(0, 0));
    EXPECT_EQ(patterns_reversed(1, 4), patterns(1, 0));
    EXPECT_EQ(patterns_reversed(0, 5), patterns(0, 1));
    EXPECT_EQ(patterns_reversed(1, 5), patterns(1, 1));
    EXPECT_EQ(patterns_reversed(0, 6), patterns(0, 2));
    EXPECT_EQ(patterns_reversed(1, 6), patterns(1, 2));
    EXPECT_EQ(patterns_reversed(0, 7), patterns(0, 3));
    EXPECT_EQ(patterns_reversed(1, 7), patterns(1, 3));
}

TEST(HirschbergMyers, myers_get_query_pattern)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    static_assert(word_size == 32, "This test assumes word_size = 32bit.");
    using test::compute_myers_preprocess_matrix;
    using test::myers_get_query_pattern_test;
    std::string query =
        "AACCGGTTACGTACGT"
        "AAACCCGGGTTTACGT"
        "AAACCCGGGTTTACG";
    std::vector<WordType> patterns_0 = myers_get_query_pattern_test(query, 0, 'A', false);
    std::vector<WordType> patterns_1 = myers_get_query_pattern_test(query, 1, 'A', false);
    ASSERT_EQ(get_size(patterns_0), get_size(patterns_1));
    int32_t const n = get_size(patterns_0);
    for (int32_t i = 0; i < n; ++i)
    {
        std::string shifted_query  = std::string(query.begin() + i, query.end());
        matrix<WordType> shifted_p = compute_myers_preprocess_matrix(shifted_query);
        EXPECT_EQ(patterns_0[i], shifted_p(0, 0)) << "for shift:" << i << std::endl;
        if (get_size(shifted_query) > word_size)
        {
            ASSERT_EQ(shifted_p.num_rows(), 2);
            EXPECT_EQ(patterns_1[i], shifted_p(1, 0)) << "for shift:" << i << std::endl;
        }
        else
        {
            EXPECT_EQ(patterns_1[i], WordType(0)) << "for shift:" << i << std::endl;
        }
    }
}

TEST(HirschbergMyers, myers_get_query_pattern_reverse)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    static_assert(word_size == 32, "This test assumes word_size = 32bit.");
    using test::compute_myers_preprocess_matrix;
    using test::myers_get_query_pattern_test;
    std::string query =
        "AACCGGTTACGTACGT"
        "AAACCCGGGTTTACGT"
        "AAACCCGGGTTTACG";
    std::vector<WordType> reverse_patterns_0 = myers_get_query_pattern_test(query, 0, 'A', true);
    std::vector<WordType> reverse_patterns_1 = myers_get_query_pattern_test(query, 1, 'A', true);
    int32_t const n                          = get_size(reverse_patterns_0);
    ASSERT_EQ(get_size(reverse_patterns_0), get_size(reverse_patterns_1));
    for (int32_t i = 0; i < n; ++i)
    {
        std::string shifted_end_query = std::string(query.begin(), query.end() - i);
        matrix<WordType> shifted_p    = compute_myers_preprocess_matrix(shifted_end_query);
        EXPECT_EQ(reverse_patterns_0[i], shifted_p(0, 4)) << "for shift:" << i << std::endl;
        if (get_size(shifted_end_query) > word_size)
        {
            ASSERT_EQ(shifted_p.num_rows(), 2);
            EXPECT_EQ(reverse_patterns_1[i], shifted_p(1, 4)) << "for shift:" << i << std::endl;
        }
        else
        {
            EXPECT_EQ(reverse_patterns_1[i], WordType(0)) << "for shift:" << i << std::endl;
        }
    }
}

} // namespace cudaaligner
} // namespace claragenomics
