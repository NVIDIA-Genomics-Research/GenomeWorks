#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "myers_gpu.cuh"
#include "batched_device_matrices.cuh"

#include <claragenomics/cudaaligner/aligner.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>
#include <claragenomics/utils/mathutils.hpp>
#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/device_buffer.cuh>

#include <cassert>
#include <climits>
#include <vector>
#include <numeric>

namespace claragenomics
{

namespace cudaaligner
{

namespace myers
{

inline __device__ WordType warp_leftshift_sync(uint32_t warp_mask, WordType v)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    const WordType x                = __shfl_up_sync(warp_mask, v >> (word_size - 1), 1);
    v <<= 1;
    if (threadIdx.x != 0)
        v |= x;
    return v;
}

inline __device__ WordType warp_add_sync(uint32_t warp_mask, WordType a, WordType b)
{
    static_assert(sizeof(WordType) == 4, "This function assumes WordType to have 4 bytes.");
    static_assert(CHAR_BIT == 8, "This function assumes a char width of 8 bit.");
    const uint64_t ax = a;
    const uint64_t bx = b;
    uint64_t r        = ax + bx;
    uint32_t carry    = static_cast<uint32_t>(r >> 32);
    r &= 0xffff'ffffull;
    // TODO: I think due to the structure of the Myer blocks,
    // a carry cannot propagate over more than a single block.
    // I.e. a single carry propagation without the loop should be sufficient.
    while (__any_sync(warp_mask, carry))
    {
        uint32_t x = __shfl_up_sync(warp_mask, carry, 1);
        if (threadIdx.x != 0)
            r += x;
        carry = static_cast<uint32_t>(r >> 32);
        r &= 0xffff'ffffull;
    }
    return static_cast<WordType>(r);
}

__device__ int32_t myers_advance_block(uint32_t warp_mask, WordType highest_bit, WordType eq, WordType& pv, WordType& mv, int32_t carry_in)
{
    assert((pv & mv) == WordType(0));

    // Stage 1
    WordType xv = eq | mv;
    if (carry_in < 0)
        eq |= WordType(1);
    WordType xh = warp_add_sync(warp_mask, eq & pv, pv);
    xh          = (xh ^ pv) | eq;
    WordType ph = mv | (~(xh | pv));
    WordType mh = pv & xh;

    int32_t carry_out = ((ph & highest_bit) == WordType(0) ? 0 : 1) - ((mh & highest_bit) == WordType(0) ? 0 : 1);

    ph = warp_leftshift_sync(warp_mask, ph);
    mh = warp_leftshift_sync(warp_mask, mh);

    if (carry_in < 0)
        mh |= WordType(1);

    if (carry_in > 0)
        ph |= WordType(1);

    // Stage 2
    pv = mh | (~(xv | ph));
    mv = ph & xv;

    return carry_out;
}

__device__ WordType myers_preprocess(char x, char const* query, int32_t query_size, int32_t offset)
{
    // Sets a 1 bit at the position of every matching character
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    assert(offset < query_size);
    const int32_t max_i = min(query_size - offset, word_size);
    WordType r          = 0;
    for (int32_t i = 0; i < max_i; ++i)
    {
        if (x == query[i + offset])
            r = r | (WordType(1) << i);
    }
    return r;
}

inline __device__ int32_t get_myers_score(int32_t i, int32_t j, device_matrix_view<WordType> const& pv, device_matrix_view<WordType> const& mv, device_matrix_view<int32_t> const& score, WordType last_entry_mask)
{
    assert(i > 0); // row 0 is implicit, NW matrix is shifted by i -> i-1
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    const int32_t word_idx          = (i - 1) / word_size;
    const int32_t bit_idx           = (i - 1) % word_size;
    int32_t s                       = score(word_idx, j);
    WordType mask                   = (~WordType(1)) << bit_idx;
    if (word_idx == score.num_rows() - 1)
        mask &= last_entry_mask;
    s -= __popc(mask & pv(word_idx, j));
    s += __popc(mask & mv(word_idx, j));
    return s;
}

__device__ void myers_backtrace(int8_t* paths_base, int32_t* lengths, int32_t max_path_length, device_matrix_view<WordType> const& pv, device_matrix_view<WordType> const& mv, device_matrix_view<int32_t> const& score, int32_t query_size, int32_t id)
{
    using nw_score_t                = int32_t;
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    assert(pv.num_rows() == score.num_rows());
    assert(mv.num_rows() == score.num_rows());
    assert(pv.num_cols() == score.num_cols());
    assert(mv.num_cols() == score.num_cols());
    int32_t i = query_size;
    int32_t j = score.num_cols() - 1;

    int8_t* path = paths_base + id * static_cast<ptrdiff_t>(max_path_length);

    const WordType last_entry_mask = query_size % word_size != 0 ? (WordType(1) << (query_size % word_size)) - 1 : ~WordType(0);

    nw_score_t myscore = score((i - 1) / word_size, j);
    int32_t pos        = 0;
    while (i > 0 && j > 0)
    {
        int8_t r               = 0;
        nw_score_t const above = i == 1 ? j : get_myers_score(i - 1, j, pv, mv, score, last_entry_mask);
        nw_score_t const diag  = i == 1 ? j - 1 : get_myers_score(i - 1, j - 1, pv, mv, score, last_entry_mask);
        nw_score_t const left  = get_myers_score(i, j - 1, pv, mv, score, last_entry_mask);
        if (left + 1 == myscore)
        {
            r       = static_cast<int8_t>(AlignmentState::insertion);
            myscore = left;
            --j;
        }
        else if (above + 1 == myscore)
        {
            r       = static_cast<int8_t>(AlignmentState::deletion);
            myscore = above;
            --i;
        }
        else
        {
            r       = (diag == myscore ? static_cast<int8_t>(AlignmentState::match) : static_cast<int8_t>(AlignmentState::mismatch));
            myscore = diag;
            --i;
            --j;
        }
        path[pos] = r;
        ++pos;
    }
    while (i > 0)
    {
        path[pos] = static_cast<int8_t>(AlignmentState::deletion);
        ++pos;
        --i;
    }
    while (j > 0)
    {
        path[pos] = static_cast<int8_t>(AlignmentState::insertion);
        ++pos;
        --j;
    }
    lengths[id] = pos;
}

__global__ void myers_backtrace_kernel(int8_t* paths_base, int32_t* lengths, int32_t max_path_length,
                                       batched_device_matrices<WordType>::device_interface* pvi,
                                       batched_device_matrices<WordType>::device_interface* mvi,
                                       batched_device_matrices<int32_t>::device_interface* scorei,
                                       int32_t const* sequence_lengths_d,
                                       int32_t n_alignments)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_alignments)
        return;
    CGA_CONSTEXPR int32_t word_size         = sizeof(WordType) * CHAR_BIT;
    const int32_t query_size                = sequence_lengths_d[2 * idx];
    const int32_t target_size               = sequence_lengths_d[2 * idx + 1];
    const int32_t n_words                   = (query_size + word_size - 1) / word_size;
    const device_matrix_view<WordType> pv   = pvi->get_matrix_view(idx, n_words, target_size + 1);
    const device_matrix_view<WordType> mv   = mvi->get_matrix_view(idx, n_words, target_size + 1);
    const device_matrix_view<int32_t> score = scorei->get_matrix_view(idx, n_words, target_size + 1);
    myers_backtrace(paths_base, lengths, max_path_length, pv, mv, score, query_size, idx);
}

__global__ void myers_convert_to_full_score_matrix_kernel(batched_device_matrices<int32_t>::device_interface* fullscorei,
                                                          batched_device_matrices<WordType>::device_interface* pvi,
                                                          batched_device_matrices<WordType>::device_interface* mvi,
                                                          batched_device_matrices<int32_t>::device_interface* scorei,
                                                          int32_t const* sequence_lengths_d,
                                                          int32_t i)
{
    CGA_CONSTEXPR int32_t word_size       = sizeof(WordType) * CHAR_BIT;
    const int32_t query_size              = sequence_lengths_d[2 * i];
    const int32_t target_size             = sequence_lengths_d[2 * i + 1];
    const int32_t n_words                 = (query_size + word_size - 1) / word_size;
    device_matrix_view<WordType> pv       = pvi->get_matrix_view(0, n_words, target_size + 1);
    device_matrix_view<WordType> mv       = mvi->get_matrix_view(0, n_words, target_size + 1);
    device_matrix_view<int32_t> score     = scorei->get_matrix_view(0, n_words, target_size + 1);
    device_matrix_view<int32_t> fullscore = fullscorei->get_matrix_view(0, query_size + 1, target_size + 1);

    assert(query_size > 0);
    const WordType last_entry_mask = query_size % word_size != 0 ? (WordType(1) << (query_size % word_size)) - 1 : ~WordType(0);

    for (int32_t j = 0; j < target_size + 1; ++j)
    {
        fullscore(0, j) = j;
        for (int32_t i = 1; i < query_size + 1; ++i) // should be query_size + 1
        {
            fullscore(i, j) = get_myers_score(i, j, pv, mv, score, last_entry_mask);
        }
    }
}

__global__ void myers_compute_score_matrix_kernel(
    batched_device_matrices<WordType>::device_interface* pvi,
    batched_device_matrices<WordType>::device_interface* mvi,
    batched_device_matrices<int32_t>::device_interface* scorei,
    batched_device_matrices<WordType>::device_interface* query_patternsi,
    char const* sequences_d, int32_t const* sequence_lengths_d,
    int32_t max_sequence_length,
    int32_t n_alignments)
{
    CGA_CONSTEXPR int32_t word_size = sizeof(WordType) * CHAR_BIT;
    CGA_CONSTEXPR int32_t warp_size = 32;
    assert(warpSize == warp_size);
    assert(threadIdx.x < warp_size);
    assert(blockIdx.x == 0);

    const int32_t alignment_idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (alignment_idx >= n_alignments)
        return;
    const int32_t query_size  = sequence_lengths_d[2 * alignment_idx];
    const int32_t target_size = sequence_lengths_d[2 * alignment_idx + 1];
    const char* const query   = sequences_d + 2 * alignment_idx * max_sequence_length;
    const char* const target  = sequences_d + (2 * alignment_idx + 1) * max_sequence_length;
    const int32_t n_words     = (query_size + word_size - 1) / word_size;

    assert(query_size > 0);

    device_matrix_view<WordType> pv            = pvi->get_matrix_view(alignment_idx, n_words, target_size + 1);
    device_matrix_view<WordType> mv            = mvi->get_matrix_view(alignment_idx, n_words, target_size + 1);
    device_matrix_view<int32_t> score          = scorei->get_matrix_view(alignment_idx, n_words, target_size + 1);
    device_matrix_view<WordType> query_pattern = query_patternsi->get_matrix_view(alignment_idx, n_words, 4);

    for (int32_t idx = threadIdx.x; idx < n_words; idx += warp_size)
    {
        pv(idx, 0)    = ~WordType(0);
        mv(idx, 0)    = 0;
        score(idx, 0) = min((idx + 1) * word_size, query_size);
        // TODO query load is inefficient
        query_pattern(idx, 0) = myers_preprocess('A', query, query_size, idx * word_size);
        query_pattern(idx, 1) = myers_preprocess('C', query, query_size, idx * word_size);
        query_pattern(idx, 2) = myers_preprocess('G', query, query_size, idx * word_size);
        query_pattern(idx, 3) = myers_preprocess('T', query, query_size, idx * word_size);
    }

    for (int32_t t = 1; t <= target_size; ++t)
    {
        int32_t warp_carry = 0;
        if (threadIdx.x == 0)
            warp_carry = 1; // for global alignment the (implicit) first row has to be 0,1,2,3,... -> carry 1
        for (int32_t idx = threadIdx.x; idx < n_words; idx += warp_size)
        {
            const uint32_t warp_mask = idx / warp_size < n_words / warp_size ? 0xffff'ffffu : (1u << (n_words % warp_size)) - 1;

            WordType pv_local = pv(idx, t - 1);
            WordType mv_local = mv(idx, t - 1);
            // TODO these might be cached or only computed for the specific t at hand.
            const WordType highest_bit = WordType(1) << (idx == (n_words - 1) ? query_size - (n_words - 1) * word_size - 1 : word_size - 1);

            const WordType eq = query_pattern(idx, [](char x) -> WordType {
                assert(x == 'A' || x == 'C' || x == 'G' || x == 'T');
                switch (x)
                {
                case 'A':
                    return 0;
                case 'C':
                    return 1;
                case 'G':
                    return 2;
                case 'T':
                    return 3;
                default:
                    return 0;
                }
            }(target[t - 1]));

            warp_carry    = myers_advance_block(warp_mask, highest_bit, eq, pv_local, mv_local, warp_carry);
            score(idx, t) = score(idx, t - 1) + warp_carry;
            if (threadIdx.x == 0)
                warp_carry = 0;
            if (warp_mask == 0xffff'ffffu)
                warp_carry = __shfl_down_sync(0x8000'0001u, warp_carry, warp_size - 1);
            if (threadIdx.x != 0)
                warp_carry = 0;
            pv(idx, t) = pv_local;
            mv(idx, t) = mv_local;
        }
    }
}

} // namespace myers

int32_t myers_compute_edit_distance(std::string const& target, std::string const& query)
{
    constexpr int32_t warp_size = 32;
    constexpr int32_t word_size = sizeof(myers::WordType) * CHAR_BIT;
    if (get_size(query) == 0)
        return get_size(target);

    hipStream_t stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&stream));

    int32_t max_sequence_length = std::max(get_size(target), get_size(query));
    device_buffer<char> sequences_d(2 * max_sequence_length);
    device_buffer<int32_t> sequence_lengths_d(2);

    const int32_t n_words = (get_size(query) + word_size - 1) / word_size;
    batched_device_matrices<myers::WordType> pv(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<myers::WordType> mv(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<int32_t> score(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<myers::WordType> query_patterns(1, n_words * 4, stream);

    std::array<int32_t, 2> lengths = {static_cast<int32_t>(get_size(query)), static_cast<int32_t>(get_size(target))};
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequences_d.data(), query.data(), sizeof(char) * get_size(query), hipMemcpyHostToDevice, stream));
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequences_d.data() + max_sequence_length, target.data(), sizeof(char) * get_size(target), hipMemcpyHostToDevice, stream));
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequence_lengths_d.data(), lengths.data(), sizeof(int32_t) * 2, hipMemcpyHostToDevice, stream));

    myers::myers_compute_score_matrix_kernel<<<1, warp_size, 0, stream>>>(pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), query_patterns.get_device_interface(), sequences_d.data(), sequence_lengths_d.data(), max_sequence_length, 1);

    matrix<int32_t> score_host = score.get_matrix(0, n_words, get_size(target) + 1, stream);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(stream));
    return score_host(n_words - 1, get_size(target));
}

matrix<int32_t> myers_get_full_score_matrix(std::string const& target, std::string const& query)
{
    constexpr int32_t warp_size = 32;
    constexpr int32_t word_size = sizeof(myers::WordType) * CHAR_BIT;

    if (get_size(target) == 0)
    {
        matrix<int32_t> r(get_size(query) + 1, 1);
        std::iota(r.data(), r.data() + get_size(query) + 1, 0);
        return r;
    }
    if (get_size(query) == 0)
    {
        matrix<int32_t> r(1, get_size(target) + 1);
        std::iota(r.data(), r.data() + get_size(target) + 1, 0);
        return r;
    }

    hipStream_t stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&stream));

    int32_t max_sequence_length = std::max(get_size(target), get_size(query));
    device_buffer<char> sequences_d(2 * max_sequence_length);
    device_buffer<int32_t> sequence_lengths_d(2);

    const int32_t n_words = (get_size(query) + word_size - 1) / word_size;
    batched_device_matrices<myers::WordType> pv(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<myers::WordType> mv(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<int32_t> score(1, n_words * (get_size(target) + 1), stream);
    batched_device_matrices<myers::WordType> query_patterns(1, n_words * 4, stream);

    batched_device_matrices<int32_t> fullscore(1, (get_size(query) + 1) * (get_size(target) + 1), stream);

    std::array<int32_t, 2> lengths = {static_cast<int32_t>(get_size(query)), static_cast<int32_t>(get_size(target))};
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequences_d.data(), query.data(), sizeof(char) * get_size(query), hipMemcpyHostToDevice, stream));
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequences_d.data() + max_sequence_length, target.data(), sizeof(char) * get_size(target), hipMemcpyHostToDevice, stream));
    CGA_CU_CHECK_ERR(hipMemcpyAsync(sequence_lengths_d.data(), lengths.data(), sizeof(int32_t) * 2, hipMemcpyHostToDevice, stream));

    myers::myers_compute_score_matrix_kernel<<<1, warp_size, 0, stream>>>(pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), query_patterns.get_device_interface(), sequences_d.data(), sequence_lengths_d.data(), max_sequence_length, 1);
    myers::myers_convert_to_full_score_matrix_kernel<<<1, 1, 0, stream>>>(fullscore.get_device_interface(), pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), sequence_lengths_d.data(), 0);

    matrix<int32_t> fullscore_host = fullscore.get_matrix(0, get_size(query) + 1, get_size(target) + 1, stream);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(stream));
    return fullscore_host;
}

void myers_gpu(int8_t* paths_d, int32_t* path_lengths_d, int32_t max_path_length,
               char const* sequences_d,
               int32_t const* sequence_lengths_d,
               int32_t max_sequence_length,
               int32_t n_alignments,
               batched_device_matrices<myers::WordType>& pv,
               batched_device_matrices<myers::WordType>& mv,
               batched_device_matrices<int32_t>& score,
               batched_device_matrices<myers::WordType>& query_patterns,
               hipStream_t stream)
{
    constexpr int32_t warp_size = 32;

    {
        const dim3 threads(warp_size, 1, 1);
        const dim3 blocks(1, ceiling_divide<int32_t>(n_alignments, threads.y), 1);
        myers::myers_compute_score_matrix_kernel<<<blocks, threads, 0, stream>>>(pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), query_patterns.get_device_interface(), sequences_d, sequence_lengths_d, max_sequence_length, n_alignments);
    }
    {
        const dim3 threads(128, 1, 1);
        const dim3 blocks(ceiling_divide<int32_t>(n_alignments, threads.x), 1, 1);
        myers::myers_backtrace_kernel<<<blocks, threads, 0, stream>>>(paths_d, path_lengths_d, max_path_length, pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), sequence_lengths_d, n_alignments);
    }
}

} // namespace cudaaligner
} // namespace claragenomics
