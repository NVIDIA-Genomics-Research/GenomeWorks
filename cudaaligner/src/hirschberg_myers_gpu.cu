#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "hirschberg_myers_gpu.cuh"
#include <cassert>
#include "batched_device_matrices.cuh"
#include <claraparabricks/genomeworks/cudaaligner/aligner.hpp>
#include <claraparabricks/genomeworks/utils/cudautils.hpp>
#include <claraparabricks/genomeworks/utils/mathutils.hpp>
#include <cuda/std/limits>
#include <cstring>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudaaligner
{

namespace hirschbergmyers
{

constexpr int32_t warp_size = 32;
constexpr int32_t word_size = sizeof(WordType) * CHAR_BIT;

inline __device__ WordType warp_leftshift_sync(uint32_t warp_mask, WordType v)
{
    assert(((warp_mask >> threadIdx.x) & 1u) == 1);
    const WordType x = __shfl_up_sync(warp_mask, v >> (word_size - 1), 1);
    v <<= 1;
    if (threadIdx.x != 0)
        v |= x;
    return v;
}

inline __device__ WordType warp_add_sync(uint32_t warp_mask, WordType a, WordType b)
{
    assert(((warp_mask >> threadIdx.x) & 1u) == 1);
    static_assert(sizeof(WordType) == 4, "This function assumes WordType to have 4 bytes.");
    static_assert(CHAR_BIT == 8, "This function assumes a char width of 8 bit.");
    const uint64_t ax = a;
    const uint64_t bx = b;
    uint64_t r        = ax + bx;
    uint32_t carry    = static_cast<uint32_t>(r >> 32);
    if (warp_mask == 1)
    {
        return static_cast<WordType>(r);
    }
    r &= 0xffff'ffffull;
    // TODO: I think due to the structure of the Myer blocks,
    // a carry cannot propagate over more than a single block.
    // I.e. a single carry propagation without the loop should be sufficient.
    while (__any_sync(warp_mask, carry))
    {
        uint32_t x = __shfl_up_sync(warp_mask, carry, 1);
        if (threadIdx.x != 0)
            r += x;
        carry = static_cast<uint32_t>(r >> 32);
        r &= 0xffff'ffffull;
    }
    return static_cast<WordType>(r);
}

__device__ int32_t myers_advance_block(uint32_t warp_mask, WordType highest_bit, WordType eq, WordType& pv, WordType& mv, int32_t carry_in)
{
    assert((pv & mv) == WordType(0));

    // Stage 1
    WordType xv = eq | mv;
    if (carry_in < 0)
        eq |= WordType(1);
    WordType xh = warp_add_sync(warp_mask, eq & pv, pv);
    xh          = (xh ^ pv) | eq;
    WordType ph = mv | (~(xh | pv));
    WordType mh = pv & xh;

    int32_t carry_out = ((ph & highest_bit) == WordType(0) ? 0 : 1) - ((mh & highest_bit) == WordType(0) ? 0 : 1);

    ph = warp_leftshift_sync(warp_mask, ph);
    mh = warp_leftshift_sync(warp_mask, mh);

    if (carry_in < 0)
        mh |= WordType(1);

    if (carry_in > 0)
        ph |= WordType(1);

    // Stage 2
    pv = mh | (~(xv | ph));
    mv = ph & xv;

    return carry_out;
}

inline __device__ int32_t get_myers_score(int32_t i, int32_t j, device_matrix_view<WordType> const& pv, device_matrix_view<WordType> const& mv, device_matrix_view<int32_t> const& score, WordType last_entry_mask)
{
    assert(i > 0); // row 0 is implicit, NW matrix is shifted by i -> i-1
    const int32_t word_idx = (i - 1) / word_size;
    const int32_t bit_idx  = (i - 1) % word_size;
    int32_t s              = score(word_idx, j);
    WordType mask          = (~WordType(1)) << bit_idx;
    if (word_idx == score.num_rows() - 1)
        mask &= last_entry_mask;
    s -= __popc(mask & pv(word_idx, j));
    s += __popc(mask & mv(word_idx, j));
    return s;
}

__device__ int32_t append_myers_backtrace(int8_t* path, device_matrix_view<WordType> const& pv, device_matrix_view<WordType> const& mv, device_matrix_view<int32_t> const& score, int32_t query_size)
{
    assert(threadIdx.x == 0);
    using nw_score_t = int32_t;
    assert(pv.num_rows() == score.num_rows());
    assert(mv.num_rows() == score.num_rows());
    assert(pv.num_cols() == score.num_cols());
    assert(mv.num_cols() == score.num_cols());
    assert(score.num_rows() == ceiling_divide(query_size, word_size));
    int32_t i = query_size;
    int32_t j = score.num_cols() - 1;

    const WordType last_entry_mask = query_size % word_size != 0 ? (WordType(1) << (query_size % word_size)) - 1 : ~WordType(0);

    nw_score_t myscore = score((i - 1) / word_size, j);
    int32_t pos        = 0;
    while (i > 0 && j > 0)
    {
        int8_t r               = 0;
        nw_score_t const above = i == 1 ? j : get_myers_score(i - 1, j, pv, mv, score, last_entry_mask);
        nw_score_t const diag  = i == 1 ? j - 1 : get_myers_score(i - 1, j - 1, pv, mv, score, last_entry_mask);
        nw_score_t const left  = get_myers_score(i, j - 1, pv, mv, score, last_entry_mask);
        if (left + 1 == myscore)
        {
            r       = static_cast<int8_t>(AlignmentState::insertion);
            myscore = left;
            --j;
        }
        else if (above + 1 == myscore)
        {
            r       = static_cast<int8_t>(AlignmentState::deletion);
            myscore = above;
            --i;
        }
        else
        {
            r       = (diag == myscore ? static_cast<int8_t>(AlignmentState::match) : static_cast<int8_t>(AlignmentState::mismatch));
            myscore = diag;
            --i;
            --j;
        }
        path[pos] = r;
        ++pos;
    }
    while (i > 0)
    {
        path[pos] = static_cast<int8_t>(AlignmentState::deletion);
        ++pos;
        --i;
    }
    while (j > 0)
    {
        path[pos] = static_cast<int8_t>(AlignmentState::insertion);
        ++pos;
        --j;
    }
    return pos;
}

inline __device__ void hirschberg_myers_fill_path_warp(int8_t*& path, int32_t* path_length, int32_t n, int8_t value)
{
    // TODO parallelize
    if (threadIdx.x == 0)
    {
        int8_t const* const path_end = path + n;
        while (path != path_end)
        {
            *path = value;
            ++path;
        }
        *path_length += n;
    }
}

__device__ WordType myers_generate_query_pattern(char x, char const* query, int32_t query_size, int32_t offset)
{
    // Sets a 1 bit at the position of every matching character
    assert(offset < query_size);
    const int32_t max_i = min(query_size - offset, word_size);
    WordType r          = 0;
    for (int32_t i = 0; i < max_i; ++i)
    {
        if (x == query[i + offset])
            r = r | (WordType(1) << i);
    }
    return r;
}

__device__ WordType myers_generate_query_pattern_reverse(char x, char const* query, int32_t query_size, int32_t offset)
{
    // Sets a 1 bit at the position of every matching character
    assert(offset < query_size);
    const int32_t max_i = min(query_size - offset, word_size);
    WordType r          = 0;
    // TODO make this a forward loop
    for (int32_t i = 0; i < max_i; ++i)
    {
        if (x == query[query_size - 1 - (i + offset)])
            r = r | (WordType(1) << i);
    }
    return r;
}

__device__ void myers_preprocess(device_matrix_view<WordType>& query_pattern, char const* query, int32_t query_size)
{
    const int32_t n_words = ceiling_divide(query_size, word_size);
    for (int32_t idx = threadIdx.x; idx < n_words; idx += warp_size)
    {
        // TODO query load is inefficient
        query_pattern(idx, 0) = myers_generate_query_pattern('A', query, query_size, idx * word_size);
        query_pattern(idx, 1) = myers_generate_query_pattern('C', query, query_size, idx * word_size);
        query_pattern(idx, 2) = myers_generate_query_pattern('T', query, query_size, idx * word_size);
        query_pattern(idx, 3) = myers_generate_query_pattern('G', query, query_size, idx * word_size);
        query_pattern(idx, 4) = myers_generate_query_pattern_reverse('A', query, query_size, idx * word_size);
        query_pattern(idx, 5) = myers_generate_query_pattern_reverse('C', query, query_size, idx * word_size);
        query_pattern(idx, 6) = myers_generate_query_pattern_reverse('T', query, query_size, idx * word_size);
        query_pattern(idx, 7) = myers_generate_query_pattern_reverse('G', query, query_size, idx * word_size);
    }
}

inline __device__ WordType get_query_pattern(device_matrix_view<WordType>& query_patterns, int32_t idx, int32_t query_begin_offset, char x, bool reverse)
{
    static_assert(std::is_unsigned<WordType>::value, "WordType has to be an unsigned type for well-defined >> operations.");
    const int32_t char_idx = [](char x) -> int32_t {
        int32_t r = x;
        return (r >> 1) & 0x3;
    }(x) + (reverse ? 4 : 0);

    // 4-bit word example:
    // query_patterns contains character match bit patterns "XXXX" for the full query string.
    // we want the bit pattern "yyyy" for a view of on the query string starting at eg. character 11:
    //       4    3    2     1      0 (pattern index)
    //    XXXX XXXX XXXX [XXXX] [XXXX]
    //     YYY Yyyy y
    //         1    0 (idx)
    //
    // query_begin_offset = 11
    // => idx_offset = 11/4 = 2, shift = 11%4 = 3

    const int32_t idx_offset = query_begin_offset / word_size;
    const int32_t shift      = query_begin_offset % word_size;

    WordType r = query_patterns(idx + idx_offset, char_idx);
    if (shift != 0)
    {
        r >>= shift;
        if (idx + idx_offset + 1 < query_patterns.num_rows())
        {
            r |= query_patterns(idx + idx_offset + 1, char_idx) << (word_size - shift);
        }
    }
    return r;
}

__device__ void
myers_compute_scores(
    device_matrix_view<WordType>& pv,
    device_matrix_view<WordType>& mv,
    device_matrix_view<int32_t>& score,
    device_matrix_view<WordType>& query_patterns,
    char const* target_begin,
    char const* target_end,
    char const* query_begin,
    char const* query_end,
    int32_t const pattern_idx_offset,
    bool full_score_matrix,
    bool reverse)
{
    assert(warpSize == warp_size);
    assert(threadIdx.x < warp_size);
    assert(blockIdx.x == 0);

    assert(query_end - query_begin > 0);
    assert(target_begin < target_end);

    const int32_t n_words     = ceiling_divide<int32_t>(query_end - query_begin, word_size);
    const int32_t target_size = target_end - target_begin;

    assert(pv.num_rows() == n_words);
    assert(mv.num_rows() == n_words);
    assert(score.num_rows() == full_score_matrix ? n_words : target_size + 1);
    assert(pv.num_cols() == full_score_matrix ? target_size + 1 : 1);
    assert(mv.num_cols() == full_score_matrix ? target_size + 1 : 1);
    assert(score.num_cols() == full_score_matrix ? target_size + 1 : 2);

    {
        for (int32_t idx = threadIdx.x; idx < n_words; idx += warp_size)
        {
            pv(idx, 0) = ~WordType(0);
            mv(idx, 0) = 0;
        }

        const int32_t query_size = query_end - query_begin;
        if (full_score_matrix)
        {
            for (int32_t idx = threadIdx.x; idx < n_words; idx += warp_size)
                score(idx, 0) = min((idx + 1) * word_size, query_size);
        }
        else
        {
            if (threadIdx.x == 0)
                score(0, reverse ? 1 : 0) = query_size;
        }
    }

    const int32_t n_warp_iterations = ceiling_divide(n_words, warp_size) * warp_size;
    __syncwarp();
    for (int32_t t = 1; t <= target_size; ++t)
    {
        int32_t warp_carry = 0;
        if (threadIdx.x == 0)
        {
            warp_carry = 1; // for global alignment the (implicit) first row has to be 0,1,2,3,... -> carry 1
        }
        for (int32_t idx = threadIdx.x; idx < n_warp_iterations; idx += warp_size)
        {
            if (idx < n_words)
            {
                const uint32_t warp_mask = idx / warp_size < n_words / warp_size ? 0xffff'ffffu : (1u << (n_words % warp_size)) - 1;

                WordType pv_local = pv(idx, full_score_matrix ? t - 1 : 0);
                WordType mv_local = mv(idx, full_score_matrix ? t - 1 : 0);
                // TODO these might be cached or only computed for the specific t at hand.
                const WordType highest_bit = WordType(1) << (idx == (n_words - 1) ? (query_end - query_begin) - (n_words - 1) * word_size - 1 : word_size - 1);

                const WordType eq = get_query_pattern(query_patterns, idx, pattern_idx_offset, target_begin[reverse ? target_size - t : t - 1], reverse);

                warp_carry = myers_advance_block(warp_mask, highest_bit, eq, pv_local, mv_local, warp_carry);
                if (full_score_matrix)
                {
                    score(idx, t) = score(idx, t - 1) + warp_carry;
                }
                else
                {
                    if (idx + 1 == n_words)
                    {
                        score(t, reverse ? 1 : 0) = score(t - 1, reverse ? 1 : 0) + warp_carry;
                    }
                }
                if (threadIdx.x == 0)
                {
                    warp_carry = 0;
                }
                if (warp_mask == 0xffff'ffffu && (threadIdx.x == 31 || threadIdx.x == 0))
                {
                    warp_carry = __shfl_down_sync(0x8000'0001u, warp_carry, warp_size - 1);
                }
                if (threadIdx.x != 0)
                {
                    warp_carry = 0;
                }
                pv(idx, full_score_matrix ? t : 0) = pv_local;
                mv(idx, full_score_matrix ? t : 0) = mv_local;
            }
            __syncwarp();
        }
    }
}

__device__ void hirschberg_myers_compute_path(
    int8_t*& path,
    int32_t* path_length,
    batched_device_matrices<WordType>::device_interface* pvi,
    batched_device_matrices<WordType>::device_interface* mvi,
    batched_device_matrices<int32_t>::device_interface* scorei,
    device_matrix_view<WordType>& query_patterns,
    char const* target_begin,
    char const* target_end,
    char const* query_begin,
    char const* query_end,
    char const* query_begin_absolute,
    int32_t alignment_idx)
{
    assert(query_begin < query_end);
    const int32_t n_words             = ceiling_divide<int32_t>(query_end - query_begin, word_size);
    device_matrix_view<int32_t> score = scorei->get_matrix_view(alignment_idx, n_words, target_end - target_begin + 1);
    device_matrix_view<WordType> pv   = pvi->get_matrix_view(alignment_idx, n_words, target_end - target_begin + 1);
    device_matrix_view<WordType> mv   = mvi->get_matrix_view(alignment_idx, n_words, target_end - target_begin + 1);
    myers_compute_scores(pv, mv, score, query_patterns, target_begin, target_end, query_begin, query_end, query_begin - query_begin_absolute, true, false);
    __syncwarp();
    if (threadIdx.x == 0)
    {
        int32_t len = append_myers_backtrace(path, pv, mv, score, query_end - query_begin);
        path += len;
        *path_length += len;
    }
}

__device__ const char* hirschberg_myers_compute_target_mid_warp(
    batched_device_matrices<WordType>::device_interface* pvi,
    batched_device_matrices<WordType>::device_interface* mvi,
    batched_device_matrices<int32_t>::device_interface* scorei,
    device_matrix_view<WordType>& query_patterns,
    char const* target_begin,
    char const* target_end,
    char const* query_begin,
    char const* query_mid,
    char const* query_end,
    char const* query_begin_absolute,
    char const* query_end_absolute,
    int32_t alignment_idx)
{
    assert(query_begin <= query_mid);
    assert(query_mid < query_end);
    assert(target_begin < target_end);

    device_matrix_view<int32_t> score = scorei->get_matrix_view(alignment_idx, target_end - target_begin + 1, 2);

    if (query_begin < query_mid)
    {
        const int32_t n_words           = ceiling_divide<int32_t>(query_mid - query_begin, word_size);
        device_matrix_view<WordType> pv = pvi->get_matrix_view(alignment_idx, n_words, 2);
        device_matrix_view<WordType> mv = mvi->get_matrix_view(alignment_idx, n_words, 2);
        myers_compute_scores(pv, mv, score, query_patterns, target_begin, target_end, query_begin, query_mid, query_begin - query_begin_absolute, false, false);
    }
    else
    {
        const int32_t target_size = (target_end - target_begin);
        for (int32_t t = threadIdx.x; t <= target_size; t += warp_size)
        {
            score(t, 0) = t;
        }
        __syncwarp();
    }

    {
        const int32_t n_words           = ceiling_divide<int32_t>(query_end - query_mid, word_size);
        device_matrix_view<WordType> pv = pvi->get_matrix_view(alignment_idx, n_words, 2);
        device_matrix_view<WordType> mv = mvi->get_matrix_view(alignment_idx, n_words, 2);
        myers_compute_scores(pv, mv, score, query_patterns, target_begin, target_end, query_mid, query_end, query_end_absolute - query_end, false, true);
    }

    const int32_t target_size = (target_end - target_begin);
    int32_t midpoint          = 0;
    nw_score_t cur_min        = cuda::std::numeric_limits<nw_score_t>::max();
    for (int32_t t = threadIdx.x; t <= target_size; t += warp_size)
    {
        nw_score_t sum = score(t, 0) + score(target_size - t, 1);
        if (sum < cur_min)
        {
            cur_min  = sum;
            midpoint = t;
        }
    }
#pragma unroll
    for (int32_t i = 16; i > 0; i >>= 1)
    {
        const int32_t mv = __shfl_down_sync(0xffff'ffffu, cur_min, i);
        const int32_t mp = __shfl_down_sync(0xffff'ffffu, midpoint, i);
        if (mv < cur_min)
        {
            cur_min  = mv;
            midpoint = mp;
        }
    }
    __shfl_sync(0xffff'ffffu, midpoint, 0);
    return target_begin + midpoint;
}

__device__ void hirschberg_myers_single_char_warp(int8_t*& path, int32_t* path_length, char query_char, char const* target_begin, char const* target_end)
{
    // TODO parallelize
    if (threadIdx.x == 0)
    {
        char const* t = target_end - 1;
        while (t >= target_begin)
        {
            if (*t == query_char)
            {
                *path = static_cast<int8_t>(AlignmentState::match);
                ++path;
                --t;
                break;
            }
            *path = static_cast<int8_t>(AlignmentState::insertion);
            ++path;
            --t;
        }
        if (*(path - 1) != static_cast<int8_t>(AlignmentState::match))
        {
            *(path - 1) = static_cast<int8_t>(AlignmentState::mismatch);
        }
        while (t >= target_begin)
        {
            *path = static_cast<int8_t>(AlignmentState::insertion);
            ++path;
            --t;
        }
        *path_length += target_end - target_begin;
    }
}

template <typename T>
class warp_shared_stack
{
public:
    __device__ warp_shared_stack(T* buffer_begin, T* buffer_end)
        : buffer_begin_(buffer_begin)
        , cur_end_(buffer_begin)
        , buffer_end_(buffer_end)
    {
        assert(buffer_begin_ < buffer_end_);
    }

    __device__ bool inline push(T const& t, unsigned warp_mask = 0xffff'ffffu)
    {
        if (buffer_end_ - cur_end_ >= 1)
        {
            __syncwarp(warp_mask);
            if (threadIdx.x == 0)
            {
                *cur_end_ = t;
            }
            __syncwarp(warp_mask);
            ++cur_end_;
            return true;
        }
        else
        {
            if (threadIdx.x == 0)
            {
                printf("ERROR: stack full!");
            }
            return false;
        }
    }

    __device__ inline void pop()
    {
        assert(cur_end_ > buffer_begin_);
        if (cur_end_ - 1 >= buffer_begin_)
            --cur_end_;
    }

    __device__ inline T back() const
    {
        assert(cur_end_ - 1 >= buffer_begin_);
        return *(cur_end_ - 1);
    }

    __device__ inline bool empty() const
    {
        return buffer_begin_ == cur_end_;
    }

private:
    T* buffer_begin_;
    T* cur_end_;
    T* buffer_end_;
};

__device__ void hirschberg_myers(
    query_target_range* stack_buffer_begin,
    query_target_range* stack_buffer_end,
    int8_t*& path,
    int32_t* path_length,
    int32_t full_myers_threshold,
    batched_device_matrices<WordType>::device_interface* pvi,
    batched_device_matrices<WordType>::device_interface* mvi,
    batched_device_matrices<int32_t>::device_interface* scorei,
    device_matrix_view<WordType>& query_patterns,
    char const* target_begin_absolute,
    char const* target_end_absolute,
    char const* query_begin_absolute,
    char const* query_end_absolute,
    int32_t alignment_idx)
{
    assert(blockDim.x == warp_size);
    assert(blockDim.z == 1);
    assert(query_begin_absolute <= query_end_absolute);
    assert(target_begin_absolute <= target_end_absolute);

    warp_shared_stack<query_target_range> stack(stack_buffer_begin, stack_buffer_end);
    stack.push({query_begin_absolute, query_end_absolute, target_begin_absolute, target_end_absolute});

    assert(pvi->get_max_elements_per_matrix(alignment_idx) == mvi->get_max_elements_per_matrix(alignment_idx));
    assert(scorei->get_max_elements_per_matrix(alignment_idx) >= pvi->get_max_elements_per_matrix(alignment_idx));

    bool success   = true;
    int32_t length = 0;
    while (success && !stack.empty())
    {
        query_target_range e = stack.back();
        stack.pop();
        assert(e.query_begin <= e.query_end);
        assert(e.target_begin <= e.target_end);
        if (e.target_begin == e.target_end)
        {
            hirschberg_myers_fill_path_warp(path, &length, e.query_end - e.query_begin, static_cast<int8_t>(AlignmentState::deletion));
        }
        else if (e.query_begin == e.query_end)
        {
            hirschberg_myers_fill_path_warp(path, &length, e.target_end - e.target_begin, static_cast<int8_t>(AlignmentState::insertion));
        }
        else if (e.query_begin + 1 == e.query_end)
        {
            hirschberg_myers_single_char_warp(path, &length, *e.query_begin, e.target_begin, e.target_end);
        }
        else
        {
            if (e.query_end - e.query_begin < full_myers_threshold && e.query_end != e.query_begin)
            {
                const int32_t n_words = ceiling_divide<int32_t>(e.query_end - e.query_begin, word_size);
                if ((e.target_end - e.target_begin + 1) * n_words <= pvi->get_max_elements_per_matrix(alignment_idx))
                {
                    hirschberg_myers_compute_path(path, &length, pvi, mvi, scorei, query_patterns, e.target_begin, e.target_end, e.query_begin, e.query_end, query_begin_absolute, alignment_idx);
                    continue;
                }
            }

            const char* query_mid  = e.query_begin + (e.query_end - e.query_begin) / 2;
            const char* target_mid = hirschberg_myers_compute_target_mid_warp(pvi, mvi, scorei, query_patterns, e.target_begin, e.target_end, e.query_begin, query_mid, e.query_end, query_begin_absolute, query_end_absolute, alignment_idx);
            success                = success && stack.push({e.query_begin, query_mid, e.target_begin, target_mid});
            success                = success && stack.push({query_mid, e.query_end, target_mid, e.target_end});
        }
    }
    if (!success)
        length = 0;
    if (threadIdx.x == 0)
        *path_length = length;
}

__global__ void hirschberg_myers_compute_alignment(
    query_target_range* stack_buffer_base,
    int32_t stack_buffer_size_per_alignment,
    int32_t full_myers_threshold,
    int8_t* paths_base,
    int32_t* path_lengths,
    int32_t max_path_length,
    batched_device_matrices<WordType>::device_interface* pvi,
    batched_device_matrices<WordType>::device_interface* mvi,
    batched_device_matrices<int32_t>::device_interface* scorei,
    batched_device_matrices<WordType>::device_interface* query_patternsi,
    char const* sequences_d, int32_t const* sequence_lengths_d,
    int32_t max_sequence_length,
    int32_t n_alignments)
{
    assert(blockDim.x == warp_size);
    assert(blockDim.z == 1);

    const int32_t alignment_idx = blockIdx.z;
    if (alignment_idx >= n_alignments)
        return;

    const char* const query_begin               = sequences_d + 2 * alignment_idx * max_sequence_length;
    const char* const target_begin              = sequences_d + (2 * alignment_idx + 1) * max_sequence_length;
    const char* const query_end                 = query_begin + sequence_lengths_d[2 * alignment_idx];
    const char* const target_end                = target_begin + sequence_lengths_d[2 * alignment_idx + 1];
    int8_t* path                                = paths_base + alignment_idx * max_path_length;
    query_target_range* stack_buffer_begin      = stack_buffer_base + alignment_idx * stack_buffer_size_per_alignment;
    device_matrix_view<WordType> query_patterns = query_patternsi->get_matrix_view(alignment_idx, ceiling_divide<int32_t>(query_end - query_begin, word_size), 8);
    myers_preprocess(query_patterns, query_begin, query_end - query_begin);
    hirschberg_myers(stack_buffer_begin, stack_buffer_begin + stack_buffer_size_per_alignment, path, path_lengths + alignment_idx, full_myers_threshold, pvi, mvi, scorei, query_patterns, target_begin, target_end, query_begin, query_end, alignment_idx);
}

} // namespace hirschbergmyers

void hirschberg_myers_gpu(device_buffer<hirschbergmyers::query_target_range>& stack_buffer,
                          int32_t stack_buffer_size_per_alignment,
                          int8_t* paths_d, int32_t* path_lengths_d, int32_t max_path_length,
                          char const* sequences_d,
                          int32_t const* sequence_lengths_d,
                          int32_t max_sequence_length,
                          int32_t n_alignments,
                          batched_device_matrices<hirschbergmyers::WordType>& pv,
                          batched_device_matrices<hirschbergmyers::WordType>& mv,
                          batched_device_matrices<int32_t>& score,
                          batched_device_matrices<hirschbergmyers::WordType>& query_patterns,
                          int32_t switch_to_myers_threshold,
                          hipStream_t stream)
{
    using hirschbergmyers::warp_size;

    const dim3 threads(warp_size, 1, 1);
    const dim3 blocks(1, 1, ceiling_divide<int32_t>(n_alignments, threads.z));
    hirschbergmyers::hirschberg_myers_compute_alignment<<<blocks, threads, 0, stream>>>(stack_buffer.data(), stack_buffer_size_per_alignment, switch_to_myers_threshold, paths_d, path_lengths_d, max_path_length, pv.get_device_interface(), mv.get_device_interface(), score.get_device_interface(), query_patterns.get_device_interface(), sequences_d, sequence_lengths_d, max_sequence_length, n_alignments);
    GW_CU_CHECK_ERR(hipPeekAtLastError());
}

} // namespace cudaaligner

} // namespace genomeworks

} // namespace claraparabricks
