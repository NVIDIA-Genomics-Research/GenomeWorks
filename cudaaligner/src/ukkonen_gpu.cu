#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "ukkonen_gpu.cuh"
#include "batched_device_matrices.cuh"
#include <claragenomics/cudaaligner/cudaaligner.hpp>
#include <claragenomics/utils/limits.cuh>

#include <limits>
#include <cstdint>
#include <algorithm>
#include <cassert>
#include <thrust/tuple.h>

#define CGA_UKKONEN_MAX_THREADS_PER_BLOCK 1024

namespace claragenomics
{
namespace cudaaligner
{
namespace kernels
{

template <typename T>
__device__ T min3(T t1, T t2, T t3)
{
    return min(t1, min(t2, t3));
}

__device__ thrust::tuple<int, int> to_matrix_indices(int k, int l, int p)
{
    int const j = k - (p + l) / 2 + l;
    int const i = l - j;
    return thrust::make_tuple(i, j);
}

__device__ thrust::tuple<int, int> to_band_indices(int i, int j, int p)
{
    int const k = (j - i + p) / 2;
    int const l = (j + i);
    return thrust::make_tuple(k, l);
}

#ifndef NDEBUG
__launch_bounds__(CGA_UKKONEN_MAX_THREADS_PER_BLOCK) // Workaround for a register allocation problem when compiled with -g
#endif
    __global__ void ukkonen_backtrace_kernel(int8_t* paths_base, int32_t* lengths, int32_t max_path_length, batched_device_matrices<nw_score_t>::device_interface* s, int32_t const* sequence_lengths_d, int32_t n_alignments, int32_t p)
{
    // Using scoring schema from cudaaligner.hpp
    // Match = 0
    // Mismatch = 1
    // Insertion = 2
    // Deletion = 3

    using thrust::swap;
    using thrust::tie;
    int32_t const id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= n_alignments)
        return;

    CGA_CONSTEXPR nw_score_t max = numeric_limits<nw_score_t>::max() - 1;

    int32_t m        = sequence_lengths_d[2 * id] + 1;
    int32_t n        = sequence_lengths_d[2 * id + 1] + 1;
    int8_t insertion = static_cast<int8_t>(AlignmentState::insertion);
    int8_t deletion  = static_cast<int8_t>(AlignmentState::deletion);
    if (m > n)
    {
        swap(n, m);
        swap(insertion, deletion);
    }
    int8_t* path = paths_base + id * static_cast<ptrdiff_t>(max_path_length);
    assert(p >= 0);
    assert(n >= m);
    int32_t const bw                      = (1 + n - m + 2 * p + 1) / 2;
    device_matrix_view<nw_score_t> scores = s->get_matrix_view(id, bw, n + m);

    int32_t i = m - 1;
    int32_t j = n - 1;

    nw_score_t myscore = [scores, i, j, p] {
        int k, l;
        tie(k, l) = to_band_indices(i, j, p);
        return scores(k, l);
    }();
    int32_t pos = 0;
    while (i > 0 && j > 0)
    {
        int8_t r = 0;
        int k, l;
        tie(k, l)              = to_band_indices(i - 1, j, p);
        nw_score_t const above = k < 0 || k >= scores.num_rows() || l < 0 || l >= scores.num_cols() ? max : scores(k, l);
        tie(k, l)              = to_band_indices(i - 1, j - 1, p);
        nw_score_t const diag  = k < 0 || k >= scores.num_rows() || l < 0 || l >= scores.num_cols() ? max : scores(k, l);
        tie(k, l)              = to_band_indices(i, j - 1, p);
        nw_score_t const left  = k < 0 || k >= scores.num_rows() || l < 0 || l >= scores.num_cols() ? max : scores(k, l);

        if (left + 1 == myscore)
        {
            r       = insertion;
            myscore = left;
            --j;
        }
        else if (above + 1 == myscore)
        {
            r       = deletion;
            myscore = above;
            --i;
        }
        else
        {
            r       = (diag == myscore ? static_cast<int8_t>(AlignmentState::match) : static_cast<int8_t>(AlignmentState::mismatch));
            myscore = diag;
            --i;
            --j;
        }
        path[pos] = r;
        ++pos;
    }
    while (i > 0)
    {
        path[pos] = deletion;
        ++pos;
        --i;
    }
    while (j > 0)
    {
        path[pos] = insertion;
        ++pos;
        --j;
    }
    lengths[id] = pos;
}

__device__ void ukkonen_compute_score_matrix_odd(device_matrix_view<nw_score_t>& scores, int32_t kmax, int32_t k, int32_t m, int32_t n, char const* query, char const* target, int32_t max_target_query_length, int32_t p, int32_t l)
{
    CGA_CONSTEXPR nw_score_t max = numeric_limits<nw_score_t>::max() - 1;
    while (k < kmax)
    {
        int32_t const lmin = abs(2 * k + 1 - p);
        int32_t const lmax = 2 * k + 1 <= p ? 2 * (m - p + 2 * k + 1) + lmin : (2 * min(m, n - (2 * k + 1) + p) + lmin);
        if (lmin + 1 <= l && l < lmax)
        {
            int32_t const j        = k - (p + l) / 2 + l;
            int32_t const i        = l - j;
            nw_score_t const diag  = l - 2 < 0 ? max : scores(k, l - 2) + (query[i - 1] == target[j - 1] ? 0 : 1);
            nw_score_t const left  = l - 1 < 0 ? max : scores(k, l - 1) + 1;
            nw_score_t const above = l - 1 < 0 || k + 1 >= scores.num_rows() ? max : scores(k + 1, l - 1) + 1;
            scores(k, l)           = min3(diag, left, above);
        }
        k += blockDim.x;
    }
}

__device__ void ukkonen_compute_score_matrix_even(device_matrix_view<nw_score_t>& scores, int32_t kmax, int32_t k, int32_t m, int32_t n, char const* query, char const* target, int32_t max_target_query_length, int32_t p, int32_t l)
{
    CGA_CONSTEXPR nw_score_t max = numeric_limits<nw_score_t>::max() - 1;
    while (k < kmax)
    {
        int32_t const lmin = abs(2 * k - p);
        int32_t const lmax = 2 * k <= p ? 2 * (m - p + 2 * k) + lmin : (2 * min(m, n - 2 * k + p) + lmin);
        if (lmin + 1 <= l && l < lmax)
        {
            int32_t const j        = k - (p + l) / 2 + l;
            int32_t const i        = l - j;
            nw_score_t const left  = k - 1 < 0 || l - 1 < 0 ? max : scores(k - 1, l - 1) + 1;
            nw_score_t const diag  = l - 2 < 0 ? max : scores(k, l - 2) + (query[i - 1] == target[j - 1] ? 0 : 1);
            nw_score_t const above = l - 1 < 0 ? max : scores(k, l - 1) + 1;
            scores(k, l)           = min3(left, diag, above);
        }
        k += blockDim.x;
    }
}

__device__ void ukkonen_init_score_matrix(device_matrix_view<nw_score_t>& scores, int32_t k, int32_t p)
{
    CGA_CONSTEXPR nw_score_t max = numeric_limits<nw_score_t>::max() - 1;
    while (k < scores.num_rows())
    {
        for (int32_t l = 0; l < scores.num_cols(); ++l)
        {
            nw_score_t init_value = max;
            int32_t i, j;
            thrust::tie(i, j) = to_matrix_indices(k, l, p);

            if (i == 0)
                init_value = j;
            else if (j == 0)
                init_value = i;

            scores(k, l) = init_value;
        }
        k += blockDim.x;
    }
}

#ifndef NDEBUG
__launch_bounds__(CGA_UKKONEN_MAX_THREADS_PER_BLOCK) // Workaround for a register allocation problem when compiled with -g
#endif
    __global__ void ukkonen_compute_score_matrix(batched_device_matrices<nw_score_t>::device_interface* s, char const* sequences_d, int32_t const* sequence_lengths_d, int32_t max_target_query_length, int32_t p, int32_t max_cols)
{
    using thrust::swap;
    int32_t const k  = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t const id = blockIdx.y * blockDim.y + threadIdx.y;

    int32_t m          = sequence_lengths_d[2 * id] + 1;
    int32_t n          = sequence_lengths_d[2 * id + 1] + 1;
    char const* query  = sequences_d + (2 * id) * max_target_query_length;
    char const* target = sequences_d + (2 * id + 1) * max_target_query_length;
    if (m > n)
    {
        swap(n, m);
        swap(query, target);
    }
    assert(p >= 0);
    int32_t const bw        = (1 + n - m + 2 * p + 1) / 2;
    int32_t const kmax_odd  = (n - m + 2 * p - 1) / 2 + 1;
    int32_t const kmax_even = (n - m + 2 * p) / 2 + 1;

    device_matrix_view<nw_score_t> scores = s->get_matrix_view(id, bw, n + m);
    ukkonen_init_score_matrix(scores, k, p);
    __syncthreads();
    if (p % 2 == 0)
    {
        for (int lx = 0; lx < 2 * max_cols; ++lx)
        {
            ukkonen_compute_score_matrix_even(scores, kmax_even, k, m, n, query, target, max_target_query_length, p, 2 * lx);
            __syncthreads();
            ukkonen_compute_score_matrix_odd(scores, kmax_odd, k, m, n, query, target, max_target_query_length, p, 2 * lx + 1);
            __syncthreads();
        }
    }
    else
    {
        for (int lx = 0; lx < 2 * max_cols; ++lx)
        {
            ukkonen_compute_score_matrix_odd(scores, kmax_odd, k, m, n, query, target, max_target_query_length, p, 2 * lx);
            __syncthreads();
            ukkonen_compute_score_matrix_even(scores, kmax_even, k, m, n, query, target, max_target_query_length, p, 2 * lx + 1);
            __syncthreads();
        }
    }
}

} // end namespace kernels

dim3 calc_blocks(dim3 const& n_threads, dim3 const& blocksize)
{
    dim3 r;
    r.x = (n_threads.x + blocksize.x - 1) / blocksize.x;
    r.y = (n_threads.y + blocksize.y - 1) / blocksize.y;
    r.z = (n_threads.z + blocksize.z - 1) / blocksize.z;
    return r;
}

constexpr int32_t calc_good_blockdim(int32_t n)
{
    constexpr int32_t warpsize = 32;
    int32_t i                  = n + (warpsize - n % warpsize);
    return i > CGA_UKKONEN_MAX_THREADS_PER_BLOCK ? CGA_UKKONEN_MAX_THREADS_PER_BLOCK : i;
}

void ukkonen_compute_score_matrix_gpu(batched_device_matrices<nw_score_t>& score_matrices, char const* sequences_d, int32_t const* sequence_lengths_d, int32_t max_length_difference, int32_t max_target_query_length, int32_t n_alignments, int32_t p, hipStream_t stream)
{
    using kernels::ukkonen_compute_score_matrix;
    assert(p >= 0);
    assert(max_length_difference >= 0);
    assert(max_target_query_length >= 0);

    int32_t const max_bw   = (1 + max_length_difference + 2 * p + 1) / 2;
    int32_t const max_cols = 2 * (max_target_query_length + 1);

    // Transform to diagonal coordinates
    // (i,j) -> (k=(j-i+p)/2, l=(j+i))
    dim3 const compute_blockdims(calc_good_blockdim(max_bw), 1, 1);
    dim3 const blocks = dim3(1, n_alignments, 1);

    ukkonen_compute_score_matrix<<<blocks, compute_blockdims, 0, stream>>>(score_matrices.get_device_interface(), sequences_d, sequence_lengths_d, max_target_query_length, p, max_cols);
    CGA_CU_CHECK_ERR(hipPeekAtLastError());
}

void ukkonen_backtrace_gpu(int8_t* paths_d, int32_t* path_lengths_d, int32_t max_path_length, batched_device_matrices<nw_score_t>& scores, int32_t const* sequence_lengths_d, int32_t n_alignments, int32_t p, hipStream_t stream)
{
    kernels::ukkonen_backtrace_kernel<<<n_alignments, 1, 0, stream>>>(paths_d, path_lengths_d, max_path_length, scores.get_device_interface(), sequence_lengths_d, n_alignments, p);
    CGA_CU_CHECK_ERR(hipPeekAtLastError());
}

void ukkonen_gpu(int8_t* paths_d, int32_t* path_lengths_d, int32_t max_path_length,
                 char const* sequences_d,
                 int32_t const* sequence_lengths_d,
                 int32_t max_length_difference,
                 int32_t max_target_query_length,
                 int32_t n_alignments,
                 batched_device_matrices<nw_score_t>* score_matrices,
                 int32_t ukkonen_p,
                 hipStream_t stream)
{
    if (score_matrices == nullptr)
        return;

    ukkonen_compute_score_matrix_gpu(*score_matrices, sequences_d, sequence_lengths_d, max_length_difference, max_target_query_length, n_alignments, ukkonen_p, stream);
    ukkonen_backtrace_gpu(paths_d, path_lengths_d, max_path_length, *score_matrices, sequence_lengths_d, n_alignments, ukkonen_p, stream);
}

size_t ukkonen_max_score_matrix_size(int32_t max_query_length, int32_t max_target_length, int32_t max_length_difference, int32_t max_p)
{
    assert(max_target_length >= 0);
    assert(max_query_length >= 0);
    assert(max_p >= 0);
    assert(max_length_difference >= 0);
    size_t const max_target_query_length = std::max(max_target_length, max_query_length);
    size_t const n                       = max_target_query_length + 1;
    size_t const m                       = max_target_query_length + 1;
    size_t const bw                      = (1 + max_length_difference + 2 * max_p + 1) / 2;
    return bw * (n + m);
}

} // end namespace cudaaligner
} // end namespace claragenomics
