#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <limits>
#include "minimizer.hpp"

#include <thrust/scan.h>
#include <thrust/execution_policy.h>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

Minimizer::Minimizer(representation_t representation, position_in_read_t position_in_read, DirectionOfRepresentation direction, read_id_t read_id)
    : representation_(representation)
    , position_in_read_(position_in_read)
    , direction_(direction)
    , read_id_(read_id)
{
}

representation_t Minimizer::representation() const
{
    return representation_;
}

position_in_read_t Minimizer::position_in_read() const
{
    return position_in_read_;
}

read_id_t Minimizer::read_id() const
{
    return read_id_;
}

/// \brief Apply a hash function to a representation
///
/// Because of the non-Poisson distribuition of DNA, some common sequences with common kmer-content (e.g long poly-A runs)
/// may be over-represented in sketches. By applying a hash function, kmers are mapped to representations over
/// a more uniform space. The hash function implemented here was developed by Thomas Wang and is described
/// [here](https://gist.github.com/badboy/6267743). A mask is applied to the output so that all representations are mapped
/// to a 32 bit space.
///
/// \param key the input representation
__device__ representation_t wang_hash64(representation_t key)
{
    uint64_t mask = (uint64_t(1) << 32) - 1;
    key           = (~key + (key << 21)) & mask;
    key           = key ^ key >> 24;
    key           = ((key + (key << 3)) + (key << 8)) & mask;
    key           = key ^ key >> 14;
    key           = ((key + (key << 2)) + (key << 4)) & mask;
    key           = key ^ key >> 28;
    key           = (key + (key << 31)) & mask;
    return key;
}

Minimizer::DirectionOfRepresentation Minimizer::direction() const
{
    return direction_;
}

/// \brief finds front end minimizers
///
/// Finds the minimizers of windows starting at position 0 and having window size range from 1 to window_size-1
///
/// \param minimizer_size kmer length
/// \param window_size number of kmers in one central minimizer window, kmers being shifted by one basepair each (for front end minimizers window size actually varies from 1 to window_size-1)
/// \param basepairs array of basepairs, first come basepairs for read 0, then read 1 and so on
/// \param read_id_to_basepairs_section index of the first basepair of every read (in basepairs array) and the number of basepairs in that read
/// \param window_minimizers_representation output array of representations of minimizers, grouped by reads
/// \param window_minimizers_direction output array of directions of minimizers, grouped by reads (0 - forward, 1 - reverse)
/// \param window_minimizers_position_in_read output array of positions in read of minimizers, grouped by reads
/// \param read_id_to_window_section index of first element dedicated to that read in output arrays and the number of dedicated elements (enough elements are allocated to each read to support cases where each window has a different minimizer, no need to check that condition)
/// \param read_id_to_minimizers_written how many minimizers have been written for this read already (initially zero)
__global__ void find_front_end_minimizers(const std::uint64_t minimizer_size,
                                          const std::uint64_t window_size,
                                          const char* const basepairs,
                                          const ArrayBlock* const read_id_to_basepairs_section,
                                          representation_t* const window_minimizers_representation,
                                          char* const window_minimizers_direction,
                                          position_in_read_t* const window_minimizers_position_in_read,
                                          const ArrayBlock* const read_id_to_windows_section,
                                          std::int64_t* const read_id_to_minimizers_written,
                                          const bool hash_representations)
{
    // TODO: simplify this method similarly to find_back_end_minimizers

    if (1 == window_size)
    {
        // if 1 == window_size there are no end minimizer
        return;
    }

    const auto input_array_first_element = read_id_to_basepairs_section[blockIdx.x].first_element_;
    const auto output_arrays_offset      = read_id_to_windows_section[blockIdx.x].first_element_;

    // Dynamically allocating shared memory and assigning parts of it to different pointers
    // Everything is 8-byte alligned
    extern __shared__ std::uint64_t sm[];
    // TODO: not all arrays are needed at the same time -> reduce shared memory requirements by reusing parts of the memory
    // TODO: use sizeof to get the number of bytes
    std::uint32_t shared_memory_64_bit_elements_already_taken = 0;
    char* forward_basepair_hashes                             = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // blockDim.x elements
    shared_memory_64_bit_elements_already_taken += (blockDim.x + 7) / 8;

    char* reverse_basepair_hashes = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // blockDim.x elements
    shared_memory_64_bit_elements_already_taken += (blockDim.x + 7) / 8;

    representation_t* minimizers_representation = reinterpret_cast<representation_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // blockDim.x - (minimizer_size - 1) elements
    shared_memory_64_bit_elements_already_taken += blockDim.x - (minimizer_size - 1);

    char* minimizers_direction = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // blockDim.x - (minimizer_size - 1) elements; 0 - forward, 1 - reverse
    shared_memory_64_bit_elements_already_taken += (blockDim.x - (minimizer_size - 1) + 7) / 8;

    position_in_read_t* minimizers_position_in_read = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]);
    shared_memory_64_bit_elements_already_taken += (blockDim.x - (minimizer_size - 1) + 1) / 2;

    position_in_read_t* different_minimizer_than_neighbors = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // blockDim.x - (minimizer_size - 1) elements; 0 - same, 1 - different
    shared_memory_64_bit_elements_already_taken += (blockDim.x - (minimizer_size - 1) + 1) / 2;

    representation_t* minimizer_representation_of_largest_window_from_previous_step = (&sm[shared_memory_64_bit_elements_already_taken]); // 1 element
    shared_memory_64_bit_elements_already_taken += 1;

    position_in_read_t* minimizer_position_in_read_of_largest_window_from_previous_step = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // 1 element
    shared_memory_64_bit_elements_already_taken += (1 + 1) / 2;

    // local index = index in section of the output array dedicated to this read
    position_in_read_t* local_index_to_write_next_minimizer_to = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // 1 element
    shared_memory_64_bit_elements_already_taken += (1 + 1) / 2;

    // TODO: Move to constant memory
    char* forward_to_reverse_complement = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // 8 elements

    if (0 == threadIdx.x)
    {
        forward_to_reverse_complement[0b000] = 0b0000;
        forward_to_reverse_complement[0b001] = 0b0100; // A -> T (0b1 -> 0b10100)
        forward_to_reverse_complement[0b010] = 0b0000;
        forward_to_reverse_complement[0b011] = 0b0111; // C -> G (0b11 -> 0b111)
        forward_to_reverse_complement[0b100] = 0b0001; // T -> A (0b10100 -> 0b1)
        forward_to_reverse_complement[0b101] = 0b0000;
        forward_to_reverse_complement[0b110] = 0b0000;
        forward_to_reverse_complement[0b111] = 0b0011; // G -> C (0b111 -> 0b11)
    }
    __syncthreads();

    // Each thread loads one basepair, making it blockDim.x basepairs. Each kmer has minimizer_size elements
    // Number of windows is equal to the number of kmers for end minimizer
    // This means a total of blockDim.x - (minimizer_size - 1) kmer can be processed in one block, where each kmer is shifted by one one basepair compared to the previous kmer
    // For blockDim.x = 6 and minimizer_size = 3 there are 6 - (3 - 1) = 4  kmers
    // 0 1 2
    //   1 2 3
    //     2 3 4
    //       3 4 5
    // If more minimizers have to be processed a new step is needed, in this case meaning
    //         4 5 6
    //           5 6 7
    //             6 7 8
    //               7 8 9
    // This means that a number of basepairs is loaded twice, but this a tradeoff for less complex code
    const std::uint16_t windows_per_loop_step = blockDim.x - (minimizer_size - 1);

    *minimizer_representation_of_largest_window_from_previous_step   = 0;
    *minimizer_position_in_read_of_largest_window_from_previous_step = 0;
    *local_index_to_write_next_minimizer_to                          = 0;

    for (std::uint32_t first_element_in_step = 0; first_element_in_step < window_size - 1; first_element_in_step += windows_per_loop_step)
    {

        // load basepairs into shared memory and calculate the lexical ordering hash
        if (first_element_in_step + threadIdx.x < window_size - 1 + minimizer_size - 1)
        { // window_size - 1 + minimizer_size - 1 -> total number of basepairs needed for all front minimizers
            const char bp                        = basepairs[input_array_first_element + first_element_in_step + threadIdx.x];
            forward_basepair_hashes[threadIdx.x] = 0b11 & (bp >> 2 ^ bp >> 1);
            reverse_basepair_hashes[threadIdx.x] = 0b11 & (forward_to_reverse_complement[0b111 & bp] >> 2 ^ forward_to_reverse_complement[0b111 & bp] >> 1);
        }
        __syncthreads();

        // First front end window covers only one minimizer (the one starting at positon 0), second minimizers starting at 0 and 1 and so one until the window which covers window_size-1 minimizers
        // For window_size = 7 and minimize_size = 3 this means:
        // window 0: 0 1 2                  (0 1 2)
        // window 1: 0 1 2 3                (0 1 2; 1 2 3)
        // widnow 2: 0 1 2 3 4              (0 1 2; 1 2 3; 2 3 4)
        // window 3: 0 1 2 3 4 5            (0 1 2; 1 2 3; 2 3 4; 3 4 5)
        // window 4: 0 1 2 3 4 5 6          (0 1 2; 1 2 3; 2 3 4; 3 4 5; 4 5 6)
        // window 5: 0 1 2 3 4 5 6 7        (0 1 2; 1 2 3; 2 3 4; 3 4 5; 4 5 6; 5 6 7)
        // If window_size > windows_per_loop_step the other windows have to be processed in other loop steps
        // For example, for blockDim.x = 6, minimizer_size = 3 (=> windows_per_loop_step = 4) and window_size = 7:
        // step 0 (first_element_in_step = 0):
        // window 0: 0 1 2                  (0 1 2)
        // window 1: 0 1 2 3                (0 1 2; 1 2 3)
        // widnow 2: 0 1 2 3 4              (0 1 2; 1 2 3; 2 3 4)
        // window 3: 0 1 2 3 4 5            (0 1 2; 1 2 3; 2 3 4; 3 4 5)
        // step 1 (first_element_in_step = 4):
        // window 4: 0 1 2 3 4 5 6          (take results for window 3 and add: 4 5 6)
        // window 5: 0 1 2 3 4 5 6 7        (take results for window 3 and add: 4 5 6; 5 6 7)
        // This means that a thread has a window assigned to it when thraedIdx.x < minimizers_per_loop (for all loops other than the last one) and
        // when first_element_in_step + threadIdx.x < window_size - 1
        const bool thread_assigned_to_a_window = first_element_in_step + threadIdx.x < window_size - 1 && threadIdx.x < windows_per_loop_step;

        // calculate minimizer for each kmer in front end windows
        if (thread_assigned_to_a_window)
        { // largest front minimizer window starts at basepar 0 and goes up to window_size -1
            representation_t forward_representation = 0;
            representation_t reverse_representation = 0;
            // TODO: It's not necessary to fully build both representations in order to determine which one is smaller. In most cases there is going to be a difference already at the first element
            for (std::uint16_t i = 0; i < minimizer_size; ++i)
            {
                forward_representation |= forward_basepair_hashes[threadIdx.x + i] << 2 * (minimizer_size - i - 1);
                reverse_representation |= reverse_basepair_hashes[threadIdx.x + i] << 2 * i;
            }

            if (hash_representations)
            {
                forward_representation = wang_hash64(forward_representation);
                reverse_representation = wang_hash64(reverse_representation);
            }

            if (forward_representation <= reverse_representation)
            {
                minimizers_representation[threadIdx.x] = forward_representation;
                minimizers_direction[threadIdx.x]      = 0;
            }
            else
            {
                minimizers_representation[threadIdx.x] = reverse_representation;
                minimizers_direction[threadIdx.x]      = 1;
            }
        }
        __syncthreads();

        representation_t window_minimizer_representation     = 0;
        position_in_read_t window_minimizer_position_in_read = 0;
        // calculate minimizer for each window
        // Start by the value of the first minimizer and iteratively compare it with the other minimizers in the window
        // If first_element_in_step != 0 there is no need to go through all minimizers in the window. One can take the minimizer of window first_element_in_step-1
        // as the current window would check exaclty the same minimizers before checking minimizer first_element_in_step
        if (thread_assigned_to_a_window)
        {
            if (first_element_in_step != 0)
            {
                window_minimizer_representation   = *minimizer_representation_of_largest_window_from_previous_step;
                window_minimizer_position_in_read = *minimizer_position_in_read_of_largest_window_from_previous_step;
                if (minimizers_representation[0] <= window_minimizer_representation)
                {
                    window_minimizer_representation   = minimizers_representation[0];
                    window_minimizer_position_in_read = first_element_in_step;
                }
            }
            else
            {
                window_minimizer_representation   = minimizers_representation[0];
                window_minimizer_position_in_read = 0;
            }
            // All threads have to wait for the largest block to finish. Probably no better solution without big restructuring
            // If there are several minimizers with the same representation only save the latest one (thus <=), others will be covered by smaller windows
            for (std::uint16_t i = 1; i <= threadIdx.x; ++i)
            {
                if (minimizers_representation[i] <= window_minimizer_representation)
                {
                    window_minimizer_representation   = minimizers_representation[i];
                    window_minimizer_position_in_read = first_element_in_step + i;
                }
            }
            minimizers_position_in_read[threadIdx.x] = window_minimizer_position_in_read;
        }
        __syncthreads();

        // only write first occurence of each minimizer to the output array
        // Hash of the last kmer in a window can be a minimizer only if it is smaller or equal than the minimizer of the previous window
        // That means that the minimizer of the current window should only be written if it is different than the one of the previous window
        // Otherwise it it the same minimizer and there is no need write to the the output array
        // Imagine that hash representation of windows are are follows (the number in the parentheses marks the position of the last occurance of the minimizer with that representation):
        // 8, 89, 898, 8987, 89878, 898785, 8987856, 89878562
        // Minimizers of these windows are
        // 8(0) 8(0) 8(2) 7(3) 7(3) 5(5) 5(5) 2(7)
        // If we use 1 to indicate the first occurence of minimizer and 0 for repretition we get
        // 1 0 1 1 0 1 0 1
        // If we do an an inclusive scan on this array we get the indices to which the unique minimizers should be written to (plus one)
        // 1 1 2 3 3 4 4 5
        // From this it's clear that only the windows whose value is larger than the one of its neighbor should write its minimizer and it should write to the element with the index of value-1
        if (first_element_in_step + threadIdx.x < window_size - 1 && threadIdx.x < windows_per_loop_step)
        {
            if (0 == first_element_in_step && 0 == threadIdx.x)
            {
                // minimizer of first window is unique for sure as it has no left neighbor
                different_minimizer_than_neighbors[0] = 1;
            }
            else
            {
                representation_t neighbors_minimizers_position_in_read = 0;
                // find left neighbor's window minimizer's position in read
                if (0 == threadIdx.x)
                {
                    neighbors_minimizers_position_in_read = *minimizer_position_in_read_of_largest_window_from_previous_step;
                }
                else
                {
                    // TODO: consider using warp shuffle instead of shared memory
                    neighbors_minimizers_position_in_read = minimizers_position_in_read[threadIdx.x - 1];
                }
                // check if it's the same minimizer
                if (neighbors_minimizers_position_in_read == minimizers_position_in_read[threadIdx.x])
                {
                    different_minimizer_than_neighbors[threadIdx.x] = 0;
                }
                else
                {
                    different_minimizer_than_neighbors[threadIdx.x] = 1;
                }
            }
        }
        __syncthreads();

        // if there are more loop steps to follow write the value and position of minimizer of the largest window
        if (first_element_in_step + windows_per_loop_step < window_size - 1 && threadIdx.x == windows_per_loop_step - 1)
        {
            *minimizer_representation_of_largest_window_from_previous_step   = window_minimizer_representation;
            *minimizer_position_in_read_of_largest_window_from_previous_step = window_minimizer_position_in_read;
        }
        // no need to sync, these two values are not used before the next sync

        // perform inclusive scan
        // different_minimizer_than_neighbors changes meaning an becomes more like "output_array_index_to_write_the_value_plus_one"
        // TODO: implement it using warp shuffle or use CUB
        if (0 == threadIdx.x)
        {
            std::uint16_t i = 0;
            different_minimizer_than_neighbors[i] += *local_index_to_write_next_minimizer_to;
            for (i = 1; i < blockDim.x - (minimizer_size - 1); ++i)
            {
                different_minimizer_than_neighbors[i] += different_minimizer_than_neighbors[i - 1];
            }
        }
        __syncthreads();

        // now save minimizers to output array
        if (first_element_in_step + threadIdx.x < window_size - 1 && threadIdx.x < windows_per_loop_step)
        {
            const std::uint32_t neighbors_write_index = 0 == threadIdx.x ? *local_index_to_write_next_minimizer_to : different_minimizer_than_neighbors[threadIdx.x - 1];
            if (neighbors_write_index < different_minimizer_than_neighbors[threadIdx.x])
            {
                const std::uint64_t output_index                 = output_arrays_offset + different_minimizer_than_neighbors[threadIdx.x] - 1;
                window_minimizers_representation[output_index]   = minimizers_representation[minimizers_position_in_read[threadIdx.x] - first_element_in_step];
                window_minimizers_direction[output_index]        = minimizers_direction[minimizers_position_in_read[threadIdx.x] - first_element_in_step];
                window_minimizers_position_in_read[output_index] = minimizers_position_in_read[threadIdx.x];
            }
        }
        __syncthreads();

        // index (plus one) to which the last window minimizer was written is the number of all unique front end window minimizers
        if (first_element_in_step + threadIdx.x == window_size - 1 - 1)
        {
            // "plus one" is already included in different_minimizer_than_neighbors as it was created by an inclusive scan
            read_id_to_minimizers_written[blockIdx.x] = different_minimizer_than_neighbors[threadIdx.x];
        }

        // if there are more loop steps to follow write the output array index of last minimizer in this loop step
        if (first_element_in_step + windows_per_loop_step <= window_size - 1 && threadIdx.x == windows_per_loop_step - 1)
        {
            *local_index_to_write_next_minimizer_to = different_minimizer_than_neighbors[threadIdx.x];
        }
    }
}

/// \brief finds central minimizers
///
/// Finds the minimizers of windows of size window_size starting at position 0 and moving by one basepair at a time
///
/// \param minimizer_size kmer length
/// \param window_size number of kmers in one window, kmers being shifted by one one basepair each
/// \param basepairs array of basepairs, first come basepairs for read 0, then read 1 and so on
/// \param read_id_to_basepairs_section index of the first basepair of every read (in basepairs array) and the number of basepairs in that read
/// \param window_minimizers_representation output array of representations of minimizers, grouped by reads
/// \param window_minimizers_direction output array of directions of minimizers, grouped by reads (0 - forward, 1 - reverse)
/// \param window_minimizers_position_in_read output array of positions in read of minimizers, grouped by reads
/// \param read_id_to_window_section index of first element dedicated to that read in output arrays and the number of dedicated elements (enough elements are allocated to each read to support cases where each window has a different minimizer, no need to check that condition)
/// \param read_id_to_minimizers_written how many minimizers have been written for this read already (initially number of front end minimizers)
__global__ void find_central_minimizers(const std::uint64_t minimizer_size,
                                        const std::uint64_t window_size,
                                        const std::uint32_t basepairs_per_thread,
                                        const char* const basepairs,
                                        const ArrayBlock* const read_id_to_basepairs_section,
                                        representation_t* const window_minimizers_representation,
                                        char* const window_minimizers_direction,
                                        position_in_read_t* const window_minimizers_position_in_read,
                                        const ArrayBlock* const read_id_to_windows_section,
                                        std::int64_t* const read_id_to_minimizers_written,
                                        const bool hash_representations)
{
    // See find_front_end_minimizers for more details about the algorithm

    const std::uint64_t index_of_first_element_to_process_global = read_id_to_basepairs_section[blockIdx.x].first_element_;
    // Index of the element to which the first central minimizer of this read should be written. Index refers to the positions withing the whole array dedicated to all reads
    const std::uint64_t output_index_to_write_the_first_minimizer_global = read_id_to_windows_section[blockIdx.x].first_element_ + read_id_to_minimizers_written[blockIdx.x];
    const std::uint32_t basepairs_in_read                                = read_id_to_basepairs_section[blockIdx.x].block_size_;
    const std::uint32_t kmers_in_read                                    = basepairs_in_read - (minimizer_size - 1);
    const std::uint32_t windows_in_read                                  = kmers_in_read - (window_size - 1);
    const std::uint16_t basepairs_per_loop_step                          = blockDim.x * basepairs_per_thread;
    const std::uint16_t kmers_per_loop_step                              = basepairs_per_loop_step - (minimizer_size - 1);
    const std::uint16_t windows_per_loop_step                            = kmers_per_loop_step - (window_size - 1);

    // Dynamically allocating shared memory and assigning parts of it to different pointers
    // Everything is 8-byte alligned
    extern __shared__ std::uint64_t sm[];
    // TODO: not all arrays are needed at the same time -> reduce shared memory requirements by reusing parts of the memory
    // TODO: use sizeof to get the number of bytes
    std::uint32_t shared_memory_64_bit_elements_already_taken = 0;
    char* forward_basepair_hashes                             = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // basepairs_per_loop_step elements
    shared_memory_64_bit_elements_already_taken += (basepairs_per_loop_step + 7) / 8;

    char* reverse_basepair_hashes = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // basepairs_per_loop_step elements
    shared_memory_64_bit_elements_already_taken += (basepairs_per_loop_step + 7) / 8;

    representation_t* minimizers_representation = reinterpret_cast<representation_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // kmers_loop_step elements
    shared_memory_64_bit_elements_already_taken += kmers_per_loop_step;

    char* minimizers_direction = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // windows_per_loop_step elements; 0 - forward, 1 - reverse
    shared_memory_64_bit_elements_already_taken += (windows_per_loop_step + 7) / 8;

    position_in_read_t* minimizers_position_in_read = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]);
    shared_memory_64_bit_elements_already_taken += (windows_per_loop_step + 1) / 2;

    position_in_read_t* different_minimizer_than_neighbors = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // windows_per_loop_step elements; 0 - same, 1 - different
    shared_memory_64_bit_elements_already_taken += (windows_per_loop_step + 1) / 2;

    position_in_read_t* minimizer_position_in_read_of_largest_window_from_previous_step = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // 1 element
    shared_memory_64_bit_elements_already_taken += (1 + 1) / 2;

    position_in_read_t* local_index_to_write_next_minimizer_to = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // 1 element
    shared_memory_64_bit_elements_already_taken += (1 + 1) / 2;

    // TODO: Move to constant memory
    char* forward_to_reverse_complement = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // 8 elements

    if (0 == threadIdx.x)
    {
        forward_to_reverse_complement[0b000] = 0b0000;
        forward_to_reverse_complement[0b001] = 0b0100; // A -> T (0b1 -> 0b10100)
        forward_to_reverse_complement[0b010] = 0b0000;
        forward_to_reverse_complement[0b011] = 0b0111; // C -> G (0b11 -> 0b111)
        forward_to_reverse_complement[0b100] = 0b0001; // T -> A (0b10100 -> 0b1)
        forward_to_reverse_complement[0b101] = 0b0000;
        forward_to_reverse_complement[0b110] = 0b0000;
        forward_to_reverse_complement[0b111] = 0b0011; // G -> C (0b111 -> 0b11)
    }
    __syncthreads();

    // if there are front minimizers take them into account
    if (0 != read_id_to_minimizers_written[blockIdx.x])
    {
        *minimizer_position_in_read_of_largest_window_from_previous_step = window_minimizers_position_in_read[output_index_to_write_the_first_minimizer_global - 1];
        *local_index_to_write_next_minimizer_to                          = read_id_to_minimizers_written[blockIdx.x];
    }
    else
    {
        *minimizer_position_in_read_of_largest_window_from_previous_step = 0; // N/A
        *local_index_to_write_next_minimizer_to                          = 0;
    }

    for (std::uint32_t first_element_in_step = 0; first_element_in_step < windows_in_read; first_element_in_step += windows_per_loop_step)
    {
        // load basepairs into shared memory and calculate the lexical ordering hash
        for (std::uint32_t basepair_index = threadIdx.x; basepair_index < basepairs_per_loop_step && first_element_in_step + basepair_index < basepairs_in_read; basepair_index += blockDim.x)
        {
            const char bp                           = basepairs[index_of_first_element_to_process_global + first_element_in_step + basepair_index];
            forward_basepair_hashes[basepair_index] = 0b11 & (bp >> 2 ^ bp >> 1);
            reverse_basepair_hashes[basepair_index] = 0b11 & (forward_to_reverse_complement[0b111 & bp] >> 2 ^ forward_to_reverse_complement[0b111 & bp] >> 1);
        }
        __syncthreads();

        // calculate kmer minimizers
        for (std::uint32_t kmer_index = threadIdx.x; kmer_index < kmers_per_loop_step && first_element_in_step + kmer_index < kmers_in_read; kmer_index += blockDim.x)
        {
            representation_t forward_representation = 0;
            representation_t reverse_representation = 0;
            // TODO: It's not necessary to fully build both representations in order to determine which one is smaller. In most cases there is going to be a difference already at the first element
            for (std::uint16_t i = 0; i < minimizer_size; ++i)
            {
                forward_representation |= forward_basepair_hashes[kmer_index + i] << 2 * (minimizer_size - i - 1);
                reverse_representation |= reverse_basepair_hashes[kmer_index + i] << 2 * i;
            }

            if (hash_representations)
            {
                forward_representation = wang_hash64(forward_representation);
                reverse_representation = wang_hash64(reverse_representation);
            }
            if (forward_representation <= reverse_representation)
            {
                minimizers_representation[kmer_index] = forward_representation;
                minimizers_direction[kmer_index]      = 0;
            }
            else
            {
                minimizers_representation[kmer_index] = reverse_representation;
                minimizers_direction[kmer_index]      = 1;
            }
        }
        __syncthreads();

        position_in_read_t window_minimizer_position_in_read = 0;
        // find window minimizer
        for (std::uint32_t window_index = threadIdx.x; window_index < windows_per_loop_step && first_element_in_step + window_index < windows_in_read; window_index += blockDim.x)
        {
            // assume that the minimizer of the first kmer in step is the window minimizer
            representation_t window_minimizer_representation = minimizers_representation[window_index];
            window_minimizer_position_in_read                = first_element_in_step + window_index;
            // now check the minimizers of all other windows
            for (std::uint16_t i = 1; i < window_size; ++i)
            {
                if (minimizers_representation[window_index + i] <= window_minimizer_representation)
                {
                    window_minimizer_representation   = minimizers_representation[window_index + i];
                    window_minimizer_position_in_read = first_element_in_step + window_index + i;
                }
            }
            minimizers_position_in_read[window_index] = window_minimizer_position_in_read;
        }
        __syncthreads();

        // check if the window to the left has a the same minimizer
        for (std::uint32_t window_index = threadIdx.x; window_index < windows_per_loop_step && first_element_in_step + window_index < windows_in_read; window_index += blockDim.x)
        {
            // if this is the first window in read and there were no front end minimizers than this is the first occurence of this minimizer
            if (0 == first_element_in_step + window_index && 0 == read_id_to_minimizers_written[blockIdx.x])
            {
                different_minimizer_than_neighbors[0] = 1;
            }
            else
            {
                representation_t neighbors_minimizers_position_in_read = 0;
                // find left neighbor's window minimizer's position in read
                if (0 == window_index)
                {
                    neighbors_minimizers_position_in_read = *minimizer_position_in_read_of_largest_window_from_previous_step;
                }
                else
                {
                    // TODO: consider using warp shuffle instead of shared memory
                    neighbors_minimizers_position_in_read = minimizers_position_in_read[window_index - 1];
                }
                // check if it's the same minimizer
                if (neighbors_minimizers_position_in_read == minimizers_position_in_read[window_index])
                {
                    different_minimizer_than_neighbors[window_index] = 0;
                }
                else
                {
                    different_minimizer_than_neighbors[window_index] = 1;
                }
            }
        }
        __syncthreads();

        // if there are more loop steps to follow write the position of minimizer of the last window
        // "windows_per_loop_step % blockDim.x - 1" determines the thread which processes the last window
        if (first_element_in_step + windows_per_loop_step < windows_in_read && threadIdx.x == windows_per_loop_step % blockDim.x - 1)
        {
            *minimizer_position_in_read_of_largest_window_from_previous_step = window_minimizer_position_in_read;
        }

        // perform inclusive scan
        // different_minimizer_than_neighbors changes meaning an becomes more like "output_array_index_to_write_the_value_plus_one"
        // TODO: implement it using warp shuffle or use CUB
        if (0 == threadIdx.x)
        {
            std::uint16_t i = 0;
            different_minimizer_than_neighbors[i] += *local_index_to_write_next_minimizer_to;
            for (i = 1; i < windows_per_loop_step && first_element_in_step + i < windows_in_read; ++i)
            {
                different_minimizer_than_neighbors[i] += different_minimizer_than_neighbors[i - 1];
            }
        }
        __syncthreads();

        // now save minimizers to output array
        for (std::uint32_t window_index = threadIdx.x; window_index < windows_per_loop_step && first_element_in_step + window_index < windows_in_read; window_index += blockDim.x)
        {
            // if first_element_in_loop == 0 and window_index == 0 then *local_index_to_write_next_minimizer_to is set to 0 before entering the loop
            const std::uint32_t neighbors_write_index = 0 == window_index ? *local_index_to_write_next_minimizer_to : different_minimizer_than_neighbors[window_index - 1];
            if (neighbors_write_index < different_minimizer_than_neighbors[window_index])
            {
                // output array offset added in inclusive sum
                const auto output_index                          = read_id_to_windows_section[blockIdx.x].first_element_ + different_minimizer_than_neighbors[window_index] - 1;
                window_minimizers_representation[output_index]   = minimizers_representation[minimizers_position_in_read[window_index] - first_element_in_step];
                window_minimizers_direction[output_index]        = minimizers_direction[minimizers_position_in_read[window_index] - first_element_in_step];
                window_minimizers_position_in_read[output_index] = minimizers_position_in_read[window_index];
            }
        }
        __syncthreads();

        // increase the number of written minimizers by the number of central minimizers
        // the value is increased by the write index of the last window in read
        if (first_element_in_step + windows_per_loop_step >= windows_in_read && 0 == threadIdx.x)
        {                                                                                                                                // only do it when there is not going to be new loop step
            read_id_to_minimizers_written[blockIdx.x] = different_minimizer_than_neighbors[windows_in_read - first_element_in_step - 1]; // write the index of the last window
        }

        // if there are more loop steps to follow write the output array index of the last minimizer in this loop step
        if (first_element_in_step + windows_per_loop_step < windows_in_read && 0 == threadIdx.x)
        {
            *local_index_to_write_next_minimizer_to = different_minimizer_than_neighbors[windows_per_loop_step - 1]; // index of last written minimizer + 1
        }
    }
}

/// \brief finds back end minimizers
///
/// Finds the minimizers of windows ending end the last basepair and having window size range from 1 to window_size-1
///
/// \param minimizer_size kmer length
/// \param window_size number of kmers in one central minimizer window, kmers being shifted by one basepair each (for back end minimizers window size actually varies from 1 to window_size-1)
/// \param basepairs array of basepairs, first come basepairs for read 0, then read 1 and so on
/// \param read_id_to_basepairs_section index of the first basepair of every read (in basepairs array) and the number of basepairs in that read
/// \param window_minimizers_representation output array of representations of minimizers, grouped by reads
/// \param window_minimizers_direction output array of directions of minimizers, grouped by reads (0 - forward, 1 - reverse)
/// \param window_minimizers_position_in_read output array of positions in read of minimizers, grouped by reads
/// \param read_id_to_window_section index of first element dedicated to that read in output arrays and the number of dedicated elements (enough elements are allocated to each read to support cases where each window has a different minimizer, no need to check that condition)
/// \param read_id_to_minimizers_written how many minimizers have been written for this read already (initially number of front end and central minimizers)
__global__ void find_back_end_minimizers(const std::uint64_t minimizer_size,
                                         const std::uint64_t window_size,
                                         const char* const basepairs,
                                         const ArrayBlock* const read_id_to_basepairs_section,
                                         representation_t* const window_minimizers_representation,
                                         char* const window_minimizers_direction,
                                         position_in_read_t* const window_minimizers_position_in_read,
                                         const ArrayBlock* const read_id_to_windows_section,
                                         std::int64_t* const read_id_to_minimizers_written,
                                         const bool hash_representations)
{
    // See find_front_end_minimizers for more details about the algorithm

    if (1 == window_size)
    {
        // if 1 == window_size there are no end minimizer
        return;
    }

    // Index of first basepair which belongs to the largest back end minimizers. Index of that basepair within the read
    const auto index_of_first_element_to_process_local = read_id_to_basepairs_section[blockIdx.x].block_size_ - (window_size - 1 + minimizer_size - 1);
    // Index of first basepair which belongs to the largest back end minimizers. Index of that basepair within the whole array of basepairs for all reads
    const auto index_of_first_element_to_process_global = read_id_to_basepairs_section[blockIdx.x].first_element_ + index_of_first_element_to_process_local;
    // Index of the element to which the first back end minimizer of this read should be written. Index refers to the positions withing the section dedicate to this read
    const auto output_index_to_write_the_first_minimizer_local = read_id_to_minimizers_written[blockIdx.x];
    // Index of the element to which the first back end minimizer of this read should be written. Index refers to the positions withing the whole array dedicated to all reads
    const auto output_index_to_write_the_first_minimizer_global = read_id_to_windows_section[blockIdx.x].first_element_ + output_index_to_write_the_first_minimizer_local;

    // Dynamically allocating shared memory and assigning parts of it to different pointers
    // Everything is 8-byte alligned
    extern __shared__ std::uint64_t sm[];
    // TODO: not all arrays are needed at the same time -> reduce shared memory requirements by reusing parts of the memory
    // TODO: use sizeof to get the number of bytes
    std::uint32_t shared_memory_64_bit_elements_already_taken = 0;
    char* forward_basepair_hashes                             = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // basepairs_to_process elements
    shared_memory_64_bit_elements_already_taken += (window_size - 1 + minimizer_size - 1 + 7) / 8;

    char* reverse_basepair_hashes = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // basepairs_to_process elements
    shared_memory_64_bit_elements_already_taken += (window_size - 1 + minimizer_size - 1 + 7) / 8;

    representation_t* minimizers_representation = reinterpret_cast<representation_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // kmers_to_process elements
    shared_memory_64_bit_elements_already_taken += window_size - 1;

    char* minimizers_direction = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // kmers_to_process elements; 0 - forward, 1 - reverse
    shared_memory_64_bit_elements_already_taken += (window_size - 1 + 7) / 8;

    position_in_read_t* minimizers_position_in_read = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // windows_to_process elements
    shared_memory_64_bit_elements_already_taken += (window_size - 1 + 1) / 2;

    position_in_read_t* different_minimizer_than_neighbors = reinterpret_cast<position_in_read_t*>(&sm[shared_memory_64_bit_elements_already_taken]); // windows_to_process elements; 0 - same, 1 - different
    shared_memory_64_bit_elements_already_taken += (window_size - 1 + 1) / 2;

    // TODO: Move to constant memory
    char* forward_to_reverse_complement = reinterpret_cast<char*>(&sm[shared_memory_64_bit_elements_already_taken]); // 8 elements

    if (0 == threadIdx.x)
    {
        forward_to_reverse_complement[0b000] = 0b0000;
        forward_to_reverse_complement[0b001] = 0b0100; // A -> T (0b1 -> 0b10100)
        forward_to_reverse_complement[0b010] = 0b0000;
        forward_to_reverse_complement[0b011] = 0b0111; // C -> G (0b11 -> 0b111)
        forward_to_reverse_complement[0b100] = 0b0001; // T -> A (0b10100 -> 0b1)
        forward_to_reverse_complement[0b101] = 0b0000;
        forward_to_reverse_complement[0b110] = 0b0000;
        forward_to_reverse_complement[0b111] = 0b0011; // G -> C (0b111 -> 0b11)
    }
    __syncthreads();

    // There are only window_size-1 back end windows. window_size usually has the value of a few dozens
    // Having windows_size so large that it does not fit the shared memory is unlikely
    // If that happens implement this method similarly to find_central_minimizers

    // load basepairs into shared memory and calculate the lexical ordering hash
    for (std::uint16_t basepair_index = threadIdx.x; basepair_index < window_size - 1 + minimizer_size - 1; basepair_index += blockDim.x)
    {
        const char bp                           = basepairs[index_of_first_element_to_process_global + basepair_index];
        forward_basepair_hashes[basepair_index] = 0b11 & (bp >> 2 ^ bp >> 1);
        reverse_basepair_hashes[basepair_index] = 0b11 & (forward_to_reverse_complement[0b111 & bp] >> 2 ^ forward_to_reverse_complement[0b111 & bp] >> 1);
    }
    __syncthreads();

    // calculate kmer minimizers
    // For back end minimizers the number of kmers is the same as the number of windows
    for (std::uint16_t kmer_index = threadIdx.x; kmer_index < window_size - 1; kmer_index += blockDim.x)
    {
        representation_t forward_representation = 0;
        representation_t reverse_representation = 0;
        // TODO: It's not necessary to fully build both representations in order to determine which one is smaller. In most cases there is going to be a difference already at the first element
        for (std::uint16_t i = 0; i < minimizer_size; ++i)
        {
            forward_representation |= forward_basepair_hashes[kmer_index + i] << 2 * (minimizer_size - i - 1);
            reverse_representation |= reverse_basepair_hashes[kmer_index + i] << 2 * i;
        }

        if (hash_representations)
        {
            forward_representation = wang_hash64(forward_representation);
            reverse_representation = wang_hash64(reverse_representation);
        }
        if (forward_representation <= reverse_representation)
        {
            minimizers_representation[kmer_index] = forward_representation;
            minimizers_direction[kmer_index]      = 0;
        }
        else
        {
            minimizers_representation[kmer_index] = reverse_representation;
            minimizers_direction[kmer_index]      = 1;
        }
    }
    __syncthreads();

    // find window minimizer
    for (std::uint16_t window_index = threadIdx.x; window_index < window_size - 1; window_index += blockDim.x)
    {
        // assume that the first kmer in the window is the minimizer
        representation_t window_minimizer_representation     = minimizers_representation[window_index];
        position_in_read_t window_minimizer_position_in_read = index_of_first_element_to_process_local + window_index;
        // now check other kmers in the window (note that this the back end minimizer, so not all windows have the same length)
        for (std::uint16_t i = 1; window_index + i < window_size - 1; ++i)
        {
            if (minimizers_representation[window_index + i] <= window_minimizer_representation)
            {
                window_minimizer_representation   = minimizers_representation[window_index + i];
                window_minimizer_position_in_read = index_of_first_element_to_process_local + window_index + i;
            }
        }
        minimizers_position_in_read[window_index] = window_minimizer_position_in_read;
    }
    __syncthreads();

    // check if the window to the left has a the same minimizer
    for (std::uint16_t window_index = threadIdx.x; window_index < window_size - 1; window_index += blockDim.x)
    {
        representation_t neighbors_minimizers_position_in_read = 0;
        // find left neighbor's window minimizer's position in read
        if (0 == window_index)
        {
            // if this is the first window take the position of the minimizer of the last central minimizer
            neighbors_minimizers_position_in_read = window_minimizers_position_in_read[output_index_to_write_the_first_minimizer_global - 1];
        }
        else
        {
            // TODO: consider using warp shuffle instead of shared memory
            neighbors_minimizers_position_in_read = minimizers_position_in_read[window_index - 1];
        }
        // check if it's the same minimizer
        if (neighbors_minimizers_position_in_read == minimizers_position_in_read[window_index])
        {
            different_minimizer_than_neighbors[window_index] = 0;
        }
        else
        {
            different_minimizer_than_neighbors[window_index] = 1;
        }
    }
    __syncthreads();

    // perform inclusive scan
    // different_minimizer_than_neighbors changes meaning an becomes more like "output_array_index_to_write_the_value_plus_one"
    // TODO: implement it using warp shuffle or use CUB
    if (0 == threadIdx.x)
    {
        // read_id_to_minimizers_written[blockIdx.x] is the index of the last written plus one
        different_minimizer_than_neighbors[0] += output_index_to_write_the_first_minimizer_local;
        for (std::uint16_t i = 1; i < window_size - 1; ++i)
        {
            different_minimizer_than_neighbors[i] += different_minimizer_than_neighbors[i - 1];
        }
    }
    __syncthreads();

    // now save minimizers to output array
    for (std::uint16_t window_index = threadIdx.x; window_index < window_size - 1; window_index += blockDim.x)
    {
        // different_minimizer_than_neighbors contians an inclusive scan, i.e. it's index_to_write_to + 1
        const std::uint32_t neighbors_write_index = 0 == window_index ? output_index_to_write_the_first_minimizer_local : different_minimizer_than_neighbors[window_index - 1];
        if (neighbors_write_index < different_minimizer_than_neighbors[window_index])
        {
            // to get the actual index to write to do -1 to different_minimizer_than_neighbors
            const auto output_index = read_id_to_windows_section[blockIdx.x].first_element_ + different_minimizer_than_neighbors[window_index] - 1;
            // substract index_of_first_element_to_process_local to get the index in shared memory
            window_minimizers_representation[output_index]   = minimizers_representation[minimizers_position_in_read[window_index] - index_of_first_element_to_process_local];
            window_minimizers_direction[output_index]        = minimizers_direction[minimizers_position_in_read[window_index] - index_of_first_element_to_process_local];
            window_minimizers_position_in_read[output_index] = minimizers_position_in_read[window_index];
        }
    }
    __syncthreads();

    // save the write index of the last written minimizer
    if (0 == threadIdx.x)
    {
        read_id_to_minimizers_written[blockIdx.x] = different_minimizer_than_neighbors[window_size - 1 - 1];
    }
}

/// \brief packs minimizers of different reads together
///
/// window_minimizers_representation, window_minimizers_position_in_read and window_minimizers_direction all allocate one element for each window in the read.
/// Many windows share the same minimizer and each minimizer is written only once, meaning many elements do not contain minimizers.
/// This function creates new arrays where such elements do not exist.
/// Note that in the input arrays all minimizers of one read are written consecutively, i.e. [read_0 minimizers], [read_0 junk], [read_1 minimizers], [read_1 junk], [read_2 minimizers]...
///
/// \param window_minimizers_representation array of representations of minimizers, grouped by reads
/// \param window_minimizers_position_in_read array of positions in read of minimizers, grouped by reads
/// \param window_minimizers_direction array of directions of minimizers, grouped by reads (0 - forward, 1 - reverse)
/// \param read_id_to_windows_section index of first element dedicated to that read in input arrays and the number of dedicated elements
/// \param representations_compressed array of representations of minimizers, grouped by reads, without invalid elements between the reads
/// \param rest_compressed array of read_ids, positions_in_read and directions of reads, grouped by reads, without invalid elements between the reads
/// \param read_id_to_compressed_minimizers index of the first minimizer of the next read (array comes from an inclusive scan, hence all indices are shifted by one)
/// \param read_id_of_first_read
__global__ void compress_minimizers(const representation_t* const window_minimizers_representation,
                                    const position_in_read_t* const window_minimizers_position_in_read,
                                    const char* const window_minimizers_direction,
                                    const ArrayBlock* const read_id_to_windows_section,
                                    representation_t* const representations_compressed,
                                    Minimizer::ReadidPositionDirection* const rest_compressed,
                                    const std::int64_t* const read_id_to_compressed_minimizers,
                                    std::uint32_t read_id_of_first_read)
{
    const auto first_input_minimizer = read_id_to_windows_section[blockIdx.x].first_element_;
    // elements have the index of read_id+1, i.e. everything is shifted by one
    const auto first_output_minimizer = blockIdx.x == 0 ? 0 : read_id_to_compressed_minimizers[blockIdx.x - 1];
    const auto number_of_minimizers   = read_id_to_compressed_minimizers[blockIdx.x] - first_output_minimizer;

    for (std::uint32_t i = threadIdx.x; i < number_of_minimizers; i += blockDim.x)
    {
        representations_compressed[first_output_minimizer + i]        = window_minimizers_representation[first_input_minimizer + i];
        rest_compressed[first_output_minimizer + i].read_id_          = blockIdx.x + read_id_of_first_read;
        rest_compressed[first_output_minimizer + i].position_in_read_ = window_minimizers_position_in_read[first_input_minimizer + i];
        rest_compressed[first_output_minimizer + i].direction_        = window_minimizers_direction[first_input_minimizer + i];
    }
}

Minimizer::GeneratedSketchElements Minimizer::generate_sketch_elements(DefaultDeviceAllocator allocator,
                                                                       const std::uint64_t number_of_reads_to_add,
                                                                       const std::uint64_t minimizer_size,
                                                                       const std::uint64_t window_size,
                                                                       const std::uint64_t read_id_of_first_read,
                                                                       const device_buffer<char>& merged_basepairs_d,
                                                                       const std::vector<ArrayBlock>& read_id_to_basepairs_section_h,
                                                                       const device_buffer<ArrayBlock>& read_id_to_basepairs_section_d,
                                                                       const bool hash_representations,
                                                                       const hipStream_t hip_stream)
{
    GW_NVTX_RANGE(profiler, "Minimizer::generate_sketch_elements");

    // for each read find the maximum number of minimizers (one per window), determine their section in the minimizer arrays and allocate the arrays
    std::uint64_t total_windows = 0;
    std::vector<ArrayBlock> read_id_to_windows_section_h(number_of_reads_to_add, {0, 0});
    for (read_id_t read_id = 0; read_id < number_of_reads_to_add; ++read_id)
    {
        read_id_to_windows_section_h[read_id].first_element_ = total_windows;
        std::uint32_t windows                                = window_size - 1;                                  // front end minimizers
        windows += read_id_to_basepairs_section_h[read_id].block_size_ - (minimizer_size + window_size - 1) + 1; // central minimizers
        windows += window_size - 1;
        read_id_to_windows_section_h[read_id].block_size_ = windows;
        total_windows += windows;
    }

    device_buffer<decltype(read_id_to_windows_section_h)::value_type> read_id_to_windows_section_d(read_id_to_windows_section_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(read_id_to_windows_section_h.data(),
                             read_id_to_windows_section_h.size(),
                             read_id_to_windows_section_d.data(),
                             hip_stream); // H2D

    device_buffer<representation_t> window_minimizers_representation_d(total_windows, allocator, hip_stream);
    device_buffer<char> window_minimizers_direction_d(total_windows, allocator, hip_stream);
    device_buffer<position_in_read_t> window_minimizers_position_in_read_d(total_windows, allocator, hip_stream);
    device_buffer<std::int64_t> read_id_to_minimizers_written_d(number_of_reads_to_add, allocator, hip_stream);
    // initially there are no minimizers written to the output arrays
    // TODO: is this needed?
    GW_CU_CHECK_ERR(hipMemsetAsync(read_id_to_minimizers_written_d.data(), 0, number_of_reads_to_add * sizeof(std::int64_t), hip_stream));

    // *** front end minimizers ***
    std::uint32_t num_of_basepairs_for_front_minimizers = (window_size - 1) + minimizer_size - 1;
    std::uint32_t num_of_threads                        = std::min(num_of_basepairs_for_front_minimizers, 64u);
    // largest window in end minimizers has the size of window_size-1, meaning it covers window_size-1 + minimizer_size - 1 basepairs
    const std::uint32_t basepairs_for_end_minimizers = (window_size - 1 + minimizer_size - 1);
    const std::uint32_t kmers_for_end_minimizers     = window_size - 1; // for end minimizers number of kmers is the as the number of windows because the last window has only one kmer
    const std::uint32_t windows_for_end_minimizers   = window_size - 1;
    // determine total ammount for shared memory needed (see kernel for clarification)
    // shared memeory is alligned to 8 bytes, so for 1-byte variables (x+7)/8 values are allocate (for 10 1-byte elements (10+7)/8=17/8=2 8-byte elements are allocated, instead of 10/1=1 which would be wrong)
    // the final number of allocated 8-byte values is multiplied by 8 at the end in order to get number of bytes needed
    std::uint32_t shared_memory_for_kernel = 0;
    shared_memory_for_kernel += (basepairs_for_end_minimizers + 7) / 8; // forward basepairs (char)
    shared_memory_for_kernel += (basepairs_for_end_minimizers + 7) / 8; // reverse basepairs (char)
    shared_memory_for_kernel += (kmers_for_end_minimizers);             // representations of minimizers (representation_t)
    shared_memory_for_kernel += (windows_for_end_minimizers + 7) / 8;   // directions of representations of minimizers (char)
    shared_memory_for_kernel += (windows_for_end_minimizers + 1) / 2;   // position_in_read of minimizers (position_in_read_t)
    shared_memory_for_kernel += (windows_for_end_minimizers + 1) / 2;   // does the window have a different minimizer than its left neighbor (position_in_read_t)
    shared_memory_for_kernel += 1;                                      // representation from previous step
    shared_memory_for_kernel += (1 + 1) / 2;                            // position from previous step (char)
    shared_memory_for_kernel += (1 + 1) / 2;                            // inclusive sum from previous step (position_in_read_t)
    shared_memory_for_kernel += 8 / 8;                                  // forward -> reverse complement conversion (char)

    shared_memory_for_kernel *= 8; // before it the number of 8-byte values, now get the number of bytes

    GW_LOG_INFO("Launching find_front_end_minimizers with {} bytes of shared memory", shared_memory_for_kernel);
    find_front_end_minimizers<<<number_of_reads_to_add, num_of_threads, shared_memory_for_kernel, hip_stream>>>(minimizer_size,
                                                                                                                 window_size,
                                                                                                                 merged_basepairs_d.data(),
                                                                                                                 read_id_to_basepairs_section_d.data(),
                                                                                                                 window_minimizers_representation_d.data(),
                                                                                                                 window_minimizers_direction_d.data(),
                                                                                                                 window_minimizers_position_in_read_d.data(),
                                                                                                                 read_id_to_windows_section_d.data(),
                                                                                                                 read_id_to_minimizers_written_d.data(),
                                                                                                                 hash_representations);

    // *** central minimizers ***
    const std::uint32_t basepairs_per_thread    = 8;  // arbitrary, tradeoff between the number of thread blocks that can be scheduled simultaneously and the number of basepairs which have to be loaded multiple times beacuse only basepairs_per_thread*num_of_threads-(window_size_ + minimizer_size_ - 1) + 1 can be processed at once, i.e. window_size+minimizer_size-2 basepairs have to be loaded again
    num_of_threads                              = 64; // arbitrary
    const std::uint32_t basepairs_in_loop_step  = num_of_threads * basepairs_per_thread;
    const std::uint32_t minimizers_in_loop_step = basepairs_in_loop_step - minimizer_size + 1;
    const std::uint32_t windows_in_loop_step    = minimizers_in_loop_step - window_size + 1;

    shared_memory_for_kernel = 0;
    shared_memory_for_kernel += (basepairs_in_loop_step + 7) / 8; // forward basepairs (char)
    shared_memory_for_kernel += (basepairs_in_loop_step + 7) / 8; // reverse basepairs (char)
    shared_memory_for_kernel += minimizers_in_loop_step;          // representations of minimizers (representation_t)
    shared_memory_for_kernel += (windows_in_loop_step + 7) / 8;   // directions of representations of minimizers (char)
    shared_memory_for_kernel += (windows_in_loop_step + 1) / 2;   // position_in_read of minimizers (position_in_read_t)
    shared_memory_for_kernel += (windows_in_loop_step + 1) / 2;   // does the window have a different minimizer than its left neighbor
    shared_memory_for_kernel += (1 + 1) / 2;                      // position from previous step (char)
    shared_memory_for_kernel += (1 + 1) / 2;                      // inclusive sum from previous step (position_in_read_t)
    shared_memory_for_kernel += 8 / 8;                            // forward -> reverse complement conversion (char)

    shared_memory_for_kernel *= 8; // before it the number of 8-byte values, now get the number of bytes

    GW_LOG_INFO("Launching find_central_minimizers with {} bytes of shared memory", shared_memory_for_kernel);
    find_central_minimizers<<<number_of_reads_to_add, num_of_threads, shared_memory_for_kernel, hip_stream>>>(minimizer_size,
                                                                                                               window_size,
                                                                                                               basepairs_per_thread,
                                                                                                               merged_basepairs_d.data(),
                                                                                                               read_id_to_basepairs_section_d.data(),
                                                                                                               window_minimizers_representation_d.data(),
                                                                                                               window_minimizers_direction_d.data(),
                                                                                                               window_minimizers_position_in_read_d.data(),
                                                                                                               read_id_to_windows_section_d.data(),
                                                                                                               read_id_to_minimizers_written_d.data(),
                                                                                                               hash_representations);

    // *** back end minimizers ***
    num_of_threads = 64;
    // largest window should fit shared memory
    shared_memory_for_kernel = 0;
    shared_memory_for_kernel += (basepairs_for_end_minimizers + 7) / 8; // forward basepairs (char)
    shared_memory_for_kernel += (basepairs_for_end_minimizers + 7) / 8; // reverse basepairs (char)
    shared_memory_for_kernel += kmers_for_end_minimizers;               // representations of minimizers (representation_t)
    shared_memory_for_kernel += (kmers_for_end_minimizers + 7) / 8;     // directions of representations of minimizers (char)
    shared_memory_for_kernel += (windows_for_end_minimizers + 1) / 2;   // position_in_read of minimizers (position_in_read_t)
    shared_memory_for_kernel += (windows_for_end_minimizers + 1) / 2;   // does the window have a different minimizer than its left neighbor
    shared_memory_for_kernel += 8 / 8;                                  // forward -> reverse complement conversion (char)

    shared_memory_for_kernel *= 8; // before it the number of 8-byte values, now get the number of bytes

    GW_LOG_INFO("Launching find_back_end_minimizers with {} bytes of shared memory", shared_memory_for_kernel);
    find_back_end_minimizers<<<number_of_reads_to_add, num_of_threads, shared_memory_for_kernel, hip_stream>>>(minimizer_size,
                                                                                                                window_size,
                                                                                                                merged_basepairs_d.data(),
                                                                                                                read_id_to_basepairs_section_d.data(),
                                                                                                                window_minimizers_representation_d.data(),
                                                                                                                window_minimizers_direction_d.data(),
                                                                                                                window_minimizers_position_in_read_d.data(),
                                                                                                                read_id_to_windows_section_d.data(),
                                                                                                                read_id_to_minimizers_written_d.data(),
                                                                                                                hash_representations);

    // *** remove unused elemets from the window minimizers arrays ***
    // In window_minimizers_representation_d and other arrays enough space was allocated to support cases in which each window has a different minimizer. In reality many neighboring windows share the same minimizer
    // As a result there are unused areas between minimizers belonging to different reads (space_allocated_for_all_possible_minimizers_of_a_read - space_needed_for_the_actual_minimizers)
    // At this point all mininizer are put together (compressed) so that the last minimizer of one read is next to the first minimizer of another read

    // after this operation read_id_to_minimizers_written_d should be interpreted as read_id_to_index_of_the_first_minimizer_of_the_next_read_d
    thrust::inclusive_scan(thrust::cuda::par(allocator).on(hip_stream),
                           read_id_to_minimizers_written_d.data(),
                           read_id_to_minimizers_written_d.data() + read_id_to_minimizers_written_d.size(),
                           read_id_to_minimizers_written_d.data());

    // last element of contains the index of first minimizer of theoretical past-the-last read, which is equal to the overall number of minimizers
    std::int64_t total_minimizers = cudautils::get_value_from_device(&(read_id_to_minimizers_written_d.data()[read_id_to_minimizers_written_d.size() - 1]), hip_stream);

    // Data is organized in two arrays in order to support usage of thrust::stable_sort_by_key. One contains representations (key) and the other the rest (values)
    device_buffer<representation_t> representations_compressed_d(total_minimizers, allocator, hip_stream);
    // rest = position_in_read, direction and read_id
    device_buffer<ReadidPositionDirection> rest_compressed_d(total_minimizers, allocator, hip_stream);

    GW_LOG_INFO("Launching compress_minimizers with {} bytes of shared memory", 0);
    compress_minimizers<<<number_of_reads_to_add, 128, 0, hip_stream>>>(window_minimizers_representation_d.data(),
                                                                         window_minimizers_position_in_read_d.data(),
                                                                         window_minimizers_direction_d.data(),
                                                                         read_id_to_windows_section_d.data(),
                                                                         representations_compressed_d.data(),
                                                                         rest_compressed_d.data(),
                                                                         read_id_to_minimizers_written_d.data(),
                                                                         read_id_of_first_read);

    // free these arrays as they are not needed anymore
    window_minimizers_representation_d.free();
    window_minimizers_direction_d.free();
    window_minimizers_position_in_read_d.free();
    read_id_to_minimizers_written_d.free();
    read_id_to_windows_section_d.free();

    return {std::move(representations_compressed_d),
            std::move(rest_compressed_d)};
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
