#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <atomic>
#include <algorithm>
#include <iostream>
#include <future>
#include <mutex>
#include <string>
#include <thread>

#include <claraparabricks/genomeworks/utils/cudautils.hpp>
#include <claraparabricks/genomeworks/utils/mathutils.hpp>
#include <claraparabricks/genomeworks/utils/signed_integer_utils.hpp>
#include <claraparabricks/genomeworks/utils/threadsafe_containers.hpp>

#include <claraparabricks/genomeworks/cudaaligner/aligner.hpp>
#include <claraparabricks/genomeworks/cudaaligner/alignment.hpp>

#include <claraparabricks/genomeworks/cudamapper/index.hpp>
#include <claraparabricks/genomeworks/cudamapper/matcher.hpp>
#include <claraparabricks/genomeworks/cudamapper/overlapper.hpp>
#include <claraparabricks/genomeworks/cudamapper/utils.hpp>

#include "application_parameters.hpp"
#include "cudamapper_utils.hpp"
#include "index_batcher.cuh"

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

namespace
{

void run_alignment_batch(DefaultDeviceAllocator allocator,
                         std::mutex& overlap_idx_mtx,
                         std::vector<Overlap>& overlaps,
                         const io::FastaParser& query_parser,
                         const io::FastaParser& target_parser,
                         int32_t& overlap_idx,
                         const int32_t max_query_size, const int32_t max_target_size,
                         std::vector<std::string>& cigars, const int32_t batch_size)
{
    int32_t device_id;
    GW_CU_CHECK_ERR(hipGetDevice(&device_id));
    CudaStream stream = make_cuda_stream();
    std::unique_ptr<cudaaligner::Aligner> batch =
        cudaaligner::create_aligner(
            max_query_size,
            max_target_size,
            batch_size,
            cudaaligner::AlignmentType::global_alignment,
            allocator,
            stream.get(),
            device_id);
    while (true)
    {
        int32_t idx_start = 0, idx_end = 0;
        // Get the range of overlaps for this batch
        {
            std::lock_guard<std::mutex> lck(overlap_idx_mtx);
            if (overlap_idx == get_size<int32_t>(overlaps))
            {
                break;
            }
            else
            {
                idx_start   = overlap_idx;
                idx_end     = std::min(idx_start + batch_size, get_size<int32_t>(overlaps));
                overlap_idx = idx_end;
            }
        }
        for (int32_t idx = idx_start; idx < idx_end; idx++)
        {
            const Overlap& overlap         = overlaps[idx];
            const io::FastaSequence query  = query_parser.get_sequence_by_id(overlap.query_read_id_);
            const io::FastaSequence target = target_parser.get_sequence_by_id(overlap.target_read_id_);
            const char* query_start        = &query.seq[overlap.query_start_position_in_read_];
            const int32_t query_length     = overlap.query_end_position_in_read_ - overlap.query_start_position_in_read_;
            const char* target_start       = &target.seq[overlap.target_start_position_in_read_];
            const int32_t target_length    = overlap.target_end_position_in_read_ - overlap.target_start_position_in_read_;
            cudaaligner::StatusType status = batch->add_alignment(query_start, query_length, target_start, target_length,
                                                                  false, overlap.relative_strand == RelativeStrand::Reverse);
            if (status != cudaaligner::success)
            {
                throw std::runtime_error("Experienced error type " + std::to_string(status));
            }
        }
        // Launch alignment on the GPU. align_all is an async call.
        batch->align_all();
        // Synchronize all alignments.
        batch->sync_alignments();
        const std::vector<std::shared_ptr<cudaaligner::Alignment>>& alignments = batch->get_alignments();
        {
            GW_NVTX_RANGE(profiler, "copy_alignments");
            for (int32_t i = 0; i < get_size<int32_t>(alignments); i++)
            {
                cigars[idx_start + i] = alignments[i]->convert_to_cigar();
            }
        }
        // Reset batch to reuse memory for new alignments.
        batch->reset();
    }
}

/// \brief performs global alignment between overlapped regions of reads
/// \param overlaps List of overlaps to align
/// \param query_parser Parser for query reads
/// \param target_parser Parser for target reads
/// \param num_alignment_engines Number of parallel alignment engines to use for alignment
/// \param cigars Output vector to store CIGAR strings for alignments
/// \param allocator The allocator to allocate memory on the device
void align_overlaps(DefaultDeviceAllocator allocator,
                    std::vector<Overlap>& overlaps,
                    const io::FastaParser& query_parser,
                    const io::FastaParser& target_parser,
                    int32_t num_alignment_engines,
                    std::vector<std::string>& cigars)
{
    // Calculate max target/query size in overlaps
    int32_t max_query_size  = 0;
    int32_t max_target_size = 0;
    for (const auto& overlap : overlaps)
    {
        int32_t query_overlap_size  = overlap.query_end_position_in_read_ - overlap.query_start_position_in_read_;
        int32_t target_overlap_size = overlap.target_end_position_in_read_ - overlap.target_start_position_in_read_;
        if (query_overlap_size > max_query_size)
            max_query_size = query_overlap_size;
        if (target_overlap_size > max_target_size)
            max_target_size = target_overlap_size;
    }

    // Heuristically calculate max alignments possible with available memory based on
    // empirical measurements of memory needed for alignment per base.
    const float memory_per_base = 0.03f; // Estimation of space per base in bytes for alignment
    float memory_per_alignment  = memory_per_base * max_query_size * max_target_size;
    size_t free, total;
    GW_CU_CHECK_ERR(hipMemGetInfo(&free, &total));
    const size_t max_alignments = (static_cast<float>(free) * 85 / 100) / memory_per_alignment; // Using 85% of available memory
    int32_t batch_size          = std::min(get_size<int32_t>(overlaps), static_cast<int32_t>(max_alignments)) / num_alignment_engines;
    std::cerr << "Aligning " << overlaps.size() << " overlaps (" << max_query_size << "x" << max_target_size << ") with batch size " << batch_size << std::endl;

    int32_t overlap_idx = 0;
    std::mutex overlap_idx_mtx;

    // Launch multiple alignment engines in separate threads to overlap D2H and H2D copies
    // with compute from concurrent engines.
    std::vector<std::future<void>> align_futures;
    for (int32_t t = 0; t < num_alignment_engines; t++)
    {
        align_futures.push_back(std::async(std::launch::async,
                                           &run_alignment_batch,
                                           allocator,
                                           std::ref(overlap_idx_mtx),
                                           std::ref(overlaps),
                                           std::ref(query_parser),
                                           std::ref(target_parser),
                                           std::ref(overlap_idx),
                                           max_query_size,
                                           max_target_size,
                                           std::ref(cigars),
                                           batch_size));
    }

    for (auto& f : align_futures)
    {
        f.get();
    }
}

/// OverlapsAndCigars - packs overlaps and cigars together so they can be passed to writer thread more easily
struct OverlapsAndCigars
{
    std::vector<Overlap> overlaps;
    std::vector<std::string> cigars;
};

/// \brief does overlapping and matching for pairs of query and target indices from device_batch
/// \param device_batch
/// \param device_cache data will be loaded into cache within the function
/// \param application_parameters
/// \param overlaps_and_cigars_to_process overlaps and cigars are output here and the then consumed by another thread
/// \param number_of_skipped_pairs_of_indices number of pairs of indices skipped due to OOM error, variable shared between all threads, each call increases the number by the number of skipped pairs
/// \param hip_stream
void process_one_device_batch(const IndexBatch& device_batch,
                              IndexCacheDevice& device_cache,
                              const ApplicationParameters& application_parameters,
                              DefaultDeviceAllocator device_allocator,
                              ThreadsafeProducerConsumer<OverlapsAndCigars>& overlaps_and_cigars_to_process,
                              std::atomic<int32_t>& number_of_skipped_pairs_of_indices,
                              hipStream_t hip_stream)
{
    GW_NVTX_RANGE(profiler, "main::process_one_device_batch");
    const std::vector<IndexDescriptor>& query_index_descriptors  = device_batch.query_indices;
    const std::vector<IndexDescriptor>& target_index_descriptors = device_batch.target_indices;

    // fetch indices for this batch from host memory
    assert(!query_index_descriptors.empty() && !target_index_descriptors.empty());
    device_cache.generate_query_cache_content(query_index_descriptors);
    device_cache.generate_target_cache_content(target_index_descriptors);

    // process pairs of query and target indices
    for (const IndexDescriptor& query_index_descriptor : query_index_descriptors)
    {
        for (const IndexDescriptor& target_index_descriptor : target_index_descriptors)
        {
            // if doing all-to-all skip pairs in which target batch has smaller id than query batch as it will be covered by symmetry
            if (!application_parameters.all_to_all || target_index_descriptor.first_read() >= query_index_descriptor.first_read())
            {
                std::shared_ptr<Index> query_index  = device_cache.get_index_from_query_cache(query_index_descriptor);
                std::shared_ptr<Index> target_index = device_cache.get_index_from_target_cache(target_index_descriptor);

                try
                {
                    // find anchors and overlaps
                    auto matcher = Matcher::create_matcher(device_allocator,
                                                           *query_index,
                                                           *target_index,
                                                           hip_stream);

                    std::vector<Overlap> overlaps;
                    auto overlapper = Overlapper::create_overlapper(device_allocator,
                                                                    hip_stream);

                    overlapper->get_overlaps(overlaps,
                                             matcher->anchors(),
                                             application_parameters.all_to_all,
                                             application_parameters.min_residues,
                                             application_parameters.min_overlap_len,
                                             application_parameters.min_bases_per_residue,
                                             application_parameters.min_overlap_fraction);

                    // free up memory taken by matcher
                    matcher.reset(nullptr);

                    // Align overlaps
                    std::vector<std::string> cigars;
                    if (application_parameters.alignment_engines > 0)
                    {
                        cigars.resize(overlaps.size());
                        GW_NVTX_RANGE(profiler, "align_overlaps");
                        align_overlaps(device_allocator,
                                       overlaps,
                                       *application_parameters.query_parser,
                                       *application_parameters.target_parser,
                                       application_parameters.alignment_engines,
                                       cigars);
                    }

                    // pass overlaps and cigars to writer thread
                    overlaps_and_cigars_to_process.add_new_element({std::move(overlaps), std::move(cigars)});
                }
                catch (device_memory_allocation_exception& oom_exception)
                {
                    // if the application ran out of memory skip this pair of indices
                    ++(number_of_skipped_pairs_of_indices);
                }
            }
        }
    }
}

/// \brief loads one batch into host memory and then processes its device batches one by one
/// \param batch
/// \param application_parameters
/// \param host_cache data will be loaded into cache within the function
/// \param device_cache data will be loaded into cache within the function
/// \param overlaps_and_cigars_to_process overlaps and cigars are output to this structure and the then consumed by another thread
/// \param number_of_skipped_pairs_of_indices number of pairs of indices skipped due to OOM error, variable shared between all threads, each call increases the number by the number of skipped pairs
/// \param hip_stream
void process_one_batch(const BatchOfIndices& batch,
                       const ApplicationParameters& application_parameters,
                       DefaultDeviceAllocator device_allocator,
                       IndexCacheHost& host_cache,
                       IndexCacheDevice& device_cache,
                       ThreadsafeProducerConsumer<OverlapsAndCigars>& overlaps_and_cigars_to_process,
                       std::atomic<int32_t>& number_of_skipped_pairs_of_indices,
                       hipStream_t hip_stream)
{
    GW_NVTX_RANGE(profiler, "main::process_one_batch");
    const IndexBatch& host_batch                  = batch.host_batch;
    const std::vector<IndexBatch>& device_batches = batch.device_batches;

    // if there is only one device batch and it is the same as host bach (which should be the case then) there is no need to copy indices to host
    // as they will be queried only once
    const bool skip_copy_to_host = 1 == device_batches.size();
    assert(!skip_copy_to_host || (host_batch.query_indices == device_batches.front().query_indices && host_batch.target_indices == device_batches.front().target_indices));

    // load indices into host memory
    {
        assert(!host_batch.query_indices.empty() && !host_batch.target_indices.empty() && !device_batches.empty());

        GW_NVTX_RANGE(profiler, "main::process_one_batch::host_indices");
        host_cache.generate_query_cache_content(host_batch.query_indices,
                                                device_batches.front().query_indices,
                                                skip_copy_to_host);
        host_cache.generate_target_cache_content(host_batch.target_indices,
                                                 device_batches.front().target_indices,
                                                 skip_copy_to_host);
    }

    // process device batches one by one
    for (const IndexBatch& device_batch : batch.device_batches)
    {
        process_one_device_batch(device_batch,
                                 device_cache,
                                 application_parameters,
                                 device_allocator,
                                 overlaps_and_cigars_to_process,
                                 number_of_skipped_pairs_of_indices,
                                 hip_stream);
    }
}

/// \brief does post-processing and writes data to output
/// \param device_id
/// \param application_parameters
/// \param overlaps_and_cigars_to_process new data is added to this structure as it gets available, also signals when there is not going to be any new data
/// \param output_mutex controls access to output to prevent race conditions
void postprocess_and_write_thread_function(const int32_t device_id,
                                           const ApplicationParameters& application_parameters,
                                           ThreadsafeProducerConsumer<OverlapsAndCigars>& overlaps_and_cigars_to_process,
                                           std::mutex& output_mutex)
{
    GW_NVTX_RANGE(profiler, ("main::postprocess_and_write_thread_for_device_" + std::to_string(device_id)).c_str());
    // This function is expected to run in a separate thread so set current device in order to avoid problems
    GW_CU_CHECK_ERR(hipSetDevice(device_id));

    // keep processing data as it arrives
    gw_optional_t<OverlapsAndCigars> data_to_write;
    while (data_to_write = overlaps_and_cigars_to_process.get_next_element()) // if optional is empty that means that there will be no more overlaps to process and the thread can finish
    {
        {
            GW_NVTX_RANGE(profiler, "main::postprocess_and_write_thread::one_set");
            std::vector<Overlap>& overlaps         = data_to_write->overlaps;
            const std::vector<std::string>& cigars = data_to_write->cigars;

            {
                GW_NVTX_RANGE(profiler, "main::postprocess_and_write_thread::postprocessing");
                // Overlap post processing - add overlaps which can be combined into longer ones.
                Overlapper::post_process_overlaps(data_to_write->overlaps, application_parameters.drop_fused_overlaps);
            }

            if (application_parameters.perform_overlap_end_rescue)
            {
                GW_NVTX_RANGE(profiler, "main::postprocess_and_write_thread::rescue_overlap_end");
                // Perform overlap-end rescue
                Overlapper::rescue_overlap_ends(data_to_write->overlaps,
                                                *application_parameters.query_parser,
                                                *application_parameters.target_parser,
                                                50,
                                                0.5);
            }

            if (application_parameters.all_to_all && application_parameters.drop_self_mappings)
            {
                GW_NVTX_RANGE(profiler, "main::postprocess_and_write_thread::remove_self_mappings");
                ::claraparabricks::genomeworks::cudamapper::details::overlapper::filter_self_mappings(overlaps,
                                                                                                      *application_parameters.query_parser,
                                                                                                      *application_parameters.target_parser,
                                                                                                      0.8);
            }

            // write to output
            {
                GW_NVTX_RANGE(profiler, "main::postprocess_and_write_thread::print_paf");
                print_paf(overlaps,
                          cigars,
                          *application_parameters.query_parser,
                          *application_parameters.target_parser,
                          application_parameters.kmer_size,
                          output_mutex);
            }
        }
    }
}

/// \brief controls one GPU
///
/// Each thread is resposible for one GPU. It takes one batch, processes it and passes it to postprocess_and_write_thread.
/// It keeps doing this as long as there are available batches. It also controls the postprocess_and_write_thread.
///
/// \param device_id
/// \param batches_of_indices
/// \param application_parameters
/// \param output_mutex
/// \param hip_stream
/// \param number_of_total_batches
/// \param number_of_skipped_pairs_of_indices
/// \param number_of_processed_batches
void worker_thread_function(const int32_t device_id,
                            ThreadsafeDataProvider<BatchOfIndices>& batches_of_indices,
                            const ApplicationParameters& application_parameters,
                            std::mutex& output_mutex,
                            hipStream_t hip_stream,
                            const int64_t number_of_total_batches,
                            std::atomic<int32_t>& number_of_skipped_pairs_of_indices,
                            std::atomic<int64_t>& number_of_processed_batches)
{
    GW_NVTX_RANGE(profiler, "main::worker_thread");

    // This function is expected to run in a separate thread so set current device in order to avoid problems
    GW_CU_CHECK_ERR(hipSetDevice(device_id));

    DefaultDeviceAllocator device_allocator = create_default_device_allocator(application_parameters.max_cached_memory_bytes);

    // create host_cache, data is not loaded at this point but later as each batch gets processed
    auto host_cache = std::make_shared<IndexCacheHost>(application_parameters.all_to_all,
                                                       device_allocator,
                                                       application_parameters.query_parser,
                                                       application_parameters.target_parser,
                                                       application_parameters.kmer_size,
                                                       application_parameters.windows_size,
                                                       true, // hash_representations
                                                       application_parameters.filtering_parameter,
                                                       hip_stream);

    // create host_cache, data is not loaded at this point but later as each batch gets processed
    IndexCacheDevice device_cache(application_parameters.all_to_all,
                                  host_cache);

    // data structure used to exchange data with postprocess_and_write_thread
    ThreadsafeProducerConsumer<OverlapsAndCigars> overlaps_and_cigars_to_process;

    // There should be at least one postprocess_and_write_thread per worker_thread. If more threads are available one thread should be reserved for
    // worker_thread and all other threads should be postprocess_and_write_threads
    const int32_t threads_per_device = ceiling_divide(static_cast<int32_t>(std::thread::hardware_concurrency()),
                                                      application_parameters.num_devices);

    const int32_t postprocess_and_write_threads_per_device = std::max(threads_per_device - 1, 1);

    // postprocess_and_write_threads run in the background and post-process and write overlaps and cigars to output as they become available in overlaps_and_cigars_to_process
    std::vector<std::thread> postprocess_and_write_threads;
    for (int32_t i = 0; i < postprocess_and_write_threads_per_device; ++i)
    {
        postprocess_and_write_threads.emplace_back(postprocess_and_write_thread_function,
                                                   device_id,
                                                   std::ref(application_parameters),
                                                   std::ref(overlaps_and_cigars_to_process),
                                                   std::ref(output_mutex));
    }

    // keep processing batches of indices until there are none left
    gw_optional_t<BatchOfIndices> batch_of_indices;
    while (batch_of_indices = batches_of_indices.get_next_element()) // if optional is empty that means that there are no more batches to process and the thread can finish
    {
        const int64_t batch_number         = number_of_processed_batches.fetch_add(1); // as this is not called atomically with get_next_element() the value does not have to be completely accurate, but this is ok as the value is only use for displaying progress
        const std::string progress_message = "Device " + std::to_string(device_id) + " took batch " + std::to_string(batch_number + 1) + " out of " + std::to_string(number_of_total_batches) + " batches in total\n";
        std::cerr << progress_message; // TODO: possible race condition, switch to logging library

        process_one_batch(batch_of_indices.value(),
                          application_parameters,
                          device_allocator,
                          *host_cache,
                          device_cache,
                          overlaps_and_cigars_to_process,
                          number_of_skipped_pairs_of_indices,
                          hip_stream);
    }

    // tell writer thread that there will be no more overlaps and it can finish once it has written all overlaps
    overlaps_and_cigars_to_process.signal_pushed_last_element();

    for (std::thread& postprocess_and_write_thread : postprocess_and_write_threads)
    {
        postprocess_and_write_thread.join();
    }

    // by this point all GPU work should anyway be done as postprocess_and_write_thread also finished and all GPU work had to be done before last values could be written
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
}

} // namespace

int main(int argc, char* argv[])
{
    logging::Init();

    const ApplicationParameters parameters(argc, argv);

    std::mutex output_mutex;

    // Program should process all combinations of query and target (if query and target are the same half of those can be skipped
    // due to symmetry). The matrix of query-target combinations is split into tiles called batches. Worker threads (one per GPU)
    // take batches one by one and process them.
    // Every batch is small enough for its indices to fit in host memory. Batches are further divided into sub-batches which are
    // small enough that all their indices fit in device memory.
    // After a worker thread has taken a batch it generates all necessary indices and saves them in host memory using IndexCacheHost.
    // It then processes sub-batches one by one but first loading indices into IndexCacheDevice from IndexCacheHost and then finding
    // the overlaps.
    // Output formatting and writing is done by a separate thread.

    // Split work into batches
    std::vector<BatchOfIndices> batches_of_indices_vect = generate_batches_of_indices(parameters.query_indices_in_host_memory,
                                                                                      parameters.query_indices_in_device_memory,
                                                                                      parameters.target_indices_in_host_memory,
                                                                                      parameters.target_indices_in_device_memory,
                                                                                      parameters.query_parser,
                                                                                      parameters.target_parser,
                                                                                      parameters.index_size * 1'000'000,        // value was in MB
                                                                                      parameters.target_index_size * 1'000'000, // value was in MB
                                                                                      parameters.all_to_all);
    const int64_t number_of_total_batches               = get_size<int64_t>(batches_of_indices_vect);
    std::atomic<int64_t> number_of_processed_batches(0);
    ThreadsafeDataProvider<BatchOfIndices> batches_of_indices(std::move(batches_of_indices_vect));

    // pairs of indices might be skipped if they cause out of memory errors
    std::atomic<int32_t> number_of_skipped_pairs_of_indices{0};

    // create worker threads (one thread per device)
    // these thread process batches_of_indices one by one
    std::vector<std::thread> worker_threads;

    // CudaStreams for each thread
    std::vector<CudaStream> cuda_streams;

    for (int32_t device_id = 0; device_id < parameters.num_devices; ++device_id)
    {
        GW_CU_CHECK_ERR(hipSetDevice(device_id));
        cuda_streams.emplace_back(make_cuda_stream());
        worker_threads.emplace_back(worker_thread_function,
                                    device_id,
                                    std::ref(batches_of_indices),
                                    std::ref(parameters),
                                    std::ref(output_mutex),
                                    cuda_streams.back().get(),
                                    number_of_total_batches,
                                    std::ref(number_of_skipped_pairs_of_indices),
                                    std::ref(number_of_processed_batches));
    }

    // wait for all work to be done
    for (auto& t : worker_threads)
    {
        // no need to sync, it should be done at the end of worker_threads
        t.join();
    }

    if (number_of_skipped_pairs_of_indices != 0)
    {
        std::cerr << "NOTE: Skipped " << number_of_skipped_pairs_of_indices << " pairs of indices due to device out of memory error" << std::endl;
    }

    return 0;
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks

/// \brief main function
/// main function cannot be in a namespace so using this function to call actual main function
int main(int argc, char* argv[])
{
    return claraparabricks::genomeworks::cudamapper::main(argc, argv);
}
