#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include <algorithm>
#include <getopt.h>
#include <iostream>
#include <string>
#include <deque>
#include <map>
#include <mutex>
#include <future>
#include <thread>
#include <atomic>

#include <claragenomics/logging/logging.hpp>
#include <claragenomics/io/fasta_parser.hpp>
#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>

#include <claragenomics/cudamapper/index.hpp>
#include <claragenomics/cudamapper/matcher.hpp>
#include <claragenomics/cudamapper/overlapper.hpp>
#include "overlapper_triggered.hpp"
#include "index_descriptor.hpp"

#include <claragenomics/cudaaligner/aligner.hpp>
#include <claragenomics/cudaaligner/alignment.hpp>

namespace claragenomics
{
namespace cudamapper
{

namespace
{

/// @brief prints help message
/// @param exit_code
void help(int32_t exit_code = 0)
{
    std::cerr <<
        R"(Usage: cudamapper [options ...] <query_sequences> <target_sequences>
     <sequences>
        Input file in FASTA/FASTQ format (can be compressed with gzip)
        containing sequences used for all-to-all overlapping
     options:
        -k, --kmer-size
            length of kmer to use for minimizers [15] (Max=)"
              << Index::maximum_kmer_size() << ")"
              << R"(
        -w, --window-size
            length of window to use for minimizers [15])"
              << R"(
        -d, --num-devices
            number of GPUs to use [1])"
              << R"(
        -c, --max-index-device-cache
            number of indices to keep in GPU memory [100])"
              << R"(
        -C, --max-index-host-cache
            number of indices to keep in host memory [0])"
              << R"(
        -m, --max-cached-memory
            maximum aggregate cached memory per device in GiB, if 0 program tries to allocate as much memory as possible [0])"
              << R"(
        -i, --index-size
            length of batch size used for query in MB [30])"
              << R"(
        -t, --target-index-size
            length of batch sized used for target in MB [30])"
              << R"(
        -F, --filtering-parameter
            filter all representations for which sketch_elements_with_that_representation/total_sketch_elements >= filtering_parameter), filtering disabled if filtering_parameter == 1.0 [1'000'000'001] (Min = 0.0, Max = 1.0))"
              << R"(
        -a, --alignment-engines
            Number of alignment engines to use (per device) for generating CIGAR strings for overlap alignments. Default value 0 = no alignment to be performed. Typically 2-4 engines per device gives best perf.)"
              << R"(
        -r, --min-residues
            Minimum number of matching residues in an overlap [10])"
              << R"(
        -l, --min-overlap-length
            Minimum length for an overlap [500].)"
              << R"(
        -b, --min-bases-per-residue
            Minimum number of bases in overlap per match [100].)"
              << R"(
        -z, --min-overlap-fraction
            Minimum ratio of overlap length to alignment length [0.95].)"
              << std::endl;

    exit(exit_code);
}

/// @brief application parameteres, default or passed through command line
struct ApplicationParameteres
{
    uint32_t k                                  = 15;   // k
    uint32_t w                                  = 15;   // w
    std::int32_t num_devices                    = 1;    // d
    std::int32_t max_index_cache_size_on_device = 100;  // c
    std::int32_t max_index_cache_size_on_host   = 0;    // C
    std::int32_t max_cached_memory              = 0;    // m
    std::int32_t index_size                     = 30;   // i
    std::int32_t target_index_size              = 30;   // t
    double filtering_parameter                  = 1.0;  // F
    std::int32_t alignment_engines              = 0;    // a
    std::int32_t min_residues                   = 10;   // r
    std::int32_t min_overlap_len                = 500;  // l
    std::int32_t min_bases_per_residue          = 100;  // b
    float min_overlap_fraction                  = 0.95; // z
    bool all_to_all                             = false;
    std::string query_filepath;
    std::string target_filepath;
};

/// @brief reads input from command line
/// @param argc
/// @param argv
/// @return application parameters passed through command line, default otherwise
ApplicationParameteres read_input(int argc, char* argv[])
{
    ApplicationParameteres parameters;

    struct option options[] = {
        {"kmer-size", required_argument, 0, 'k'},
        {"window-size", required_argument, 0, 'w'},
        {"num-devices", required_argument, 0, 'd'},
        {"max-index-device-cache", required_argument, 0, 'c'},
        {"max-index-host-cache", required_argument, 0, 'C'},
        {"max-cached-memory", required_argument, 0, 'm'},
        {"index-size", required_argument, 0, 'i'},
        {"target-index-size", required_argument, 0, 't'},
        {"filtering-parameter", required_argument, 0, 'F'},
        {"alignment-engines", required_argument, 0, 'a'},
        {"min-residues", required_argument, 0, 'r'},
        {"min-overlap-length", required_argument, 0, 'l'},
        {"min-bases-per-residue", required_argument, 0, 'b'},
        {"min-overlap-fraction", required_argument, 0, 'z'},
        {"help", no_argument, 0, 'h'},
    };

    std::string optstring = "k:w:d:c:C:m:i:t:F:h:a:r:l:b:z:";

    int32_t argument = 0;
    while ((argument = getopt_long(argc, argv, optstring.c_str(), options, nullptr)) != -1)
    {
        switch (argument)
        {
        case 'k':
            parameters.k = atoi(optarg);
            break;
        case 'w':
            parameters.w = atoi(optarg);
            break;
        case 'd':
            parameters.num_devices = atoi(optarg);
            break;
        case 'c':
            parameters.max_index_cache_size_on_device = atoi(optarg);
            break;
        case 'C':
            parameters.max_index_cache_size_on_host = atoi(optarg);
            break;
        case 'm':
#ifndef CGA_ENABLE_CACHING_ALLOCATOR
            std::cerr << "ERROR: Argument -m / --max-cached-memory cannot be used without caching allocator" << std::endl;
            exit(1);
#endif
            parameters.max_cached_memory = atoi(optarg);
            break;
        case 'i':
            parameters.index_size = atoi(optarg);
            break;
        case 't':
            parameters.target_index_size = atoi(optarg);
            break;
        case 'F':
            parameters.filtering_parameter = atof(optarg);
            break;
        case 'a':
            parameters.alignment_engines = atoi(optarg);
            throw_on_negative(parameters.alignment_engines, "Number of alignment engines should be non-negative");
            break;
        case 'r':
            parameters.min_residues = atoi(optarg);
            break;
        case 'l':
            parameters.min_overlap_len = atoi(optarg);
            break;
        case 'b':
            parameters.min_bases_per_residue = atoi(optarg);
            break;
        case 'z':
            parameters.min_overlap_fraction = atof(optarg);
            break;
        case 'h':
            help(0);
        default:
            exit(1);
        }
    }

    if (parameters.k > Index::maximum_kmer_size())
    {
        std::cerr << "kmer of size " << parameters.k << " is not allowed, maximum k = " << Index::maximum_kmer_size() << std::endl;
        exit(1);
    }

    if (parameters.filtering_parameter > 1.0 || parameters.filtering_parameter < 0.0)
    {
        std::cerr << "-F / --filtering-parameter must be in range [0.0, 1.0]" << std::endl;
        exit(1);
    }

    if (parameters.max_cached_memory < 0)
    {
        std::cerr << "-m / --max-cached-memory must not be negative" << std::endl;
        exit(1);
    }

    // Check remaining argument count.
    if ((argc - optind) < 2)
    {
        std::cerr << "Invalid inputs. Please refer to the help function." << std::endl;
        help(1);
    }

    parameters.query_filepath  = std::string(argv[optind++]);
    parameters.target_filepath = std::string(argv[optind++]);

    if (parameters.query_filepath == parameters.target_filepath)
    {
        parameters.all_to_all        = true;
        parameters.target_index_size = parameters.index_size;
        std::cerr << "NOTE - Since query and target files are same, activating all_to_all mode. Query index size used for both files." << std::endl;
    }

    return parameters;
}

void run_alignment_batch(DefaultDeviceAllocator allocator,
                         std::mutex& overlap_idx_mtx,
                         std::vector<Overlap>& overlaps,
                         const io::FastaParser& query_parser,
                         const io::FastaParser& target_parser,
                         int32_t& overlap_idx,
                         const int32_t max_query_size, const int32_t max_target_size,
                         std::vector<std::string>& cigar, const int32_t batch_size)
{
    int32_t device_id;
    CGA_CU_CHECK_ERR(hipGetDevice(&device_id));
    hipStream_t stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&stream));
    std::unique_ptr<cudaaligner::Aligner> batch =
        cudaaligner::create_aligner(
            max_query_size,
            max_target_size,
            batch_size,
            cudaaligner::AlignmentType::global_alignment,
            allocator,
            stream,
            device_id);
    while (true)
    {
        int32_t idx_start = 0, idx_end = 0;
        // Get the range of overlaps for this batch
        {
            std::lock_guard<std::mutex> lck(overlap_idx_mtx);
            if (overlap_idx == get_size<int32_t>(overlaps))
            {
                break;
            }
            else
            {
                idx_start   = overlap_idx;
                idx_end     = std::min(idx_start + batch_size, get_size<int32_t>(overlaps));
                overlap_idx = idx_end;
            }
        }
        for (int32_t idx = idx_start; idx < idx_end; idx++)
        {
            const Overlap& overlap         = overlaps[idx];
            const io::FastaSequence query  = query_parser.get_sequence_by_id(overlap.query_read_id_);
            const io::FastaSequence target = target_parser.get_sequence_by_id(overlap.target_read_id_);
            const char* query_start        = &query.seq[overlap.query_start_position_in_read_];
            const int32_t query_length     = overlap.query_end_position_in_read_ - overlap.query_start_position_in_read_;
            const char* target_start       = &target.seq[overlap.target_start_position_in_read_];
            const int32_t target_length    = overlap.target_end_position_in_read_ - overlap.target_start_position_in_read_;
            cudaaligner::StatusType status = batch->add_alignment(query_start, query_length, target_start, target_length,
                                                                  false, overlap.relative_strand == RelativeStrand::Reverse);
            if (status != cudaaligner::success)
            {
                throw std::runtime_error("Experienced error type " + std::to_string(status));
            }
        }
        // Launch alignment on the GPU. align_all is an async call.
        batch->align_all();
        // Synchronize all alignments.
        batch->sync_alignments();
        const std::vector<std::shared_ptr<cudaaligner::Alignment>>& alignments = batch->get_alignments();
        {
            CGA_NVTX_RANGE(profiler, "copy_alignments");
            for (int32_t i = 0; i < get_size<int32_t>(alignments); i++)
            {
                cigar[idx_start + i] = alignments[i]->convert_to_cigar();
            }
        }
        // Reset batch to reuse memory for new alignments.
        batch->reset();
    }
    CGA_CU_CHECK_ERR(hipStreamDestroy(stream));
}

/// \brief performs gloval alignment between overlapped regions of reads
/// \param overlaps List of overlaps to align
/// \param query_parser Parser for query reads
/// \param target_parser Parser for target reads
/// \param num_alignment_engines Number of parallel alignment engines to use for alignment
/// \param cigar Output vector to store CIGAR string for alignments
/// \param allocator The allocator to allocate memory on the device
void align_overlaps(DefaultDeviceAllocator allocator,
                    std::vector<Overlap>& overlaps,
                    const io::FastaParser& query_parser,
                    const io::FastaParser& target_parser,
                    int32_t num_alignment_engines,
                    std::vector<std::string>& cigar)
{
    // Calculate max target/query size in overlaps
    int32_t max_query_size  = 0;
    int32_t max_target_size = 0;
    for (const auto& overlap : overlaps)
    {
        int32_t query_overlap_size  = overlap.query_end_position_in_read_ - overlap.query_start_position_in_read_;
        int32_t target_overlap_size = overlap.target_end_position_in_read_ - overlap.target_start_position_in_read_;
        if (query_overlap_size > max_query_size)
            max_query_size = query_overlap_size;
        if (target_overlap_size > max_target_size)
            max_target_size = target_overlap_size;
    }

    // Heuristically calculate max alignments possible with available memory based on
    // empirical measurements of memory needed for alignment per base.
    const float memory_per_base = 0.03f; // Estimation of space per base in bytes for alignment
    float memory_per_alignment  = memory_per_base * max_query_size * max_target_size;
    size_t free, total;
    CGA_CU_CHECK_ERR(hipMemGetInfo(&free, &total));
    const size_t max_alignments = (static_cast<float>(free) * 85 / 100) / memory_per_alignment; // Using 85% of available memory
    int32_t batch_size          = std::min(get_size<int32_t>(overlaps), static_cast<int32_t>(max_alignments)) / num_alignment_engines;
    std::cerr << "Aligning " << overlaps.size() << " overlaps (" << max_query_size << "x" << max_target_size << ") with batch size " << batch_size << std::endl;

    int32_t overlap_idx = 0;
    std::mutex overlap_idx_mtx;

    // Launch multiple alignment engines in separate threads to overlap D2H and H2D copies
    // with compute from concurrent engines.
    std::vector<std::future<void>> align_futures;
    for (int32_t t = 0; t < num_alignment_engines; t++)
    {
        align_futures.push_back(std::async(std::launch::async,
                                           &run_alignment_batch,
                                           allocator,
                                           std::ref(overlap_idx_mtx),
                                           std::ref(overlaps),
                                           std::ref(query_parser),
                                           std::ref(target_parser),
                                           std::ref(overlap_idx),
                                           max_query_size,
                                           max_target_size,
                                           std::ref(cigar),
                                           batch_size));
    }

    for (auto& f : align_futures)
    {
        f.get();
    }
}

/// @brief adds read names to overlaps and writes them to output
/// This function is expected to be executed async to matcher + overlapper
/// @param overlaps_writer_mtx locked while writing the output
/// @param num_overlap_chunks_to_print increased before the function is called, decreased right before the function finishes // TODO: improve this design
/// @param filtered_overlaps overlaps to be written out, on input without read names, on output cleared
/// @param query_parser needed for read names and lenghts
/// @param target_parser needed for read names and lenghts
/// @param cigar
/// @param device_id id of device on which query and target indices were created
void writer_thread_function(std::mutex& overlaps_writer_mtx,
                            std::atomic<int>& num_overlap_chunks_to_print,
                            std::shared_ptr<std::vector<Overlap>> filtered_overlaps,
                            const io::FastaParser& query_parser,
                            const io::FastaParser& target_parser,
                            const std::vector<std::string> cigar,
                            const int device_id,
                            const int kmer_size)
{
    // This function is expected to run in a separate thread so set current device in order to avoid problems
    // with deallocating indices with different current device than the one on which they were created
    hipSetDevice(device_id);

    // Overlap post processing - add overlaps which can be combined into longer ones.
    Overlapper::post_process_overlaps(*filtered_overlaps);

    // parallel update of the query/target read names for filtered overlaps [parallel on host]
    Overlapper::update_read_names(*filtered_overlaps, query_parser, target_parser);
    std::lock_guard<std::mutex> lck(overlaps_writer_mtx);
    Overlapper::print_paf(*filtered_overlaps, cigar, kmer_size);

    //clear data
    for (auto o : *filtered_overlaps)
    {
        o.clear();
    }
    //Decrement counter which tracks number of overlap chunks to be filtered and printed
    num_overlap_chunks_to_print--;
};

} // namespace

int main(int argc, char* argv[])
{
    logging::Init();

    const ApplicationParameteres parameters = read_input(argc, argv);

    std::shared_ptr<io::FastaParser> query_parser;
    std::shared_ptr<io::FastaParser> target_parser;

    query_parser = io::create_kseq_fasta_parser(parameters.query_filepath, parameters.k + parameters.w - 1);

    if (parameters.all_to_all)
    {
        target_parser = query_parser;
    }
    else
    {
        target_parser = io::create_kseq_fasta_parser(parameters.target_filepath, parameters.k + parameters.w - 1);
    }

    std::cerr << "Query file: " << parameters.query_filepath << ", number of reads: " << query_parser->get_num_seqences() << std::endl;
    std::cerr << "Target file: " << parameters.target_filepath << ", number of reads: " << target_parser->get_num_seqences() << std::endl;

    // Data structure for holding overlaps to be written out
    std::mutex overlaps_writer_mtx;

    struct QueryTargetsRange
    {
        IndexDescriptor query_range;
        std::vector<IndexDescriptor> target_ranges;
    };

    ///Factor of 1000000 to make max cache size in MB
    std::vector<IndexDescriptor> query_index_descriptors  = group_reads_into_indices(*query_parser,
                                                                                    parameters.index_size * 1000000);
    std::vector<IndexDescriptor> target_index_descriptors = group_reads_into_indices(*target_parser,
                                                                                     parameters.target_index_size * 1000000);

    //First generate all the ranges independently, then loop over them.
    std::vector<QueryTargetsRange> query_target_ranges;

    int target_idx = 0;
    for (const IndexDescriptor& query_index_descriptor : query_index_descriptors)
    {
        QueryTargetsRange range{query_index_descriptor, {}};
        for (size_t t = target_idx; t < target_index_descriptors.size(); t++)
        {
            range.target_ranges.push_back(target_index_descriptors[t]);
        }
        query_target_ranges.push_back(range);
        // in all-to-all, for query chunk 0, we go through target chunks [target_idx = 0 , n = target_chunks.size())
        // for query chunk 1, we only need target chunks [target_idx = 1 , n), and in general for query_chunk i, we need target chunks [target_idx = i , n)
        // therefore as we're looping through query chunks, in all-to-all, will increment target_idx
        if (parameters.all_to_all)
        {
            target_idx++;
        }
    }

    // This is host cache, if it has the index it will copy it to device, if not it will generate on device and add it to host cache
    std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<IndexHostCopyBase>> host_index_cache;

    // This is a per-device cache, if it has the index it will return it, if not it will generate it, store and return it.
    std::vector<std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<Index>>> device_index_cache(parameters.num_devices);

    // The number of overlap chunks which are to be computed
    std::atomic<int> num_overlap_chunks_to_print(0);

    auto get_index = [&device_index_cache, &host_index_cache, &parameters](DefaultDeviceAllocator allocator,
                                                                           io::FastaParser& parser,
                                                                           const read_id_t start_index,
                                                                           const read_id_t end_index,
                                                                           const std::uint64_t k,
                                                                           const std::uint64_t w,
                                                                           const int device_id,
                                                                           const bool allow_cache_index,
                                                                           const double filtering_parameter,
                                                                           const hipStream_t hip_stream) {
        CGA_NVTX_RANGE(profiler, "get index");
        std::pair<uint64_t, uint64_t> key;
        key.first  = start_index;
        key.second = end_index;

        std::shared_ptr<Index> index;

        // first check if it's available on device, if not then check the host cache
        if (device_index_cache[device_id].count(key))
        {
            index = device_index_cache[device_id][key];
        }
        else if (host_index_cache.count(key))
        {
            index = host_index_cache[key]->copy_index_to_device(allocator,
                                                                hip_stream);
        }
        else
        {
            //create an index, with hashed representations (minimizers)
            index = Index::create_index(allocator,
                                        parser,
                                        start_index,
                                        end_index,
                                        k,
                                        w,
                                        true, // hash_representations
                                        filtering_parameter,
                                        hip_stream);

            // If in all-to-all mode, put this query in the cache for later use.
            // Cache eviction is handled later on by the calling thread
            // using the evict_index function.
            if (get_size<int32_t>(device_index_cache[device_id]) < parameters.max_index_cache_size_on_device && allow_cache_index)
            {
                device_index_cache[device_id][key] = index;
            }
            else if (get_size<int32_t>(host_index_cache) < parameters.max_index_cache_size_on_host && allow_cache_index && device_id == 0)
            {
                // if not cached on device, update host cache; only done on device 0 to avoid any race conditions in updating the host cache
                host_index_cache[key] = IndexHostCopyBase::create_cache(*index,
                                                                        start_index,
                                                                        k,
                                                                        w,
                                                                        hip_stream);
            }
        }
        return index;
    };

    // When performing all-to-all mapping, indices are instantitated as start-end-ranges in the reads.
    // As such, once a query index has been used it will not be needed again. For example, parsing ranges
    // [0-999], [1000-1999], [2000-2999], the caching/eviction would be as follows:
    //
    // Round 1
    // Query: [0-999] - Enter cache
    // Target: [1000-1999] - Enter cache
    // Target: [1999 - 2999] - Enter cache
    // Evict [0-999]
    // Round 2
    // Query: [1000-1999] - Use cache entry (from previous use when now query was a target)
    // Etc..
    auto evict_index = [&device_index_cache, &host_index_cache](const read_id_t query_start_index,
                                                                const read_id_t query_end_index,
                                                                const int device_id,
                                                                const int num_devices) {
        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;
        device_index_cache[device_id].erase(key);
        // host memory can be freed by removing (key) when working with 1 GPU
        // in multiple GPUs we keep (key), as it may be accessed by other GPUs depending on access pattern
        if (num_devices == 1)
            host_index_cache.erase(key);
    };

#ifdef CGA_ENABLE_CACHING_ALLOCATOR
    // uses CachingDeviceAllocator
    std::size_t max_cached_bytes = 0;
    if (parameters.max_cached_memory == 0)
    {
        std::cerr << "Programmatically looking for max cached memory" << std::endl;
        max_cached_bytes = cudautils::find_largest_contiguous_device_memory_section();
        if (max_cached_bytes == 0)
        {
            std::cerr << "No memory available for caching" << std::endl;
            exit(1);
        }
    }
    else
    {
        max_cached_bytes = parameters.max_cached_memory * 1024ull * 1024ull * 1024ull; // max_cached_memory is in GiB
    }

    std::cerr << "Using device memory cache of " << max_cached_bytes << " bytes" << std::endl;

    DefaultDeviceAllocator allocator(max_cached_bytes);
#else
    // uses CudaMallocAllocator
    DefaultDeviceAllocator allocator;
#endif

    auto compute_overlaps = [&](const QueryTargetsRange& query_target_range,
                                const int device_id,
                                const hipStream_t hip_stream) {
        auto query_start_index = query_target_range.query_range.first_read();
        auto query_end_index   = query_target_range.query_range.first_read() + query_target_range.query_range.number_of_reads();

        std::cerr << "Processing query range: (" << query_start_index << " - " << query_end_index - 1 << ")" << std::endl;

        std::shared_ptr<Index> query_index(nullptr);
        std::shared_ptr<Index> target_index(nullptr);
        std::unique_ptr<Matcher> matcher(nullptr);

        {
            CGA_NVTX_RANGE(profiler, "generate_query_index");
            query_index = get_index(allocator,
                                    *query_parser,
                                    query_start_index,
                                    query_end_index,
                                    parameters.k,
                                    parameters.w,
                                    device_id,
                                    parameters.all_to_all,
                                    parameters.filtering_parameter,
                                    hip_stream);
        }

        //Main loop
        for (const IndexDescriptor& target_range : query_target_range.target_ranges)
        {

            auto target_start_index = target_range.first_read();
            auto target_end_index   = target_range.first_read() + target_range.number_of_reads();
            {
                CGA_NVTX_RANGE(profiler, "generate_target_index");
                target_index = get_index(allocator,
                                         *target_parser,
                                         target_start_index,
                                         target_end_index,
                                         parameters.k,
                                         parameters.w,
                                         device_id,
                                         true,
                                         parameters.filtering_parameter,
                                         hip_stream);
            }
            {
                CGA_NVTX_RANGE(profiler, "generate_matcher");
                matcher = Matcher::create_matcher(allocator,
                                                  *query_index,
                                                  *target_index,
                                                  hip_stream);
            }
            {

                OverlapperTriggered overlapper(allocator, hip_stream);
                CGA_NVTX_RANGE(profiler, "generate_overlaps");

                // Get unfiltered overlaps
                auto overlaps_to_add = std::make_shared<std::vector<Overlap>>();

                overlapper.get_overlaps(*overlaps_to_add, matcher->anchors(),
                                        parameters.min_residues,
                                        parameters.min_overlap_len,
                                        parameters.min_bases_per_residue,
                                        parameters.min_overlap_fraction);

                std::vector<std::string> cigar;
                // Align overlaps
                if (parameters.alignment_engines > 0)
                {
                    cigar.resize(overlaps_to_add->size());
                    CGA_NVTX_RANGE(profiler, "align_overlaps");
                    align_overlaps(allocator, *overlaps_to_add, *query_parser, *target_parser, parameters.alignment_engines, cigar);
                }

                //Increment counter which tracks number of overlap chunks to be filtered and printed
                num_overlap_chunks_to_print++;

                std::thread t(writer_thread_function,
                              std::ref(overlaps_writer_mtx),
                              std::ref(num_overlap_chunks_to_print),
                              overlaps_to_add,
                              std::ref(*query_parser),
                              std::ref(*target_parser),
                              std::move(cigar),
                              device_id,
                              parameters.k);
                t.detach();
            }

            // reseting the matcher releases the anchor device array back to memory pool
            matcher.reset();
        }

        // If all-to-all mapping query will no longer be needed on device, remove it from the cache
        if (parameters.all_to_all)
        {
            evict_index(query_start_index, query_end_index, device_id, parameters.num_devices);
        }
    };

    // The application (File parsing, index generation, overlap generation etc) is all launched from here.
    // The main application works as follows:
    // 1. Launch a worker thread per device (GPU).
    // 2. Each worker takes target-query ranges off a queue
    // 3. Each worker pushes vector of futures (since overlap writing is dispatched to an async thread on host). All futures are waited for before the main application exits.
    std::vector<std::thread> workers;
    std::atomic<int> ranges_idx(0);

    // Each worker thread gets its own CUDA stream to work on. Currently there is only one worker thread per GPU,
    // but it is still necessary assign streams to each of then explicitly. --default-stream per-thread could
    // cause problems beacuse there are subthreads for worker threads
    std::vector<hipStream_t> cuda_streams(parameters.num_devices);

    // Launch worker threads to enable multi-GPU.
    // One worker thread is responsible for one GPU so the number
    // of worker threads launched is equal to the number of devices specified
    // by the user
    for (int device_id = 0; device_id < parameters.num_devices; ++device_id)
    {
        CGA_CU_CHECK_ERR(hipStreamCreate(&cuda_streams[device_id]));
        //Worker thread consumes query-target ranges off a queue
        workers.push_back(std::thread(
            [&, device_id]() {
                hipSetDevice(device_id);
                while (ranges_idx < get_size<int>(query_target_ranges))
                {
                    int range_idx = ranges_idx.fetch_add(1);
                    //Need to perform this check again for thread-safety
                    if (range_idx < get_size<int>(query_target_ranges))
                    {
                        //compute overlaps takes a range of read_ids and a device ID and uses
                        //that device to compute the overlaps. It prints overlaps to stdout.
                        //since multiple worker threads are running stdout is guarded
                        //by a mutex (`std::mutex overlaps_writer_mtx`)
                        compute_overlaps(query_target_ranges[range_idx],
                                         device_id,
                                         cuda_streams[device_id]);
                    }
                }
            }));
    }

    // Wait for all per-device threads to terminate
    for (auto& worker_thread : workers)
    {
        worker_thread.join();
    }

    // Wait for all futures (for overlap writing) to return
    while (num_overlap_chunks_to_print != 0)
    {
        std::this_thread::sleep_for(std::chrono::milliseconds(200));
    }

    // After last writer_thread_function has decreased num_overlap_chunks_to_print it will still take
    // some time to destroy its pointer to indices
    // TODO: this is a workaround, this part of code will be significantly changed with new index caching
    std::this_thread::sleep_for(std::chrono::milliseconds(200));

    device_index_cache.clear();

    // streams can only be destroyed once all writer threads have finished as they hold references
    // to indices which have device arrays associated with streams
    for (hipStream_t hip_stream : cuda_streams)
    {
        CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
        CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
    }

    return 0;
}

} // namespace cudamapper
} // namespace claragenomics

/// \brief main function
/// main function cannot be in a namespace so using this function to call actual main function
int main(int argc, char* argv[])
{
    return claragenomics::cudamapper::main(argc, argv);
}
