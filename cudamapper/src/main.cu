#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include <algorithm>
#include <getopt.h>
#include <iostream>
#include <string>
#include <deque>
#include <mutex>
#include <future>
#include <thread>
#include <atomic>

#include <claragenomics/logging/logging.hpp>
#include <claragenomics/io/fasta_parser.hpp>
#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>

#include <claragenomics/cudamapper/index.hpp>
#include <claragenomics/cudamapper/matcher.hpp>
#include <claragenomics/cudamapper/overlapper.hpp>
#include "overlapper_triggered.hpp"

static struct option options[] = {
    {"kmer-size", required_argument, 0, 'k'},
    {"window-size", required_argument, 0, 'w'},
    {"num-devices", required_argument, 0, 'd'},
    {"max-index-device-cache", required_argument, 0, 'c'},
    {"max-index-host-cache", required_argument, 0, 'C'},
    {"max-cached-memory", required_argument, 0, 'm'},
    {"index-size", required_argument, 0, 'i'},
    {"target-index-size", required_argument, 0, 't'},
    {"filtering-parameter", required_argument, 0, 'F'},
    {"benchmark-mode", required_argument, 0, 'b'},
    {"help", no_argument, 0, 'h'},
};

void help(int32_t exit_code);

int main(int argc, char* argv[])
{
    using claragenomics::get_size;
    claragenomics::logging::Init();

    uint32_t k                                  = 15;  // k
    uint32_t w                                  = 15;  // w
    std::int32_t num_devices                    = 1;   // d
    std::int32_t max_index_cache_size_on_device = 100; // c
    // ToDo: come up with a good heuristic to choose C and c
    std::int32_t max_index_cache_size_on_host = 0;   // C
    std::int32_t max_cached_memory            = 1;   // m
    std::int32_t index_size                   = 30;  // i
    std::int32_t target_index_size            = 30;  // t
    double filtering_parameter                = 1.0; // F
    std::int32_t benchmark_iterations         = 0;   // b
    std::string optstring                     = "k:w:d:c:C:m:i:t:F:b:h:";
    int32_t argument                          = 0;
    while ((argument = getopt_long(argc, argv, optstring.c_str(), options, nullptr)) != -1)
    {
        switch (argument)
        {
        case 'k':
            k = atoi(optarg);
            break;
        case 'w':
            w = atoi(optarg);
            break;
        case 'd':
            num_devices = atoi(optarg);
            break;
        case 'c':
            max_index_cache_size_on_device = atoi(optarg);
            break;
        case 'C':
            max_index_cache_size_on_host = atoi(optarg);
            break;
        case 'm':
            max_cached_memory = atoi(optarg);
            break;
        case 'i':
            index_size = atoi(optarg);
            break;
        case 't':
            target_index_size = atoi(optarg);
            break;
        case 'F':
            filtering_parameter = atof(optarg);
            break;
        case 'b':
            benchmark_iterations = atoi(optarg);
            break;
        case 'h':
            help(0);
        default:
            exit(1);
        }
    }

    if (k > claragenomics::cudamapper::Index::maximum_kmer_size())
    {
        std::cerr << "kmer of size " << k << " is not allowed, maximum k = " << claragenomics::cudamapper::Index::maximum_kmer_size() << std::endl;
        exit(1);
    }

    if (filtering_parameter > 1.0 || filtering_parameter < 0.0)
    {
        std::cerr << "-F / --filtering-parameter must be in range [0.0, 1.0]" << std::endl;
        exit(1);
    }

    if (max_cached_memory <= 0)
    {
        std::cerr << "-m / --max-cached-memory must be larger than zero" << std::endl;
        exit(1);
    }

    if (benchmark_iterations < 0)
    {
        std::cerr << "-b / --benchmark_iterations must be equal to or larger than zero" << std::endl;
        exit(1);
    }

    // Check remaining argument count.
    if ((argc - optind) < 2)
    {
        std::cerr << "Invalid inputs. Please refer to the help function." << std::endl;
        help(1);
    }

    std::string query_filepath  = std::string(argv[optind++]);
    std::string target_filepath = std::string(argv[optind++]);

    bool all_to_all = false;
    if (query_filepath == target_filepath)
    {
        all_to_all        = true;
        target_index_size = index_size;
        std::cerr << "NOTE - Since query and target files are same, activating all_to_all mode. Query index size used for both files." << std::endl;
    }

    std::shared_ptr<claragenomics::io::FastaParser> query_parser = claragenomics::io::create_kseq_fasta_parser(query_filepath);
    int32_t queries                                              = query_parser->get_num_seqences();

    std::shared_ptr<claragenomics::io::FastaParser> target_parser;
    if (all_to_all)
    {
        target_parser = query_parser;
    }
    else
    {
        target_parser = claragenomics::io::create_kseq_fasta_parser(target_filepath);
    }

    int32_t targets = target_parser->get_num_seqences();

    std::cerr << "Query " << query_filepath << " index " << queries << std::endl;
    std::cerr << "Target " << target_filepath << " index " << targets << std::endl;

    // Data structure for holding overlaps to be written out
    std::mutex overlaps_writer_mtx;

    struct QueryTargetsRange
    {
        std::pair<std::int32_t, int32_t> query_range;
        std::vector<std::pair<std::int32_t, int32_t>> target_ranges;
    };

    ///Factor of 1000000 to make max cache size in MiB
    auto query_chunks  = query_parser->get_read_chunks(index_size * 1000000);
    auto target_chunks = target_parser->get_read_chunks(target_index_size * 1000000);

    //First generate all the ranges independently, then loop over them.
    std::vector<QueryTargetsRange> query_target_ranges;

    int target_idx = 0;
    for (auto const& query_chunk : query_chunks)
    {
        QueryTargetsRange range;
        range.query_range = query_chunk;
        for (size_t t = target_idx; t < target_chunks.size(); t++)
        {
            range.target_ranges.push_back(target_chunks[t]);
        }
        query_target_ranges.push_back(range);
        // in all-to-all, for query chunk 0, we go through target chunks [target_idx = 0 , n = target_chunks.size())
        // for query chunk 1, we only need target chunks [target_idx = 1 , n), and in general for query_chunk i, we need target chunks [target_idx = i , n)
        // therefore as we're looping through query chunks, in all-to-all, will increment target_idx
        if (all_to_all)
        {
            target_idx++;
        }
    }

    // This is host cache, if it has the index it will copy it to device, if not it will generate on device and add it to host cache
    std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::IndexHostCopy>> host_index_cache;

    // This is a per-device cache, if it has the index it will return it, if not it will generate it, store and return it.
    std::vector<std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::Index>>> device_index_cache(num_devices);

    // The number of overlap chunks which are to be computed
    std::atomic<int> num_overlap_chunks_to_print(0);

    // benchmark data per device
    std::vector<claragenomics::cudamapper::BenchMarkData> benchmark_log(num_devices);
    // flag indicating benchmark mode is enabled
    const bool benchmark_mode = benchmark_iterations > 0;

    auto get_index = [&device_index_cache, &host_index_cache, max_index_cache_size_on_device,
                      max_index_cache_size_on_host, &benchmark_log](claragenomics::DefaultDeviceAllocator allocator,
                                                                    claragenomics::io::FastaParser& parser,
                                                                    const claragenomics::cudamapper::read_id_t start_index,
                                                                    const claragenomics::cudamapper::read_id_t end_index,
                                                                    const std::uint64_t k,
                                                                    const std::uint64_t w,
                                                                    const int device_id,
                                                                    const bool allow_cache_index,
                                                                    const double filtering_parameter) {
        CGA_NVTX_RANGE(profiler, "get index");
        std::pair<uint64_t, uint64_t> key;
        key.first  = start_index;
        key.second = end_index;

        std::shared_ptr<claragenomics::cudamapper::Index> index;

        // first check if it's available on device, if not then check the host cache
        if (device_index_cache[device_id].count(key))
        {
            index = device_index_cache[device_id][key];
        }
        else if (host_index_cache.count(key))
        {
            index = host_index_cache[key]->copy_index_to_device(allocator);
        }
        else
        {
            //create an index, with hashed representations (minimizers)
            index = std::move(claragenomics::cudamapper::Index::create_index(allocator, parser, start_index, end_index, k, w, true, filtering_parameter));

            // If in all-to-all mode, put this query in the cache for later use.
            // Cache eviction is handled later on by the calling thread
            // using the evict_index function.
            if (get_size<int32_t>(device_index_cache[device_id]) < max_index_cache_size_on_device && allow_cache_index)
            {
                device_index_cache[device_id][key] = index;
            }

            // update host cache, only done by device 0 to avoid any race conditions in updating the host cache
            if (get_size<int32_t>(host_index_cache) < max_index_cache_size_on_host && allow_cache_index && device_id == 0)
            {
                host_index_cache[key] = std::move(claragenomics::cudamapper::IndexHostCopy::create_cache(*index, start_index, k, w));
            }
        }
        return index;
    };

    // When performing all-to-all mapping, indices are instantitated as start-end-ranges in the reads.
    // As such, once a query index has been used it will not be needed again. For example, parsing ranges
    // [0-999], [1000-1999], [2000-2999], the caching/eviction would be as follows:
    //
    // Round 1
    // Query: [0-999] - Enter cache
    // Target: [1000-1999] - Enter cache
    // Target: [1999 - 2999] - Enter cache
    // Evict [0-999]
    // Round 2
    // Query: [1000-1999] - Use cache entry (from previous use when now query was a target)
    // Etc..
    auto evict_index = [&device_index_cache, &host_index_cache](const claragenomics::cudamapper::read_id_t query_start_index,
                                                                const claragenomics::cudamapper::read_id_t query_end_index,
                                                                const int device_id,
                                                                const int num_devices) {
        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;
        device_index_cache[device_id].erase(key);
        // host memory can be freed by removing (key) when working with 1 GPU
        // in multiple GPUs we keep (key), as it may be accessed by other GPUs depending on access pattern
        if (num_devices == 1)
            host_index_cache.erase(key);
    };

#ifdef CGA_ENABLE_ALLOCATOR
    // uses CachingDeviceAllocator
    auto max_cached_bytes = max_cached_memory * 1e9; // max_cached_memory is in GB
    claragenomics::DefaultDeviceAllocator allocator(max_cached_bytes);
#else
    // uses CudaMallocAllocator
    claragenomics::DefaultDeviceAllocator allocator;
#endif

    auto compute_overlaps = [&](const QueryTargetsRange& query_target_range, const int device_id) {
        hipSetDevice(device_id);

        auto query_start_index = query_target_range.query_range.first;
        auto query_end_index   = query_target_range.query_range.second;

        std::cerr << "Processing query range: (" << query_start_index << " - " << query_end_index - 1 << ")" << std::endl;

        std::shared_ptr<claragenomics::cudamapper::Index> query_index(nullptr);
        std::shared_ptr<claragenomics::cudamapper::Index> target_index(nullptr);
        std::unique_ptr<claragenomics::cudamapper::Matcher> matcher(nullptr);

        {
            CGA_NVTX_RANGE(profiler, "generate_query_index");
            benchmark_log[device_id].start_timer(benchmark_mode);
            query_index = get_index(allocator, *query_parser, query_start_index, query_end_index, k, w, device_id, all_to_all, filtering_parameter);
            benchmark_log[device_id].stop_timer_and_gather_data('i', benchmark_mode);
        }

        //Main loop
        for (const auto target_range : query_target_range.target_ranges)
        {

            auto target_start_index = target_range.first;
            auto target_end_index   = target_range.second;
            {
                CGA_NVTX_RANGE(profiler, "generate_target_index");
                benchmark_log[device_id].start_timer(benchmark_mode);
                target_index = get_index(allocator, *target_parser, target_start_index, target_end_index, k, w, device_id, true, filtering_parameter);
                benchmark_log[device_id].stop_timer_and_gather_data('i', benchmark_mode);
            }
            {
                CGA_NVTX_RANGE(profiler, "generate_matcher");
                benchmark_log[device_id].start_timer(benchmark_mode);
                matcher = claragenomics::cudamapper::Matcher::create_matcher(allocator, *query_index, *target_index);
                benchmark_log[device_id].stop_timer_and_gather_data('m', benchmark_mode);
            }
            {

                claragenomics::cudamapper::OverlapperTriggered overlapper(allocator);
                CGA_NVTX_RANGE(profiler, "generate_overlaps");

                // Get unfiltered overlaps
                auto overlaps_to_add = std::make_shared<std::vector<claragenomics::cudamapper::Overlap>>();

                benchmark_log[device_id].start_timer(benchmark_mode);
                overlapper.get_overlaps(*overlaps_to_add, matcher->anchors());
                benchmark_log[device_id].stop_timer_and_gather_data('o', benchmark_mode);

                //Increment counter which tracks number of overlap chunks to be filtered and printed
                num_overlap_chunks_to_print++;
                auto filter_and_print_overlaps = [&overlaps_writer_mtx, &num_overlap_chunks_to_print](std::shared_ptr<std::vector<claragenomics::cudamapper::Overlap>> overlaps,
                                                                                                      std::shared_ptr<claragenomics::cudamapper::Index> query_index,
                                                                                                      std::shared_ptr<claragenomics::cudamapper::Index> target_index) {
                    std::vector<claragenomics::cudamapper::Overlap> filtered_overlaps;
                    claragenomics::cudamapper::Overlapper::filter_overlaps(filtered_overlaps, *overlaps, 50);
                    // parallel update of the query/target read names for filtered overlaps [parallel on host]
                    claragenomics::cudamapper::Overlapper::update_read_names(filtered_overlaps, *query_index, *target_index);
                    std::lock_guard<std::mutex> lck(overlaps_writer_mtx);
                    claragenomics::cudamapper::Overlapper::print_paf(filtered_overlaps);

                    //clear data
                    for (auto o : *overlaps)
                    {
                        o.clear();
                    }
                    //Decrement counter which tracks number of overlap chunks to be filtered and printed
                    num_overlap_chunks_to_print--;
                };

                std::thread t(filter_and_print_overlaps, overlaps_to_add, query_index, target_index);
                t.detach();
            }
            // reseting the matcher releases the anchor device array back to memory pool
            matcher.reset();
        }

        // If all-to-all mapping query will no longer be needed on device, remove it from the cache
        if (all_to_all)
        {
            evict_index(query_start_index, query_end_index, device_id, num_devices);
        }

        benchmark_log[device_id].update_iteration_data(benchmark_mode);
    };

    // The application (File parsing, index generation, overlap generation etc) is all launched from here.
    // The main application works as follows:
    // 1. Launch a worker thread per device (GPU).
    // 2. Each worker takes target-query ranges off a queue
    // 3. Each worker pushes vector of futures (since overlap writing is dispatched to an async thread on host). All futures are waited for before the main application exits.
    std::vector<std::thread> workers;
    std::atomic<int> ranges_idx(0);

    // Launch worker threads to enable multi-GPU.
    // One worker thread is responsible for one GPU so the number
    // of worker threads launched is equal to the number of devices specified
    // by the user
    for (int device_id = 0; device_id < num_devices; device_id++)
    {
        //Worker thread consumes query-target ranges off a queue
        workers.push_back(std::thread(
            [&, device_id]() {
                while (ranges_idx < get_size<int>(query_target_ranges))
                {
                    int range_idx = ranges_idx.fetch_add(1);

                    // if benchmark-mode is activated by entering a positive integer for -b,
                    // limit iterations to benchmark_iterations
                    if (benchmark_iterations > 0 && range_idx > benchmark_iterations)
                    {
                        break;
                    }

                    //Need to perform this check again for thread-safety
                    if (range_idx < get_size<int>(query_target_ranges))
                    {
                        //compute overlaps takes a range of read_ids and a device ID and uses
                        //that device to compute the overlaps. It prints overlaps to stdout.
                        //since multiple worker threads are running stdout is guarded
                        //by a mutex (`std::mutex overlaps_writer_mtx`)
                        compute_overlaps(query_target_ranges[range_idx], device_id);
                    }
                }
                if (benchmark_iterations > 0)
                {
                    benchmark_log[device_id].display();
                }
            }));
    }

    // Wait for all per-device threads to terminate
    std::for_each(workers.begin(), workers.end(), [](std::thread& t) {
        t.join();
    });

    // Wait for all futures (for overlap filtering and writing) to return
    while (num_overlap_chunks_to_print != 0)
    {
        std::this_thread::sleep_for(std::chrono::milliseconds(200));
    }

    return 0;
}

void help(int32_t exit_code = 0)
{
    std::cerr <<
        R"(Usage: cudamapper [options ...] <query_sequences> <target_sequences>
     <sequences>
        Input file in FASTA/FASTQ format (can be compressed with gzip)
        containing sequences used for all-to-all overlapping
     options:
        -k, --kmer-size
            length of kmer to use for minimizers [15] (Max=)"
              << claragenomics::cudamapper::Index::maximum_kmer_size() << ")"
              << R"(
        -w, --window-size
            length of window to use for minimizers [15])"
              << R"(
        -d, --num-devices
            number of GPUs to use [1])"
              << R"(
        -c, --max-index-device-cache
            number of indices to keep in GPU memory [100])"
              << R"(
        -C, --max-index-host-cache
            number of indices to keep in host memory [0])"
              << R"(
        -m, --max-cached-memory
            maximum aggregate cached memory per device in GB [1])"
              << R"(
        -i, --index-size
            length of batch size used for query in MB [30])"
              << R"(
        -t --target-index-size
            length of batch sized used for target in MB [30])"
              << R"(
        -b, --benchmark-iterations
            number of query index batches used in benchmarking [0])"
              << R"(
        -F --filtering-parameter
            filter all representations for which sketch_elements_with_that_representation/total_sketch_elements >= filtering_parameter), filtering disabled if filtering_parameter == 1.0 [1'000'000'001] (Min = 0.0, Max = 1.0))"
              << std::endl;

    exit(exit_code);
}
