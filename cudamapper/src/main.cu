#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include <algorithm>
#include <getopt.h>
#include <iostream>
#include <string>
#include <deque>
#include <mutex>
#include <future>
#include <thread>
#include <atomic>
#include <fstream>

#include <claragenomics/logging/logging.hpp>
#include <claragenomics/io/fasta_parser.hpp>
#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>

#include <claragenomics/cudamapper/index.hpp>
#include <claragenomics/cudamapper/matcher.hpp>
#include <claragenomics/cudamapper/overlapper.hpp>
#include "overlapper_triggered.hpp"

static struct option options[] = {
    {"kmer-size", required_argument, 0, 'k'},
    {"window-size", required_argument, 0, 'w'},
    {"num-devices", required_argument, 0, 'd'},
    {"max-index-device-cache", required_argument, 0, 'c'},
    {"max-index-host-cache", required_argument, 0, 'C'},
    {"max-cached-memory", required_argument, 0, 'm'},
    {"index-size", required_argument, 0, 'i'},
    {"target-index-size", required_argument, 0, 't'},
    {"filtering-parameter", required_argument, 0, 'F'},
    {"alignment-engines", required_argument, 0, 'a'},
    {"help", no_argument, 0, 'h'},
};

void help(int32_t exit_code);
std::size_t find_largest_contiguous_device_memory_section();

int main(int argc, char* argv[])
{
    using claragenomics::get_size;
    claragenomics::logging::Init();

    uint32_t k                                  = 15;  // k
    uint32_t w                                  = 15;  // w
    std::int32_t num_devices                    = 1;   // d
    std::int32_t max_index_cache_size_on_device = 100; // c
    // ToDo: come up with a good heuristic to choose C and c
    std::int32_t max_index_cache_size_on_host = 0;   // C
    std::int32_t max_cached_memory            = 0;   // m
    std::int32_t index_size                   = 30;  // i
    std::int32_t target_index_size            = 30;  // t
    double filtering_parameter                = 1.0; // F
    std::int32_t alignment_engines            = 0;   // a
    std::string optstring                     = "k:w:d:c:C:m:i:t:F:h:a:";
    int32_t argument                          = 0;
    while ((argument = getopt_long(argc, argv, optstring.c_str(), options, nullptr)) != -1)
    {
        switch (argument)
        {
        case 'k':
            k = atoi(optarg);
            break;
        case 'w':
            w = atoi(optarg);
            break;
        case 'd':
            num_devices = atoi(optarg);
            break;
        case 'c':
            max_index_cache_size_on_device = atoi(optarg);
            break;
        case 'C':
            max_index_cache_size_on_host = atoi(optarg);
            break;
        case 'm':
#ifndef CGA_ENABLE_CACHING_ALLOCATOR
            std::cerr << "ERROR: Argument -m / --max-cached-memory cannot be used without caching allocator" << std::endl;
            exit(1);
#endif
            max_cached_memory = atoi(optarg);
            break;
        case 'i':
            index_size = atoi(optarg);
            break;
        case 't':
            target_index_size = atoi(optarg);
            break;
        case 'F':
            filtering_parameter = atof(optarg);
            break;
        case 'a':
            alignment_engines = atoi(optarg);
            claragenomics::throw_on_negative(alignment_engines, "Number of alignment engines should be non-negative");
            break;
        case 'h':
            help(0);
        default:
            exit(1);
        }
    }

    if (k > claragenomics::cudamapper::Index::maximum_kmer_size())
    {
        std::cerr << "kmer of size " << k << " is not allowed, maximum k = " << claragenomics::cudamapper::Index::maximum_kmer_size() << std::endl;
        exit(1);
    }

    if (filtering_parameter > 1.0 || filtering_parameter < 0.0)
    {
        std::cerr << "-F / --filtering-parameter must be in range [0.0, 1.0]" << std::endl;
        exit(1);
    }

    if (max_cached_memory < 0)
    {
        std::cerr << "-m / --max-cached-memory must not be negative" << std::endl;
        exit(1);
    }

    // Check remaining argument count.
    if ((argc - optind) < 2)
    {
        std::cerr << "Invalid inputs. Please refer to the help function." << std::endl;
        help(1);
    }

    std::string query_filepath  = std::string(argv[optind++]);
    std::ifstream query_file(query_filepath);
    if (query_file.fail())
    {
    	std::cerr << "Failed to access file: "+ query_filepath << std::endl;
    	exit(1);
    }

    std::string target_filepath = std::string(argv[optind++]);
    std::ifstream target_file(target_filepath);
    if (target_file.fail())
    {
    	std::cerr << "Failed to access file: "+ target_filepath << std::endl;
    	exit(1);
    }
    
    bool all_to_all = false;
    if (query_filepath == target_filepath)
    {
        all_to_all        = true;
        target_index_size = index_size;
        std::cerr << "NOTE - Since query and target files are same, activating all_to_all mode. Query index size used for both files." << std::endl;
    }

    std::shared_ptr<claragenomics::io::FastaParser> query_parser = claragenomics::io::create_kseq_fasta_parser(query_filepath, k + w - 1);
    int32_t queries                                              = query_parser->get_num_seqences();

    std::shared_ptr<claragenomics::io::FastaParser> target_parser;
    if (all_to_all)
    {
        target_parser = query_parser;
    }
    else
    {
        target_parser = claragenomics::io::create_kseq_fasta_parser(target_filepath, k + w - 1);
    }

    int32_t targets = target_parser->get_num_seqences();

    std::cerr << "Query " << query_filepath << " index " << queries << std::endl;
    std::cerr << "Target " << target_filepath << " index " << targets << std::endl;
    if ((queries == 0)||(targets==0))
    {
        std::cerr << "Query or target index can not be zero." << std::endl; 
        exit(1);
    }
    
    // Data structure for holding overlaps to be written out
    std::mutex overlaps_writer_mtx;

    struct QueryTargetsRange
    {
        std::pair<std::int32_t, int32_t> query_range;
        std::vector<std::pair<std::int32_t, int32_t>> target_ranges;
    };

    ///Factor of 1000000 to make max cache size in MiB
    auto query_chunks  = query_parser->get_read_chunks(index_size * 1000000);
    auto target_chunks = target_parser->get_read_chunks(target_index_size * 1000000);

    //First generate all the ranges independently, then loop over them.
    std::vector<QueryTargetsRange> query_target_ranges;

    int target_idx = 0;
    for (auto const& query_chunk : query_chunks)
    {
        QueryTargetsRange range;
        range.query_range = query_chunk;
        for (size_t t = target_idx; t < target_chunks.size(); t++)
        {
            range.target_ranges.push_back(target_chunks[t]);
        }
        query_target_ranges.push_back(range);
        // in all-to-all, for query chunk 0, we go through target chunks [target_idx = 0 , n = target_chunks.size())
        // for query chunk 1, we only need target chunks [target_idx = 1 , n), and in general for query_chunk i, we need target chunks [target_idx = i , n)
        // therefore as we're looping through query chunks, in all-to-all, will increment target_idx
        if (all_to_all)
        {
            target_idx++;
        }
    }

    // This is host cache, if it has the index it will copy it to device, if not it will generate on device and add it to host cache
    std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::IndexHostCopy>> host_index_cache;

    // This is a per-device cache, if it has the index it will return it, if not it will generate it, store and return it.
    std::vector<std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::Index>>> device_index_cache(num_devices);

    // The number of overlap chunks which are to be computed
    std::atomic<int> num_overlap_chunks_to_print(0);

    auto get_index = [&device_index_cache, &host_index_cache, max_index_cache_size_on_device, max_index_cache_size_on_host](claragenomics::DefaultDeviceAllocator allocator,
                                                                                                                            claragenomics::io::FastaParser& parser,
                                                                                                                            const claragenomics::cudamapper::read_id_t start_index,
                                                                                                                            const claragenomics::cudamapper::read_id_t end_index,
                                                                                                                            const std::uint64_t k,
                                                                                                                            const std::uint64_t w,
                                                                                                                            const int device_id,
                                                                                                                            const bool allow_cache_index,
                                                                                                                            const double filtering_parameter,
                                                                                                                            const hipStream_t hip_stream) {
        CGA_NVTX_RANGE(profiler, "get index");
        std::pair<uint64_t, uint64_t> key;
        key.first  = start_index;
        key.second = end_index;

        std::shared_ptr<claragenomics::cudamapper::Index> index;

        // first check if it's available on device, if not then check the host cache
        if (device_index_cache[device_id].count(key))
        {
            index = device_index_cache[device_id][key];
        }
        else if (host_index_cache.count(key))
        {
            index = host_index_cache[key]->copy_index_to_device(allocator,
                                                                hip_stream);
        }
        else
        {
            //create an index, with hashed representations (minimizers)
            index = std::move(claragenomics::cudamapper::Index::create_index(allocator,
                                                                             parser,
                                                                             start_index,
                                                                             end_index,
                                                                             k,
                                                                             w,
                                                                             true, // hash_representations
                                                                             filtering_parameter,
                                                                             hip_stream));

            // If in all-to-all mode, put this query in the cache for later use.
            // Cache eviction is handled later on by the calling thread
            // using the evict_index function.
            if (get_size<int32_t>(device_index_cache[device_id]) < max_index_cache_size_on_device && allow_cache_index)
            {
                device_index_cache[device_id][key] = index;
            }
            else if (get_size<int32_t>(host_index_cache) < max_index_cache_size_on_host && allow_cache_index && device_id == 0)
            {
                // if not cached on device, update host cache; only done on device 0 to avoid any race conditions in updating the host cache
                host_index_cache[key] = std::move(claragenomics::cudamapper::IndexHostCopy::create_cache(*index,
                                                                                                         start_index,
                                                                                                         k,
                                                                                                         w,
                                                                                                         hip_stream));
            }
        }
        return index;
    };

    // When performing all-to-all mapping, indices are instantitated as start-end-ranges in the reads.
    // As such, once a query index has been used it will not be needed again. For example, parsing ranges
    // [0-999], [1000-1999], [2000-2999], the caching/eviction would be as follows:
    //
    // Round 1
    // Query: [0-999] - Enter cache
    // Target: [1000-1999] - Enter cache
    // Target: [1999 - 2999] - Enter cache
    // Evict [0-999]
    // Round 2
    // Query: [1000-1999] - Use cache entry (from previous use when now query was a target)
    // Etc..
    auto evict_index = [&device_index_cache, &host_index_cache](const claragenomics::cudamapper::read_id_t query_start_index,
                                                                const claragenomics::cudamapper::read_id_t query_end_index,
                                                                const int device_id,
                                                                const int num_devices) {
        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;
        device_index_cache[device_id].erase(key);
        // host memory can be freed by removing (key) when working with 1 GPU
        // in multiple GPUs we keep (key), as it may be accessed by other GPUs depending on access pattern
        if (num_devices == 1)
            host_index_cache.erase(key);
    };

#ifdef CGA_ENABLE_CACHING_ALLOCATOR
    // uses CachingDeviceAllocator
    std::size_t max_cached_bytes = 0;
    if (max_cached_memory == 0)
    {
        std::cerr << "Programmatically looking for max cached memory" << std::endl;
        max_cached_bytes = find_largest_contiguous_device_memory_section();
        if (max_cached_bytes == 0)
        {
            std::cerr << "No memory available for caching" << std::endl;
            exit(1);
        }
    }
    else
    {
        max_cached_bytes = max_cached_memory * 1024ull * 1024ull * 1024ull; // max_cached_memory is in GiB
    }

    std::cerr << "Using device memory cache of " << max_cached_bytes << " bytes" << std::endl;

    claragenomics::DefaultDeviceAllocator allocator(max_cached_bytes);
#else
    // uses CudaMallocAllocator
    claragenomics::DefaultDeviceAllocator allocator;
#endif

    auto compute_overlaps = [&](const QueryTargetsRange& query_target_range,
                                const int device_id,
                                const hipStream_t hip_stream) {
        auto query_start_index = query_target_range.query_range.first;
        auto query_end_index   = query_target_range.query_range.second;

        std::cerr << "Processing query range: (" << query_start_index << " - " << query_end_index - 1 << ")" << std::endl;

        std::shared_ptr<claragenomics::cudamapper::Index> query_index(nullptr);
        std::shared_ptr<claragenomics::cudamapper::Index> target_index(nullptr);
        std::unique_ptr<claragenomics::cudamapper::Matcher> matcher(nullptr);

        {
            CGA_NVTX_RANGE(profiler, "generate_query_index");
            query_index = get_index(allocator,
                                    *query_parser,
                                    query_start_index,
                                    query_end_index,
                                    k,
                                    w,
                                    device_id,
                                    all_to_all,
                                    filtering_parameter,
                                    hip_stream);
        }

        //Main loop
        for (const auto target_range : query_target_range.target_ranges)
        {

            auto target_start_index = target_range.first;
            auto target_end_index   = target_range.second;
            {
                CGA_NVTX_RANGE(profiler, "generate_target_index");
                target_index = get_index(allocator,
                                         *target_parser,
                                         target_start_index,
                                         target_end_index,
                                         k,
                                         w,
                                         device_id,
                                         true,
                                         filtering_parameter,
                                         hip_stream);
            }
            {
                CGA_NVTX_RANGE(profiler, "generate_matcher");
                matcher = claragenomics::cudamapper::Matcher::create_matcher(allocator,
                                                                             *query_index,
                                                                             *target_index,
                                                                             hip_stream);
            }
            {

                claragenomics::cudamapper::OverlapperTriggered overlapper(allocator, hip_stream);
                CGA_NVTX_RANGE(profiler, "generate_overlaps");

                // Get unfiltered overlaps
                auto overlaps_to_add = std::make_shared<std::vector<claragenomics::cudamapper::Overlap>>();

                overlapper.get_overlaps(*overlaps_to_add, matcher->anchors(), 50);

                std::vector<std::string> cigar;
                // Align overlaps
                if (alignment_engines > 0)
                {
                    cigar.resize(overlaps_to_add->size());
                    CGA_NVTX_RANGE(profiler, "align_overlaps");
                    claragenomics::cudamapper::Overlapper::Overlapper::align_overlaps(*overlaps_to_add, *query_parser, *target_parser, alignment_engines, cigar);
                }

                //Increment counter which tracks number of overlap chunks to be filtered and printed
                num_overlap_chunks_to_print++;
                auto print_overlaps = [&overlaps_writer_mtx, &num_overlap_chunks_to_print](std::shared_ptr<std::vector<claragenomics::cudamapper::Overlap>> filtered_overlaps,
                                                                                           std::shared_ptr<claragenomics::cudamapper::Index> query_index,
                                                                                           std::shared_ptr<claragenomics::cudamapper::Index> target_index,
                                                                                           const std::vector<std::string>& cigar,
                                                                                           const int device_id) {
                    // This lambda is expected to run in a separate thread so set current device in order to avoid problems
                    // with deallocating indices with different current device then the one on which they were created
                    hipSetDevice(device_id);

                    // parallel update of the query/target read names for filtered overlaps [parallel on host]
                    claragenomics::cudamapper::Overlapper::update_read_names(*filtered_overlaps, *query_index, *target_index);
                    std::lock_guard<std::mutex> lck(overlaps_writer_mtx);
                    claragenomics::cudamapper::Overlapper::print_paf(*filtered_overlaps, cigar);

                    //clear data
                    for (auto o : *filtered_overlaps)
                    {
                        o.clear();
                    }
                    //Decrement counter which tracks number of overlap chunks to be filtered and printed
                    num_overlap_chunks_to_print--;
                };

                std::thread t(print_overlaps, overlaps_to_add, query_index, target_index, cigar, device_id);
                t.detach();
            }

            // reseting the matcher releases the anchor device array back to memory pool
            matcher.reset();
        }

        // If all-to-all mapping query will no longer be needed on device, remove it from the cache
        if (all_to_all)
        {
            evict_index(query_start_index, query_end_index, device_id, num_devices);
        }
    };

    // The application (File parsing, index generation, overlap generation etc) is all launched from here.
    // The main application works as follows:
    // 1. Launch a worker thread per device (GPU).
    // 2. Each worker takes target-query ranges off a queue
    // 3. Each worker pushes vector of futures (since overlap writing is dispatched to an async thread on host). All futures are waited for before the main application exits.
    std::vector<std::thread> workers;
    std::atomic<int> ranges_idx(0);

    // Each worker thread gets its own CUDA stream to work on. Currently there is only one worker thread per GPU,
    // but it is still necessary assign streams to each of then explicitly. --default-stream per-thread could
    // cause problems beacuse there are subthreads for worker threads
    std::vector<hipStream_t> cuda_streams(num_devices);

    // Launch worker threads to enable multi-GPU.
    // One worker thread is responsible for one GPU so the number
    // of worker threads launched is equal to the number of devices specified
    // by the user
    for (int device_id = 0; device_id < num_devices; ++device_id)
    {
        CGA_CU_CHECK_ERR(hipStreamCreate(&cuda_streams[device_id]));
        //Worker thread consumes query-target ranges off a queue
        workers.push_back(std::thread(
            [&, device_id]() {
                hipSetDevice(device_id);
                while (ranges_idx < get_size<int>(query_target_ranges))
                {
                    int range_idx = ranges_idx.fetch_add(1);
                    //Need to perform this check again for thread-safety
                    if (range_idx < get_size<int>(query_target_ranges))
                    {
                        //compute overlaps takes a range of read_ids and a device ID and uses
                        //that device to compute the overlaps. It prints overlaps to stdout.
                        //since multiple worker threads are running stdout is guarded
                        //by a mutex (`std::mutex overlaps_writer_mtx`)
                        compute_overlaps(query_target_ranges[range_idx],
                                         device_id,
                                         cuda_streams[device_id]);
                    }
                }
            }));
    }

    // Wait for all per-device threads to terminate
    for (auto& worker_thread : workers)
    {
        worker_thread.join();
    }

    // Wait for all futures (for overlap writing) to return
    while (num_overlap_chunks_to_print != 0)
    {
        std::this_thread::sleep_for(std::chrono::milliseconds(200));
    }

    // streams can only be destroyed once all writer threads have finished as they hold references
    // to indices which have device arrays associated with streams
    for (hipStream_t hip_stream : cuda_streams)
    {
        CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
        CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
    }

    return 0;
}

void help(int32_t exit_code = 0)
{
    std::cerr <<
        R"(Usage: cudamapper [options ...] <query_sequences> <target_sequences>
     <sequences>
        Input file in FASTA/FASTQ format (can be compressed with gzip)
        containing sequences used for all-to-all overlapping
     options:
        -k, --kmer-size
            length of kmer to use for minimizers [15] (Max=)"
              << claragenomics::cudamapper::Index::maximum_kmer_size() << ")"
              << R"(
        -w, --window-size
            length of window to use for minimizers [15])"
              << R"(
        -d, --num-devices
            number of GPUs to use [1])"
              << R"(
        -c, --max-index-device-cache
            number of indices to keep in GPU memory [100])"
              << R"(
        -C, --max-index-host-cache
            number of indices to keep in host memory [0])"
              << R"(
        -m, --max-cached-memory
            maximum aggregate cached memory per device in GiB, if 0 program tries to allocate as much memory as possible [0])"
              << R"(
        -i, --index-size
            length of batch size used for query in MB [30])"
              << R"(
        -t, --target-index-size
            length of batch sized used for target in MB [30])"
              << R"(
        -F, --filtering-parameter
            filter all representations for which sketch_elements_with_that_representation/total_sketch_elements >= filtering_parameter), filtering disabled if filtering_parameter == 1.0 [1'000'000'001] (Min = 0.0, Max = 1.0))"
              << R"(
        -a, --alignment-engines
            Number of alignment engines to use (per device) for generating CIGAR strings for overlap alignments. Default value 0 = no alignment to be performed. Typically 2-4 engines per device gives best perf.)"
              << std::endl;

    exit(exit_code);
}

/// @brief finds largest section of contiguous memory on device
/// @return number of bytes
std::size_t find_largest_contiguous_device_memory_section()
{
    // find the largest block of contiguous memory
    size_t free;
    size_t total;
    hipMemGetInfo(&free, &total);
    const size_t memory_decrement = free / 100;              // decrease requested memory one by one percent
    size_t size_to_try            = free - memory_decrement; // do not go for all memory
    while (true)
    {
        void* dummy_ptr    = nullptr;
        hipError_t status = hipMalloc(&dummy_ptr, size_to_try);
        // if it was able to allocate memory free the memory and return the size
        if (status == hipSuccess)
        {
            hipFree(dummy_ptr);
            return size_to_try;
        }

        if (status == hipErrorOutOfMemory)
        {
            // if it was not possible to allocate the memory because there was not enough of it
            // try allocating less memory in next iteration
            if (size_to_try > memory_decrement)
            {
                size_to_try -= memory_decrement;
            }
            else
            { // a very small amount of memory left, report an error
                CGA_CU_CHECK_ERR(hipErrorOutOfMemory);
                return 0;
            }
        }
        else
        {
            // if hipMalloc failed because of error other than hipErrorOutOfMemory process the error
            CGA_CU_CHECK_ERR(status);
        }
    }

    // this point should actually never be reached (loop either finds memory or causes an error)
    assert(false);
    CGA_CU_CHECK_ERR(hipErrorOutOfMemory);
    return 0;
}