#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "overlapper_anchmer.hpp"

#include <fstream>
#include <cstdlib>

// Needed for accumulate - remove when ported to cuda
#include <numeric>
#include <limits>

#include <hipcub/hipcub.hpp>
#include <thrust/execution_policy.h>

#include <claraparabricks/genomeworks/utils/cudautils.hpp>

#ifndef NDEBUG // only needed to check if input is sorted in assert
#include <algorithm>
#include <thrust/host_vector.h>
#endif

//
//          36a32532-4135-4ffe-a346-0b1b08c1b747   6370    173     683     -       fedde900-1485-42a2-8adb-b7a30dcf82fe    10014   91      620     105     529     0       minimap2        not_in_cm
//         da4230aa-e79e-4a5f-9738-c272aad98a82   8554    74      293     +       ebb4ceeb-76f1-454a-84d9-a65cfcb3a9fa    1824    1572    1801    114     231     0       minimap2        not_in_cm

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

#define MAX_ANCHMER_WINDOW 10
#define MAX_OVERLAPMER_WINDOW 20

struct Anchmer
{
    std::int8_t n_chained_anchors[MAX_ANCHMER_WINDOW] = {0};
    std::int8_t chain_id[MAX_ANCHMER_WINDOW]          = {0};
    std::int8_t n_chains                              = 0;
    std::int32_t n_anchors                            = 0;
};

struct Overlapmer
{
    std::int8_t n_chained_anchors[MAX_OVERLAPMER_WINDOW] = {0};
    std::int8_t chain_id[MAX_OVERLAPMER_WINDOW]          = {0};
    std::int8_t n_chains                                 = 0;
    std::int8_t n_overlaps                               = 0;
};

struct ChainPiece
{
    Overlap overlap;
    __device__ ChainPiece() {}
};

struct OverlapToChainPieceOp
{

    __device__ __forceinline__ ChainPiece operator()(const Overlap& a) const
    {
        ChainPiece c;
        c.overlap = a;
        return c;
    }
};

struct QueryTargetPair
{
    Overlap overlap;
    __device__ QueryTargetPair() {}
};

struct OverlapToQueryTargetPairOp
{
    __device__ __forceinline__ QueryTargetPair operator()(const Overlap& a) const
    {
        QueryTargetPair p;
        p.overlap = a;
        return p;
    }
};

__device__ bool operator==(const QueryTargetPair& a, const QueryTargetPair& b)
{
    return a.overlap.query_read_id_ == b.overlap.query_read_id_ && a.overlap.target_read_id_ == b.overlap.target_read_id_;
}

__device__ bool
operator==(const ChainPiece& a, const ChainPiece& b)
{
    const bool q_adjacent = abs(int(b.overlap.query_start_position_in_read_) - int(a.overlap.query_end_position_in_read_)) < 5000;
    const bool t_adjacent = a.overlap.relative_strand == RelativeStrand::Forward ? abs(int(a.overlap.target_end_position_in_read_) - int(b.overlap.target_start_position_in_read_)) < 5000 : abs(int(b.overlap.target_start_position_in_read_) - int(a.overlap.target_end_position_in_read_)) < 5000;
    return a.overlap.query_read_id_ == b.overlap.query_read_id_ &&
           a.overlap.target_read_id_ == b.overlap.target_read_id_ &&
           a.overlap.relative_strand == b.overlap.relative_strand &&
           t_adjacent && q_adjacent;
}
__device__ bool operator==(const Overlap& a,
                           const Overlap& b)
{
    bool same_strand   = a.relative_strand == b.relative_strand;
    bool identical_ids = a.query_read_id_ == b.query_read_id_ && a.target_read_id_ == b.target_read_id_;
    // bool q_ends_overlap;
    // bool t_end_overlap;
    position_in_read_t q_gap = abs((int)b.query_start_position_in_read_ - (int)a.query_end_position_in_read_);
    position_in_read_t t_gap = abs((int)b.target_start_position_in_read_ - (int)a.target_end_position_in_read_);
    bool gap_match           = q_gap < 150 && t_gap < 150;
    bool gap_ratio_okay      = float(min(q_gap, t_gap) / max(q_gap, t_gap)) < 0.8;

    //return identical_ids && same_strand && (gap_match || gap_ratio_okay);

    return identical_ids && same_strand && (gap_match);
}

__device__ bool operator==(const Anchor& lhs,
                           const Anchor& rhs)
{
    auto score_threshold = 1;

    // Very simple scoring function to quantify quality of overlaps.
    auto score = 1;

    if (abs(int(rhs.query_position_in_read_) - int(lhs.query_position_in_read_)) <= 50 and
        abs(int(rhs.target_position_in_read_) - int(lhs.target_position_in_read_)) <= 50)
        score = 2;
    return ((lhs.query_read_id_ == rhs.query_read_id_) &&
            (lhs.target_read_id_ == rhs.target_read_id_) &&
            score > score_threshold);
}

struct AnchmerCountChainsOp
{

    AnchmerCountChainsOp()
    {
    }

    __host__ __device__ __forceinline__
        std::int32_t
        operator()(const Anchmer& a) const
    {
        return static_cast<int32_t>(a.n_chains);
    }
};

struct MergeOverlapRunOp
{
    __device__ __forceinline__ Overlap operator()(const Overlap& a, const Overlap& b)
    {
        Overlap c;
        c.query_read_id_                 = a.query_read_id_;
        c.target_read_id_                = a.target_read_id_;
        c.relative_strand                = b.relative_strand;
        c.query_start_position_in_read_  = min(a.query_start_position_in_read_, b.query_start_position_in_read_);
        c.query_end_position_in_read_    = max(a.query_end_position_in_read_, b.query_end_position_in_read_);
        c.target_start_position_in_read_ = min(a.target_start_position_in_read_, b.target_start_position_in_read_);
        c.target_end_position_in_read_   = max(a.target_end_position_in_read_, b.target_end_position_in_read_);
        c.num_residues_                  = a.num_residues_ + b.num_residues_;
        return c;
    }
};

struct MergeChainPiecesOp
{
    __device__ __forceinline__ ChainPiece operator()(const ChainPiece& a, const ChainPiece& b)
    {
        Overlap c;
        c.query_read_id_                 = a.overlap.query_read_id_;
        c.target_read_id_                = a.overlap.target_read_id_;
        c.relative_strand                = a.overlap.relative_strand;
        c.query_start_position_in_read_  = min(a.overlap.query_start_position_in_read_, b.overlap.query_start_position_in_read_);
        c.query_end_position_in_read_    = max(a.overlap.query_end_position_in_read_, b.overlap.query_end_position_in_read_);
        c.target_start_position_in_read_ = min(a.overlap.target_start_position_in_read_, b.overlap.target_start_position_in_read_);
        c.target_end_position_in_read_   = max(a.overlap.target_end_position_in_read_, b.overlap.target_end_position_in_read_);
        c.num_residues_                  = a.overlap.num_residues_ + b.overlap.num_residues_;
        ChainPiece c_p;
        c_p.overlap = c;
        return c_p;
    }
};

struct DecrementerOp
{
    __host__ __device__ __forceinline__ std::size_t operator()(const std::size_t& val)
    {
        return val - 1;
    }
};

__global__ void mask_overlaps(Overlap* overlaps, std::size_t n_overlaps, bool* mask,
                              const std::size_t min_overlap_length,
                              const std::size_t min_residues,
                              const std::size_t min_bases_per_residue,
                              const bool all_to_all,
                              const bool filter_self_mappings)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        position_in_read_t overlap_query_length  = overlaps[d_tid].query_end_position_in_read_ - overlaps[d_tid].query_start_position_in_read_;
        position_in_read_t overlap_target_length = overlaps[d_tid].target_end_position_in_read_ - overlaps[d_tid].target_start_position_in_read_;
        const bool mask_self_self                = overlaps[d_tid].query_read_id_ == overlaps[d_tid].target_read_id_ && all_to_all && filter_self_mappings;
        auto query_bases_per_residue             = overlap_query_length / overlaps[d_tid].num_residues_;
        auto target_bases_per_residue            = overlap_target_length / overlaps[d_tid].num_residues_;
        mask[d_tid]                              = overlap_query_length >= min_overlap_length & overlap_target_length >= min_overlap_length;
        mask[d_tid] &= overlaps[d_tid].num_residues_ >= min_residues;
        mask[d_tid] &= !mask_self_self;
        //mask[d_tid] &= (query_bases_per_residue < min_bases_per_residue || target_bases_per_residue < min_bases_per_residue);
    }
}

__global__ void finalize_overlaps(Overlap* overlaps, const std::size_t n_overlaps)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        if (overlaps[d_tid].target_start_position_in_read_ > overlaps[d_tid].target_end_position_in_read_)
        {
            overlaps[d_tid].relative_strand                = RelativeStrand::Reverse;
            auto tmp                                       = overlaps[d_tid].target_start_position_in_read_;
            overlaps[d_tid].target_start_position_in_read_ = overlaps[d_tid].target_end_position_in_read_;
            overlaps[d_tid].target_end_position_in_read_   = tmp;
        }
    }
};

__global__ void convert_offsets_to_ends(std::int32_t* starts, std::int32_t* lengths, std::int32_t* ends, std::int32_t n_starts)
{
    std::int32_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_starts)
    {
        ends[d_tid] = starts[d_tid] + lengths[d_tid] - 1;
    }
}

__device__ __forceinline__ Overlap merge_helper(Overlap& a, Overlap& b)
{
    Overlap c;
    c.query_read_id_                 = a.query_read_id_;
    c.target_read_id_                = a.target_read_id_;
    c.relative_strand                = a.relative_strand;
    c.query_start_position_in_read_  = min(a.query_start_position_in_read_, b.query_start_position_in_read_);
    c.query_end_position_in_read_    = max(a.query_end_position_in_read_, b.query_end_position_in_read_);
    c.target_start_position_in_read_ = min(a.target_start_position_in_read_, b.target_start_position_in_read_);
    c.target_end_position_in_read_   = max(a.target_end_position_in_read_, b.target_end_position_in_read_);
    c.num_residues_                  = a.num_residues_ + b.num_residues_;
    return c;
}

__global__ void merge_overlap_runs(Overlap* overlaps,
                                   std::int32_t* starts, std::int32_t* ends, std::size_t n_runs,
                                   Overlap* fused_overlaps)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_runs)
    {
        fused_overlaps[d_tid] = overlaps[starts[d_tid]];
        if (starts[d_tid] != ends[d_tid])
        {
            for (std::size_t i = starts[d_tid] + 1; i < ends[d_tid]; ++i)
            {
                fused_overlaps[d_tid] = merge_helper(fused_overlaps[d_tid], overlaps[i]);
            }
        }
    }
}

__device__ __forceinline__ void add_anchor_to_overlap(const Anchor& anchor, Overlap& overlap)
{
    overlap.query_read_id_                = anchor.query_read_id_;
    overlap.target_read_id_               = anchor.target_read_id_;
    overlap.query_start_position_in_read_ = min(anchor.query_position_in_read_, overlap.query_start_position_in_read_);
    overlap.query_end_position_in_read_   = max(anchor.query_position_in_read_, overlap.query_end_position_in_read_);

    // Handles whether the match is on the forward or reverse strand.
    // Requires anchors to be sorted in order
    // First by query_read_id, then target_read_id, then query_pos, then target_pos.
    // If these are sorted, adding an anchor to an overlap with at least one
    // anchor in it already will indicate whether the overlap is increasing on its start or end.
    // If the anchor falls before the target_start, we are on the reverse strand and we should use it to extend
    // the target_end_.
    // If the anchor falls after the target_start, we should extend the target_start
    // Since the anchors will be monotonically increasing or monotonically decreasing, this
    // should consistently head in the correct direction.

    if (overlap.num_residues_ == 0)
    {
        overlap.target_start_position_in_read_ = anchor.target_position_in_read_;
        overlap.target_end_position_in_read_   = anchor.target_position_in_read_;
    }
    else
    {
        bool on_forward_strand                 = anchor.target_position_in_read_ > overlap.target_start_position_in_read_;
        overlap.target_start_position_in_read_ = on_forward_strand ? overlap.target_start_position_in_read_ : anchor.target_position_in_read_;
        overlap.target_end_position_in_read_   = on_forward_strand ? anchor.target_position_in_read_ : overlap.target_end_position_in_read_;
        overlap.relative_strand                = on_forward_strand ? RelativeStrand::Forward : RelativeStrand::Reverse;
    }

    ++overlap.num_residues_;
}

__device__ __forceinline__ void init_overlap(Overlap& overlap)
{
    overlap.query_start_position_in_read_  = 4294967295;
    overlap.query_end_position_in_read_    = 0;
    overlap.target_start_position_in_read_ = 4294967295;
    overlap.target_end_position_in_read_   = 0;
    overlap.relative_strand                = RelativeStrand::Forward;
    overlap.num_residues_                  = 0;
}

__global__ void initialize_overlaps_array(Overlap* overlaps, const size_t n_overlaps)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        init_overlap(overlaps[d_tid]);
    }
}

__global__ void anchmers_to_overlaps(const Anchmer* anchmers, const int32_t* overlap_ends, const size_t n_anchmers, const Anchor* anchors, const size_t n_anchors, Overlap* overlaps, const size_t n_overlaps)
{
    // thread ID, which is used to index into the anchmers array
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (d_tid < n_anchmers)
    {
        for (std::size_t i = 0; i < anchmers[d_tid].n_anchors; ++i)
        {
            std::size_t overlap_index = overlap_ends[d_tid] - anchmers[d_tid].n_chains + anchmers[d_tid].chain_id[i];
            add_anchor_to_overlap(anchors[d_tid * MAX_ANCHMER_WINDOW + i], overlaps[overlap_index]);
        }
    }
}

__global__ void
generate_anchmers(const Anchor* d_anchors, const size_t n_anchors, Anchmer* anchmers, const uint8_t anchmer_size)
{

    // thread ID, which is used to index into the Anchors array
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;

    // First index within the anchors array for this Anchmer
    std::size_t first_anchor_index = d_tid * anchmer_size;

    // Initialize Anchmer fields
    anchmers[d_tid].n_anchors  = 0;
    anchmers[d_tid].n_chains   = 0;
    std::int32_t current_chain = 1;
    for (int i = 0; i < MAX_ANCHMER_WINDOW; ++i)
    {
        anchmers[d_tid].chain_id[i] = 0;
    }
    anchmers[d_tid].chain_id[0] = current_chain;
    anchmers[d_tid].n_chains    = 1;
    // end intialization

    /**
    * Iterate through the anchors within this thread's range (first_anchor_index -> first_anchor_index + anchmer_size (or the end of the Anchors array))
    * For each anchor
    *   if the anchor has not been chained to another anchor, create a new chain (by incrementing the chain ID) and increment the number of chains in the Anchmer
    *   
    */
    for (std::size_t i = 0; i < anchmer_size; ++i)
    {
        std::size_t global_anchor_index = first_anchor_index + i;
        if (global_anchor_index < n_anchors)
        {
            ++(anchmers[d_tid].n_anchors);
            anchmers[d_tid].n_chains = anchmers[d_tid].chain_id[i] == 0 ? anchmers[d_tid].n_chains + 1 : anchmers[d_tid].n_chains;
            //Label the anchor with its chain ID
            anchmers[d_tid].chain_id[i] = anchmers[d_tid].chain_id[i] == 0 ? ++current_chain : anchmers[d_tid].chain_id[i];

            std::size_t j = i + 1;
            while (j < anchmer_size && j + first_anchor_index < n_anchors)
            {
                if (d_anchors[global_anchor_index] == d_anchors[first_anchor_index + j])
                {
                    anchmers[d_tid].chain_id[j] = anchmers[d_tid].chain_id[i];
                }
                ++j;
            }
        }
    }
}

__global__ void decrementer_kernel(std::size_t* vals, const std::size_t n_vals)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_vals)
    {
        vals[d_tid] = vals[d_tid] - 1;
    }
}

__device__ __forceinline__ bool overlaps_mergable(const Overlap& a, const Overlap& b, std::int32_t max_dist)
{
    const bool q_adjacent = abs(int(b.query_start_position_in_read_) - int(a.query_end_position_in_read_)) <= max_dist;
    const bool t_adjacent = abs(int(a.target_end_position_in_read_) - int(b.target_start_position_in_read_)) <= max_dist || abs(int(a.target_start_position_in_read_) - int(b.target_end_position_in_read_)) <= max_dist;
    return a.query_read_id_ == b.query_read_id_ &&
           a.target_read_id_ == b.target_read_id_ &&
           a.relative_strand == b.relative_strand &&
           t_adjacent && q_adjacent;
}

__global__ void chain_overlaps_in_window(Overlap* overlaps,
                                         const std::int32_t n_overlaps,
                                         const std::int32_t n_overlapmers,
                                         bool* overlap_mask,
                                         const std::int32_t overlapmer_size, const std::int32_t max_dist)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlapmers)
    {
        std::int32_t first_overlap_index = d_tid * overlapmer_size;
        for (std::size_t i = 0; i < overlapmer_size && i < n_overlaps; ++i)
        {
            std::size_t global_overlap_index   = first_overlap_index + i;
            overlap_mask[global_overlap_index] = true;
            if (global_overlap_index < n_overlaps)
            {

                std::size_t j = i + 1;
                while (j < overlapmer_size && j + first_overlap_index < n_overlaps)
                {
                    // if (overlaps[global_overlap_index] == overlaps[first_overlap_index + j])
                    if (overlaps_mergable(overlaps[global_overlap_index], overlaps[first_overlap_index + j], max_dist))
                    {
                        overlaps[first_overlap_index + j]     = merge_helper(overlaps[first_overlap_index + j], overlaps[first_overlap_index + i]);
                        overlap_mask[first_overlap_index + i] = false;
                    }
                    ++j;
                }
            }
        }
    }
}

// void chain_overlaps(Overlap* overlaps,
//                     bool* overlap_mask,
//                     Overlap* chains,
//                     std::size_t* d_num_filtered_overlaps,
//                     std::int32_t n_overlaps,
//                     const std::int32_t overlapmer_size,
//                     std::int32_t max_dist)
// {
//     std::int32_t num_overlapmers = (n_overlaps / overlapmer_size) + 1;
//     chain_overlaps_in_window<<<(n_chains / overlapmer_window_size) + 1, block_size, 0, _cuda_stream>>>(overlaps.data(),
//                                                                                                        n_overlaps,
//                                                                                                        (n_overlaps / overlapmer_size) + 1,
//                                                                                                        overlap_mask.data(),
//                                                                                                        overlapmer_size, max_dist);

//     void* d_temp_storage      = nullptr;
//     size_t temp_storage_bytes = 0;
//     hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, overlaps.data(),
//                                d_initial_overlap_mask.data(),
//                                d_filtered_overlaps.data(),
//                                d_num_filtered_overlaps,
//                                n_chains,
//                                _cuda_stream);
//     d_temp_buf.clear_and_resize(temp_storage_bytes);
//     d_temp_storage = d_temp_buf.data();
//     hipcub::DeviceSelect::Flagged(d_temp_storage,
//                                temp_storage_bytes,
//                                d_fused_chains.data(),
//                                d_initial_overlap_mask.data(),
//                                d_filtered_overlaps.data(),
//                                d_num_filtered_overlaps,
//                                n_chains,
//                                _cuda_stream);
// }

void OverlapperAnchmer::get_overlaps(std::vector<Overlap>& fused_overlaps,
                                     const device_buffer<Anchor>& d_anchors,
                                     bool all_to_all,
                                     int64_t min_residues,
                                     int64_t min_overlap_len,
                                     int64_t min_bases_per_residue,
                                     float min_overlap_fraction)
{

    // const std::int32_t anchmer_generation_rounds = 1;
    // const std::int32_t chain_filter_min_anchors  = 2;
    // const std::int32_t anchor_merge_min_dist     = 150;
    const std::int32_t anchors_per_anchmer    = MAX_ANCHMER_WINDOW;
    const std::int32_t overlapmer_window_size = MAX_OVERLAPMER_WINDOW;
    std::size_t n_anchors                     = d_anchors.size();
    std::size_t n_anchmers                    = (d_anchors.size() / anchors_per_anchmer) + 1;
    std::int32_t block_size                   = 32;

    //std::vector<Anchmer> anchmers(n_anchmers);
    device_buffer<Anchmer> d_anchmers(n_anchmers, _allocator, _cuda_stream);

    // Stage one: generate anchmers
    generate_anchmers<<<(n_anchmers / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchors.data(), n_anchors, d_anchmers.data(), anchors_per_anchmer);

#ifdef DEBUG
    cudautils::device_copy_n(d_anchmers.data(), d_anchmers.size(), anchmers.data(), _cuda_stream);

    for (auto a : anchmers)
    {
        std::cout << a.n_anchors << " " << static_cast<int16_t>(a.n_chains) << std::endl;
        for (std::size_t i = 0; i < a.n_anchors; ++i)
        {
            std::cout << static_cast<int16_t>(a.chain_id[i]) << " ";
        }
        std::cout << std::endl;
    }
#endif

    // Stage 2: Given a buffer of anchmers, generate overlaps within each anchmer.
    // Anchmers may contain between 1 and anchors_per_anchmer overlaps

    // Calculate the number of overlaps needed for the initial generation.
    // This is equal to the sum of each anchmer's n_chains value.
    // Transform each anchmer's n_chains value into a device vector so we can calculate a prefix
    // sum (which will give us the mapping between anchmer -> index in overlaps array)
    AnchmerCountChainsOp anchmer_chain_count_op;
    hipcub::TransformInputIterator<int32_t, AnchmerCountChainsOp, Anchmer*> d_chain_counts(d_anchmers.data(), anchmer_chain_count_op);

    device_buffer<int32_t> d_overlap_ends(n_anchmers, _allocator, _cuda_stream);

    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_chain_counts,
                                  d_overlap_ends.data(),
                                  n_anchmers,
                                  _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_chain_counts,
                                  d_overlap_ends.data(),
                                  n_anchmers,
                                  _cuda_stream);

    // Holds the last prefix sum in the overlap_ends vector.
    // This value is the total number of overlaps
    int32_t n_initial_overlaps = cudautils::get_value_from_device(d_overlap_ends.data() + n_anchmers - 1, _cuda_stream);

    std::cerr << "Generating " << n_initial_overlaps << " initial overlaps from " << n_anchmers << " anchmers..." << std::endl;

    // Initialize overlaps to hold default values
    device_buffer<Overlap> d_initial_overlaps(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<Overlap> d_filtered_overlaps(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<bool> d_initial_overlap_mask(n_initial_overlaps, _allocator, _cuda_stream);

    initialize_overlaps_array<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_initial_overlaps.data(), n_initial_overlaps);

    // Generate overlaps within each anchmer, filling the overlaps buffer
    anchmers_to_overlaps<<<(n_anchmers / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchmers.data(), d_overlap_ends.data(), n_anchmers,
                                                                                         d_anchors.data(), n_anchors, d_initial_overlaps.data(), n_initial_overlaps);

    /** Initial overlaps have now been generated **/
    device_buffer<Overlap> d_init_chains(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<int32_t> d_init_starts(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<int32_t> d_init_chain_lengths(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<int32_t> d_init_ends(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<std::size_t> d_init_num_chains(1, _allocator, _cuda_stream);
    // Perform a round of RLE on the overlaps
    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       d_initial_overlaps.data(),
                                       d_init_chains.data(),
                                       d_init_chain_lengths.data(),
                                       d_init_num_chains.data(),
                                       n_initial_overlaps);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       d_initial_overlaps.data(),
                                       d_init_chains.data(),
                                       d_init_chain_lengths.data(),
                                       d_init_num_chains.data(),
                                       n_initial_overlaps);
    std::size_t n_chains = cudautils::get_value_from_device(d_init_num_chains.data(), _cuda_stream);
    d_temp_storage       = nullptr;
    temp_storage_bytes   = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_init_chain_lengths.data(), d_init_starts.data(),
                                  n_chains, _cuda_stream);
    // allocate temporary storage
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_init_chain_lengths.data(), d_init_starts.data(),
                                  n_chains, _cuda_stream);

    convert_offsets_to_ends<<<(n_chains / block_size) + 1, block_size, 0, _cuda_stream>>>(d_init_starts.data(),
                                                                                          d_init_chain_lengths.data(),
                                                                                          d_init_ends.data(), n_chains);
    d_initial_overlap_mask.clear_and_resize(n_chains);
    device_buffer<Overlap> d_fused_chains(n_chains, _allocator, _cuda_stream);
    merge_overlap_runs<<<(n_chains / block_size) + 1, block_size, 0, _cuda_stream>>>(d_initial_overlaps.data(), d_init_starts.data(), d_init_ends.data(), n_chains, d_fused_chains.data());

    std::cerr << "Merged " << n_chains << " initial chains." << std::endl;
    d_initial_overlap_mask.clear_and_resize(n_chains);
    /*** First round of local overlap chaining ***/
    chain_overlaps_in_window<<<(n_chains / overlapmer_window_size) + 1, block_size, 0, _cuda_stream>>>(d_fused_chains.data(),
                                                                                                       n_chains,
                                                                                                       (n_chains / overlapmer_window_size) + 1,
                                                                                                       d_initial_overlap_mask.data(),
                                                                                                       overlapmer_window_size, 150);
    std::cerr << "First round of superchaining complete." << std::endl;

    // // Remove overlaps that were masked during chaining
    device_buffer<size_t> d_num_filtered_overlaps(1, _allocator, _cuda_stream);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_filtered_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_chains,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage,
                               temp_storage_bytes,
                               d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_filtered_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_chains,
                               _cuda_stream);

    std::size_t n_filtered_overlaps = cudautils::get_value_from_device(d_num_filtered_overlaps.data(), _cuda_stream);
    std::cerr << n_filtered_overlaps << " remain after initial chaining round." << std::endl;

    d_initial_overlap_mask.clear_and_resize(n_filtered_overlaps);
    /*** Second round of local overlap chaining ***/
    chain_overlaps_in_window<<<(n_chains / overlapmer_window_size) + 1, block_size, 0, _cuda_stream>>>(d_filtered_overlaps.data(),
                                                                                                       n_filtered_overlaps,
                                                                                                       (n_filtered_overlaps / overlapmer_window_size) + 1,
                                                                                                       d_initial_overlap_mask.data(),
                                                                                                       overlapmer_window_size, 1000);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_filtered_overlaps.data(),
                               d_initial_overlap_mask.data(),
                               d_fused_chains.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_filtered_overlaps.data(),
                               d_initial_overlap_mask.data(),
                               d_fused_chains.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    n_filtered_overlaps = cudautils::get_value_from_device(d_num_filtered_overlaps.data(), _cuda_stream);
    std::cerr << n_filtered_overlaps << " remain after second chaining round." << std::endl;

    d_initial_overlap_mask.clear_and_resize(n_filtered_overlaps);
    mask_overlaps<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_fused_chains.data(), n_filtered_overlaps, d_initial_overlap_mask.data(), 10, 2, 0, all_to_all, false);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_filtered_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_filtered_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    n_filtered_overlaps = cudautils::get_value_from_device(d_num_filtered_overlaps.data(), _cuda_stream);
    std::cerr << n_filtered_overlaps << " remain after post-secondary filtering." << std::endl;

    /*** Tertiary chaining ***/
    d_initial_overlap_mask.clear_and_resize(n_filtered_overlaps);
    d_fused_chains.clear_and_resize(n_filtered_overlaps);
    chain_overlaps_in_window<<<(n_chains / overlapmer_window_size) + 1, block_size, 0, _cuda_stream>>>(d_filtered_overlaps.data(),
                                                                                                       n_filtered_overlaps,
                                                                                                       (n_filtered_overlaps / overlapmer_window_size) + 1,
                                                                                                       d_initial_overlap_mask.data(),
                                                                                                       overlapmer_window_size, 5000);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_filtered_overlaps.data(),
                               d_initial_overlap_mask.data(),
                               d_fused_chains.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_filtered_overlaps.data(),
                               d_initial_overlap_mask.data(),
                               d_fused_chains.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    n_filtered_overlaps = cudautils::get_value_from_device(d_num_filtered_overlaps.data(), _cuda_stream);
    std::cerr << "After long-chaining, " << n_filtered_overlaps << " remain." << std::endl;

    device_buffer<Overlap> d_final_overlaps(n_filtered_overlaps, _allocator, _cuda_stream);
    d_initial_overlap_mask.clear_and_resize(n_filtered_overlaps);
    mask_overlaps<<<(n_filtered_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_fused_chains.data(), n_chains, d_initial_overlap_mask.data(), min_overlap_len, min_residues, min_bases_per_residue, all_to_all, true);
    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_final_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
                               d_initial_overlap_mask.data(),
                               d_final_overlaps.data(),
                               d_num_filtered_overlaps.data(),
                               n_filtered_overlaps,
                               _cuda_stream);
    n_filtered_overlaps = cudautils::get_value_from_device(d_num_filtered_overlaps.data(), _cuda_stream);
    std::cerr << "After final filtering, " << n_filtered_overlaps << " remain." << std::endl;

    fused_overlaps.resize(n_filtered_overlaps);
    cudautils::device_copy_n(d_final_overlaps.data(), n_filtered_overlaps, fused_overlaps.data(), _cuda_stream);

    // d_temp_storage     = nullptr;
    // temp_storage_bytes = 0;
    // hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
    //                               d_chain_lengths.data(), d_chain_starts.data(),
    //                               n_chains, _cuda_stream);

    // // allocate temporary storage
    // d_temp_buf.clear_and_resize(temp_storage_bytes);
    // d_temp_storage = d_temp_buf.data();

    // hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
    //                               d_chain_lengths.data(), d_chain_starts.data(),
    //                               n_chains, _cuda_stream);

    // device_buffer<Overlap> d_fused_chains(n_chains, _allocator, _cuda_stream);
    // device_buffer<std::size_t> d_chain_ends(n_overlap_runs, _allocator, _cuda_stream);
    // convert_offsets_to_ends<<<(n_overlap_runs / block_size) + 1, block_size, 0, _cuda_stream>>>(d_chain_starts.data(), d_chain_lengths.data(), d_chain_ends.data(), n_overlap_runs);
    // merge_overlap_runs<<<(n_chains) + 1, block_size, 0, _cuda_stream>>>(d_fused_overlaps.data(), d_chain_starts.data(), d_chain_ends.data(), n_chains, d_fused_chains.data());

    // d_initial_overlap_mask.clear_and_resize(n_chains);
    // mask_overlaps<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_fused_chains.data(), n_chains, d_initial_overlap_mask.data(), min_overlap_len, min_residues, min_bases_per_residue, all_to_all, true);

    // device_buffer<Overlap> d_final_overlaps(n_chains, _allocator, _cuda_stream);
    // device_buffer<size_t> d_num_final_overlaps(1, _allocator, _cuda_stream);

    // d_temp_storage     = nullptr;
    // temp_storage_bytes = 0;
    // hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
    //                            d_initial_overlap_mask.data(),
    //                            d_final_overlaps.data(),
    //                            d_num_final_overlaps.data(),
    //                            n_chains,
    //                            _cuda_stream);
    // d_temp_buf.clear_and_resize(temp_storage_bytes);
    // d_temp_storage = d_temp_buf.data();
    // hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_fused_chains.data(),
    //                            d_initial_overlap_mask.data(),
    //                            d_final_overlaps.data(),
    //                            d_num_final_overlaps.data(),
    //                            n_chains,
    //                            _cuda_stream);
    // std::size_t n_final_overlaps = cudautils::get_value_from_device(d_num_final_overlaps.data(), _cuda_stream);
    // std::cerr << "Produced " << n_final_overlaps << " final overlaps; " << n_overlap_runs - n_final_overlaps << " overlaps removed by filtering." << std::endl;

    // This is not completely necessary, but if removed one has to make sure that the next step
    // uses the same stream or that sync is done in caller
    GW_CU_CHECK_ERR(hipStreamSynchronize(_cuda_stream));
}

OverlapperAnchmer::OverlapperAnchmer(DefaultDeviceAllocator allocator,
                                     const hipStream_t hip_stream)
    : _allocator(allocator)
    , _cuda_stream(hip_stream)
{
}

} // namespace cudamapper
} // namespace genomeworks
} // namespace claraparabricks
