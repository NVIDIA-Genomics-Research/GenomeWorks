#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "overlapper_anchmer.hpp"

#include <fstream>
#include <cstdlib>

// Needed for accumulate - remove when ported to cuda
#include <numeric>
#include <limits>

#include <hipcub/hipcub.hpp>
#include <thrust/execution_policy.h>

#include <claraparabricks/genomeworks/utils/cudautils.hpp>

#ifndef NDEBUG // only needed to check if input is sorted in assert
#include <algorithm>
#include <thrust/host_vector.h>
#endif

//
//          36a32532-4135-4ffe-a346-0b1b08c1b747   6370    173     683     -       fedde900-1485-42a2-8adb-b7a30dcf82fe    10014   91      620     105     529     0       minimap2        not_in_cm
//         da4230aa-e79e-4a5f-9738-c272aad98a82   8554    74      293     +       ebb4ceeb-76f1-454a-84d9-a65cfcb3a9fa    1824    1572    1801    114     231     0       minimap2        not_in_cm

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

#define MAX_ANCHMER_WINDOW 10
#define MAX_OVERLAPMER_WINDOW 10
#define INT32_INFINITY 1000000

struct Anchmer
{
    std::int8_t n_chained_anchors[MAX_ANCHMER_WINDOW] = {0};
    std::int8_t chain_id[MAX_ANCHMER_WINDOW]          = {0};
    std::int8_t n_chains                              = 0;
    std::int32_t n_anchors                            = 0;
};

struct Overlapmer
{
    std::int8_t n_chained_anchors[MAX_OVERLAPMER_WINDOW] = {0};
    std::int8_t chain_id[MAX_OVERLAPMER_WINDOW]          = {0};
    std::int8_t n_chains                                 = 0;
    std::int8_t n_overlaps                               = 0;
};

struct ChainPiece
{
    Overlap overlap;
    __device__ ChainPiece() {}
};

struct OverlapToChainPieceOp
{

    __device__ __forceinline__ ChainPiece operator()(const Overlap& a) const
    {
        ChainPiece c;
        c.overlap = a;
        return c;
    }
};

struct QueryTargetPair
{
    Overlap overlap;
    __device__ QueryTargetPair() {}
};

struct OverlapToQueryTargetPairOp
{
    __device__ __forceinline__ QueryTargetPair operator()(const Overlap& a) const
    {
        QueryTargetPair p;
        p.overlap = a;
        return p;
    }
};

struct AnchorToPosStruct
{

    __host__ __device__ __forceinline__ std::int32_t operator()(const Anchor* anchor) const
    {
        return static_cast<int32_t>(anchor->query_position_in_read_);
    }

    __host__ __device__ __forceinline__ std::int32_t operator()(const Anchor& anchor) const
    {
        return static_cast<int32_t>(anchor.query_position_in_read_);
    }
};

__device__ bool operator==(const QueryTargetPair& a, const QueryTargetPair& b)
{
    return a.overlap.query_read_id_ == b.overlap.query_read_id_ && a.overlap.target_read_id_ == b.overlap.target_read_id_;
}

__device__ bool
operator==(const ChainPiece& a, const ChainPiece& b)
{
    const bool q_adjacent = abs(int(b.overlap.query_start_position_in_read_) - int(a.overlap.query_end_position_in_read_)) < 5000;
    const bool t_adjacent = a.overlap.relative_strand == RelativeStrand::Forward ? abs(int(a.overlap.target_end_position_in_read_) - int(b.overlap.target_start_position_in_read_)) < 5000 : abs(int(b.overlap.target_start_position_in_read_) - int(a.overlap.target_end_position_in_read_)) < 5000;
    return a.overlap.query_read_id_ == b.overlap.query_read_id_ &&
           a.overlap.target_read_id_ == b.overlap.target_read_id_ &&
           a.overlap.relative_strand == b.overlap.relative_strand &&
           t_adjacent && q_adjacent;
}
__device__ bool operator==(const Overlap& a,
                           const Overlap& b)
{
    bool same_strand   = a.relative_strand == b.relative_strand;
    bool identical_ids = a.query_read_id_ == b.query_read_id_ && a.target_read_id_ == b.target_read_id_;
    // bool q_ends_overlap;
    // bool t_end_overlap;
    position_in_read_t q_gap = abs((int)b.query_start_position_in_read_ - (int)a.query_end_position_in_read_);
    position_in_read_t t_gap = abs((int)b.target_start_position_in_read_ - (int)a.target_end_position_in_read_);
    bool gap_match           = q_gap < 150 && t_gap < 150;
    bool gap_ratio_okay      = float(min(q_gap, t_gap) / max(q_gap, t_gap)) < 0.8;

    //return identical_ids && same_strand && (gap_match || gap_ratio_okay);

    return identical_ids && same_strand && (gap_match);
}

__device__ bool operator==(const Anchor& lhs,
                           const Anchor& rhs)
{
    auto score_threshold = 1;

    // Very simple scoring function to quantify quality of overlaps.
    auto score = 1;

    if (abs(int(rhs.query_position_in_read_) - int(lhs.query_position_in_read_)) <= 50 and
        abs(int(rhs.target_position_in_read_) - int(lhs.target_position_in_read_)) <= 50)
        score = 2;
    if (lhs.query_position_in_read_ == rhs.query_position_in_read_)
        score = 0;
    return ((lhs.query_read_id_ == rhs.query_read_id_) &&
            (lhs.target_read_id_ == rhs.target_read_id_) &&
            score > score_threshold);
}

struct AnchmerCountChainsOp
{

    AnchmerCountChainsOp()
    {
    }

    __host__ __device__ __forceinline__ std::int32_t operator()(const Anchmer& a) const
    {
        return static_cast<int32_t>(a.n_chains);
    }
};

struct MergeOverlapRunOp
{
    __device__ __forceinline__ Overlap operator()(const Overlap& a, const Overlap& b)
    {
        Overlap c;
        c.query_read_id_                 = a.query_read_id_;
        c.target_read_id_                = a.target_read_id_;
        c.relative_strand                = b.relative_strand;
        c.query_start_position_in_read_  = min(a.query_start_position_in_read_, b.query_start_position_in_read_);
        c.query_end_position_in_read_    = max(a.query_end_position_in_read_, b.query_end_position_in_read_);
        c.target_start_position_in_read_ = min(a.target_start_position_in_read_, b.target_start_position_in_read_);
        c.target_end_position_in_read_   = max(a.target_end_position_in_read_, b.target_end_position_in_read_);
        c.num_residues_                  = a.num_residues_ + b.num_residues_;
        return c;
    }
};

struct MergeChainPiecesOp
{
    __device__ __forceinline__ ChainPiece operator()(const ChainPiece& a, const ChainPiece& b)
    {
        Overlap c;
        c.query_read_id_                 = a.overlap.query_read_id_;
        c.target_read_id_                = a.overlap.target_read_id_;
        c.relative_strand                = a.overlap.relative_strand;
        c.query_start_position_in_read_  = min(a.overlap.query_start_position_in_read_, b.overlap.query_start_position_in_read_);
        c.query_end_position_in_read_    = max(a.overlap.query_end_position_in_read_, b.overlap.query_end_position_in_read_);
        c.target_start_position_in_read_ = min(a.overlap.target_start_position_in_read_, b.overlap.target_start_position_in_read_);
        c.target_end_position_in_read_   = max(a.overlap.target_end_position_in_read_, b.overlap.target_end_position_in_read_);
        c.num_residues_                  = a.overlap.num_residues_ + b.overlap.num_residues_;
        ChainPiece c_p;
        c_p.overlap = c;
        return c_p;
    }
};

struct DecrementerOp
{
    __host__ __device__ __forceinline__ std::size_t operator()(const std::size_t& val)
    {
        return val - 1;
    }
};

__global__ void mask_overlaps(Overlap* overlaps, std::size_t n_overlaps, bool* mask,
                              const std::size_t min_overlap_length,
                              const std::size_t min_residues,
                              const std::size_t min_bases_per_residue,
                              const bool all_to_all,
                              const bool filter_self_mappings)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        position_in_read_t overlap_query_length  = overlaps[d_tid].query_end_position_in_read_ - overlaps[d_tid].query_start_position_in_read_;
        position_in_read_t overlap_target_length = overlaps[d_tid].target_end_position_in_read_ - overlaps[d_tid].target_start_position_in_read_;
        const bool mask_self_self                = overlaps[d_tid].query_read_id_ == overlaps[d_tid].target_read_id_ && all_to_all && filter_self_mappings;
        //auto query_bases_per_residue             = overlap_query_length / overlaps[d_tid].num_residues_;
        //auto target_bases_per_residue            = overlap_target_length / overlaps[d_tid].num_residues_;
        mask[d_tid] &= overlap_query_length >= min_overlap_length & overlap_target_length >= min_overlap_length;
        mask[d_tid] &= overlaps[d_tid].num_residues_ >= min_residues;
        mask[d_tid] &= !mask_self_self;
        //mask[d_tid] &= (query_bases_per_residue < min_bases_per_residue || target_bases_per_residue < min_bases_per_residue);
    }
}

__global__ void finalize_overlaps(Overlap* overlaps, const std::size_t n_overlaps)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        if (overlaps[d_tid].target_start_position_in_read_ > overlaps[d_tid].target_end_position_in_read_)
        {
            overlaps[d_tid].relative_strand                = RelativeStrand::Reverse;
            auto tmp                                       = overlaps[d_tid].target_start_position_in_read_;
            overlaps[d_tid].target_start_position_in_read_ = overlaps[d_tid].target_end_position_in_read_;
            overlaps[d_tid].target_end_position_in_read_   = tmp;
        }
    }
};

__global__ void convert_offsets_to_ends(std::int32_t* starts, std::int32_t* lengths, std::int32_t* ends, std::int32_t n_starts)
{
    std::int32_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_starts)
    {
        ends[d_tid] = starts[d_tid] + lengths[d_tid] - 1;
    }
}

__device__ __forceinline__ Overlap merge_helper(Overlap& a, Overlap& b)
{
    Overlap c;
    c.query_read_id_                 = a.query_read_id_;
    c.target_read_id_                = a.target_read_id_;
    c.relative_strand                = a.num_residues_ > b.num_residues_ ? a.relative_strand : b.relative_strand;
    c.query_start_position_in_read_  = min(a.query_start_position_in_read_, b.query_start_position_in_read_);
    c.query_end_position_in_read_    = max(a.query_end_position_in_read_, b.query_end_position_in_read_);
    c.target_start_position_in_read_ = min(a.target_start_position_in_read_, b.target_start_position_in_read_);
    c.target_end_position_in_read_   = max(a.target_end_position_in_read_, b.target_end_position_in_read_);
    c.num_residues_                  = a.num_residues_ + b.num_residues_;
    return c;
}

__global__ void merge_overlap_runs(Overlap* overlaps,
                                   std::int32_t* starts, std::int32_t* ends, std::size_t n_runs,
                                   Overlap* fused_overlaps)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_runs)
    {
        fused_overlaps[d_tid] = overlaps[starts[d_tid]];
        if (starts[d_tid] != ends[d_tid])
        {
            for (std::size_t i = starts[d_tid] + 1; i < ends[d_tid]; ++i)
            {
                fused_overlaps[d_tid] = merge_helper(fused_overlaps[d_tid], overlaps[i]);
            }
        }
    }
}

__device__ __forceinline__ bool point_contained(const position_in_read_t point, const position_in_read_t interval_start, const position_in_read_t interval_end)
{
    return point >= interval_start && point <= interval_end;
}

__device__ __forceinline__ position_in_read_t abs_diff_position(const position_in_read_t first, const position_in_read_t second)
{
    return abs(int(second) - int(first));
}

__device__ __forceinline__ bool point_adjacent(position_in_read_t point, position_in_read_t interval_start, position_in_read_t interval_end, position_in_read_t max_dist)
{
    return abs_diff_position(point, interval_start) < max_dist || abs_diff_position(point, interval_end) < max_dist;
}

__device__ __forceinline__ bool gaps_match(const Overlap& a, const Overlap& b, const double threshold)
{
    position_in_read_t q_gap = abs((int)b.query_start_position_in_read_ - (int)a.query_end_position_in_read_);
    position_in_read_t t_gap = abs((int)b.target_start_position_in_read_ - (int)a.target_end_position_in_read_);

    return min(q_gap, t_gap) / max(q_gap, t_gap) < threshold;
}

__device__ __forceinline__ bool exhaustive_overlap_compare(const Overlap& a, const Overlap& b)
{

    const bool same_strand = a.relative_strand == b.relative_strand;
    const bool query_same  = a.query_read_id_ == b.query_read_id_;
    const bool target_same = a.target_read_id_ == b.target_read_id_;

    const bool query_contained  = point_contained(b.query_start_position_in_read_, a.query_start_position_in_read_, a.query_end_position_in_read_) || point_contained(b.query_end_position_in_read_, a.query_start_position_in_read_, a.query_end_position_in_read_);
    const bool target_contained = point_contained(b.target_start_position_in_read_, a.target_start_position_in_read_, a.target_end_position_in_read_) || point_contained(b.target_end_position_in_read_, a.target_start_position_in_read_, a.target_end_position_in_read_);
    const bool query_adjacent   = point_adjacent(b.query_start_position_in_read_, a.query_start_position_in_read_, a.query_end_position_in_read_, 300) || point_adjacent(b.query_end_position_in_read_, a.query_start_position_in_read_, a.query_end_position_in_read_, 300);
    const bool target_adjacent  = point_adjacent(b.target_start_position_in_read_, a.target_start_position_in_read_, a.target_end_position_in_read_, 300) || point_adjacent(b.target_start_position_in_read_, a.target_start_position_in_read_, a.target_end_position_in_read_, 300);

    const bool query_target_gaps_match = gaps_match(a, b, 0.8);

    const bool positions_mergable = query_target_gaps_match || ((query_contained || query_adjacent) && (target_contained || target_adjacent));

    return query_same && target_same && same_strand && positions_mergable;
}

__global__ void merge_overlaps_in_query_target_pairs(Overlap* overlaps, std::int32_t* starts, std::int32_t* ends, bool* mask, const std::size_t n_qt_runs)
{
    std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_qt_runs)
    {
        if (starts[d_tid] != ends[d_tid])
        {
            for (std::int32_t ind = starts[d_tid]; ind < ends[d_tid]; ++ind)
            {
                mask[ind] = true;
            }
            for (std::int32_t i = starts[d_tid]; i < ends[d_tid]; ++i)
            {
                std::int32_t j = i + 1;
                while (j < ends[d_tid])
                {
                    if (mask[j] && exhaustive_overlap_compare(overlaps[i], overlaps[j]))
                    {
                        overlaps[i] = merge_helper(overlaps[i], overlaps[j]);
                        mask[j]     = false;
                    }
                    ++j;
                }
            }
        }
    }
}

__device__ __forceinline__ void init_overlap(Overlap& overlap)
{
    overlap.query_start_position_in_read_  = 4294967295;
    overlap.query_end_position_in_read_    = 0;
    overlap.target_start_position_in_read_ = 4294967295;
    overlap.target_end_position_in_read_   = 0;
    overlap.relative_strand                = RelativeStrand::Forward;
    overlap.num_residues_                  = 0;
}

__global__ void mask_anchor_repeat_runs(const Anchor* anchors,
                                        bool* anchor_mask,
                                        int32_t* run_starts,
                                        int32_t* run_lengths,
                                        const int32_t n_anchors,
                                        const int32_t n_runs,
                                        const int32_t min_repeat_length)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_runs)
    {
        if (run_lengths[d_tid] > min_repeat_length)
        {
            for (int32_t i = run_starts[d_tid]; i < run_starts[d_tid] + run_lengths[d_tid]; ++i)
            {
                anchor_mask[i] = false;
            }
        }
    }
}

void mask_repetitive_anchors_by_RLE(const device_buffer<Anchor>& d_anchors,
                                    device_buffer<bool>& d_mask,
                                    const size_t n_anchors,
                                    const int32_t max_placements,
                                    DefaultDeviceAllocator& _allocator,
                                    hipStream_t& _cuda_stream,
                                    int32_t block_size = 32)
{
    AnchorToPosStruct anchor_to_pos_op;
    hipcub::TransformInputIterator<int32_t, AnchorToPosStruct, const Anchor*> anchor_q_pos(d_anchors.data(), anchor_to_pos_op);

    device_buffer<int32_t> d_query_start_runs(n_anchors, _allocator, _cuda_stream);
    device_buffer<int32_t> d_query_start_run_lengths(n_anchors, _allocator, _cuda_stream);
    device_buffer<int32_t> d_query_start_run_starts(n_anchors, _allocator, _cuda_stream);
    device_buffer<int32_t> d_num_runs(1, _allocator, _cuda_stream);

    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       anchor_q_pos,
                                       d_query_start_runs.data(),
                                       d_query_start_run_lengths.data(),
                                       d_num_runs.data(),
                                       n_anchors);

    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       anchor_q_pos,
                                       d_query_start_runs.data(),
                                       d_query_start_run_lengths.data(),
                                       d_num_runs.data(),
                                       n_anchors);

    std::size_t n_runs = cudautils::get_value_from_device(d_num_runs.data(), _cuda_stream);
    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_query_start_run_lengths.data(),
                                  d_query_start_run_starts.data(),
                                  n_runs,
                                  _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_query_start_run_lengths.data(),
                                  d_query_start_run_starts.data(),
                                  n_runs,
                                  _cuda_stream);

    mask_anchor_repeat_runs<<<(n_runs / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchors.data(),
                                                                                        d_mask.data(),
                                                                                        d_query_start_run_starts.data(),
                                                                                        d_query_start_run_lengths.data(),
                                                                                        n_anchors,
                                                                                        n_runs,
                                                                                        max_placements);
}

__global__ void mask_repetitive_anchors(const Anchor* anchors,
                                        bool* anchor_mask,
                                        const int32_t n_anchors,
                                        const int32_t window_size,
                                        const int32_t max_acceptable_anchor_run)
{
    // thread ID, which is used to index into the Anchors array
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_anchors)
    {
        // First index within the anchors array
        std::size_t anchor_index_start = d_tid * window_size;
        std::size_t anchor_index_end   = min(int(anchor_index_start + window_size), int(n_anchors));
        /**
        */
        int32_t max_run_length     = 0;
        int32_t current_run_length = 1;
        int32_t max_run_end        = 0;
        for (std::size_t i = anchor_index_start + 1; i < anchor_index_end; ++i)
        {
            anchor_mask[i - 1] = true;
            anchor_mask[i]     = true;
            if (anchors[i].query_position_in_read_ == anchors[i - 1].query_position_in_read_)
            {
                current_run_length += 1;
                max_run_length = max(max_run_length, current_run_length);
            }

            else
            {
                max_run_end        = current_run_length == max_run_length ? i : max_run_end;
                current_run_length = 0;
            }
        }
        if (max_run_length > max_acceptable_anchor_run)
        {
            for (int32_t i = max_run_end; i >= max_run_end - max_run_length; --i)
            {
                anchor_mask[i] = false;
            }
        }
    }
}

__global__ void initalize_anchors_mask(bool* anchors_mask, const size_t n_anchors, bool val)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_anchors)
    {
        anchors_mask[d_tid] = val;
    }
}

__global__ void
initialize_overlaps_array(Overlap* overlaps, const size_t n_overlaps)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        init_overlap(overlaps[d_tid]);
    }
}

__global__ void init_overlap_scores(const Overlap* overlaps, double* scores, const int32_t n_overlaps, const double exp)

{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        scores[d_tid] = pow(double(overlaps[d_tid].num_residues_), exp);
    }
}

__global__ void init_overlap_mask(bool* mask, const int32_t n_overlaps, const bool value)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        mask[d_tid] = value;
    }
}

__device__ __forceinline__ void add_anchor_to_overlap(const Anchor& anchor, Overlap& overlap)
{
    overlap.query_read_id_                = anchor.query_read_id_;
    overlap.target_read_id_               = anchor.target_read_id_;
    overlap.query_start_position_in_read_ = min(anchor.query_position_in_read_, overlap.query_start_position_in_read_);
    overlap.query_end_position_in_read_   = max(anchor.query_position_in_read_, overlap.query_end_position_in_read_);

    // Handles whether the match is on the forward or reverse strand.
    // Requires anchors to be sorted in order
    // First by query_read_id, then target_read_id, then query_pos, then target_pos.
    // If these are sorted, adding an anchor to an overlap with at least one
    // anchor in it already will indicate whether the overlap is increasing on its start or end.
    // If the anchor falls before the target_start, we are on the reverse strand and we should use it to extend
    // the target_end_.
    // If the anchor falls after the target_start, we should extend the target_start
    // Since the anchors will be monotonically increasing or monotonically decreasing, this
    // should consistently head in the correct direction.

    if (overlap.num_residues_ == 0)
    {
        overlap.target_start_position_in_read_ = anchor.target_position_in_read_;
        overlap.target_end_position_in_read_   = anchor.target_position_in_read_;
    }
    else
    {
        bool on_forward_strand                 = anchor.target_position_in_read_ > overlap.target_start_position_in_read_;
        overlap.target_start_position_in_read_ = on_forward_strand ? overlap.target_start_position_in_read_ : anchor.target_position_in_read_;
        overlap.target_end_position_in_read_   = on_forward_strand ? anchor.target_position_in_read_ : overlap.target_end_position_in_read_;
        overlap.relative_strand                = on_forward_strand ? RelativeStrand::Forward : RelativeStrand::Reverse;
    }

    ++overlap.num_residues_;
}
__global__ void anchmers_to_overlaps(const Anchmer* anchmers,
                                     const int32_t* overlap_ends,
                                     const size_t n_anchmers,
                                     const Anchor* anchors,
                                     const bool* anchor_mask,
                                     const size_t n_anchors,
                                     Overlap* overlaps,
                                     bool* overlap_mask,
                                     const size_t n_overlaps)
{
    // thread ID, which is used to index into the anchmers array
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (d_tid < n_anchmers)
    {
        for (std::size_t i = 0; i < anchmers[d_tid].n_anchors; ++i)
        {
            //if (anchor_mask[i])
            if (true)
            {
                std::size_t overlap_index = overlap_ends[d_tid] - anchmers[d_tid].n_chains + anchmers[d_tid].chain_id[i];
                std::size_t anchor_index  = d_tid * MAX_ANCHMER_WINDOW + i;

                add_anchor_to_overlap(anchors[anchor_index], overlaps[overlap_index]);
            }
        }
    }
}

__global__ void
generate_anchmers(const Anchor* d_anchors,
                  const bool* d_anchor_select_mask,
                  const size_t n_anchors,
                  Anchmer* anchmers,
                  const uint8_t anchmer_size)
{

    // thread ID, which is used to index into the Anchors array
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;

    // First index within the anchors array for this Anchmer
    std::size_t first_anchor_index = d_tid * anchmer_size;

    // Initialize Anchmer fields
    anchmers[d_tid].n_anchors  = 0;
    anchmers[d_tid].n_chains   = 0;
    std::int32_t current_chain = 1;
    for (int i = 0; i < MAX_ANCHMER_WINDOW; ++i)
    {
        anchmers[d_tid].chain_id[i] = 0;
    }
    anchmers[d_tid].chain_id[0] = current_chain;
    anchmers[d_tid].n_chains    = 0;
    // end intialization

    /**
    * Iterate through the anchors within this thread's range (first_anchor_index -> first_anchor_index + anchmer_size (or the end of the Anchors array))
    * For each anchor
    *   if the anchor has not been chained to another anchor, create a new chain (by incrementing the chain ID) and increment the number of chains in the Anchmer
    *   
    */
    for (std::size_t i = 0; i < anchmer_size; ++i)
    {
        std::size_t global_anchor_index = first_anchor_index + i;
        std::size_t max_ind             = min(int(n_anchors - first_anchor_index), int(anchmer_size));
        if (global_anchor_index < n_anchors)
        {
            ++(anchmers[d_tid].n_anchors);
            if (!d_anchor_select_mask[global_anchor_index])
            {
                continue;
            }
            //anchmers[d_tid].n_chains = anchmers[d_tid].chain_id[i] == 0 ? anchmers[d_tid].n_chains + 1 : anchmers[d_tid].n_chains;
            //Label the anchor with its chain ID
            anchmers[d_tid].chain_id[i] = anchmers[d_tid].chain_id[i] == 0 ? ++current_chain : anchmers[d_tid].chain_id[i];
            anchmers[d_tid].n_chains    = max(anchmers[d_tid].chain_id[i], anchmers[d_tid].n_chains);
            for (std::size_t j = i + 1; j < max_ind; ++j)
            {
                if (d_anchors[global_anchor_index] == d_anchors[first_anchor_index + j])
                {
                    anchmers[d_tid].chain_id[j] = anchmers[d_tid].chain_id[i];
                }
            }
        }
    }
}

__global__ void decrementer_kernel(std::size_t* vals, const std::size_t n_vals)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_vals)
    {
        vals[d_tid] = vals[d_tid] - 1;
    }
}

__device__ __forceinline__ bool overlaps_mergable(const Overlap& a, const Overlap& b, std::int32_t max_dist, float min_gap_ratio)
{
    const int q_diff      = abs(int(b.query_start_position_in_read_) - int(a.query_end_position_in_read_));
    const int t_diff      = abs(int(a.target_end_position_in_read_) - int(b.target_start_position_in_read_));
    const bool q_adjacent = q_diff <= max_dist;
    const bool t_adjacent = t_diff <= max_dist;
    const float gap_ratio = float(min(t_diff, q_diff)) / float(max(t_diff, q_diff));
    return a.query_read_id_ == b.query_read_id_ &&
           a.target_read_id_ == b.target_read_id_ &&
           (a.relative_strand == b.relative_strand || min(a.num_residues_, b.num_residues_) == 1) &&
           ((t_adjacent && q_adjacent) || (gap_ratio > min_gap_ratio && t_diff < 5000 && q_diff < 5000));
    //(t_adjacent && q_adjacent);
}

__global__ void chain_overlaps_in_window(Overlap* overlaps,
                                         bool* overlap_mask,
                                         double* scores,
                                         const std::int32_t n_overlaps,
                                         const int32_t n_overlapmers,
                                         const std::int32_t overlapmer_size,
                                         const std::int32_t max_dist,
                                         const float max_gap_ratio)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlapmers)
    {
        std::int32_t first_overlap_index = d_tid * overlapmer_size;
        std::int32_t max_ind             = min(overlapmer_size, n_overlaps - first_overlap_index);
        for (std::size_t i = 0; i < overlapmer_size && i < n_overlaps; ++i)
        {
            std::size_t global_overlap_index = first_overlap_index + i;
            if (global_overlap_index < n_overlaps)
            {
                for (std::int32_t j = i + 1; j < max_ind; ++j)
                {
                    if (overlap_mask[first_overlap_index + j] && overlaps_mergable(overlaps[global_overlap_index], overlaps[first_overlap_index + j], max_dist, 0.8))
                    {
                        overlaps[first_overlap_index + i]     = merge_helper(overlaps[first_overlap_index + i], overlaps[first_overlap_index + j]);
                        overlap_mask[first_overlap_index + j] = false;
                        scores[first_overlap_index + i]       = scores[first_overlap_index + i] + scores[first_overlap_index + j];
                    }
                }
            }
        }
    }
}

__global__ void init_predecessor_and_score_arrays(int32_t* predecessors,
                                                  double* scores,
                                                  bool* mask,
                                                  int32_t n_overlaps)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        predecessors[d_tid] = d_tid;
    }
}

__device__ __forceinline__ int32_t exp_gap_cost(Overlap& a, Overlap& b)
{

    int32_t id_match_cost               = a.query_read_id_ == b.query_read_id_ && a.target_read_id_ == b.target_read_id_ ? 0 : INT32_INFINITY;
    int32_t q_diff                      = abs(int(b.query_start_position_in_read_) - int(a.query_end_position_in_read_));
    const bool relative_strands_reverse = (a.relative_strand == claraparabricks::genomeworks::cudamapper::RelativeStrand::Reverse) && (b.relative_strand == claraparabricks::genomeworks::cudamapper::RelativeStrand::Reverse);

    int32_t t_diff      = relative_strands_reverse ? abs(int(a.target_start_position_in_read_) - int(b.target_end_position_in_read_)) : abs(int(b.target_start_position_in_read_) - int(a.target_end_position_in_read_));
    int32_t strand_cost = a.relative_strand == b.relative_strand || a.num_residues_ == 1 || b.num_residues_ == 1 ? 0 : INT32_INFINITY;

    float gap_ratio = min(float(q_diff), float(t_diff)) / max(float(q_diff), float(t_diff));

    int32_t ratio_cost = min(int(100), int(1.0 / pow(gap_ratio, 5)));
    int32_t diff_cost  = max(int(q_diff), int(t_diff)) < 5000 ? 0 : 100;
    return ratio_cost + diff_cost + strand_cost + id_match_cost;
}

__global__ void chain_overlaps_by_score(Overlap* overlaps,
                                        double* scores,
                                        bool* select_mask,
                                        int32_t* predecessors,
                                        const int32_t n_overlaps,
                                        const int32_t max_distance,
                                        const int32_t max_iter)
{
    // Expects one thread per overlap
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        int32_t global_overlap_index = d_tid;
        int32_t end_index            = min(int(global_overlap_index + max_iter), int(n_overlaps));
        for (int32_t i = global_overlap_index; i < end_index; ++i)
        {
            int32_t i_score = scores[i];
            for (int32_t j = i + 1; j < i + 20; ++j)
            {
                int32_t score_addition  = min(int(scores[j]), int(100));
                int32_t gap_cost        = exp_gap_cost(overlaps[i], overlaps[j]);
                int32_t marginal_score  = min(int(score_addition), int(100)) - int(gap_cost);
                int32_t tentative_score = min(i_score + marginal_score, 100);
                if (tentative_score > scores[j])
                {
                    scores[j]       = i_score + score_addition - gap_cost;
                    predecessors[j] = i;
                    select_mask[i]  = false;
                }
            }
        }
    }
}

__global__ void produce_chains(Overlap* overlaps,
                               double* scores,
                               int32_t* predecessors,
                               bool* select_mask,
                               int32_t n_overlaps,
                               double min_score)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < n_overlaps)
    {
        int32_t global_overlap_index = d_tid;

        if (true)
        {
            int32_t index = global_overlap_index;
            Overlap final_overlap;
            final_overlap      = overlaps[index];
            double final_score = scores[index];
            while (index != predecessors[index])
            {
                int32_t pred      = predecessors[index];
                final_overlap     = merge_helper(final_overlap, overlaps[pred]);
                select_mask[pred] = false;
                index             = pred;
            }
            overlaps[global_overlap_index] = final_overlap;
            //select_mask[global_overlap_index] = true;
            scores[global_overlap_index] = final_score;
            // printf("Chain start: %d %d %d %d %d %d %d %f\n",
            //        final_overlap.query_read_id_,
            //        final_overlap.query_start_position_in_read_,
            //        final_overlap.query_end_position_in_read_,
            //        final_overlap.target_read_id_,
            //        final_overlap.target_start_position_in_read_,
            //        final_overlap.target_end_position_in_read_,
            //        final_overlap.num_residues_,
            //        final_score);
        }
    }
}

__global__ void flip_adjacent_sign(Overlap* overlaps, const int32_t n_overlaps)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid > 0 && d_tid < n_overlaps - 1 && overlaps[d_tid].num_residues_ == 1 && overlaps[d_tid - 1].num_residues_ > 1)
    {
        RelativeStrand left_strand      = overlaps[d_tid - 1].relative_strand;
        overlaps[d_tid].relative_strand = left_strand;
    }
    else if (d_tid > 0 && d_tid < n_overlaps - 1 && overlaps[d_tid].num_residues_ == 1 && overlaps[d_tid + 1].num_residues_ > 1)
    {
        RelativeStrand right_strand     = overlaps[d_tid + 1].relative_strand;
        overlaps[d_tid].relative_strand = right_strand;
    }
}

__global__ void drop_single_anchor_overlaps(Overlap* overlaps, bool* mask, const int32_t num_overlaps)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < num_overlaps)
    {
        mask[d_tid] = overlaps[d_tid].query_start_position_in_read_ != overlaps[d_tid].query_end_position_in_read_ && overlaps[d_tid].target_start_position_in_read_ != overlaps[d_tid].target_end_position_in_read_;
    }
}

/**
* Mark primary chains with a score of 1 and secondaries with a score of 0.
* TODO: implement secondary chaining check.
*/
__global__ void primary_chains_in_query_target_pairs(Overlap* overlaps,
                                                     bool* select_mask,
                                                     double* scores,
                                                     const int32_t* qt_starts,
                                                     const int32_t* qt_ends,
                                                     const int32_t num_overlaps,
                                                     const int32_t num_qt_chains,
                                                     const int32_t max_lookback,
                                                     const int32_t minimum_chain_score,
                                                     const int32_t max_dist)
{
    const std::size_t d_tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (d_tid < num_qt_chains)
    {
        //int16_t current_chain = 1;
        const int32_t start = qt_starts[d_tid];
        const int32_t end   = qt_ends[d_tid];
        if (start != end)
        {

            // for (int32_t i = start + 1; i < end; ++i)
            // {
            //     const bool strand_equal = overlaps[i].relative_strand == overlaps[i - 1].relative_strand;
            //     double tmp_score        = scores[i] + scores[i - 1];
            //     const int32_t q_gap     = overlaps[i].query_start_position_in_read_ - overlaps[i - 1].query_end_position_in_read_;
            //     const int32_t t_gap     = overlaps[i].relative_strand == RelativeStrand::Forward ? overlaps[i].target_start_position_in_read_ - overlaps[i - 1].target_end_position_in_read_ : overlaps[i].target_end_position_in_read_ - overlaps[i - 1].target_start_position_in_read_;
            //     const int32_t gap_cost  = max(q_gap, t_gap);
            //     if (strand_equal && gap_cost < max_dist && (min(int(tmp_score), int(100)) - (int(gap_cost) / 100)) > 50)
            //     {
            //         overlaps[i]        = merge_helper(overlaps[i - 1], overlaps[i]);
            //         select_mask[i - 1] = false;
            //         scores[i]          = tmp_score;
            //     }
            // }

            for (int32_t i = start; i < end; ++i)
            {
                if (select_mask[i])
                {
                    for (int32_t j = start; j < i; ++j)
                    {
                        const bool strand_equal = overlaps[i].relative_strand == overlaps[j].relative_strand;
                        double tmp_score        = scores[i] + scores[j];
                        const int32_t q_gap     = overlaps[i].query_start_position_in_read_ - overlaps[j].query_end_position_in_read_;
                        const int32_t t_gap     = overlaps[i].relative_strand == RelativeStrand::Forward ? overlaps[i].target_start_position_in_read_ - overlaps[j].target_end_position_in_read_ : overlaps[i].target_end_position_in_read_ - overlaps[j].target_start_position_in_read_;
                        const int32_t gap_cost  = max(q_gap, t_gap);
                        if (select_mask[j] && gap_cost < max_dist && strand_equal && (min(int(tmp_score), int(100)) - (int(gap_cost) / 100)) > minimum_chain_score)
                        {
                            overlaps[i]    = merge_helper(overlaps[i], overlaps[j]);
                            select_mask[j] = false;
                            scores[j]      = tmp_score;
                        }
                    }
                }
            }

            for (int32_t i = start; i < end; ++i)
            {
                select_mask[i] &= scores[i] > minimum_chain_score;
            }
        }
    }
}

void drop_scores_by_mask(device_buffer<double>& d_scores,
                         device_buffer<bool>& d_mask,
                         const std::int32_t n_overlaps,
                         device_buffer<double>& d_dest,
                         device_buffer<int32_t>& d_filtered_count,
                         DefaultDeviceAllocator& _allocator,
                         hipStream_t& _cuda_stream)
{
    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage,
                               temp_storage_bytes,
                               d_scores.data(),
                               d_mask.data(),
                               d_dest.data(),
                               d_filtered_count.data(),
                               n_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage,
                               temp_storage_bytes,
                               d_scores.data(),
                               d_mask.data(),
                               d_dest.data(),
                               d_filtered_count.data(),
                               n_overlaps,
                               _cuda_stream);
}

void drop_overlaps_by_mask(device_buffer<Overlap>& d_overlaps,
                           device_buffer<bool>& d_mask,
                           const std::int32_t n_overlaps,
                           device_buffer<Overlap>& d_dest,
                           device_buffer<int32_t>& d_filtered_count,
                           DefaultDeviceAllocator& _allocator,
                           hipStream_t& _cuda_stream)
{
    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_overlaps.data(),
                               d_mask.data(),
                               d_dest.data(),
                               d_filtered_count.data(),
                               n_overlaps,
                               _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_overlaps.data(),
                               d_mask.data(),
                               d_dest.data(),
                               d_filtered_count.data(),
                               n_overlaps,
                               _cuda_stream);
}

void encode_query_target_pairs(Overlap* overlaps,
                               int32_t n_overlaps,
                               device_buffer<int32_t>& query_target_starts,
                               device_buffer<int32_t>& query_target_lengths,
                               device_buffer<int32_t>& query_target_ends,
                               int32_t& n_query_target_pairs,
                               DefaultDeviceAllocator& _allocator,
                               hipStream_t& _cuda_stream,
                               int32_t block_size = 32)
{
    OverlapToQueryTargetPairOp qt_pair_op;
    hipcub::TransformInputIterator<QueryTargetPair, OverlapToQueryTargetPairOp, Overlap*> d_query_target_pairs(overlaps, qt_pair_op);
    device_buffer<QueryTargetPair> d_qt_pairs(n_overlaps, _allocator, _cuda_stream);
    device_buffer<int32_t> d_num_query_target_pairs(1, _allocator, _cuda_stream);

    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       d_query_target_pairs,
                                       d_qt_pairs.data(),
                                       query_target_lengths.data(),
                                       d_num_query_target_pairs.data(),
                                       n_overlaps);

    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    hipcub::DeviceRunLengthEncode::Encode(d_temp_storage,
                                       temp_storage_bytes,
                                       d_query_target_pairs,
                                       d_qt_pairs.data(),
                                       query_target_lengths.data(),
                                       d_num_query_target_pairs.data(),
                                       n_overlaps);

    n_query_target_pairs = cudautils::get_value_from_device(d_num_query_target_pairs.data(), _cuda_stream);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  query_target_lengths.data(),
                                  query_target_starts.data(),
                                  n_query_target_pairs, _cuda_stream);

    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  query_target_lengths.data(),
                                  query_target_starts.data(),
                                  n_query_target_pairs, _cuda_stream);

    convert_offsets_to_ends<<<(n_query_target_pairs / block_size) + 1, block_size, 0, _cuda_stream>>>(query_target_starts.data(), query_target_lengths.data(), query_target_ends.data(), n_query_target_pairs);
}

void OverlapperAnchmer::get_overlaps(std::vector<Overlap>& fused_overlaps,
                                     const device_buffer<Anchor>& d_anchors,
                                     bool all_to_all,
                                     int64_t min_residues,
                                     int64_t min_overlap_len,
                                     int64_t min_bases_per_residue,
                                     float min_overlap_fraction)
{

    // const std::int32_t anchmer_generation_rounds = 1;
    // const std::int32_t chain_filter_min_anchors  = 2;
    // const std::int32_t anchor_merge_min_dist     = 150;
    const std::int32_t anchors_per_anchmer    = MAX_ANCHMER_WINDOW;
    const std::int32_t overlapmer_window_size = MAX_OVERLAPMER_WINDOW;
    const std::size_t n_anchors               = d_anchors.size();
    const std::size_t n_anchmers              = (d_anchors.size() / anchors_per_anchmer) + 1;
    const std::int32_t block_size             = 32;

    //const std::int32_t max_gap = 5000;

    //std::vector<Anchmer> anchmers(n_anchmers);
    device_buffer<Anchmer> d_anchmers(n_anchmers, _allocator, _cuda_stream);
    device_buffer<bool> d_anchor_select_mask(n_anchors, _allocator, _cuda_stream);

    initalize_anchors_mask<<<(n_anchors / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchor_select_mask.data(),
                                                                                          n_anchors,
                                                                                          true);

    //mask_repetitive_anchors<<<(n_anchors / 20) + 1, block_size, 0, _cuda_stream>>>(d_anchors.data(), d_anchor_select_mask.data(), n_anchors, 12, 5);
    //mask_repetitive_anchors_by_RLE(d_anchors, d_anchor_select_mask, n_anchors, 5, _allocator, _cuda_stream, block_size);
    // Stage one: generate anchmers
    generate_anchmers<<<(n_anchmers / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchors.data(),
                                                                                      d_anchor_select_mask.data(),
                                                                                      n_anchors,
                                                                                      d_anchmers.data(),
                                                                                      anchors_per_anchmer);

    // #define DEBUG
    // #ifdef DEBUG
    //     std::vector<Anchor> anchors;
    //     //std::vector<bool> host_masks;
    //     //host_masks.resize(n_anchors);
    //     anchors.resize(n_anchors);
    //     cudautils::device_copy_n(d_anchors.data(), n_anchors, anchors.data(), _cuda_stream);
    //     //hipMemcpy(masks.data(), d_anchor_select_mask.data(), n_anchors * sizeof(bool), hipMemcpyDefault);
    //     //cudautils::device_copy_n(d_anchor_select_mask.data(), n_anchors, host_masks.data(), _cuda_stream);
    //     for (std::size_t i = 0; i < n_anchors; ++i)
    //     {
    //         Anchor a = anchors[i];
    //         std::cerr << a.query_position_in_read_;
    //         std::cerr << " " << a.target_position_in_read_ << std::endl;
    //         // std::cerr << (host_masks[i] ? "true" : "false") << std::endl;
    //     }
    // #endif
    // #ifdef DEBUG
    //     std::vector<Anchmer> anchmers;
    //     std::vector<Anchor> anchors;
    //     anchmers.resize(d_anchmers.size());
    //     anchors.resize(n_anchors);
    //     cudautils::device_copy_n(d_anchmers.data(), d_anchmers.size(), anchmers.data(), _cuda_stream);
    //     cudautils::device_copy_n(d_anchors.data(), n_anchors, anchors.data(), _cuda_stream);
    //     for (std::size_t i = 0; i < anchmers.size(); ++i)
    //     {
    //         Anchmer a = anchmers[i];
    //         std::cerr << a.n_anchors << " " << static_cast<int16_t>(a.n_chains) << std::endl;
    //         for (std::size_t j = 0; j < a.n_anchors; ++j)
    //         {
    //             std::cerr << static_cast<int16_t>(a.chain_id[j]) << " ";
    //         }
    //         std::cerr << std::endl;
    //         for (std::size_t j = 0; j < a.n_anchors; ++j)
    //         {
    //             std::cerr << static_cast<int16_t>(anchors[(MAX_ANCHMER_WINDOW * i) + j].query_position_in_read_) << " ";
    //         }

    //         std::cerr << std::endl;
    //     }
    // #endif

    // Stage 2: Given a buffer of anchmers, generate overlaps within each anchmer.
    // Anchmers may contain between 1 and anchors_per_anchmer overlaps

    // Calculate the number of overlaps needed for the initial generation.
    // This is equal to the sum of each anchmer's n_chains value.
    // Transform each anchmer's n_chains value into a device vector so we can calculate a prefix
    // sum (which will give us the mapping between anchmer -> index in overlaps array)
    AnchmerCountChainsOp anchmer_chain_count_op;
    hipcub::TransformInputIterator<int32_t, AnchmerCountChainsOp, Anchmer*> d_chain_counts(d_anchmers.data(), anchmer_chain_count_op);

    device_buffer<int32_t> d_overlap_ends(n_anchmers, _allocator, _cuda_stream);

    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);
    void* d_temp_storage           = nullptr;
    std::size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_chain_counts,
                                  d_overlap_ends.data(),
                                  n_anchmers,
                                  _cuda_stream);
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();
    hipcub::DeviceScan::InclusiveSum(d_temp_storage,
                                  temp_storage_bytes,
                                  d_chain_counts,
                                  d_overlap_ends.data(),
                                  n_anchmers,
                                  _cuda_stream);

    // Holds the last prefix sum in the overlap_ends vector.
    // This value is the total number of overlaps
    int32_t n_initial_overlaps = cudautils::get_value_from_device(d_overlap_ends.data() + n_anchmers - 1, _cuda_stream);

    //std::cerr << "Generating " << n_initial_overlaps << " initial overlaps from " << n_anchmers << " anchmers..." << std::endl;

    // Create device buffers needed to hold all future values.
    // Overlaps_SRC and Overlaps_DEST provide two vectors for overlaps so that filtering can be done in rounds.
    // Overlaps_SELECT_MASK provides a single boolean mask for all overlaps
    // query_target_starts provides the start indicies of the query-target pairs in the overlaps vector.
    device_buffer<Overlap> d_overlaps_source(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<Overlap> d_overlaps_dest(n_initial_overlaps, _allocator, _cuda_stream);

    device_buffer<bool> d_overlaps_select_mask(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<bool> d_chain_max(n_initial_overlaps, _allocator, _cuda_stream);

    device_buffer<double> d_overlap_scores(n_initial_overlaps, _allocator, _cuda_stream);
    device_buffer<double> d_overlap_scores_dest(n_initial_overlaps, _allocator, _cuda_stream);

    device_buffer<int32_t> d_predecessors(n_initial_overlaps, _allocator, _cuda_stream);

    initialize_overlaps_array<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                                      n_initial_overlaps);
    init_overlap_mask<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_select_mask.data(),
                                                                                              n_initial_overlaps,
                                                                                              true);

    anchmers_to_overlaps<<<(n_anchmers / block_size) + 1, block_size, 0, _cuda_stream>>>(d_anchmers.data(),
                                                                                         d_overlap_ends.data(),
                                                                                         n_anchmers,
                                                                                         d_anchors.data(),
                                                                                         d_anchor_select_mask.data(),
                                                                                         n_anchors,
                                                                                         d_overlaps_source.data(),
                                                                                         d_overlaps_select_mask.data(),
                                                                                         n_initial_overlaps);
    flip_adjacent_sign<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                               n_initial_overlaps);

    init_overlap_scores<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                                d_overlap_scores.data(),
                                                                                                n_initial_overlaps,
                                                                                                2.0);

    d_anchmers.clear_and_resize(0);
    d_anchor_select_mask.clear_and_resize(0);

//#define DEBUG
#ifdef DEBUG
    // std::size_t num_overlaps = n_initial_overlaps;
    // std::vector<Overlap>
    //     intermediate_overlaps;
    // std::vector<double> intermediate_scores;
    // intermediate_scores.resize(num_overlaps);
    // cudautils::device_copy_n(d_overlap_scores.data(), num_overlaps, intermediate_scores.data(), _cuda_stream);
    // intermediate_overlaps.resize(num_overlaps);
    // cudautils::device_copy_n(d_overlaps_source.data(), num_overlaps, intermediate_overlaps.data(), _cuda_stream);
    // for (std::size_t i = 0; i < intermediate_overlaps.size(); ++i)
    // {
    //     Overlap o    = intermediate_overlaps[i];
    //     double score = intermediate_scores[i];
    //     std::cerr << o.query_read_id_ << " " << o.query_start_position_in_read_;
    //     std::cerr << " " << o.query_end_position_in_read_ << " ";
    //     std::cerr << static_cast<char>(o.relative_strand) << " " << char(o.relative_strand);
    //     std::cerr << " " << o.target_read_id_ << " " << o.target_start_position_in_read_ << " ";
    //     std::cerr << o.target_end_position_in_read_ << " " << o.num_residues_ << " " << score << std::endl;
    // }
    // std::cerr << std::endl
    //           << std::endl;
#endif

    // chain_overlaps_in_window<<<(n_initial_overlaps / overlapmer_window_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
    //                                                                                                              d_overlaps_select_mask.data(),
    //                                                                                                              d_overlap_scores.data(),
    //                                                                                                              n_initial_overlaps, (n_initial_overlaps / overlapmer_window_size) + 1,
    //                                                                                                              overlapmer_window_size,
    //                                                                                                              500,
    //                                                                                                              0.8);

    init_predecessor_and_score_arrays<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_predecessors.data(), d_overlap_scores.data(), d_overlaps_select_mask.data(), n_initial_overlaps);

    init_overlap_mask<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_select_mask.data(),
                                                                                              n_initial_overlaps,
                                                                                              true);

    chain_overlaps_by_score<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                                    d_overlap_scores.data(),
                                                                                                    d_overlaps_select_mask.data(),
                                                                                                    d_predecessors.data(), n_initial_overlaps,
                                                                                                    5000,
                                                                                                    32);

    produce_chains<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                           d_overlap_scores.data(),
                                                                                           d_predecessors.data(),
                                                                                           d_overlaps_select_mask.data(),
                                                                                           n_initial_overlaps,
                                                                                           50);

    device_buffer<int32_t> d_n_filtered_overlaps(1, _allocator, _cuda_stream);

    // device_buffer<int32_t> d_query_target_pair_starts(n_initial_overlaps, _allocator, _cuda_stream);
    // device_buffer<int32_t> d_query_target_pair_lengths(n_initial_overlaps, _allocator, _cuda_stream);
    // device_buffer<int32_t> d_query_target_pair_ends(n_initial_overlaps, _allocator, _cuda_stream);
    // int32_t n_query_target_pairs = 0;

    // encode_query_target_pairs(d_overlaps_source.data(),
    //                           n_initial_overlaps,
    //                           d_query_target_pair_starts,
    //                           d_query_target_pair_lengths,
    //                           d_query_target_pair_ends,
    //                           n_query_target_pairs,
    //                           _allocator,
    //                           _cuda_stream,
    //                           block_size);

#ifdef DEBUG
    // num_overlaps = n_initial_overlaps;
    // intermediate_scores.resize(num_overlaps);
    // cudautils::device_copy_n(d_overlap_scores.data(), num_overlaps, intermediate_scores.data(), _cuda_stream);

    // intermediate_overlaps.resize(num_overlaps);
    // cudautils::device_copy_n(d_overlaps_source.data(), num_overlaps, intermediate_overlaps.data(), _cuda_stream);

    // std::vector<int32_t> intermediate_preds;
    // intermediate_preds.resize(num_overlaps);
    // cudautils::device_copy_n(d_predecessors.data(), num_overlaps, intermediate_preds.data(), _cuda_stream);
    // for (std::size_t i = 0; i < intermediate_overlaps.size(); ++i)
    // {
    //     Overlap o    = intermediate_overlaps[i];
    //     double score = intermediate_scores[i];
    //     int32_t pred = intermediate_preds[i];
    //     std::cerr << o.query_read_id_ << " " << o.query_start_position_in_read_;
    //     std::cerr << " " << o.query_end_position_in_read_ << " ";
    //     std::cerr << static_cast<char>(o.relative_strand) << " ";
    //     std::cerr << " " << o.target_read_id_ << " " << o.target_start_position_in_read_ << " ";
    //     // std::cerr << o.target_end_position_in_read_ << " " << o.num_residues_ << std::endl;
    //     std::cerr << o.target_end_position_in_read_ << " " << o.num_residues_;
    //     std::cerr << " " << score << " " << i << " " << pred << std::endl;
    // }
#endif

    // primary_chains_in_query_target_pairs<<<(n_query_target_pairs / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
    //                                                                                                                d_overlaps_select_mask.data(),
    //                                                                                                                d_overlap_scores.data(),
    //                                                                                                                d_query_target_pair_starts.data(),
    //                                                                                                                d_query_target_pair_ends.data(),
    //                                                                                                                n_initial_overlaps,
    //                                                                                                                n_query_target_pairs,
    //                                                                                                                0,
    //                                                                                                                20,
    //                                                                                                                5000);

    mask_overlaps<<<(n_initial_overlaps / block_size) + 1, block_size, 0, _cuda_stream>>>(d_overlaps_source.data(),
                                                                                          n_initial_overlaps,
                                                                                          d_overlaps_select_mask.data(),
                                                                                          min_overlap_len,
                                                                                          min_residues,
                                                                                          min_bases_per_residue,
                                                                                          all_to_all,
                                                                                          true);

    drop_overlaps_by_mask(d_overlaps_source,
                          d_overlaps_select_mask,
                          n_initial_overlaps,
                          d_overlaps_dest,
                          d_n_filtered_overlaps,
                          _allocator,
                          _cuda_stream);
    drop_scores_by_mask(d_overlap_scores,
                        d_overlaps_select_mask,
                        n_initial_overlaps,
                        d_overlap_scores_dest,
                        d_n_filtered_overlaps,
                        _allocator,
                        _cuda_stream);
    int32_t n_filtered_overlaps = cudautils::get_value_from_device(d_n_filtered_overlaps.data(), _cuda_stream);

    fused_overlaps.resize(n_filtered_overlaps);
    cudautils::device_copy_n(d_overlaps_dest.data(), n_filtered_overlaps, fused_overlaps.data(), _cuda_stream);

    // This is not completely necessary, but if removed one has to make sure that the next step
    // uses the same stream or that sync is done in caller
    GW_CU_CHECK_ERR(hipStreamSynchronize(_cuda_stream));
}

OverlapperAnchmer::OverlapperAnchmer(DefaultDeviceAllocator allocator,
                                     const hipStream_t hip_stream)
    : _allocator(allocator)
    , _cuda_stream(hip_stream)
{
}

} // namespace cudamapper
} // namespace genomeworks
} // namespace claraparabricks
