#include "hip/hip_runtime.h"
/*
* Copyright 2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "chainer_utils.cuh"

#include <hipcub/hipcub.hpp>
#include <claraparabricks/genomeworks/utils/cudautils.hpp>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{
namespace chainerutils
{

struct OverlapToNumResiduesOp
{
    __device__ __forceinline__ int32_t operator()(const Overlap& overlap) const
    {
        return overlap.num_residues_;
    }
};

__host__ __device__ Overlap create_overlap(const Anchor& start, const Anchor& end, const int32_t num_anchors)
{
    Overlap overlap;
    overlap.num_residues_ = num_anchors;

    overlap.query_read_id_  = start.query_read_id_;
    overlap.target_read_id_ = start.target_read_id_;
    assert(start.query_read_id_ == end.query_read_id_ && start.target_read_id_ == end.target_read_id_);

    overlap.query_start_position_in_read_ = min(start.query_position_in_read_, end.query_position_in_read_);
    overlap.query_end_position_in_read_   = max(start.query_position_in_read_, end.query_position_in_read_);
    const bool is_negative_strand         = end.target_position_in_read_ < start.target_position_in_read_;
    if (is_negative_strand)
    {
        overlap.relative_strand                = RelativeStrand::Reverse;
        overlap.target_start_position_in_read_ = end.target_position_in_read_;
        overlap.target_end_position_in_read_   = start.target_position_in_read_;
    }
    else
    {
        overlap.relative_strand                = RelativeStrand::Forward;
        overlap.target_start_position_in_read_ = start.target_position_in_read_;
        overlap.target_end_position_in_read_   = end.target_position_in_read_;
    }
    return overlap;
}

__global__ void backtrace_anchors_to_overlaps(const Anchor* const anchors,
                                              Overlap* overlaps,
                                              double* const scores,
                                              bool* const max_select_mask,
                                              int32_t* const predecessors,
                                              const int64_t n_anchors,
                                              const int32_t min_score)
{
    const int64_t tid    = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n_anchors; i += stride)
    {
        if (scores[i] >= min_score)
        {
            int32_t index                = i;
            int32_t first_index          = index;
            int32_t num_anchors_in_chain = 0;
            Anchor final_anchor          = anchors[i];

            while (index != -1)
            {
                first_index  = index;
                int32_t pred = predecessors[index];
                if (pred != -1)
                {
                    max_select_mask[pred] = false;
                }
                num_anchors_in_chain++;
                index = predecessors[index];
            }
            Anchor first_anchor = anchors[first_index];
            overlaps[i]         = create_overlap(first_anchor, final_anchor, num_anchors_in_chain);
        }
        else
        {
            max_select_mask[i] = false;
        }
    }
}

void allocate_anchor_chains(const device_buffer<Overlap>& overlaps,
                            device_buffer<int32_t>& unrolled_anchor_chains,
                            device_buffer<int32_t>& anchor_chain_starts,
                            int64_t& num_total_anchors,
                            DefaultDeviceAllocator allocator,
                            hipStream_t hip_stream)
{
    // sum the number of chains across all overlaps

    void* temp_storage_d           = nullptr;
    std::size_t temp_storage_bytes = 0;
    OverlapToNumResiduesOp overlap_residue_count_op;
    hipcub::TransformInputIterator<int32_t, OverlapToNumResiduesOp, const Overlap*> residue_counts_d(overlaps.data(),
                                                                                                  overlap_residue_count_op);

    device_buffer<int64_t> num_total_anchors_d(1, allocator, hip_stream);

    hipcub::DeviceReduce::Sum(temp_storage_d,
                           temp_storage_bytes,
                           residue_counts_d,
                           num_total_anchors_d.data(),
                           overlaps.size(),
                           hip_stream);

    device_buffer<char> temp_buf_d(temp_storage_bytes, allocator, hip_stream);
    temp_storage_d = temp_buf_d.data();

    hipcub::DeviceReduce::Sum(temp_storage_d,
                           temp_storage_bytes,
                           residue_counts_d,
                           num_total_anchors_d.data(),
                           overlaps.size(),
                           hip_stream);

    temp_storage_d     = nullptr;
    temp_storage_bytes = 0;

    num_total_anchors = cudautils::get_value_from_device(num_total_anchors_d.data(), hip_stream);

    unrolled_anchor_chains.clear_and_resize(num_total_anchors);
    anchor_chain_starts.clear_and_resize(overlaps.size());

    hipcub::DeviceScan::ExclusiveSum(temp_storage_d,
                                  temp_storage_bytes,
                                  residue_counts_d,
                                  anchor_chain_starts.data(),
                                  overlaps.size(),
                                  hip_stream);

    temp_buf_d.clear_and_resize(temp_storage_bytes);
    temp_storage_d = temp_buf_d.data();

    hipcub::DeviceScan::ExclusiveSum(temp_storage_d,
                                  temp_storage_bytes,
                                  residue_counts_d,
                                  anchor_chain_starts.data(),
                                  overlaps.size(),
                                  hip_stream);
}

__global__ void output_overlap_chains_by_backtrace(const Overlap* const overlaps,
                                                   const Anchor* const anchors,
                                                   const bool* const select_mask,
                                                   const int32_t* const predecessors,
                                                   int32_t* const anchor_chains,
                                                   int32_t* const anchor_chain_starts,
                                                   const int32_t num_overlaps,
                                                   const bool check_mask)
{
    const int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride    = blockDim.x * gridDim.x;

    // Processes one overlap per iteration,
    // "i" corresponds to an overlap
    for (int i = thread_id; i < num_overlaps; i += stride)
    {
        // index within this chain of anchors (i.e., the anchors within a single overlap)

        if (!check_mask || (check_mask & select_mask[i]))
        {
            int32_t anchor_chain_index = 0;
            // As chaining proceeds backwards (i.e., it's a backtrace),
            // we need to fill the new anchor chain array in in reverse order.
            int32_t index = anchor_chain_starts[i];
            while (index != -1)
            {
                anchor_chains[anchor_chain_starts[i] + (overlaps[i].num_residues_ - anchor_chain_index)] = index;
                index                                                                                    = predecessors[index];
                ++anchor_chain_index;
            }
        }
    }
}

__global__ void output_overlap_chains_by_RLE(const Overlap* const overlaps,
                                             const Anchor* const anchors,
                                             const int32_t* const chain_starts,
                                             const int32_t* const chain_lengths,
                                             int32_t* const anchor_chains,
                                             int32_t* const anchor_chain_starts,
                                             const uint32_t num_overlaps)
{
    const int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride    = blockDim.x * gridDim.x;
    for (uint32_t i = thread_id; i < num_overlaps; i += stride)
    {
        int32_t chain_start  = chain_starts[i];
        int32_t chain_length = chain_lengths[i];
        for (int32_t ind = chain_start; ind < chain_start + chain_length; ++i)
        {
            anchor_chains[ind] = ind;
        }
    }
}

} // namespace chainerutils
} // namespace cudamapper
} // namespace genomeworks
} // namespace claraparabricks