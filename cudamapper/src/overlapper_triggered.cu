#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "overlapper_triggered.hpp"

#include <fstream>
#include <cstdlib>

#include <hipcub/hipcub.hpp>
#include <thrust/execution_policy.h>

#include <claraparabricks/genomeworks/utils/cudautils.hpp>

#include <algorithm>

#ifndef NDEBUG
#include <thrust/host_vector.h>
#endif
namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

__host__ __device__ bool operator==(const Anchor& lhs,
                                    const Anchor& rhs)
{
    auto score_threshold = 1;

    // Very simple scoring function to quantify quality of overlaps.
    auto score = 1;

    if ((rhs.query_position_in_read_ - lhs.query_position_in_read_) < 150 and abs(int(rhs.target_position_in_read_) - int(lhs.target_position_in_read_)) < 150)
        score = 2;
    return ((lhs.query_read_id_ == rhs.query_read_id_) &&
            (lhs.target_read_id_ == rhs.target_read_id_) &&
            score > score_threshold);
}

struct cuOverlapKey
{
    const Anchor* anchor;
};

struct cuOverlapKey_transform
{
    const Anchor* d_anchors;
    const int32_t* d_chain_start;

    cuOverlapKey_transform(const Anchor* anchors, const int32_t* chain_start)
        : d_anchors(anchors)
        , d_chain_start(chain_start)
    {
    }

    __host__ __device__ __forceinline__ cuOverlapKey
    operator()(const int32_t& idx) const
    {
        auto anchor_idx = d_chain_start[idx];

        cuOverlapKey key;
        key.anchor = &d_anchors[anchor_idx];
        return key;
    }
};

__host__ __device__ bool operator==(const cuOverlapKey& key0,
                                    const cuOverlapKey& key1)
{
    const Anchor* a = key0.anchor;
    const Anchor* b = key1.anchor;

    int distance_difference = abs(abs(int(a->query_position_in_read_) - int(b->query_position_in_read_)) -
                                  abs(int(a->target_position_in_read_) - int(b->target_position_in_read_)));

    bool equal = (a->target_read_id_ == b->target_read_id_) &&
                 (a->query_read_id_ == b->query_read_id_) &&
                 distance_difference < 300;

    return equal;
}

struct cuOverlapArgs
{
    int32_t overlap_end;
    int32_t num_residues;
    int32_t overlap_start;
};

struct cuOverlapArgs_transform
{
    const int32_t* d_chain_start;
    const int32_t* d_chain_length;

    cuOverlapArgs_transform(const int32_t* chain_start, const int32_t* chain_length)
        : d_chain_start(chain_start)
        , d_chain_length(chain_length)
    {
    }

    __host__ __device__ __forceinline__ cuOverlapArgs
    operator()(const int32_t& idx) const
    {
        cuOverlapArgs overlap;
        auto overlap_start    = d_chain_start[idx];
        auto overlap_length   = d_chain_length[idx];
        overlap.overlap_end   = overlap_start + overlap_length;
        overlap.num_residues  = overlap_length;
        overlap.overlap_start = overlap_start;
        return overlap;
    }
};

struct FuseOverlapOp
{
    __host__ __device__ cuOverlapArgs operator()(const cuOverlapArgs& a,
                                                 const cuOverlapArgs& b) const
    {
        cuOverlapArgs fused_overlap;
        fused_overlap.num_residues = a.num_residues + b.num_residues;
        fused_overlap.overlap_end =
            a.overlap_end > b.overlap_end ? a.overlap_end : b.overlap_end;
        fused_overlap.overlap_start =
            a.overlap_start < b.overlap_start ? a.overlap_start : b.overlap_start;
        return fused_overlap;
    }
};

struct FilterOverlapOp
{
    size_t min_residues;
    size_t min_overlap_len;
    size_t min_bases_per_residue;
    float min_overlap_fraction;
    bool indexes_identical;

    __host__ __device__ __forceinline__ FilterOverlapOp(size_t min_residues,
                                                        size_t min_overlap_len,
                                                        size_t min_bases_per_residue,
                                                        float min_overlap_fraction,
                                                        bool indexes_identical)
        : min_residues(min_residues)
        , min_overlap_len(min_overlap_len)
        , min_bases_per_residue(min_bases_per_residue)
        , min_overlap_fraction(min_overlap_fraction)
        , indexes_identical(indexes_identical)
    {
    }

    __host__ __device__ __forceinline__ bool operator()(const Overlap& overlap) const
    {

        const auto target_overlap_length = overlap.target_end_position_in_read_ - overlap.target_start_position_in_read_;
        const auto query_overlap_length  = overlap.query_end_position_in_read_ - overlap.query_start_position_in_read_;
        const auto overlap_length        = max(target_overlap_length, query_overlap_length);
        const bool self_mapping          = (overlap.query_read_id_ == overlap.target_read_id_) && indexes_identical;

        return ((overlap.num_residues_ >= min_residues) &&
                ((overlap_length / overlap.num_residues_) < min_bases_per_residue) &&
                (query_overlap_length >= min_overlap_len) &&
                (target_overlap_length >= min_overlap_len) &&
                (!self_mapping) &&
                ((static_cast<float>(target_overlap_length) / static_cast<float>(overlap_length)) > min_overlap_fraction) &&
                ((static_cast<float>(query_overlap_length) / static_cast<float>(overlap_length)) > min_overlap_fraction));
    }
};

struct CreateOverlap
{
    const Anchor* d_anchors;

    __host__ __device__ __forceinline__ CreateOverlap(const Anchor* anchors_ptr)
        : d_anchors(anchors_ptr)
    {
    }

    __host__ __device__ __forceinline__ Overlap
    operator()(cuOverlapArgs overlap)
    {
        Anchor overlap_start_anchor = d_anchors[overlap.overlap_start];
        Anchor overlap_end_anchor   = d_anchors[overlap.overlap_end - 1];

        Overlap new_overlap;

        new_overlap.query_read_id_  = overlap_end_anchor.query_read_id_;
        new_overlap.target_read_id_ = overlap_end_anchor.target_read_id_;
        new_overlap.num_residues_   = overlap.num_residues;
        new_overlap.target_end_position_in_read_ =
            overlap_end_anchor.target_position_in_read_;
        new_overlap.target_start_position_in_read_ =
            overlap_start_anchor.target_position_in_read_;
        new_overlap.query_end_position_in_read_ =
            overlap_end_anchor.query_position_in_read_;
        new_overlap.query_start_position_in_read_ =
            overlap_start_anchor.query_position_in_read_;
        new_overlap.overlap_complete = true;

        // If the target start position is greater than the target end position
        // We can safely assume that the query and target are template and
        // complement reads. TODO: Incorporate sketchelement direction value when
        // this is implemented
        if (new_overlap.target_start_position_in_read_ >
            new_overlap.target_end_position_in_read_)
        {
            new_overlap.relative_strand = RelativeStrand::Reverse;
            auto tmp                    = new_overlap.target_end_position_in_read_;
            new_overlap.target_end_position_in_read_ =
                new_overlap.target_start_position_in_read_;
            new_overlap.target_start_position_in_read_ = tmp;
        }
        else
        {
            new_overlap.relative_strand = RelativeStrand::Forward;
        }
        return new_overlap;
    };
};

OverlapperTriggered::OverlapperTriggered(DefaultDeviceAllocator allocator,
                                         const hipStream_t hip_stream)
    : _allocator(allocator)
    , _cuda_stream(hip_stream)
{
}

void OverlapperTriggered::get_overlaps(std::vector<Overlap>& fused_overlaps,
                                       const device_buffer<Anchor>& d_anchors,
                                       bool all_to_all,
                                       int64_t min_residues,
                                       int64_t min_overlap_len,
                                       int64_t min_bases_per_residue,
                                       float min_overlap_fraction)
{
    GW_NVTX_RANGE(profiler, "OverlapperTriggered::get_overlaps");
    const auto tail_length_for_chain = 3;
    auto n_anchors                   = d_anchors.size();

#ifndef NDEBUG
    // check if anchors are sorted properly

    // TODO: Copying data to host and doing the check there as using thrust::is_sorted
    //       leads to a compilaiton error. It is probably a bug in device_buffer implementation

    thrust::host_vector<Anchor> h_anchors(d_anchors.size());
    cudautils::device_copy_n(d_anchors.data(), d_anchors.size(), h_anchors.data()); // D2H

    auto comp_anchors = [](const Anchor& i, const Anchor& j) { return (i.query_read_id_ < j.query_read_id_) ||
                                                                      ((i.query_read_id_ == j.query_read_id_) &&
                                                                       (i.target_read_id_ < j.target_read_id_)) ||
                                                                      ((i.query_read_id_ == j.query_read_id_) &&
                                                                       (i.target_read_id_ == j.target_read_id_) &&
                                                                       (i.query_position_in_read_ < j.query_position_in_read_)) ||
                                                                      ((i.query_read_id_ == j.query_read_id_) &&
                                                                       (i.target_read_id_ == j.target_read_id_) &&
                                                                       (i.query_position_in_read_ == j.query_position_in_read_) &&
                                                                       (i.target_position_in_read_ < j.target_position_in_read_)); };

    assert(std::is_sorted(std::begin(h_anchors),
                          std::end(h_anchors),
                          comp_anchors));
#endif

    // temporary workspace buffer on device
    device_buffer<char> d_temp_buf(_allocator, _cuda_stream);

    // Do run length encode to compute the chains
    // note - identifies the start and end anchor of the chain without moving the anchors
    // >>>>>>>>>

    // d_start_anchor[i] contains the starting anchor of chain i
    device_buffer<Anchor> d_start_anchor(n_anchors, _allocator, _cuda_stream);

    // d_chain_length[i] contains the length of chain i
    device_buffer<int32_t> d_chain_length(n_anchors, _allocator, _cuda_stream);

    // total number of chains found
    device_buffer<int32_t> d_nchains(1, _allocator, _cuda_stream);

    //The equality of two anchors has been overriden, such that they are equal (members of the same chain) if their QID,TID are equal and they fall within a fixed distance of one another
    void* d_temp_storage      = nullptr;
    size_t temp_storage_bytes = 0;
    // calculate storage requirement for run length encoding
    hipcub::DeviceRunLengthEncode::Encode(
        d_temp_storage, temp_storage_bytes, d_anchors.data(), d_start_anchor.data(),
        d_chain_length.data(), d_nchains.data(), n_anchors, _cuda_stream);

    // allocate temporary storage
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    // run encoding
    hipcub::DeviceRunLengthEncode::Encode(
        d_temp_storage, temp_storage_bytes, d_anchors.data(), d_start_anchor.data(),
        d_chain_length.data(), d_nchains.data(), n_anchors, _cuda_stream);

    // <<<<<<<<<<

    // memcpy D2H
    auto n_chains = cudautils::get_value_from_device(d_nchains.data(), _cuda_stream); //We now know the number of chains we are working with.

    // use prefix sum to calculate the starting index position of all the chains
    // >>>>>>>>>>>>
    // for a chain i, d_chain_start[i] contains the index of starting anchor from d_anchors array
    device_buffer<int32_t> d_chain_start(n_chains, _allocator, _cuda_stream);

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_chain_length.data(), d_chain_start.data(),
                                  n_chains, _cuda_stream);

    // allocate temporary storage
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes,
                                  d_chain_length.data(), d_chain_start.data(),
                                  n_chains, _cuda_stream);

    // <<<<<<<<<<<<

    // calculate overlaps where overlap is a chain with length > tail_length_for_chain
    // >>>>>>>>>>>>

    auto thrust_exec_policy = thrust::cuda::par(_allocator).on(_cuda_stream);

    // d_overlaps[j] contains index to d_chain_length/d_chain_start where
    // d_chain_length[d_overlaps[j]] and d_chain_start[d_overlaps[j]] corresponds
    // to length and index to starting anchor of the chain-d_overlaps[j] (also referred as overlap j)
    device_buffer<int32_t> d_overlaps(n_chains, _allocator, _cuda_stream);
    auto indices_end =
        thrust::copy_if(thrust_exec_policy, thrust::make_counting_iterator<int32_t>(0),
                        thrust::make_counting_iterator<int32_t>(n_chains),
                        d_chain_length.data(), d_overlaps.data(),
                        [=] __host__ __device__(const int32_t& len) -> bool {
                            return (len >= tail_length_for_chain);
                        });

    auto n_overlaps = indices_end - d_overlaps.data();

    // <<<<<<<<<<<<<

    // >>>>>>>>>>>>
    // fuse overlaps using reduce by key operations

    // key is a minimal data structure that is required to compare the overlaps
    cuOverlapKey_transform key_op(d_anchors.data(),
                                  d_chain_start.data());
    hipcub::TransformInputIterator<cuOverlapKey, cuOverlapKey_transform, int32_t*>
        d_keys_in(d_overlaps.data(),
                  key_op);

    // value is a minimal data structure that represents a overlap
    cuOverlapArgs_transform value_op(d_chain_start.data(),
                                     d_chain_length.data());

    hipcub::TransformInputIterator<cuOverlapArgs, cuOverlapArgs_transform, int32_t*>
        d_values_in(d_overlaps.data(),
                    value_op);

    device_buffer<cuOverlapKey> d_fusedoverlap_keys(n_overlaps, _allocator, _cuda_stream);
    device_buffer<cuOverlapArgs> d_fusedoverlaps_args(n_overlaps, _allocator, _cuda_stream);
    device_buffer<int32_t> d_nfused_overlaps(1, _allocator, _cuda_stream);

    FuseOverlapOp reduction_op;

    d_temp_storage     = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(d_temp_storage,
                                   temp_storage_bytes,
                                   d_keys_in,
                                   d_fusedoverlap_keys.data(), d_values_in,
                                   d_fusedoverlaps_args.data(), d_nfused_overlaps.data(),
                                   reduction_op,
                                   n_overlaps,
                                   _cuda_stream);

    // allocate temporary storage
    d_temp_buf.clear_and_resize(temp_storage_bytes);
    d_temp_storage = d_temp_buf.data();

    hipcub::DeviceReduce::ReduceByKey(d_temp_storage,
                                   temp_storage_bytes,
                                   d_keys_in,
                                   d_fusedoverlap_keys.data(), //Write out the unique keys here
                                   d_values_in,
                                   d_fusedoverlaps_args.data(), //Write out the values here
                                   d_nfused_overlaps.data(),
                                   reduction_op,
                                   n_overlaps,
                                   _cuda_stream);

    // memcpyD2H
    auto n_fused_overlap = cudautils::get_value_from_device(d_nfused_overlaps.data(), _cuda_stream);

    // construct overlap from the overlap args
    CreateOverlap fuse_op(d_anchors.data());
    device_buffer<Overlap> d_fused_overlaps(n_fused_overlap, _allocator, _cuda_stream); //Overlaps written here

    thrust::transform(thrust_exec_policy, d_fusedoverlaps_args.data(),
                      d_fusedoverlaps_args.data() + n_fused_overlap,
                      d_fused_overlaps.data(), fuse_op);

    device_buffer<Overlap> d_filtered_overlaps(n_fused_overlap, _allocator, _cuda_stream);

    FilterOverlapOp filterOp(min_residues, min_overlap_len, min_bases_per_residue, min_overlap_fraction, all_to_all);
    auto filtered_overlaps_end =
        thrust::copy_if(thrust_exec_policy,
                        d_fused_overlaps.data(), d_fused_overlaps.data() + n_fused_overlap,
                        d_filtered_overlaps.data(),
                        filterOp);

    auto n_filtered_overlaps = filtered_overlaps_end - d_filtered_overlaps.data();

    // memcpyD2H - move fused and filtered overlaps to host
    fused_overlaps.resize(n_filtered_overlaps);
    cudautils::device_copy_n(d_filtered_overlaps.data(), n_filtered_overlaps, fused_overlaps.data(), _cuda_stream);

    // This is not completely necessary, but if removed one has to make sure that the next step
    // uses the same stream or that sync is done in caller
    GW_CU_CHECK_ERR(hipStreamSynchronize(_cuda_stream));
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
