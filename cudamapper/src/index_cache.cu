/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "index_cache.cuh"

#include "index_host_copy.cuh"

#include <unordered_set>

#include <claraparabricks/genomeworks/cudamapper/index.hpp>
#include <claraparabricks/genomeworks/io/fasta_parser.hpp>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

IndexCacheHost::IndexCacheHost(const bool same_query_and_target,
                               genomeworks::DefaultDeviceAllocator allocator,
                               std::shared_ptr<genomeworks::io::FastaParser> query_parser,
                               std::shared_ptr<genomeworks::io::FastaParser> target_parser,
                               const std::uint64_t kmer_size,
                               const std::uint64_t window_size,
                               const bool hash_representations,
                               const double filtering_parameter,
                               const hipStream_t hip_stream)
    : same_query_and_target_(same_query_and_target)
    , allocator_(allocator)
    , query_parser_(query_parser)
    , target_parser_(target_parser)
    , kmer_size_(kmer_size)
    , window_size_(window_size)
    , hash_representations_(hash_representations)
    , filtering_parameter_(filtering_parameter)
    , cuda_stream_(hip_stream)
{
}

void IndexCacheHost::generate_query_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                                  const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                                  const bool skip_copy_to_host)
{
    generate_cache_content(descriptors_of_indices_to_cache,
                           descriptors_of_indices_to_keep_on_device,
                           skip_copy_to_host,
                           CacheSelector::query_cache);
}

void IndexCacheHost::generate_target_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                                   const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                                   const bool skip_copy_to_host)
{
    generate_cache_content(descriptors_of_indices_to_cache,
                           descriptors_of_indices_to_keep_on_device,
                           skip_copy_to_host,
                           CacheSelector::target_cache);
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_query_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    return get_index_from_cache(descriptor_of_index_to_cache,
                                CacheSelector::query_cache);
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_target_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    return get_index_from_cache(descriptor_of_index_to_cache,
                                CacheSelector::target_cache);
}

void IndexCacheHost::generate_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                            const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                            const bool skip_copy_to_host,
                                            const CacheSelector which_cache)
{
    // skip_copy_to_host only makes sense if descriptors_of_indices_to_cache and descriptors_of_indices_to_keep_on_device are the same
    // otherwise some indices would be created and not saved on either host or device
    assert(!skip_copy_to_host || (descriptors_of_indices_to_cache == descriptors_of_indices_to_keep_on_device));

    cache_type_t& cache_to_edit                           = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    const cache_type_t& cache_to_check                    = (CacheSelector::query_cache == which_cache) ? target_cache_ : query_cache_;
    device_cache_type_t& temp_device_cache_to_edit        = (CacheSelector::query_cache == which_cache) ? query_temp_device_cache_ : target_temp_device_cache_;
    const device_cache_type_t& temp_device_cache_to_check = (CacheSelector::query_cache == which_cache) ? target_temp_device_cache_ : query_temp_device_cache_;
    const genomeworks::io::FastaParser* parser            = (CacheSelector::query_cache == which_cache) ? query_parser_.get() : target_parser_.get();

    // convert descriptors_of_indices_to_keep_on_device into set for faster search
    std::unordered_set<IndexDescriptor, IndexDescriptorHash> descriptors_of_indices_to_keep_on_device_set(begin(descriptors_of_indices_to_keep_on_device),
                                                                                                          end(descriptors_of_indices_to_keep_on_device));

    cache_type_t new_cache;
    temp_device_cache_to_edit.clear(); // this should be empty by now anyway

    for (const IndexDescriptor& descriptor_of_index_to_cache : descriptors_of_indices_to_cache)
    {
        // check if this index should be kept on device in addition to copying it to host
        const bool keep_on_device = descriptors_of_indices_to_keep_on_device_set.count(descriptor_of_index_to_cache) != 0;

        std::shared_ptr<const IndexHostCopyBase> index_on_host = nullptr;
        std::shared_ptr<Index> index_on_device                 = nullptr;

        if (same_query_and_target_)
        {
            // check if the same index already exists in the other cache
            auto existing_cache = cache_to_check.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_check.end())
            {
                index_on_host = existing_cache->second;
                if (keep_on_device)
                {
                    auto existing_device_cache = temp_device_cache_to_check.find(descriptor_of_index_to_cache);
                    if (existing_device_cache != temp_device_cache_to_check.end())
                    {
                        index_on_device = existing_device_cache->second;
                    }
                    else
                    {
                        index_on_device = index_on_host->copy_index_to_device(allocator_, cuda_stream_);
                    }
                }
            }
        }

        if (nullptr == index_on_host)
        {
            // check if this index is already cached in this cache
            auto existing_cache = cache_to_edit.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_edit.end())
            {
                // index already cached
                index_on_host = existing_cache->second;
                if (keep_on_device)
                {
                    index_on_device = index_on_host->copy_index_to_device(allocator_, cuda_stream_);
                }
            }
            else
            {
                // create index
                index_on_device = Index::create_index(allocator_,
                                                      *parser,
                                                      descriptor_of_index_to_cache,
                                                      kmer_size_,
                                                      window_size_,
                                                      hash_representations_,
                                                      filtering_parameter_,
                                                      cuda_stream_);
                // copy it to host memory
                if (!skip_copy_to_host)
                {
                    index_on_host = IndexHostCopy::create_cache(*index_on_device,
                                                                descriptor_of_index_to_cache.first_read(),
                                                                kmer_size_,
                                                                window_size_,
                                                                cuda_stream_);
                }
            }
        }

        // save pointer to cached index
        if (!skip_copy_to_host)
        {
            assert(nullptr != index_on_host);
            new_cache[descriptor_of_index_to_cache] = index_on_host;
        }

        if (keep_on_device)
        {
            temp_device_cache_to_edit[descriptor_of_index_to_cache] = index_on_device;
        }
    }

    std::swap(new_cache, cache_to_edit);
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_cache(const IndexDescriptor& descriptor_of_index_to_cache,
                                                            const CacheSelector which_cache)
{
    std::shared_ptr<Index> index;

    const cache_type_t& host_cache               = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    device_cache_type_t& temp_device_index_cache = (CacheSelector::query_cache == which_cache) ? query_temp_device_cache_ : target_temp_device_cache_;

    auto temp_device_index_cache_iter = temp_device_index_cache.find(descriptor_of_index_to_cache);
    // check if index is present in device memory, copy from host if not
    if (temp_device_index_cache_iter != temp_device_index_cache.end())
    {
        index = temp_device_index_cache_iter->second;
        // indices are removed from device cache after they have been used for the first time
        temp_device_index_cache.erase(temp_device_index_cache_iter);
    }
    else
    {
        // TODO: throw custom exception if index not found
        index = host_cache.at(descriptor_of_index_to_cache)->copy_index_to_device(allocator_, cuda_stream_);
    }

    return index;
}

IndexCacheDevice::IndexCacheDevice(const bool same_query_and_target,
                                   std::shared_ptr<IndexCacheHost> index_cache_host)
    : same_query_and_target_(same_query_and_target)
    , index_cache_host_(index_cache_host)
{
}

void IndexCacheDevice::generate_query_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache)
{
    generate_cache_content(descriptors_of_indices_to_cache, CacheSelector::query_cache);
}

void IndexCacheDevice::generate_target_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache)
{
    generate_cache_content(descriptors_of_indices_to_cache, CacheSelector::target_cache);
}

std::shared_ptr<Index> IndexCacheDevice::get_index_from_query_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    // TODO: throw custom exception if index not found
    return query_cache_.at(descriptor_of_index_to_cache);
}

std::shared_ptr<Index> IndexCacheDevice::get_index_from_target_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    // TODO: throw custom exception if index not found
    return target_cache_.at(descriptor_of_index_to_cache);
}

void IndexCacheDevice::generate_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                              const CacheSelector which_cache)
{
    cache_type_t& cache_to_edit        = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    const cache_type_t& cache_to_check = (CacheSelector::query_cache == which_cache) ? target_cache_ : query_cache_;

    cache_type_t new_cache;

    for (const IndexDescriptor& descriptor_of_index_to_cache : descriptors_of_indices_to_cache)
    {

        std::shared_ptr<Index> index = nullptr;

        if (same_query_and_target_)
        {
            // check if the same index already exists in the other cache
            auto existing_cache = cache_to_check.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_check.end())
            {
                index = existing_cache->second;
            }
        }

        if (nullptr == index)
        {
            // check if this index is already cached in this cache
            auto existing_cache = cache_to_edit.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_edit.end())
            {
                // index already cached
                index = existing_cache->second;
            }
            else
            {
                // index not already cached -> fetch it from index_cache_host_
                if (CacheSelector::query_cache == which_cache)
                {
                    index = index_cache_host_->get_index_from_query_cache(descriptor_of_index_to_cache);
                }
                else
                {
                    index = index_cache_host_->get_index_from_target_cache(descriptor_of_index_to_cache);
                }
            }
        }

        assert(nullptr != index);

        // save pointer to cached index
        new_cache[descriptor_of_index_to_cache] = index;
    }

    std::swap(new_cache, cache_to_edit);
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
