/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "gtest/gtest.h"

#include "cudamapper_file_location.hpp"
#include "mock_index.cuh"

#include <algorithm>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <claraparabricks/genomeworks/utils/cudautils.hpp>
#include <claraparabricks/genomeworks/utils/signed_integer_utils.hpp>

#include "../src/matcher_gpu.cuh"

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{
void test_find_query_target_matches(const thrust::host_vector<representation_t>& query_representations_h,
                                    const thrust::host_vector<representation_t>& target_representations_h,
                                    const thrust::host_vector<std::int64_t>& expected_found_target_indices_h)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();

    CudaStream hip_stream = make_cuda_stream();

    device_buffer<representation_t> query_representations_d(query_representations_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(query_representations_h.data(), query_representations_h.size(), query_representations_d.data(), hip_stream.get()); // H2D
    device_buffer<representation_t> target_representations_d(target_representations_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(target_representations_h.data(), target_representations_h.size(), target_representations_d.data(), hip_stream.get()); // H2D
    device_buffer<int64_t> found_target_indices_d(query_representations_d.size(), allocator, hip_stream.get());

    details::matcher_gpu::find_query_target_matches(found_target_indices_d, query_representations_d, target_representations_d, hip_stream.get());

    thrust::host_vector<int64_t> found_target_indices_h(found_target_indices_d.size());
    cudautils::device_copy_n(found_target_indices_d.data(), found_target_indices_d.size(), found_target_indices_h.data(), hip_stream.get()); // D2H
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(found_target_indices_h.size(), expected_found_target_indices_h.size());

    for (int32_t i = 0; i < get_size(found_target_indices_h); ++i)
    {
        EXPECT_EQ(found_target_indices_h[i], expected_found_target_indices_h[i]) << "index: " << i;
    }
}

TEST(TestCudamapperMatcherGPU, test_find_query_target_matches_small_example)
{
    thrust::host_vector<representation_t> query_representations_h;
    query_representations_h.push_back(0);
    query_representations_h.push_back(12);
    query_representations_h.push_back(23);
    query_representations_h.push_back(32);
    query_representations_h.push_back(46);
    thrust::host_vector<representation_t> target_representations_h;
    target_representations_h.push_back(5);
    target_representations_h.push_back(12);
    target_representations_h.push_back(16);
    target_representations_h.push_back(23);
    target_representations_h.push_back(24);
    target_representations_h.push_back(25);
    target_representations_h.push_back(46);

    thrust::host_vector<int64_t> expected_found_target_indices_h;
    expected_found_target_indices_h.push_back(-1);
    expected_found_target_indices_h.push_back(1);
    expected_found_target_indices_h.push_back(3);
    expected_found_target_indices_h.push_back(-1);
    expected_found_target_indices_h.push_back(6);

    test_find_query_target_matches(query_representations_h, target_representations_h, expected_found_target_indices_h);
}

TEST(TestCudamapperMatcherGPU, test_find_query_target_matches_large_example)
{
    const std::int64_t total_query_representations = 1000000;

    thrust::host_vector<representation_t> query_representations_h;
    thrust::host_vector<representation_t> target_representations_h;

    for (std::int64_t i = 0; i < total_query_representations; ++i)
    {
        query_representations_h.push_back(i * 3);
    }

    thrust::host_vector<std::int64_t> expected_found_target_indices_h(query_representations_h.size(), -1);

    const representation_t max_representation = query_representations_h.back();
    for (representation_t r = 0; r < max_representation; r += 2)
    {
        target_representations_h.push_back(r);
        if (r % 3 == 0)
        {
            if (r / 3 < expected_found_target_indices_h.size())
            {
                expected_found_target_indices_h[r / 3] = get_size(target_representations_h) - 1;
            }
        }
    }

    test_find_query_target_matches(query_representations_h, target_representations_h, expected_found_target_indices_h);
}

void test_compute_number_of_anchors(const thrust::host_vector<std::uint32_t>& query_starting_index_of_each_representation_h,
                                    const thrust::host_vector<std::int64_t>& found_target_indices_h,
                                    const thrust::host_vector<std::uint32_t>& target_starting_index_of_each_representation_h,
                                    const thrust::host_vector<std::int64_t>& expected_anchor_starting_indices_h)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();

    CudaStream hip_stream = make_cuda_stream();

    device_buffer<std::uint32_t> query_starting_index_of_each_representation_d(query_starting_index_of_each_representation_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(query_starting_index_of_each_representation_h.data(), query_starting_index_of_each_representation_h.size(), query_starting_index_of_each_representation_d.data(), hip_stream.get()); //H2D
    device_buffer<std::uint32_t> target_starting_index_of_each_representation_d(target_starting_index_of_each_representation_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(target_starting_index_of_each_representation_h.data(), target_starting_index_of_each_representation_h.size(), target_starting_index_of_each_representation_d.data(), hip_stream.get()); //H2D
    device_buffer<std::int64_t> found_target_indices_d(found_target_indices_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(found_target_indices_h.data(), found_target_indices_h.size(), found_target_indices_d.data(), hip_stream.get()); // H2D
    device_buffer<std::int64_t> anchor_starting_indices_d(found_target_indices_h.size(), allocator);
    cudautils::device_copy_n(found_target_indices_h.data(), found_target_indices_h.size(), found_target_indices_d.data(), hip_stream.get()); // H2D

    details::matcher_gpu::compute_anchor_starting_indices(anchor_starting_indices_d, query_starting_index_of_each_representation_d, found_target_indices_d, target_starting_index_of_each_representation_d, hip_stream.get());

    thrust::host_vector<std::int64_t> anchor_starting_indices_h(anchor_starting_indices_d.size());
    cudautils::device_copy_n(anchor_starting_indices_d.data(), anchor_starting_indices_d.size(), anchor_starting_indices_h.data(), hip_stream.get()); // D2H
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    for (int32_t i = 0; i < get_size(found_target_indices_h); ++i)
    {
        EXPECT_EQ(anchor_starting_indices_h[i], expected_anchor_starting_indices_h[i]);
    }
}

TEST(TestCudamapperMatcherGPU, test_compute_number_of_anchors_small_example)
{
    thrust::host_vector<representation_t> query_starting_index_of_each_representation_h;
    query_starting_index_of_each_representation_h.push_back(0);
    query_starting_index_of_each_representation_h.push_back(4);
    query_starting_index_of_each_representation_h.push_back(10);
    query_starting_index_of_each_representation_h.push_back(13);
    query_starting_index_of_each_representation_h.push_back(18);
    query_starting_index_of_each_representation_h.push_back(21);

    thrust::host_vector<representation_t> target_starting_index_of_each_representation_h;
    target_starting_index_of_each_representation_h.push_back(0);
    target_starting_index_of_each_representation_h.push_back(3);
    target_starting_index_of_each_representation_h.push_back(7);
    target_starting_index_of_each_representation_h.push_back(9);
    target_starting_index_of_each_representation_h.push_back(13);
    target_starting_index_of_each_representation_h.push_back(16);
    target_starting_index_of_each_representation_h.push_back(18);
    target_starting_index_of_each_representation_h.push_back(21);

    thrust::host_vector<int64_t> found_target_indices_h;
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(1);
    found_target_indices_h.push_back(3);
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(6);

    thrust::host_vector<int64_t> expected_anchor_starting_indices;
    expected_anchor_starting_indices.push_back(0);
    expected_anchor_starting_indices.push_back(24);
    expected_anchor_starting_indices.push_back(36);
    expected_anchor_starting_indices.push_back(36);
    expected_anchor_starting_indices.push_back(45);

    test_compute_number_of_anchors(query_starting_index_of_each_representation_h,
                                   found_target_indices_h,
                                   target_starting_index_of_each_representation_h,
                                   expected_anchor_starting_indices);
}

TEST(TestCudamapperMatcherGPU, test_compute_number_of_anchors_large_example)
{
    const std::int64_t length = 100000;

    thrust::host_vector<representation_t> query_starting_index_of_each_representation_h;
    thrust::host_vector<representation_t> target_starting_index_of_each_representation_h;
    thrust::host_vector<std::int64_t> found_target_indices_h(length - 1, -1);
    thrust::host_vector<std::int64_t> expected_anchor_starting_indices_h;
    std::int64_t expected_n_anchors = 0;
    for (std::int64_t i = 0; i < length; ++i)
    {
        query_starting_index_of_each_representation_h.push_back(2 * i);
        target_starting_index_of_each_representation_h.push_back(10 * i + i % 10);
        if (i % 3 == 0 && i < length - 1)
        {
            found_target_indices_h[i] = i;
            expected_n_anchors += 2 * (10 + (i + 1) % 10 - i % 10);
        }
        if (i < length - 1)
            expected_anchor_starting_indices_h.push_back(expected_n_anchors);
    }

    test_compute_number_of_anchors(query_starting_index_of_each_representation_h,
                                   found_target_indices_h,
                                   target_starting_index_of_each_representation_h,
                                   expected_anchor_starting_indices_h);
}

void test_generate_anchors(
    const thrust::host_vector<Anchor>& expected_anchors_h,
    const thrust::host_vector<std::int64_t>& anchor_starting_indices_h,
    const thrust::host_vector<std::uint32_t>& query_starting_index_of_each_representation_h,
    const thrust::host_vector<std::int64_t>& found_target_indices_h,
    const thrust::host_vector<std::uint32_t>& target_starting_index_of_each_representation_h,
    const thrust::host_vector<read_id_t>& query_read_ids_h,
    const thrust::host_vector<position_in_read_t>& query_positions_in_read_h,
    const thrust::host_vector<read_id_t>& target_read_ids_h,
    const thrust::host_vector<position_in_read_t>& target_positions_in_read_h,
    const read_id_t smallest_query_read_id,
    const read_id_t smallest_target_read_id,
    const read_id_t number_of_query_reads,
    const read_id_t number_of_target_reads,
    const position_in_read_t max_basepairs_in_query_reads,
    const position_in_read_t max_basepairs_in_target_reads)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();

    CudaStream hip_stream = make_cuda_stream();

    device_buffer<std::int64_t> anchor_starting_indices_d(anchor_starting_indices_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchor_starting_indices_h.data(), anchor_starting_indices_h.size(), anchor_starting_indices_d.data(), hip_stream.get()); // H2D
    device_buffer<std::uint32_t> query_starting_index_of_each_representation_d(query_starting_index_of_each_representation_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(query_starting_index_of_each_representation_h.data(), query_starting_index_of_each_representation_h.size(), query_starting_index_of_each_representation_d.data(), hip_stream.get()); // H2D
    device_buffer<std::int64_t> found_target_indices_d(found_target_indices_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(found_target_indices_h.data(), found_target_indices_h.size(), found_target_indices_d.data(), hip_stream.get()); //H2D
    device_buffer<std::uint32_t> target_starting_index_of_each_representation_d(target_starting_index_of_each_representation_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(target_starting_index_of_each_representation_h.data(), target_starting_index_of_each_representation_h.size(), target_starting_index_of_each_representation_d.data(), hip_stream.get()); // H2D
    device_buffer<read_id_t> query_read_ids_d(query_read_ids_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(query_read_ids_h.data(), query_read_ids_h.size(), query_read_ids_d.data(), hip_stream.get()); // H2D
    device_buffer<position_in_read_t> query_positions_in_read_d(query_positions_in_read_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(query_positions_in_read_h.data(), query_positions_in_read_h.size(), query_positions_in_read_d.data(), hip_stream.get()); // H2D
    device_buffer<read_id_t> target_read_ids_d(target_read_ids_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(target_read_ids_h.data(), target_read_ids_h.size(), target_read_ids_d.data(), hip_stream.get()); //H2D
    device_buffer<position_in_read_t> target_positions_in_read_d(target_positions_in_read_h.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(target_positions_in_read_h.data(), target_positions_in_read_h.size(), target_positions_in_read_d.data(), hip_stream.get()); //H2D

    device_buffer<Anchor> anchors_d(anchor_starting_indices_h.back(), allocator, hip_stream.get());

    MockIndex query_index(allocator);
    EXPECT_CALL(query_index, first_occurrence_of_representations).WillRepeatedly(testing::ReturnRef(query_starting_index_of_each_representation_d));
    EXPECT_CALL(query_index, read_ids).WillRepeatedly(testing::ReturnRef(query_read_ids_d));
    EXPECT_CALL(query_index, positions_in_reads).WillRepeatedly(testing::ReturnRef(query_positions_in_read_d));
    EXPECT_CALL(query_index, smallest_read_id).WillRepeatedly(testing::Return(smallest_query_read_id));
    EXPECT_CALL(query_index, number_of_reads).WillRepeatedly(testing::Return(number_of_query_reads));
    EXPECT_CALL(query_index, number_of_basepairs_in_longest_read).WillRepeatedly(testing::Return(max_basepairs_in_query_reads));

    MockIndex target_index(allocator);
    EXPECT_CALL(target_index, first_occurrence_of_representations).WillRepeatedly(testing::ReturnRef(target_starting_index_of_each_representation_d));
    EXPECT_CALL(target_index, read_ids).WillRepeatedly(testing::ReturnRef(target_read_ids_d));
    EXPECT_CALL(target_index, positions_in_reads).WillRepeatedly(testing::ReturnRef(target_positions_in_read_d));
    EXPECT_CALL(target_index, smallest_read_id).WillRepeatedly(testing::Return(smallest_target_read_id));
    EXPECT_CALL(target_index, number_of_reads).WillRepeatedly(testing::Return(number_of_target_reads));
    EXPECT_CALL(target_index, number_of_basepairs_in_longest_read).WillRepeatedly(testing::Return(max_basepairs_in_target_reads));

    details::matcher_gpu::generate_anchors_dispatcher(anchors_d,
                                                      anchor_starting_indices_d,
                                                      found_target_indices_d,
                                                      query_index,
                                                      target_index);

    thrust::host_vector<Anchor> anchors_h(anchors_d.size());
    cudautils::device_copy_n(anchors_d.data(), anchors_d.size(), anchors_h.data(), hip_stream.get()); // D2H
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(anchors_h.size(), expected_anchors_h.size());

    for (int64_t i = 0; i < get_size(anchors_h); ++i)
    {
        EXPECT_EQ(anchors_h[i].query_read_id_, expected_anchors_h[i].query_read_id_) << " index: " << i;
        EXPECT_EQ(anchors_h[i].query_position_in_read_, expected_anchors_h[i].query_position_in_read_) << " index: " << i;
        EXPECT_EQ(anchors_h[i].target_read_id_, expected_anchors_h[i].target_read_id_) << " index: " << i;
        EXPECT_EQ(anchors_h[i].target_position_in_read_, expected_anchors_h[i].target_position_in_read_) << " index: " << i;
    }
}

TEST(TestCudamapperMatcherGPU, test_generate_anchors_small_example_32_bit_positions)
{
    thrust::host_vector<representation_t> query_starting_index_of_each_representation_h;
    query_starting_index_of_each_representation_h.push_back(0);  // query_section_0, 4 elements
    query_starting_index_of_each_representation_h.push_back(4);  // query_section_1, 6 elements, paired with target_section_1
    query_starting_index_of_each_representation_h.push_back(10); // query_section_2, 3 elements, paired with target_section_3
    query_starting_index_of_each_representation_h.push_back(13); // query_section_3, 5 elements
    query_starting_index_of_each_representation_h.push_back(18); // query_section_4, 3 elements, paired with target_section_6
    query_starting_index_of_each_representation_h.push_back(21); // past_the_last

    thrust::host_vector<representation_t> target_starting_index_of_each_representation_h;
    target_starting_index_of_each_representation_h.push_back(0);  // target_section_0, 3 elements
    target_starting_index_of_each_representation_h.push_back(3);  // target_section_1, 4 elements, paired with query_section_1
    target_starting_index_of_each_representation_h.push_back(7);  // target_section_2, 2 elements
    target_starting_index_of_each_representation_h.push_back(9);  // target_section_3, 4 elements, paired with query_section_2
    target_starting_index_of_each_representation_h.push_back(13); // target_section_4, 3 elements
    target_starting_index_of_each_representation_h.push_back(16); // target_section_5, 2 elements
    target_starting_index_of_each_representation_h.push_back(18); // target_section_6, 3 elements, paired with query_section_4
    target_starting_index_of_each_representation_h.push_back(21); // past_the_last

    // pairing of representation is deifned here
    thrust::host_vector<int64_t> found_target_indices_h;
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(1);
    found_target_indices_h.push_back(3);
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(6);

    thrust::host_vector<int64_t> anchor_starting_indices_h;
    anchor_starting_indices_h.push_back(0);  // no pair for query_section_0
    anchor_starting_indices_h.push_back(24); // 24 anchors = 6 * 4
    anchor_starting_indices_h.push_back(36); // 12 anchors = 3 * 4
    anchor_starting_indices_h.push_back(36); // no pair for query_section_3
    anchor_starting_indices_h.push_back(45); // 9 anchors = 3 * 3

    const read_id_t smallest_query_read_id                 = 500;
    const read_id_t smallest_target_read_id                = 10000;
    const read_id_t number_of_query_reads                  = 20;
    const read_id_t number_of_target_reads                 = 2000;
    const position_in_read_t max_basepairs_in_query_reads  = 200;
    const position_in_read_t max_basepairs_in_target_reads = 20000;

    // query read_ids range from smallest_query_read_id to smallest_query_read_id + 20
    // query positions_in_reads range from 0 to 200
    thrust::host_vector<read_id_t> query_read_ids_h;
    thrust::host_vector<position_in_read_t> query_positions_in_read_h;
    for (std::uint32_t i = 0; i < query_starting_index_of_each_representation_h.back(); ++i)
    {
        query_read_ids_h.push_back(smallest_query_read_id + i);
        query_positions_in_read_h.push_back(10 * i);
    }

    // target read_ids range from smallest_target_read_id to smallest_target_read_id + 2000
    // target positions_in_read range from 0 to 20000
    thrust::host_vector<read_id_t> target_read_ids_h;
    thrust::host_vector<position_in_read_t> target_positions_in_read_h;
    for (std::uint32_t i = 0; i < target_starting_index_of_each_representation_h.back(); ++i)
    {
        target_read_ids_h.push_back(smallest_target_read_id + 100 * i);
        target_positions_in_read_h.push_back(1000 * i);
    }

    thrust::host_vector<Anchor> expected_anchors(anchor_starting_indices_h.back());
    for (int32_t i = 0; i < 6; ++i)
        for (int32_t j = 0; j < 4; ++j)
        {
            Anchor a;
            a.query_read_id_            = smallest_query_read_id + 4 + i;
            a.query_position_in_read_   = 10 * (4 + i);
            a.target_read_id_           = smallest_target_read_id + 100 * (j + 3);
            a.target_position_in_read_  = 1000 * (j + 3);
            expected_anchors[i * 4 + j] = a;
        }

    for (int32_t i = 0; i < 3; ++i)
        for (int32_t j = 0; j < 4; ++j)
        {
            Anchor a;
            a.query_read_id_                 = smallest_query_read_id + 10 + i;
            a.query_position_in_read_        = 10 * (10 + i);
            a.target_read_id_                = smallest_target_read_id + 100 * (j + 9);
            a.target_position_in_read_       = 1000 * (j + 9);
            expected_anchors[i * 4 + j + 24] = a;
        }

    for (int32_t i = 0; i < 3; ++i)
        for (int32_t j = 0; j < 3; ++j)
        {
            Anchor a;
            a.query_read_id_                 = smallest_query_read_id + 18 + i;
            a.query_position_in_read_        = 10 * (18 + i);
            a.target_read_id_                = smallest_target_read_id + 100 * (j + 18);
            a.target_position_in_read_       = 1000 * (j + 18);
            expected_anchors[i * 3 + j + 36] = a;
        }

    std::sort(std::begin(expected_anchors),
              std::end(expected_anchors),
              [](const Anchor& i, const Anchor& j) -> bool {
                  return (i.query_read_id_ < j.query_read_id_) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ < j.target_read_id_)) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ == j.target_read_id_) &&
                          (i.query_position_in_read_ < j.query_position_in_read_)) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ == j.target_read_id_) &&
                          (i.query_position_in_read_ == j.query_position_in_read_) &&
                          (i.target_position_in_read_ < j.target_position_in_read_));
              });

    test_generate_anchors(
        expected_anchors,
        anchor_starting_indices_h,
        query_starting_index_of_each_representation_h,
        found_target_indices_h,
        target_starting_index_of_each_representation_h,
        query_read_ids_h,
        query_positions_in_read_h,
        target_read_ids_h,
        target_positions_in_read_h,
        smallest_query_read_id,
        smallest_target_read_id,
        number_of_query_reads,
        number_of_target_reads,
        max_basepairs_in_query_reads,
        max_basepairs_in_target_reads);
}

TEST(TestCudamapperMatcherGPU, test_generate_anchors_small_example_64_bit_positions)
{
    thrust::host_vector<representation_t> query_starting_index_of_each_representation_h;
    query_starting_index_of_each_representation_h.push_back(0);  // query_section_0, 4 elements
    query_starting_index_of_each_representation_h.push_back(4);  // query_section_1, 6 elements, paired with target_section_1
    query_starting_index_of_each_representation_h.push_back(10); // query_section_2, 3 elements, paired with target_section_3
    query_starting_index_of_each_representation_h.push_back(13); // query_section_3, 5 elements
    query_starting_index_of_each_representation_h.push_back(18); // query_section_4, 3 elements, paired with target_section_6
    query_starting_index_of_each_representation_h.push_back(21); // past_the_last

    thrust::host_vector<representation_t> target_starting_index_of_each_representation_h;
    target_starting_index_of_each_representation_h.push_back(0);  // target_section_0, 3 elements
    target_starting_index_of_each_representation_h.push_back(3);  // target_section_1, 4 elements, paired with query_section_1
    target_starting_index_of_each_representation_h.push_back(7);  // target_section_2, 2 elements
    target_starting_index_of_each_representation_h.push_back(9);  // target_section_3, 4 elements, paired with query_section_2
    target_starting_index_of_each_representation_h.push_back(13); // target_section_4, 3 elements
    target_starting_index_of_each_representation_h.push_back(16); // target_section_5, 2 elements
    target_starting_index_of_each_representation_h.push_back(18); // target_section_6, 3 elements, paired with query_section_4
    target_starting_index_of_each_representation_h.push_back(21); // past_the_last

    // pairing of representation is deifned here
    thrust::host_vector<int64_t> found_target_indices_h;
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(1);
    found_target_indices_h.push_back(3);
    found_target_indices_h.push_back(-1);
    found_target_indices_h.push_back(6);

    thrust::host_vector<int64_t> anchor_starting_indices_h;
    anchor_starting_indices_h.push_back(0);  // no pair for query_section_0
    anchor_starting_indices_h.push_back(24); // 24 anchors = 6 * 4
    anchor_starting_indices_h.push_back(36); // 12 anchors = 3 * 4
    anchor_starting_indices_h.push_back(36); // no pair for query_section_3
    anchor_starting_indices_h.push_back(45); // 9 anchors = 3 * 3

    thrust::host_vector<read_id_t> query_read_ids_h;
    query_read_ids_h.push_back(5000); // query_section_0
    query_read_ids_h.push_back(6000);
    query_read_ids_h.push_back(7000);
    query_read_ids_h.push_back(8000);
    query_read_ids_h.push_back(2000); // query_section_1
    query_read_ids_h.push_back(3000);
    query_read_ids_h.push_back(4000);
    query_read_ids_h.push_back(4000);
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(1000); // query_section_2
    query_read_ids_h.push_back(2000);
    query_read_ids_h.push_back(8000);
    query_read_ids_h.push_back(4000); // query_section_3
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(5000);
    query_read_ids_h.push_back(4000); // query_section_4
    query_read_ids_h.push_back(6000);
    query_read_ids_h.push_back(7000);
    thrust::host_vector<position_in_read_t> query_positions_in_read_h;
    query_positions_in_read_h.push_back(100700); // query_section_0
    query_positions_in_read_h.push_back(100800);
    query_positions_in_read_h.push_back(100200);
    query_positions_in_read_h.push_back(100400);
    query_positions_in_read_h.push_back(100500); // query_section_1
    query_positions_in_read_h.push_back(100400);
    query_positions_in_read_h.push_back(100100);
    query_positions_in_read_h.push_back(100300);
    query_positions_in_read_h.push_back(100800);
    query_positions_in_read_h.push_back(100900);
    query_positions_in_read_h.push_back(100100); // query_section_2
    query_positions_in_read_h.push_back(100200);
    query_positions_in_read_h.push_back(100400);
    query_positions_in_read_h.push_back(100500); // query_section_3
    query_positions_in_read_h.push_back(100600);
    query_positions_in_read_h.push_back(100700);
    query_positions_in_read_h.push_back(100800);
    query_positions_in_read_h.push_back(100900);
    query_positions_in_read_h.push_back(100200); // query_section_4
    query_positions_in_read_h.push_back(100400);
    query_positions_in_read_h.push_back(100800);

    thrust::host_vector<read_id_t> target_read_ids_h;
    target_read_ids_h.push_back(7006); // target_section_0
    target_read_ids_h.push_back(7008);
    target_read_ids_h.push_back(7009);
    target_read_ids_h.push_back(7001); // target_section_1
    target_read_ids_h.push_back(7001);
    target_read_ids_h.push_back(7005);
    target_read_ids_h.push_back(7006);
    target_read_ids_h.push_back(7008); // target_section_2
    target_read_ids_h.push_back(7009);
    target_read_ids_h.push_back(7004); // target_section_3
    target_read_ids_h.push_back(7004);
    target_read_ids_h.push_back(7004);
    target_read_ids_h.push_back(7005);
    target_read_ids_h.push_back(7002); // target_section_4
    target_read_ids_h.push_back(7002);
    target_read_ids_h.push_back(7008);
    target_read_ids_h.push_back(7005); // target_section_5
    target_read_ids_h.push_back(7006);
    target_read_ids_h.push_back(7006); // target_section_6
    target_read_ids_h.push_back(7006);
    target_read_ids_h.push_back(7006);
    thrust::host_vector<position_in_read_t> target_positions_in_read_h;
    target_positions_in_read_h.push_back(2540000080); // target_section_0
    target_positions_in_read_h.push_back(2540000090);
    target_positions_in_read_h.push_back(2540000040);
    target_positions_in_read_h.push_back(2540000020); // target_section_1
    target_positions_in_read_h.push_back(2540000040);
    target_positions_in_read_h.push_back(2540000050);
    target_positions_in_read_h.push_back(2540000040);
    target_positions_in_read_h.push_back(2540000030); // target_section_2
    target_positions_in_read_h.push_back(2540000020);
    target_positions_in_read_h.push_back(2540000020); // target_section_3
    target_positions_in_read_h.push_back(2540000080);
    target_positions_in_read_h.push_back(2540000060);
    target_positions_in_read_h.push_back(2540000070);
    target_positions_in_read_h.push_back(2540000080); // target_section_4
    target_positions_in_read_h.push_back(2540000040);
    target_positions_in_read_h.push_back(2540000010);
    target_positions_in_read_h.push_back(2540000020); // target_section_5
    target_positions_in_read_h.push_back(2540000050);
    target_positions_in_read_h.push_back(2540000040); // target_section_6
    target_positions_in_read_h.push_back(2540000050);
    target_positions_in_read_h.push_back(2540000080);

    const read_id_t smallest_query_read_id                 = 1000;
    const read_id_t smallest_target_read_id                = 7001;
    const read_id_t number_of_query_reads                  = 8000 - 1000;
    const read_id_t number_of_target_reads                 = 7009 - 7001;
    const position_in_read_t max_basepairs_in_query_reads  = 100900;
    const position_in_read_t max_basepairs_in_target_reads = 2540000090;

    // generate anchors

    thrust::host_vector<Anchor> expected_anchors;

    // query_section_1 * target_section_1
    for (std::uint32_t query_idx = query_starting_index_of_each_representation_h[1]; query_idx < query_starting_index_of_each_representation_h[2]; ++query_idx)
    {
        for (std::uint32_t target_idx = target_starting_index_of_each_representation_h[1]; target_idx < target_starting_index_of_each_representation_h[2]; ++target_idx)
        {
            Anchor a;
            a.query_read_id_           = query_read_ids_h[query_idx];
            a.query_position_in_read_  = query_positions_in_read_h[query_idx];
            a.target_read_id_          = target_read_ids_h[target_idx];
            a.target_position_in_read_ = target_positions_in_read_h[target_idx];
            expected_anchors.push_back(a);
        }
    }

    // query_section_2 * target_section_3
    for (std::uint32_t query_idx = query_starting_index_of_each_representation_h[2]; query_idx < query_starting_index_of_each_representation_h[3]; ++query_idx)
    {
        for (std::uint32_t target_idx = target_starting_index_of_each_representation_h[3]; target_idx < target_starting_index_of_each_representation_h[4]; ++target_idx)
        {
            Anchor a;
            a.query_read_id_           = query_read_ids_h[query_idx];
            a.query_position_in_read_  = query_positions_in_read_h[query_idx];
            a.target_read_id_          = target_read_ids_h[target_idx];
            a.target_position_in_read_ = target_positions_in_read_h[target_idx];
            expected_anchors.push_back(a);
        }
    }

    // query_section_4 * target_section_6
    for (std::uint32_t query_idx = query_starting_index_of_each_representation_h[4]; query_idx < query_starting_index_of_each_representation_h[5]; ++query_idx)
    {
        for (std::uint32_t target_idx = target_starting_index_of_each_representation_h[6]; target_idx < target_starting_index_of_each_representation_h[7]; ++target_idx)
        {
            Anchor a;
            a.query_read_id_           = query_read_ids_h[query_idx];
            a.query_position_in_read_  = query_positions_in_read_h[query_idx];
            a.target_read_id_          = target_read_ids_h[target_idx];
            a.target_position_in_read_ = target_positions_in_read_h[target_idx];
            expected_anchors.push_back(a);
        }
    }

    // sort anchors
    std::sort(std::begin(expected_anchors),
              std::end(expected_anchors),
              [](const Anchor& i, const Anchor& j) -> bool {
                  return (i.query_read_id_ < j.query_read_id_) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ < j.target_read_id_)) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ == j.target_read_id_) &&
                          (i.query_position_in_read_ < j.query_position_in_read_)) ||
                         ((i.query_read_id_ == j.query_read_id_) &&
                          (i.target_read_id_ == j.target_read_id_) &&
                          (i.query_position_in_read_ == j.query_position_in_read_) &&
                          (i.target_position_in_read_ < j.target_position_in_read_));
              });

    test_generate_anchors(
        expected_anchors,
        anchor_starting_indices_h,
        query_starting_index_of_each_representation_h,
        found_target_indices_h,
        target_starting_index_of_each_representation_h,
        query_read_ids_h,
        query_positions_in_read_h,
        target_read_ids_h,
        target_positions_in_read_h,
        smallest_query_read_id,
        smallest_target_read_id,
        number_of_query_reads,
        number_of_target_reads,
        max_basepairs_in_query_reads,
        max_basepairs_in_target_reads);
}

TEST(TestCudamapperMatcherGPU, OneReadOneMinimizer)
{
    DefaultDeviceAllocator allocator        = create_default_device_allocator();
    std::unique_ptr<io::FastaParser> parser = io::create_kseq_fasta_parser(std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/gatt.fasta");
    std::unique_ptr<Index> query_index      = Index::create_index(allocator, *parser, 0, parser->get_num_seqences(), 4, 1);
    std::unique_ptr<Index> target_index     = Index::create_index(allocator, *parser, 0, parser->get_num_seqences(), 4, 1);
    MatcherGPU matcher(allocator, *query_index, *target_index);

    thrust::host_vector<Anchor> anchors(matcher.anchors().size());
    cudautils::device_copy_n(matcher.anchors().data(), matcher.anchors().size(), anchors.data()); // D2H
    ASSERT_EQ(get_size(anchors), 1);
}

TEST(TestCudamapperMatcherGPU, AtLeastOneIndexEmpty)
{
    DefaultDeviceAllocator allocator        = create_default_device_allocator();
    std::unique_ptr<io::FastaParser> parser = io::create_kseq_fasta_parser(std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/gatt.fasta");
    std::unique_ptr<Index> index_full       = Index::create_index(allocator, *parser, 0, parser->get_num_seqences(), 4, 1);
    std::unique_ptr<Index> index_empty      = Index::create_index(allocator, *parser, 0, parser->get_num_seqences(), 5, 1); // kmer longer than read

    {
        MatcherGPU matcher(allocator, *index_full, *index_empty);
        thrust::host_vector<Anchor> anchors(matcher.anchors().size());
        cudautils::device_copy_n(matcher.anchors().data(), matcher.anchors().size(), anchors.data()); // D2H
        EXPECT_EQ(get_size(anchors), 0);
    }
    {
        MatcherGPU matcher(allocator, *index_empty, *index_full);
        thrust::host_vector<Anchor> anchors(matcher.anchors().size());
        cudautils::device_copy_n(matcher.anchors().data(), matcher.anchors().size(), anchors.data()); // D2H
        EXPECT_EQ(get_size(anchors), 0);
    }
    {
        MatcherGPU matcher(allocator, *index_empty, *index_empty);
        thrust::host_vector<Anchor> anchors(matcher.anchors().size());
        cudautils::device_copy_n(matcher.anchors().data(), matcher.anchors().size(), anchors.data()); // D2H
        EXPECT_EQ(get_size(anchors), 0);
    }
    {
        MatcherGPU matcher(allocator, *index_full, *index_full);
        thrust::host_vector<Anchor> anchors(matcher.anchors().size());
        cudautils::device_copy_n(matcher.anchors().data(), matcher.anchors().size(), anchors.data()); // D2H
        EXPECT_EQ(get_size(anchors), 1);
    }
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
