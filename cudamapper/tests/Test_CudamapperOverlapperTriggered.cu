/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "gtest/gtest.h"

#include "../src/overlapper_triggered.hpp"
#include "../src/cudamapper_utils.hpp"

#include "mock_fasta_parser.hpp"

#include <algorithm>
#include <numeric>
#include <random>

#include "cudamapper_file_location.hpp"

namespace claragenomics
{
namespace cudamapper
{

TEST(TestCudamapperOverlapperTriggerred, FuseTwoOverlaps)
{
    std::vector<Overlap> unfused_overlaps;

    Overlap overlap1;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    unfused_overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 10000;
    overlap2.query_end_position_in_read_    = 12000;
    overlap2.target_start_position_in_read_ = 14000;
    overlap2.target_end_position_in_read_   = 15000;

    unfused_overlaps.push_back(overlap2);
    std::vector<Overlap> fused_overlaps;
    fuse_overlaps(fused_overlaps, unfused_overlaps);

    ASSERT_EQ(fused_overlaps.size(), 1u);
}

TEST(TestCudamapperOverlapperTriggerred, DoNotuseTwoOverlaps)
{
    std::vector<Overlap> unfused_overlaps;

    Overlap overlap1;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 23;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    unfused_overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 10000;
    overlap2.query_end_position_in_read_    = 12000;
    overlap2.target_start_position_in_read_ = 14000;
    overlap2.target_end_position_in_read_   = 15000;

    unfused_overlaps.push_back(overlap2);
    std::vector<Overlap> fused_overlaps;
    fuse_overlaps(fused_overlaps, unfused_overlaps);

    ASSERT_EQ(fused_overlaps.size(), 2u);
}

TEST(TestCudamapperOverlapperTriggerred, OneOverlap)
{
    std::vector<Overlap> unfused_overlaps;

    Overlap overlap1;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 23;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    unfused_overlaps.push_back(overlap1);

    std::vector<Overlap> fused_overlaps;
    fuse_overlaps(fused_overlaps, unfused_overlaps);

    ASSERT_EQ(fused_overlaps.size(), 1u);
}

TEST(TestCudamapperOverlapperTriggerred, NoOverlaps)
{
    std::vector<Overlap> unfused_overlaps;

    std::vector<Overlap> fused_overlaps;
    fuse_overlaps(fused_overlaps, unfused_overlaps);

    ASSERT_EQ(fused_overlaps.size(), 0u);
}

TEST(TestCudamapperOverlapperTriggerred, Fusee3Overlapsto2)
{
    std::vector<Overlap> unfused_overlaps;

    Overlap overlap1;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 23;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    unfused_overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 23;
    overlap2.query_start_position_in_read_  = 10000;
    overlap2.query_end_position_in_read_    = 12000;
    overlap2.target_start_position_in_read_ = 14000;
    overlap2.target_end_position_in_read_   = 15000;
    unfused_overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.query_read_id_                 = 27;
    overlap3.target_read_id_                = 29;
    overlap3.query_start_position_in_read_  = 10000;
    overlap3.query_end_position_in_read_    = 12000;
    overlap3.target_start_position_in_read_ = 14000;
    overlap3.target_end_position_in_read_   = 15000;

    unfused_overlaps.push_back(overlap3);

    std::vector<Overlap> fused_overlaps;
    fuse_overlaps(fused_overlaps, unfused_overlaps);

    ASSERT_EQ(fused_overlaps.size(), 2u);
}

TEST(TestCudamapperOverlapperTriggerred, OneAchorNoOverlaps)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator, hip_stream);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    std::vector<std::string> testv;
    testv.push_back("READ0");
    testv.push_back("READ1");
    testv.push_back("READ2");
    std::vector<std::uint32_t> test_read_length(testv.size(), 1000);

    Anchor anchor1;
    anchors.push_back(anchor1);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 0u);

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsOneOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator, hip_stream);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    MockFastaParser test_parser;
    std::vector<io::FastaSequence> fasta_sequences;
    fasta_sequences.push_back({"READ0", {}});
    fasta_sequences.push_back({"READ1", {}});
    fasta_sequences.push_back({"READ2", {}});

    for (std::size_t i = 0; i < fasta_sequences.size(); ++i)
    {
        EXPECT_CALL(test_parser, get_sequence_by_id(i)).WillRepeatedly(testing::ReturnRef(fasta_sequences[i]));
    }

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1300;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0, 1000);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_EQ(overlaps[0].query_read_id_, 1u);
    ASSERT_EQ(overlaps[0].target_read_id_, 2u);
    ASSERT_EQ(overlaps[0].query_start_position_in_read_, 100u);
    ASSERT_EQ(overlaps[0].query_end_position_in_read_, 400u);
    ASSERT_EQ(overlaps[0].target_start_position_in_read_, 1000u);
    ASSERT_EQ(overlaps[0].target_end_position_in_read_, 1300u);

    overlapper.update_read_names(overlaps, test_parser, test_parser);
    ASSERT_STREQ(overlaps[0].query_read_name_, fasta_sequences[1].name.c_str());
    ASSERT_STREQ(overlaps[0].target_read_name_, fasta_sequences[2].name.c_str());

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsNoOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator, hip_stream);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 3;
    anchor2.target_read_id_          = 4;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 5;
    anchor3.target_read_id_          = 6;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 8;
    anchor4.target_read_id_          = 9;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1300;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0, 1000);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 0u);

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, FourColinearAnchorsOneOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200 * 10;
    anchor2.target_position_in_read_ = 1100 * 10;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300 * 10;
    anchor3.target_position_in_read_ = 1200 * 10;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400 * 10;
    anchor4.target_position_in_read_ = 1300 * 10;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 0u);

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsLastNotInOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    MockFastaParser test_parser;
    std::vector<io::FastaSequence> fasta_sequences;
    fasta_sequences.push_back({"READ0", {}});
    fasta_sequences.push_back({"READ1", {}});
    fasta_sequences.push_back({"READ2", {}});

    for (std::size_t i = 0; i < fasta_sequences.size(); ++i)
    {
        EXPECT_CALL(test_parser, get_sequence_by_id(i)).WillRepeatedly(testing::ReturnRef(fasta_sequences[i]));
    }

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400 + 2000;
    anchor4.target_position_in_read_ = 1300 + 2000;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0, 1000);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_EQ(overlaps[0].query_read_id_, 1u);
    ASSERT_EQ(overlaps[0].target_read_id_, 2u);
    ASSERT_EQ(overlaps[0].query_start_position_in_read_, 100u);
    ASSERT_EQ(overlaps[0].query_end_position_in_read_, 300u);
    ASSERT_EQ(overlaps[0].target_start_position_in_read_, 1000u);
    ASSERT_EQ(overlaps[0].target_end_position_in_read_, 1200u);

    overlapper.update_read_names(overlaps, test_parser, test_parser);
    ASSERT_STREQ(overlaps[0].query_read_name_, fasta_sequences[1].name.c_str());
    ASSERT_STREQ(overlaps[0].target_read_name_, fasta_sequences[2].name.c_str());

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, ReverseStrand)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    MockFastaParser test_parser;
    std::vector<io::FastaSequence> fasta_sequences;
    fasta_sequences.push_back({"READ0", {}});
    fasta_sequences.push_back({"READ1", {}});
    fasta_sequences.push_back({"READ2", {}});

    for (std::size_t i = 0; i < fasta_sequences.size(); ++i)
    {
        EXPECT_CALL(test_parser, get_sequence_by_id(i)).WillRepeatedly(testing::ReturnRef(fasta_sequences[i]));
    }

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1300;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1200;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1100;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1000;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream);
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0, 1000);
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_GT(overlaps[0].target_end_position_in_read_, overlaps[0].target_start_position_in_read_);
    ASSERT_EQ(overlaps[0].relative_strand, RelativeStrand::Reverse);
    ASSERT_EQ(char(overlaps[0].relative_strand), '-');

    overlapper.update_read_names(overlaps, test_parser, test_parser);
    ASSERT_STREQ(overlaps[0].query_read_name_, fasta_sequences[1].name.c_str());
    ASSERT_STREQ(overlaps[0].target_read_name_, fasta_sequences[2].name.c_str());

    anchors_d.free();
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingTwoForwardOverlapsTwoFusable)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Forward;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Forward;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 90;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 5100;
    overlap4.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 6u);
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingTwoForwardOverlapsOneFusable)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Forward;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Forward;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 91;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 5100;
    overlap4.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 5u);
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingOneForwardOneReverseBothFuasble)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Reverse;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Reverse;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 90;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 2900;
    overlap4.target_end_position_in_read_   = 3900;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 6u);
}

} // namespace cudamapper
} // namespace claragenomics
