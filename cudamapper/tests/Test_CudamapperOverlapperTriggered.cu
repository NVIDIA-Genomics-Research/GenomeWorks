/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "gtest/gtest.h"

#include "../src/overlapper_triggered.hpp"

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

TEST(TestCudamapperOverlapperTriggerred, OneAchorNoOverlaps)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator, hip_stream.get());

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    std::vector<std::string> testv;
    testv.push_back("READ0");
    testv.push_back("READ1");
    testv.push_back("READ2");
    std::vector<std::uint32_t> test_read_length(testv.size(), 1000);

    Anchor anchor1;
    anchors.push_back(anchor1);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 0u);
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsOneOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator, hip_stream.get());

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1300;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, false, 0, 0, 1000);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_EQ(overlaps[0].query_read_id_, 1u);
    ASSERT_EQ(overlaps[0].target_read_id_, 2u);
    ASSERT_EQ(overlaps[0].query_start_position_in_read_, 100u);
    ASSERT_EQ(overlaps[0].query_end_position_in_read_, 400u);
    ASSERT_EQ(overlaps[0].target_start_position_in_read_, 1000u);
    ASSERT_EQ(overlaps[0].target_end_position_in_read_, 1300u);
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsNoOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator, hip_stream.get());

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 3;
    anchor2.target_read_id_          = 4;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 5;
    anchor3.target_read_id_          = 6;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 8;
    anchor4.target_read_id_          = 9;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1300;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0, 1000);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 0u);
}

TEST(TestCudamapperOverlapperTriggerred, FourColinearAnchorsOneOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200 * 10;
    anchor2.target_position_in_read_ = 1100 * 10;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300 * 10;
    anchor3.target_position_in_read_ = 1200 * 10;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400 * 10;
    anchor4.target_position_in_read_ = 1300 * 10;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, 0, 0);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 0u);
}

TEST(TestCudamapperOverlapperTriggerred, FourAnchorsLastNotInOverlap)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1000;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1100;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1200;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400 + 2000;
    anchor4.target_position_in_read_ = 1300 + 2000;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, false, 0, 0, 1000);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_EQ(overlaps[0].query_read_id_, 1u);
    ASSERT_EQ(overlaps[0].target_read_id_, 2u);
    ASSERT_EQ(overlaps[0].query_start_position_in_read_, 100u);
    ASSERT_EQ(overlaps[0].query_end_position_in_read_, 300u);
    ASSERT_EQ(overlaps[0].target_start_position_in_read_, 1000u);
    ASSERT_EQ(overlaps[0].target_end_position_in_read_, 1200u);
}

TEST(TestCudamapperOverlapperTriggerred, ReverseStrand)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();
    CudaStream hip_stream           = make_cuda_stream();
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> unfused_overlaps;
    std::vector<Anchor> anchors;

    Anchor anchor1;
    anchor1.query_read_id_           = 1;
    anchor1.target_read_id_          = 2;
    anchor1.query_position_in_read_  = 100;
    anchor1.target_position_in_read_ = 1300;

    Anchor anchor2;
    anchor2.query_read_id_           = 1;
    anchor2.target_read_id_          = 2;
    anchor2.query_position_in_read_  = 200;
    anchor2.target_position_in_read_ = 1200;

    Anchor anchor3;
    anchor3.query_read_id_           = 1;
    anchor3.target_read_id_          = 2;
    anchor3.query_position_in_read_  = 300;
    anchor3.target_position_in_read_ = 1100;

    Anchor anchor4;
    anchor4.query_read_id_           = 1;
    anchor4.target_read_id_          = 2;
    anchor4.query_position_in_read_  = 400;
    anchor4.target_position_in_read_ = 1000;

    anchors.push_back(anchor1);
    anchors.push_back(anchor2);
    anchors.push_back(anchor3);
    anchors.push_back(anchor4);

    device_buffer<Anchor> anchors_d(anchors.size(), allocator, hip_stream.get());
    cudautils::device_copy_n(anchors.data(), anchors.size(), anchors_d.data(), hip_stream.get()); //H2D

    std::vector<Overlap> overlaps;
    overlapper.get_overlaps(overlaps, anchors_d, false, 0, 0, 1000);
    GW_CU_CHECK_ERR(hipStreamSynchronize(hip_stream.get()));
    ASSERT_EQ(overlaps.size(), 1u);
    ASSERT_GT(overlaps[0].target_end_position_in_read_, overlaps[0].target_start_position_in_read_);
    ASSERT_EQ(overlaps[0].relative_strand, RelativeStrand::Reverse);
    ASSERT_EQ(char(overlaps[0].relative_strand), '-');
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingTwoForwardOverlapsTwoFusable)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Forward;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Forward;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 90;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 5100;
    overlap4.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 6u);
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingTwoForwardOverlapsOneFusable)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Forward;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Forward;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 91;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 5100;
    overlap4.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 5u);
}

TEST(TestCudamapperOverlapperTriggerred, OverlapPostProcessingOneForwardOneReverseBothFuasble)
{
    DefaultDeviceAllocator allocator;
    OverlapperTriggered overlapper(allocator);

    std::vector<Overlap> overlaps;

    Overlap overlap1;
    overlap1.relative_strand                = RelativeStrand::Forward;
    overlap1.query_read_id_                 = 20;
    overlap1.target_read_id_                = 22;
    overlap1.query_start_position_in_read_  = 1000;
    overlap1.query_end_position_in_read_    = 2000;
    overlap1.target_start_position_in_read_ = 4000;
    overlap1.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap1);

    Overlap overlap2;
    overlap2.relative_strand                = RelativeStrand::Forward;
    overlap2.query_read_id_                 = 20;
    overlap2.target_read_id_                = 22;
    overlap2.query_start_position_in_read_  = 2100;
    overlap2.query_end_position_in_read_    = 3100;
    overlap2.target_start_position_in_read_ = 5100;
    overlap2.target_end_position_in_read_   = 6100;
    overlaps.push_back(overlap2);

    Overlap overlap3;
    overlap3.relative_strand                = RelativeStrand::Reverse;
    overlap3.query_read_id_                 = 55;
    overlap3.target_read_id_                = 90;
    overlap3.query_start_position_in_read_  = 1000;
    overlap3.query_end_position_in_read_    = 2000;
    overlap3.target_start_position_in_read_ = 4000;
    overlap3.target_end_position_in_read_   = 5000;
    overlaps.push_back(overlap3);

    Overlap overlap4;
    overlap4.relative_strand                = RelativeStrand::Reverse;
    overlap4.query_read_id_                 = 55;
    overlap4.target_read_id_                = 90;
    overlap4.query_start_position_in_read_  = 2100;
    overlap4.query_end_position_in_read_    = 3100;
    overlap4.target_start_position_in_read_ = 2900;
    overlap4.target_end_position_in_read_   = 3900;
    overlaps.push_back(overlap4);

    Overlapper::post_process_overlaps(overlaps);

    //2 new overlaps are added
    ASSERT_EQ(overlaps.size(), 6u);
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
