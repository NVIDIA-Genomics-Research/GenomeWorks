/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "gtest/gtest.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cudamapper_file_location.hpp"
#include "../src/index_gpu.cuh"
#include "../src/minimizer.hpp"

#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/utils/mathutils.hpp>
#include <claragenomics/utils/signed_integer_utils.hpp>

namespace claragenomics
{
namespace cudamapper
{

namespace details
{
namespace index_gpu
{

// ************ Test find_first_occurrences_of_representations_kernel **************

void test_find_first_occurrences_of_representations_kernel(const thrust::host_vector<std::uint64_t>& representation_index_mask_h,
                                                           const thrust::host_vector<representation_t>& input_representations_h,
                                                           const thrust::host_vector<std::uint32_t>& expected_starting_index_of_each_representation_h,
                                                           const thrust::host_vector<representation_t>& expected_unique_representations_h,
                                                           const std::uint32_t number_of_threads)
{
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    const thrust::device_vector<std::uint64_t> representation_index_mask_d(representation_index_mask_h);
    const thrust::device_vector<representation_t> input_representations_d(input_representations_h);
    ASSERT_EQ(expected_starting_index_of_each_representation_h.size(), representation_index_mask_h.back());
    ASSERT_EQ(expected_unique_representations_h.size(), representation_index_mask_h.back());

    const std::uint64_t number_of_unique_representations = representation_index_mask_h.back();
    ASSERT_EQ(expected_starting_index_of_each_representation_h.size(), number_of_unique_representations);
    ASSERT_EQ(expected_unique_representations_h.size(), number_of_unique_representations);
    thrust::device_vector<std::uint32_t> starting_index_of_each_representation_d(number_of_unique_representations);
    thrust::device_vector<representation_t> unique_representations_d(number_of_unique_representations);

    std::uint32_t number_of_blocks = (representation_index_mask_d.size() - 1) / number_of_threads + 1;

    find_first_occurrences_of_representations_kernel<<<number_of_blocks, number_of_threads, 0, hip_stream>>>(representation_index_mask_d.data().get(),
                                                                                                              input_representations_d.data().get(),
                                                                                                              representation_index_mask_d.size(),
                                                                                                              starting_index_of_each_representation_d.data().get(),
                                                                                                              unique_representations_d.data().get());
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    const thrust::host_vector<std::uint32_t> starting_index_of_each_representation_h(starting_index_of_each_representation_d);
    const thrust::host_vector<representation_t> unique_representations_h(unique_representations_d);

    ASSERT_EQ(starting_index_of_each_representation_h.size(), expected_starting_index_of_each_representation_h.size());
    ASSERT_EQ(unique_representations_h.size(), expected_unique_representations_h.size());
    for (std::size_t i = 0; i < expected_starting_index_of_each_representation_h.size(); ++i)
    {
        EXPECT_EQ(starting_index_of_each_representation_h[i], expected_starting_index_of_each_representation_h[i]) << "index: " << i;
        EXPECT_EQ(unique_representations_h[i], expected_unique_representations_h[i]) << "index: " << i;
    }

    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_find_first_occurrences_of_representations_kernel_small_example)
{
    thrust::host_vector<std::uint64_t> representation_index_mask_h;
    thrust::host_vector<representation_t> input_representations_h;
    thrust::host_vector<std::uint32_t> expected_starting_index_of_each_representation_h;
    thrust::host_vector<representation_t> expected_unique_representations_h;
    representation_index_mask_h.push_back(1);
    input_representations_h.push_back(10);
    expected_starting_index_of_each_representation_h.push_back(0);
    expected_unique_representations_h.push_back(10);
    representation_index_mask_h.push_back(1);
    input_representations_h.push_back(10);
    representation_index_mask_h.push_back(1);
    input_representations_h.push_back(10);
    representation_index_mask_h.push_back(1);
    input_representations_h.push_back(10);
    //
    representation_index_mask_h.push_back(2);
    input_representations_h.push_back(20);
    expected_starting_index_of_each_representation_h.push_back(4);
    expected_unique_representations_h.push_back(20);
    //
    representation_index_mask_h.push_back(3);
    input_representations_h.push_back(30);
    expected_starting_index_of_each_representation_h.push_back(5);
    expected_unique_representations_h.push_back(30);
    representation_index_mask_h.push_back(3);
    input_representations_h.push_back(30);
    representation_index_mask_h.push_back(3);
    input_representations_h.push_back(30);
    representation_index_mask_h.push_back(3);
    input_representations_h.push_back(30);
    //
    representation_index_mask_h.push_back(4);
    input_representations_h.push_back(40);
    expected_starting_index_of_each_representation_h.push_back(9);
    expected_unique_representations_h.push_back(40);
    representation_index_mask_h.push_back(4);
    input_representations_h.push_back(40);
    representation_index_mask_h.push_back(4);
    input_representations_h.push_back(40);
    //
    representation_index_mask_h.push_back(5);
    input_representations_h.push_back(50);
    expected_starting_index_of_each_representation_h.push_back(12);
    expected_unique_representations_h.push_back(50);
    //
    representation_index_mask_h.push_back(6);
    input_representations_h.push_back(60);
    expected_starting_index_of_each_representation_h.push_back(13);
    expected_unique_representations_h.push_back(60);

    std::uint32_t number_of_threads = 3;

    test_find_first_occurrences_of_representations_kernel(representation_index_mask_h,
                                                          input_representations_h,
                                                          expected_starting_index_of_each_representation_h,
                                                          expected_unique_representations_h,
                                                          number_of_threads);
}

TEST(TestCudamapperIndexGPU, test_find_first_occurrences_of_representations_kernel_large_example)
{
    const std::uint64_t total_sketch_elements                    = 10000000;
    const std::uint32_t sketch_elements_with_same_representation = 1000;

    thrust::host_vector<std::uint64_t> representation_index_mask_h;
    thrust::host_vector<representation_t> input_representations_h;
    thrust::host_vector<std::size_t> expected_starting_index_of_each_representation_h;
    thrust::host_vector<representation_t> expected_unique_representations_h;
    for (std::size_t i = 0; i < total_sketch_elements; ++i)
    {
        representation_index_mask_h.push_back(i / sketch_elements_with_same_representation + 1);
        input_representations_h.push_back(representation_index_mask_h.back() * 10);
        if (i % sketch_elements_with_same_representation == 0)
        {
            expected_starting_index_of_each_representation_h.push_back(i);
            expected_unique_representations_h.push_back(input_representations_h.back());
        }
    }

    std::uint32_t number_of_threads = 256;

    test_find_first_occurrences_of_representations_kernel(representation_index_mask_h,
                                                          input_representations_h,
                                                          expected_starting_index_of_each_representation_h,
                                                          expected_unique_representations_h,
                                                          number_of_threads);
}

// ************ Test find_first_occurrences_of_representations **************

void test_find_first_occurrences_of_representations(const thrust::host_vector<representation_t>& representations_h,
                                                    const thrust::host_vector<std::uint32_t>& expected_starting_index_of_each_representation_h,
                                                    const thrust::host_vector<representation_t>& expected_unique_representations_h)
{
    DefaultDeviceAllocator allocator = create_default_device_allocator();

    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    device_buffer<representation_t> representations_d(representations_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(representations_h.data(),
                             representations_h.size(),
                             representations_d.data(),
                             hip_stream); // H2D

    device_buffer<std::uint32_t> starting_index_of_each_representation_d(allocator, hip_stream);
    device_buffer<representation_t> unique_representations_d(allocator, hip_stream);

    find_first_occurrences_of_representations(allocator,
                                              unique_representations_d,
                                              starting_index_of_each_representation_d,
                                              representations_d,
                                              hip_stream);

    thrust::host_vector<std::uint32_t> starting_index_of_each_representation_h(starting_index_of_each_representation_d.size());
    cudautils::device_copy_n(starting_index_of_each_representation_d.data(),
                             starting_index_of_each_representation_d.size(),
                             starting_index_of_each_representation_h.data(),
                             hip_stream); // D2H
    thrust::host_vector<representation_t> unique_representations_h(unique_representations_d.size());
    cudautils::device_copy_n(unique_representations_d.data(),
                             unique_representations_d.size(),
                             unique_representations_h.data(),
                             hip_stream); //D2H
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    ASSERT_EQ(starting_index_of_each_representation_h.size(), expected_starting_index_of_each_representation_h.size());
    ASSERT_EQ(unique_representations_h.size(), expected_unique_representations_h.size());
    ASSERT_EQ(starting_index_of_each_representation_h.size(), unique_representations_h.size() + 1); // starting_index_of_each_representation_h has an additional element for the past-the-end element

    for (std::size_t i = 0; i < unique_representations_h.size(); ++i)
    {
        EXPECT_EQ(starting_index_of_each_representation_h[i], expected_starting_index_of_each_representation_h[i]) << "index: " << i;
        EXPECT_EQ(unique_representations_h[i], expected_unique_representations_h[i]) << "index: " << i;
    }
    EXPECT_EQ(starting_index_of_each_representation_h.back(), expected_starting_index_of_each_representation_h.back()) << "index: " << expected_starting_index_of_each_representation_h.size() - 1;

    representations_d.free();
    starting_index_of_each_representation_d.free();
    unique_representations_d.free();

    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_find_first_occurrences_of_representations_small_example)
{
    /// 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20
    /// 0  0  0  0 12 12 12 12 12 12 23 23 23 32 32 32 32 32 46 46 46
    /// 1  0  0  0  1  0  0  0  0  0  1  0  0  1  0  0  0  0  1  0  0
    /// 1  1  1  1  2  2  2  2  2  2  3  3  3  4  4  4  4  4  5  5  5
    /// ^           ^                 ^        ^              ^       ^
    /// 0  4 10 13 18 21

    thrust::host_vector<representation_t> representations_h;
    thrust::host_vector<std::uint32_t> expected_starting_index_of_each_representation_h;
    thrust::host_vector<representation_t> expected_unique_representations_h;
    representations_h.push_back(0);
    expected_starting_index_of_each_representation_h.push_back(0);
    expected_unique_representations_h.push_back(0);
    representations_h.push_back(0);
    representations_h.push_back(0);
    representations_h.push_back(0);
    representations_h.push_back(12);
    expected_starting_index_of_each_representation_h.push_back(4);
    expected_unique_representations_h.push_back(12);
    representations_h.push_back(12);
    representations_h.push_back(12);
    representations_h.push_back(12);
    representations_h.push_back(12);
    representations_h.push_back(12);
    representations_h.push_back(23);
    expected_starting_index_of_each_representation_h.push_back(10);
    expected_unique_representations_h.push_back(23);
    representations_h.push_back(23);
    representations_h.push_back(23);
    representations_h.push_back(32);
    expected_starting_index_of_each_representation_h.push_back(13);
    expected_unique_representations_h.push_back(32);
    representations_h.push_back(32);
    representations_h.push_back(32);
    representations_h.push_back(32);
    representations_h.push_back(32);
    representations_h.push_back(46);
    expected_starting_index_of_each_representation_h.push_back(18);
    expected_unique_representations_h.push_back(46);
    representations_h.push_back(46);
    representations_h.push_back(46);
    expected_starting_index_of_each_representation_h.push_back(21);

    test_find_first_occurrences_of_representations(representations_h,
                                                   expected_starting_index_of_each_representation_h,
                                                   expected_unique_representations_h);
}

TEST(TestCudamapperIndexGPU, test_find_first_occurrences_of_representations_large_example)
{
    const std::uint64_t total_sketch_elements                    = 10000000;
    const std::uint32_t sketch_elements_with_same_representation = 1000;

    thrust::host_vector<representation_t> representations_h;
    thrust::host_vector<std::uint32_t> expected_starting_index_of_each_representation_h;
    thrust::host_vector<representation_t> expected_unique_representations_h;

    for (std::size_t i = 0; i < total_sketch_elements; ++i)
    {
        representations_h.push_back(i / sketch_elements_with_same_representation);
        if (i % sketch_elements_with_same_representation == 0)
        {
            expected_starting_index_of_each_representation_h.push_back(i);
            expected_unique_representations_h.push_back(i / sketch_elements_with_same_representation);
        }
    }
    expected_starting_index_of_each_representation_h.push_back(total_sketch_elements);

    test_find_first_occurrences_of_representations(representations_h,
                                                   expected_starting_index_of_each_representation_h,
                                                   expected_unique_representations_h);
}

// ************ Test copy_rest_to_separate_arrays **************

template <typename ReadidPositionDirection, typename DirectionOfRepresentation>
void test_function_copy_rest_to_separate_arrays(const thrust::host_vector<ReadidPositionDirection>& rest_h,
                                                const thrust::host_vector<read_id_t>& expected_read_ids_h,
                                                const thrust::host_vector<position_in_read_t>& expected_positions_in_reads_h,
                                                const thrust::host_vector<DirectionOfRepresentation>& expected_directions_of_reads_h,
                                                const std::uint32_t threads)
{
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    ASSERT_EQ(rest_h.size(), expected_read_ids_h.size());
    ASSERT_EQ(rest_h.size(), expected_positions_in_reads_h.size());
    ASSERT_EQ(rest_h.size(), expected_directions_of_reads_h.size());
    thrust::device_vector<read_id_t> generated_read_ids_d(rest_h.size());
    thrust::device_vector<position_in_read_t> generated_positions_in_reads_d(rest_h.size());
    thrust::device_vector<DirectionOfRepresentation> generated_directions_of_reads_d(rest_h.size());

    const thrust::device_vector<ReadidPositionDirection> rest_d(rest_h);

    const std::uint32_t blocks = ceiling_divide<int64_t>(rest_h.size(), threads);

    copy_rest_to_separate_arrays<<<blocks, threads, 0, hip_stream>>>(rest_d.data().get(),
                                                                      generated_read_ids_d.data().get(),
                                                                      generated_positions_in_reads_d.data().get(),
                                                                      generated_directions_of_reads_d.data().get(),
                                                                      rest_h.size());
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    const thrust::host_vector<read_id_t>& generated_read_ids_h(generated_read_ids_d);
    const thrust::host_vector<position_in_read_t>& generated_positions_in_reads_h(generated_positions_in_reads_d);
    const thrust::host_vector<DirectionOfRepresentation>& generated_directions_of_reads_h(generated_directions_of_reads_d);

    for (std::size_t i = 0; i < rest_h.size(); ++i)
    {
        EXPECT_EQ(generated_read_ids_h[i], expected_read_ids_h[i]);
        EXPECT_EQ(generated_positions_in_reads_h[i], expected_positions_in_reads_h[i]);
        EXPECT_EQ(generated_directions_of_reads_h[i], expected_directions_of_reads_h[i]);
    }

    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_function_copy_rest_to_separate_arrays)
{
    thrust::host_vector<Minimizer::ReadidPositionDirection> rest_h;
    thrust::host_vector<read_id_t> expected_read_ids_h;
    thrust::host_vector<position_in_read_t> expected_positions_in_reads_h;
    thrust::host_vector<Minimizer::DirectionOfRepresentation> expected_directions_of_reads_h;

    rest_h.push_back({5, 8, 0});
    expected_read_ids_h.push_back(5);
    expected_positions_in_reads_h.push_back(8);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({15, 6, 0});
    expected_read_ids_h.push_back(15);
    expected_positions_in_reads_h.push_back(6);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({2, 4, 1});
    expected_read_ids_h.push_back(2);
    expected_positions_in_reads_h.push_back(4);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({18, 15, 0});
    expected_read_ids_h.push_back(18);
    expected_positions_in_reads_h.push_back(15);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({6, 4, 1});
    expected_read_ids_h.push_back(6);
    expected_positions_in_reads_h.push_back(4);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({6, 3, 1});
    expected_read_ids_h.push_back(6);
    expected_positions_in_reads_h.push_back(3);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({89, 45, 0});
    expected_read_ids_h.push_back(89);
    expected_positions_in_reads_h.push_back(45);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({547, 25, 0});
    expected_read_ids_h.push_back(547);
    expected_positions_in_reads_h.push_back(25);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({14, 16, 1});
    expected_read_ids_h.push_back(14);
    expected_positions_in_reads_h.push_back(16);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({18, 16, 0});
    expected_read_ids_h.push_back(18);
    expected_positions_in_reads_h.push_back(16);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({45, 44, 0});
    expected_read_ids_h.push_back(45);
    expected_positions_in_reads_h.push_back(44);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({65, 45, 1});
    expected_read_ids_h.push_back(65);
    expected_positions_in_reads_h.push_back(45);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({15, 20, 0});
    expected_read_ids_h.push_back(15);
    expected_positions_in_reads_h.push_back(20);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({45, 654, 1});
    expected_read_ids_h.push_back(45);
    expected_positions_in_reads_h.push_back(654);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({782, 216, 0});
    expected_read_ids_h.push_back(782);
    expected_positions_in_reads_h.push_back(216);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({255, 245, 1});
    expected_read_ids_h.push_back(255);
    expected_positions_in_reads_h.push_back(245);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({346, 579, 0});
    expected_read_ids_h.push_back(346);
    expected_positions_in_reads_h.push_back(579);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({12, 8, 0});
    expected_read_ids_h.push_back(12);
    expected_positions_in_reads_h.push_back(8);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);
    rest_h.push_back({65, 42, 1});
    expected_read_ids_h.push_back(65);
    expected_positions_in_reads_h.push_back(42);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::REVERSE);
    rest_h.push_back({566, 42, 0});
    expected_read_ids_h.push_back(566);
    expected_positions_in_reads_h.push_back(42);
    expected_directions_of_reads_h.push_back(Minimizer::DirectionOfRepresentation::FORWARD);

    const std::uint32_t threads = 8;

    test_function_copy_rest_to_separate_arrays(rest_h,
                                               expected_read_ids_h,
                                               expected_positions_in_reads_h,
                                               expected_directions_of_reads_h,
                                               threads);
}

// ************ Test compress_unique_representations_after_filtering_kernel **************

void test_compress_unique_representations_after_filtering_kernel(const thrust::host_vector<representation_t>& unique_representations_before_compression_h,
                                                                 const thrust::host_vector<std::uint32_t>& first_occurrence_of_representation_before_compression_h,
                                                                 const thrust::host_vector<std::uint32_t>& new_unique_representation_index_h,
                                                                 const thrust::host_vector<representation_t>& expected_unique_representations_after_compression_h,
                                                                 const thrust::host_vector<std::uint32_t>& expected_first_occurrence_of_representation_after_compression_h,
                                                                 const std::int32_t number_of_threads)
{
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    ASSERT_EQ(unique_representations_before_compression_h.size(), first_occurrence_of_representation_before_compression_h.size() - 1);
    ASSERT_EQ(first_occurrence_of_representation_before_compression_h.size(), new_unique_representation_index_h.size());
    const std::uint32_t number_of_unique_representations_after_compression = new_unique_representation_index_h.back(); // last element of new_unique_representation_index_h is equal to the number of unique representation after filtering
    ASSERT_EQ(number_of_unique_representations_after_compression, expected_unique_representations_after_compression_h.size());
    ASSERT_EQ(expected_unique_representations_after_compression_h.size(), expected_first_occurrence_of_representation_after_compression_h.size() - 1);

    const thrust::device_vector<representation_t> unique_representations_before_compression_d(unique_representations_before_compression_h);
    const thrust::device_vector<std::uint32_t> first_occurrence_of_representation_before_compression_d(first_occurrence_of_representation_before_compression_h);
    const thrust::device_vector<std::uint32_t> new_unique_representation_index_d(new_unique_representation_index_h);

    thrust::device_vector<representation_t> unique_representations_after_compression_d(expected_unique_representations_after_compression_h.size());
    thrust::device_vector<std::uint32_t> first_occurrence_of_representation_after_compression_d(expected_first_occurrence_of_representation_after_compression_h.size());

    const std::int32_t number_of_blocks = ceiling_divide<std::int64_t>(unique_representations_before_compression_h.size() + 1,
                                                                       number_of_threads);

    compress_unique_representations_after_filtering_kernel<<<number_of_blocks, number_of_threads, 0, hip_stream>>>(unique_representations_before_compression_d.size(),
                                                                                                                    unique_representations_before_compression_d.data().get(),
                                                                                                                    first_occurrence_of_representation_before_compression_d.data().get(),
                                                                                                                    new_unique_representation_index_d.data().get(),
                                                                                                                    unique_representations_after_compression_d.data().get(),
                                                                                                                    first_occurrence_of_representation_after_compression_d.data().get());
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    const thrust::host_vector<representation_t> unique_representations_after_compression_h(unique_representations_after_compression_d);
    const thrust::host_vector<std::uint32_t> first_occurrence_of_representation_after_compression_h(first_occurrence_of_representation_after_compression_d);

    ASSERT_EQ(unique_representations_after_compression_h.size(), first_occurrence_of_representation_after_compression_h.size() - 1);
    for (std::int32_t i = 0; i < get_size(unique_representations_after_compression_h); ++i)
    {
        EXPECT_EQ(unique_representations_after_compression_h[i], expected_unique_representations_after_compression_h[i]) << "index: " << i;
        EXPECT_EQ(first_occurrence_of_representation_after_compression_h[i], expected_first_occurrence_of_representation_after_compression_h[i]) << "index: " << i;
    }
    // first_occurrence_of_representation_after_compression_h has one more element
    EXPECT_EQ(first_occurrence_of_representation_after_compression_h.back(), expected_first_occurrence_of_representation_after_compression_h.back());

    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_compress_unique_representations_after_filtering_kernel_small_example)
{
    thrust::host_vector<representation_t> unique_representations_before_compression_h;
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_before_compression_h;
    thrust::host_vector<std::uint32_t> new_unique_representation_index_h;
    thrust::host_vector<representation_t> expected_unique_representations_after_compression_h;
    thrust::host_vector<std::uint32_t> expected_first_occurrence_of_representation_after_compression_h;

    // 4 <- filtering_threshold
    // 1  3  5  6  7    <- unique_representations_before_compression_h
    // 2  2  4  6  3  0 <- number_of_sketch_elements_with_representation_d (before filtering)
    // 2  2  0  0  3  0 <- number_of_sketch_elements_with_representation_h (after filtering)
    // 0  2  4  4  4  7 <- first_occurrence_of_representation_before_compression_h
    // 1  1  0  0  1    <- keep_representation_mask
    // 0  1  2  2  2  3 <- new_unique_representation_index_h
    //
    // 1 3 7   <- unique_representations_after_compression_h
    // 0 2 4 7 <- first_occurrence_of_representation_after_compression_h

    unique_representations_before_compression_h.push_back(1);
    unique_representations_before_compression_h.push_back(3);
    unique_representations_before_compression_h.push_back(5);
    unique_representations_before_compression_h.push_back(6);
    unique_representations_before_compression_h.push_back(7);

    first_occurrence_of_representation_before_compression_h.push_back(0);
    first_occurrence_of_representation_before_compression_h.push_back(2);
    first_occurrence_of_representation_before_compression_h.push_back(4);
    first_occurrence_of_representation_before_compression_h.push_back(4);
    first_occurrence_of_representation_before_compression_h.push_back(4);
    first_occurrence_of_representation_before_compression_h.push_back(7);

    new_unique_representation_index_h.push_back(0);
    new_unique_representation_index_h.push_back(1);
    new_unique_representation_index_h.push_back(2);
    new_unique_representation_index_h.push_back(2);
    new_unique_representation_index_h.push_back(2);
    new_unique_representation_index_h.push_back(3);

    expected_unique_representations_after_compression_h.push_back(1);
    expected_unique_representations_after_compression_h.push_back(3);
    expected_unique_representations_after_compression_h.push_back(7);

    expected_first_occurrence_of_representation_after_compression_h.push_back(0);
    expected_first_occurrence_of_representation_after_compression_h.push_back(2);
    expected_first_occurrence_of_representation_after_compression_h.push_back(4);
    expected_first_occurrence_of_representation_after_compression_h.push_back(7);

    const std::int32_t number_of_threads = 4;

    test_compress_unique_representations_after_filtering_kernel(unique_representations_before_compression_h,
                                                                first_occurrence_of_representation_before_compression_h,
                                                                new_unique_representation_index_h,
                                                                expected_unique_representations_after_compression_h,
                                                                expected_first_occurrence_of_representation_after_compression_h,
                                                                number_of_threads);
}

TEST(TestCudamapperIndexGPU, test_compress_unique_representations_after_filtering_kernel_large_example)
{
    const std::uint64_t total_unique_representations = 10000000;

    thrust::host_vector<representation_t> unique_representations_before_compression_h;
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_before_compression_h;
    thrust::host_vector<std::uint32_t> new_unique_representation_index_h;
    thrust::host_vector<representation_t> expected_unique_representations_after_compression_h;
    thrust::host_vector<std::uint32_t> expected_first_occurrence_of_representation_after_compression_h;

    // 0  1  2  3  4  5  6  7  8  9 10 11    <- unique_representations_before_compression_h
    // 0  5  5  0  5  5  0  5  5  0  5  5    <- numer_of_occurrences_after_filtering
    // 0  0  5 10 10 15 20 20 25 30 30 35 40 <- first_occurrence_of_representation_before_compression_h
    // 0  1  1  0  1  1  0  1  1  0  1  1    <- keep_representation_mask
    // 0  0  1  2  2  3  4  4  5  6  6  7  8 <- new_unique_representation_index_h
    // after compression
    // 1  2  4  5  7  8 10 11    <- expected_unique_representations_after_compression_h
    // 0  5 10 15 20 25 30 35 40 <- expected_first_occurrence_of_representation_after_compression_h

    // 0  1  2  3  4  5  6  7  8  9 10 11    <- unique_representations_before_compression_h
    // 0  0  5 10 10 15 20 20 25 30 30 35 40 <- first_occurrence_of_representation_before_compression_h
    // 0  0  1  2  2  3  4  4  5  6  6  7  8 <- new_unique_representation_index_h
    //    1  2     4  5     7  8    10 11    <- expected_unique_representations_after_compression_h
    //    0  5    10 15    20 25    30 35 40 <- expected_first_occurrence_of_representation_after_compression_h

    // 0th iteration
    unique_representations_before_compression_h.push_back(0);
    first_occurrence_of_representation_before_compression_h.push_back(0);
    new_unique_representation_index_h.push_back(0);
    // 1st iteration
    unique_representations_before_compression_h.push_back(1);
    first_occurrence_of_representation_before_compression_h.push_back(0);
    new_unique_representation_index_h.push_back(0);
    expected_unique_representations_after_compression_h.push_back(1);
    expected_first_occurrence_of_representation_after_compression_h.push_back(0);
    for (std::uint32_t i = 2; i < total_unique_representations; ++i)
    {
        unique_representations_before_compression_h.push_back(i);

        if (i % 3 == 0)
        {
            first_occurrence_of_representation_before_compression_h.push_back(first_occurrence_of_representation_before_compression_h.back() + 5);
            new_unique_representation_index_h.push_back(new_unique_representation_index_h.back() + 1);
        }
        if (i % 3 == 1)
        {
            first_occurrence_of_representation_before_compression_h.push_back(first_occurrence_of_representation_before_compression_h.back());
            new_unique_representation_index_h.push_back(new_unique_representation_index_h.back());
            expected_unique_representations_after_compression_h.push_back(expected_unique_representations_after_compression_h.back() + 2);
            expected_first_occurrence_of_representation_after_compression_h.push_back(expected_first_occurrence_of_representation_after_compression_h.back() + 5);
        }
        if (i % 3 == 2)
        {
            first_occurrence_of_representation_before_compression_h.push_back(first_occurrence_of_representation_before_compression_h.back() + 5);
            new_unique_representation_index_h.push_back(new_unique_representation_index_h.back() + 1);
            expected_unique_representations_after_compression_h.push_back(expected_unique_representations_after_compression_h.back() + 1);
            expected_first_occurrence_of_representation_after_compression_h.push_back(expected_first_occurrence_of_representation_after_compression_h.back() + 5);
        }
    }
    /// 10000000 % 3 = 1
    first_occurrence_of_representation_before_compression_h.push_back(first_occurrence_of_representation_before_compression_h.back());
    new_unique_representation_index_h.push_back(new_unique_representation_index_h.back());
    expected_first_occurrence_of_representation_after_compression_h.push_back(expected_first_occurrence_of_representation_after_compression_h.back() + 5);

    const std::int32_t number_of_threads = 256;

    test_compress_unique_representations_after_filtering_kernel(unique_representations_before_compression_h,
                                                                first_occurrence_of_representation_before_compression_h,
                                                                new_unique_representation_index_h,
                                                                expected_unique_representations_after_compression_h,
                                                                expected_first_occurrence_of_representation_after_compression_h,
                                                                number_of_threads);
}

// ************ Test compress_data_arrays_after_filtering_kernel **************

template <typename DirectionOfRepresentation>
void test_compress_data_arrays_after_filtering_kernel(const thrust::host_vector<std::uint32_t>& number_of_sketch_elements_with_representation_before_compression_h,
                                                      const thrust::host_vector<std::uint32_t>& first_occurrence_of_representation_before_filtering_h,
                                                      const thrust::host_vector<std::uint32_t>& first_occurrence_of_representation_after_compression_h,
                                                      const thrust::host_vector<std::uint32_t>& unique_representation_index_after_compression_h,
                                                      const thrust::host_vector<representation_t>& representations_before_compression_h,
                                                      const thrust::host_vector<read_id_t>& read_ids_before_compression_h,
                                                      const thrust::host_vector<position_in_read_t>& positions_in_reads_before_compression_h,
                                                      const thrust::host_vector<DirectionOfRepresentation>& directions_of_representations_before_compression_h,
                                                      const thrust::host_vector<representation_t>& expected_representations_after_compression_h,
                                                      const thrust::host_vector<read_id_t>& expected_read_ids_after_compression_h,
                                                      const thrust::host_vector<position_in_read_t>& expected_positions_in_reads_after_compression_h,
                                                      const thrust::host_vector<DirectionOfRepresentation>& expected_directions_of_representations_after_compression_h,
                                                      const std::int32_t number_of_threads)
{
    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    ASSERT_EQ(number_of_sketch_elements_with_representation_before_compression_h.size(), first_occurrence_of_representation_before_filtering_h.size());
    ASSERT_EQ(number_of_sketch_elements_with_representation_before_compression_h.size(), unique_representation_index_after_compression_h.size());

    const std::size_t number_of_sketch_elements_before_compression = first_occurrence_of_representation_before_filtering_h.back();
    ASSERT_EQ(representations_before_compression_h.size(), number_of_sketch_elements_before_compression);
    ASSERT_EQ(read_ids_before_compression_h.size(), number_of_sketch_elements_before_compression);
    ASSERT_EQ(positions_in_reads_before_compression_h.size(), number_of_sketch_elements_before_compression);
    ASSERT_EQ(directions_of_representations_before_compression_h.size(), number_of_sketch_elements_before_compression);

    const std::size_t number_of_sketch_elements_after_compression = first_occurrence_of_representation_after_compression_h.back();
    ASSERT_EQ(expected_representations_after_compression_h.size(), number_of_sketch_elements_after_compression);
    ASSERT_EQ(expected_read_ids_after_compression_h.size(), number_of_sketch_elements_after_compression);
    ASSERT_EQ(expected_positions_in_reads_after_compression_h.size(), number_of_sketch_elements_after_compression);
    ASSERT_EQ(expected_directions_of_representations_after_compression_h.size(), number_of_sketch_elements_after_compression);

    const thrust::device_vector<std::uint32_t> number_of_sketch_elements_with_representation_before_compression_d(number_of_sketch_elements_with_representation_before_compression_h);
    const thrust::device_vector<std::uint32_t> first_occurrence_of_representation_before_compression_d(first_occurrence_of_representation_before_filtering_h);
    const thrust::device_vector<std::uint32_t> first_occurrence_of_representation_after_compression_d(first_occurrence_of_representation_after_compression_h);
    const thrust::device_vector<std::uint32_t> unique_representation_index_after_compression_d(unique_representation_index_after_compression_h);
    const thrust::device_vector<representation_t> representations_before_compression_d(representations_before_compression_h);
    const thrust::device_vector<read_id_t> read_ids_before_compression_d(read_ids_before_compression_h);
    const thrust::device_vector<position_in_read_t> positions_in_reads_before_compression_d(positions_in_reads_before_compression_h);
    const thrust::device_vector<DirectionOfRepresentation> directions_of_representations_before_compression_d(directions_of_representations_before_compression_h);
    thrust::device_vector<representation_t> representations_after_compression_d(expected_representations_after_compression_h.size());
    thrust::device_vector<read_id_t> read_ids_after_compression_d(expected_read_ids_after_compression_h.size());
    thrust::device_vector<position_in_read_t> positions_in_reads_after_compression_d(expected_positions_in_reads_after_compression_h.size());
    thrust::device_vector<DirectionOfRepresentation> directions_of_representations_after_compression_d(expected_directions_of_representations_after_compression_h.size());

    const std::uint32_t number_of_unique_representations = number_of_sketch_elements_with_representation_before_compression_h.size() - 1; // array has an additional element in the end

    // launch one block per unique representation before compression
    const std::int32_t number_of_blocks = number_of_unique_representations;

    compress_data_arrays_after_filtering_kernel<<<number_of_blocks, number_of_threads, 0, hip_stream>>>(number_of_unique_representations,
                                                                                                         number_of_sketch_elements_with_representation_before_compression_d.data().get(),
                                                                                                         first_occurrence_of_representation_before_compression_d.data().get(),
                                                                                                         first_occurrence_of_representation_after_compression_d.data().get(),
                                                                                                         unique_representation_index_after_compression_d.data().get(),
                                                                                                         representations_before_compression_d.data().get(),
                                                                                                         read_ids_before_compression_d.data().get(),
                                                                                                         positions_in_reads_before_compression_d.data().get(),
                                                                                                         directions_of_representations_before_compression_d.data().get(),
                                                                                                         representations_after_compression_d.data().get(),
                                                                                                         read_ids_after_compression_d.data().get(),
                                                                                                         positions_in_reads_after_compression_d.data().get(),
                                                                                                         directions_of_representations_after_compression_d.data().get());
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    thrust::host_vector<representation_t> representations_after_compression_h(representations_after_compression_d);
    thrust::host_vector<read_id_t> read_ids_after_compression_h(read_ids_after_compression_d);
    thrust::host_vector<position_in_read_t> positions_in_reads_after_compression_h(positions_in_reads_after_compression_d);
    thrust::host_vector<DirectionOfRepresentation> directions_of_representations_after_compression_h(directions_of_representations_after_compression_d);

    ASSERT_EQ(expected_representations_after_compression_h.size(), representations_after_compression_h.size());
    ASSERT_EQ(expected_read_ids_after_compression_h.size(), read_ids_after_compression_h.size());
    ASSERT_EQ(expected_positions_in_reads_after_compression_h.size(), positions_in_reads_after_compression_h.size());
    ASSERT_EQ(expected_directions_of_representations_after_compression_h.size(), directions_of_representations_after_compression_h.size());

    for (std::int32_t i = 0; i < get_size(expected_representations_after_compression_h); ++i)
    {
        EXPECT_EQ(expected_representations_after_compression_h[i], representations_after_compression_h[i]) << "index: " << i;
        EXPECT_EQ(expected_read_ids_after_compression_h[i], read_ids_after_compression_h[i]) << "index: " << i;
        EXPECT_EQ(expected_positions_in_reads_after_compression_h[i], positions_in_reads_after_compression_h[i]) << "index: " << i;
        EXPECT_EQ(expected_directions_of_representations_after_compression_h[i], directions_of_representations_after_compression_h[i]) << "index: " << i;
    }

    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_compress_data_arrays_after_filtering_kernel_small_example)
{
    // 4 <- filtering_threshold
    // 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16
    // 1  1  3  3  5  5  5  5  6  6  6  6  6  6  7  7  7 <- representations_before_compression
    // 0  1  3  5  3  4  6  6  0  1  2  2  2  3  7  8  9 <- read_ids_before_compression
    // 0  0  1  1  4  5  8  9  3  6  7  8  9  5  4  7  3 <- positions_in_reads_before_compression
    // F  F  F  F  R  R  R  F  R  F  F  R  R  F  F  R  R <- directions_of_reads_before_compression
    // 1  3  5  6  7    <- unique_representations_before_compression
    // 2  2  4  6  3  0 <- number_of_sketch_elements_with_representation (before filtering)
    // 0  2  4  8 14 17 <- first_occurrence_of_representation_before_filtering
    // 2  2  0  0  3  0 <- number_of_sketch_elements_with_representation_before_compression (after filtering)
    // 0  2  4  4  4  7 <- first_occurrence_of_representation_before_compression (after filtering)
    // 0  2  4  7       <- first_occurrence_of_representation_after_compression
    // 1  1  0  0  1    <- keep_representation_mask
    // 0  1  2  2  2  3 <- unique_representation_index_after_compression (keep_representation_mask_d after exclusive sum)
    //
    // after compression gives:
    // 0  1  2  3  4  5  6  7
    // 1  1  3  3  7  7  7    <- representations_before_compression
    // 0  1  3  5  7  8  9    <- read_ids_before_compression
    // 0  0  1  1  4  7  3    <- positions_in_reads_before_compression
    // F  F  F  F  F  R  R    <- directions_of_reads_before_compression

    // 1  1  3  3  5  5  5  5  6  6  6  6  6  6  7  7  7 <- representations_before_compression
    std::vector<representation_t> representations_before_compression_std({1, 1, 3, 3, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 7, 7, 7});
    thrust::host_vector<representation_t> representations_before_compression_h(begin(representations_before_compression_std),
                                                                               end(representations_before_compression_std));
    // 0  1  3  5  3  4  6  6  0  1  2  2  2  3  7  8  9 <- read_ids_before_compression
    std::vector<read_id_t> read_ids_before_compression_std({0, 1, 3, 5, 3, 4, 6, 6, 0, 1, 2, 2, 2, 3, 7, 8, 9});
    thrust::host_vector<read_id_t> read_ids_before_compression_h(begin(read_ids_before_compression_std),
                                                                 end(read_ids_before_compression_std));
    // 0  0  1  1  4  5  8  9  3  6  7  8  9  5  4  7  3 <- positions_in_reads_before_compression
    std::vector<position_in_read_t> positions_in_reads_before_compression_std({0, 0, 1, 1, 4, 5, 8, 9, 3, 6, 7, 8, 9, 5, 4, 7, 3});
    thrust::host_vector<position_in_read_t> positions_in_reads_before_compression_h(begin(positions_in_reads_before_compression_std),
                                                                                    end(positions_in_reads_before_compression_std));
    // F  F  F  F  R  R  R  F  R  F  F  R  R  F  F  R  R <- directions_of_reads_before_compression
    thrust::host_vector<SketchElement::DirectionOfRepresentation> directions_of_representations_before_compression_h;
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);

    // 1  1  3  3  7  7  7 <- representations_before_compression
    std::vector<representation_t> expected_representations_after_compression_std({1, 1, 3, 3, 7, 7, 7});
    thrust::host_vector<representation_t> expected_representations_after_compression_h(begin(expected_representations_after_compression_std),
                                                                                       end(expected_representations_after_compression_std));
    // 0  1  3  5  7  8  9 <- read_ids_before_compression
    std::vector<read_id_t> expected_read_ids_after_compression_std({0, 1, 3, 5, 7, 8, 9});
    thrust::host_vector<read_id_t> expected_read_ids_after_compression_h(begin(expected_read_ids_after_compression_std),
                                                                         end(expected_read_ids_after_compression_std));
    // 0  0  1  1  4  7  3 <- positions_in_reads_before_compression
    std::vector<position_in_read_t> expected_positions_in_reads_after_compression_std({0, 0, 1, 1, 4, 7, 3});
    thrust::host_vector<position_in_read_t> expected_positions_in_reads_after_compression_h(begin(expected_positions_in_reads_after_compression_std),
                                                                                            end(expected_positions_in_reads_after_compression_std));
    // F  F  F  F  F  R  R <- directions_of_reads_before_compression
    thrust::host_vector<SketchElement::DirectionOfRepresentation> expected_directions_of_representations_after_compression_h;
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);

    // 2  2  0  0  3  0 <- number_of_sketch_elements_with_representation_before_compression
    std::vector<std::uint32_t> number_of_sketch_elements_with_representation_before_compression_std({2, 2, 0, 0, 3, 0});
    thrust::host_vector<std::uint32_t> number_of_sketch_elements_with_representation_before_compression_h(begin(number_of_sketch_elements_with_representation_before_compression_std),
                                                                                                          end(number_of_sketch_elements_with_representation_before_compression_std));
    // 0  2  4  8 14 17 <- first_occurrence_of_representation_before_filtering_h
    std::vector<std::uint32_t> first_occurrence_of_representation_before_filtering_std({0, 2, 4, 8, 14, 17});
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_before_filtering_h(begin(first_occurrence_of_representation_before_filtering_std),
                                                                                             end(first_occurrence_of_representation_before_filtering_std));
    // 0  2  4  7       <- first_occurrence_of_representation_after_compression
    std::vector<std::uint32_t> first_occurrence_of_representation_after_compression_std({0, 2, 4, 7});
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_after_compression_h(begin(first_occurrence_of_representation_after_compression_std),
                                                                                              end(first_occurrence_of_representation_after_compression_std));
    // 0  1  2  2  2  3 <- unique_representation_index_after_compression
    std::vector<std::uint32_t> unique_representation_index_after_compression_std({0, 1, 2, 2, 2, 3});
    thrust::host_vector<std::uint32_t> unique_representation_index_after_compression_h(begin(unique_representation_index_after_compression_std),
                                                                                       end(unique_representation_index_after_compression_std));

    const std::int32_t number_of_threads = 4;

    test_compress_data_arrays_after_filtering_kernel(number_of_sketch_elements_with_representation_before_compression_h,
                                                     first_occurrence_of_representation_before_filtering_h,
                                                     first_occurrence_of_representation_after_compression_h,
                                                     unique_representation_index_after_compression_h,
                                                     representations_before_compression_h,
                                                     read_ids_before_compression_h,
                                                     positions_in_reads_before_compression_h,
                                                     directions_of_representations_before_compression_h,
                                                     expected_representations_after_compression_h,
                                                     expected_read_ids_after_compression_h,
                                                     expected_positions_in_reads_after_compression_h,
                                                     expected_directions_of_representations_after_compression_h,
                                                     number_of_threads);
}

TEST(TestCudamapperIndexGPU, test_compress_data_arrays_after_filtering_kernel_large_example)
{
    //  6 <- filtering_threshold
    //  0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15|16 17 18 19 20 21 22 23 24 25 26
    //  0  0  0  0  0  0  1  1  1  1  1  2  2  2  2  2| 3  3  3  3  3  3  4  4  4  4  4... <- representations_before_compression
    //  0  1  2  3  4  5  1  2  3  4  5  2  3  4  5  6| 0  1  2  3  4  5  1  2  3  4  5... <- read_ids_before_compression
    // 10 11 12 13 14 15 11 12 13 14 15 12 13 14 15 16|10 11 12 13 14 15 11 12 13 14 15... <- positions_in_reads_before_compression
    //  F  R  F  R  F  R  F  R  F  R  F  R  F  R  F  R| F  R  F  R  F  R  F  R  F  R  F... <- directions_of_reads_before_compression
    //
    //  0  1  2| 3  4  5| 6  7  8| 9 10 11..    <- unique_representations_before_compression
    //  6  5  5| 6  5  5| 6  5  5| 6  5  5...   <- number_of_occurrences_before_filtering
    //  0  6 11|16 22 27|32 38 43|48 54 59... X <- first_occurrence_of_representation_before_filtering (with aditional element)
    //  0  5  5| 0  5  5| 0  5  5| 0  5  5... X <- number_of_occurrences_after_filtering (with aditional element)
    //  0  0  5|10 10 15|20 20 25|30 30 35... X <- first_occurrence_of_representation_before_compression (with aditional element)
    //  0  1  1| 0  1  1| 0  1  1| 0  1  1      <- keep_representation_mask
    //  0  0  1| 2  2  3| 4  4  5| 6  6  7... X <- unique_representation_index_after_compression (with aditional element)
    //  0  5  5| 0  5  5| 0  5  5| 0  5  5... 0 <- number_of_sketch_elements_with_representation_before_compression (after filtering) (with additional element)
    //
    // after compression
    //  0  1  2  3  4  5  6  7  8  9|10 11 12 13 14 15 16 17 18 19|20 21
    //  1  1  1  1  1  2  2  2  2  2| 4  4  4  4  4  5  5  5  5  5| 7  7... <- expected_representations_after_compression
    //  1  2  3  4  5  2  3  4  5  6| 1  2  3  4  5  2  3  4  5  6| 1  2... <- expected_read_ids_after_compression
    // 11 12 13 14 15 12 13 14 15 16|11 12 13 14 15 12 13 14 15 16|11 12    <- expected_positions_in_reads_after_compression
    //  F  R  F  R  F  R  F  R  F  R| F  R  F  R  F  R  F  R  F  R| F  R    <- expected_directions_of_reads_before_compression
    //

    thrust::host_vector<std::uint32_t> number_of_sketch_elements_with_representation_before_compression_h;
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_before_filtering_h;
    thrust::host_vector<std::uint32_t> first_occurrence_of_representation_after_compression_h;
    thrust::host_vector<std::uint32_t> unique_representation_index_after_compression_h;
    thrust::host_vector<representation_t> representations_before_compression_h;
    thrust::host_vector<read_id_t> read_ids_before_compression_h;
    thrust::host_vector<position_in_read_t> positions_in_reads_before_compression_h;
    thrust::host_vector<SketchElement::DirectionOfRepresentation> directions_of_representations_before_compression_h;
    thrust::host_vector<representation_t> expected_representations_after_compression_h;
    thrust::host_vector<read_id_t> expected_read_ids_after_compression_h;
    thrust::host_vector<position_in_read_t> expected_positions_in_reads_after_compression_h;
    thrust::host_vector<SketchElement::DirectionOfRepresentation> expected_directions_of_representations_after_compression_h;

    const std::uint64_t total_sketch_elements = 50000000; // = 3125000 * 16

    representation_t current_representation                                    = 0;
    std::uint32_t current_first_occurrence_of_representation_before_filtering  = 0;
    std::uint32_t current_first_occurrence_of_representation_after_compression = 0;
    std::uint32_t current_unique_representation_index_after_compression        = 0;
    for (std::uint64_t i = 0; i < total_sketch_elements;)
    {
        // first 6 elements
        for (std::uint64_t j = 0; j < 6; ++j)
        {
            representations_before_compression_h.push_back(current_representation);
            read_ids_before_compression_h.push_back(j);
            positions_in_reads_before_compression_h.push_back(j + 10);
            if (0 == j / 2)
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            else
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
        }
        ++current_representation;
        i += 6;
        number_of_sketch_elements_with_representation_before_compression_h.push_back(0); // all sketch elements would get filtered out
        first_occurrence_of_representation_before_filtering_h.push_back(current_first_occurrence_of_representation_before_filtering);
        current_first_occurrence_of_representation_before_filtering += 6;
        unique_representation_index_after_compression_h.push_back(current_unique_representation_index_after_compression);
        current_unique_representation_index_after_compression += 0;

        // next 5 elements
        for (std::uint64_t j = 0; j < 5; ++j)
        {
            representations_before_compression_h.push_back(current_representation);
            expected_representations_after_compression_h.push_back(current_representation);
            read_ids_before_compression_h.push_back(j + 1);
            expected_read_ids_after_compression_h.push_back(j + 1);
            positions_in_reads_before_compression_h.push_back(j + 11);
            expected_positions_in_reads_after_compression_h.push_back(j + 11);
            if (0 == j / 2)
            {
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
                expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
            }
            else
            {
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
                expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            }
        }
        ++current_representation;
        i += 5;
        number_of_sketch_elements_with_representation_before_compression_h.push_back(5);
        first_occurrence_of_representation_before_filtering_h.push_back(current_first_occurrence_of_representation_before_filtering);
        current_first_occurrence_of_representation_before_filtering += 5;
        first_occurrence_of_representation_after_compression_h.push_back(current_first_occurrence_of_representation_after_compression);
        current_first_occurrence_of_representation_after_compression += 5;
        unique_representation_index_after_compression_h.push_back(current_unique_representation_index_after_compression);
        current_unique_representation_index_after_compression += 1;

        // next 5 elements
        for (std::uint64_t j = 0; j < 5; ++j)
        {
            representations_before_compression_h.push_back(current_representation);
            expected_representations_after_compression_h.push_back(current_representation);
            read_ids_before_compression_h.push_back(j + 2);
            expected_read_ids_after_compression_h.push_back(j + 2);
            positions_in_reads_before_compression_h.push_back(j + 12);
            expected_positions_in_reads_after_compression_h.push_back(j + 12);
            if (0 == j / 2)
            {
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
                expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            }
            else
            {
                directions_of_representations_before_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
                expected_directions_of_representations_after_compression_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
            }
        }
        ++current_representation;
        i += 5;
        number_of_sketch_elements_with_representation_before_compression_h.push_back(5);
        first_occurrence_of_representation_before_filtering_h.push_back(current_first_occurrence_of_representation_before_filtering);
        current_first_occurrence_of_representation_before_filtering += 5;
        first_occurrence_of_representation_after_compression_h.push_back(current_first_occurrence_of_representation_after_compression);
        current_first_occurrence_of_representation_after_compression += 5;
        unique_representation_index_after_compression_h.push_back(current_unique_representation_index_after_compression);
        current_unique_representation_index_after_compression += 1;
    }
    number_of_sketch_elements_with_representation_before_compression_h.push_back(0);
    first_occurrence_of_representation_before_filtering_h.push_back(current_first_occurrence_of_representation_before_filtering);
    first_occurrence_of_representation_after_compression_h.push_back(current_first_occurrence_of_representation_after_compression);
    unique_representation_index_after_compression_h.push_back(current_unique_representation_index_after_compression);

    const std::int32_t number_of_threads = 3; // intentionally going with such small blocks

    test_compress_data_arrays_after_filtering_kernel(number_of_sketch_elements_with_representation_before_compression_h,
                                                     first_occurrence_of_representation_before_filtering_h,
                                                     first_occurrence_of_representation_after_compression_h,
                                                     unique_representation_index_after_compression_h,
                                                     representations_before_compression_h,
                                                     read_ids_before_compression_h,
                                                     positions_in_reads_before_compression_h,
                                                     directions_of_representations_before_compression_h,
                                                     expected_representations_after_compression_h,
                                                     expected_read_ids_after_compression_h,
                                                     expected_positions_in_reads_after_compression_h,
                                                     expected_directions_of_representations_after_compression_h,
                                                     number_of_threads);
}

// ************ Test filter_out_most_common_representations **************

template <typename DirectionOfRepresentation>
void test_filter_out_most_common_representations(const double filtering_parameter,
                                                 const thrust::host_vector<representation_t>& input_representations_h,
                                                 const thrust::host_vector<read_id_t>& input_read_ids_h,
                                                 const thrust::host_vector<position_in_read_t>& input_positions_in_reads_h,
                                                 const thrust::host_vector<DirectionOfRepresentation>& input_directions_of_representations_h,
                                                 const thrust::host_vector<representation_t>& input_unique_representations_h,
                                                 const thrust::host_vector<std::uint32_t>& input_first_occurrence_of_representations_h,
                                                 const thrust::host_vector<representation_t>& expected_output_representations_h,
                                                 const thrust::host_vector<read_id_t>& expected_output_read_ids_h,
                                                 const thrust::host_vector<position_in_read_t>& expected_output_positions_in_reads_h,
                                                 const thrust::host_vector<DirectionOfRepresentation>& expected_output_directions_of_representations_h,
                                                 const thrust::host_vector<representation_t>& expected_output_unique_representations_h,
                                                 const thrust::host_vector<std::uint32_t>& expected_output_first_occurrence_of_representations_h)
{
    ASSERT_GE(filtering_parameter, 0u);
    ASSERT_LE(filtering_parameter, 1'000'000'000ull);
    ASSERT_EQ(input_representations_h.size(), input_read_ids_h.size());
    ASSERT_EQ(input_representations_h.size(), input_positions_in_reads_h.size());
    ASSERT_EQ(input_representations_h.size(), input_directions_of_representations_h.size());
    ASSERT_EQ(input_unique_representations_h.size(), input_first_occurrence_of_representations_h.size() - 1);
    ASSERT_EQ(input_representations_h.size(), input_first_occurrence_of_representations_h.back());
    ASSERT_EQ(expected_output_representations_h.size(), expected_output_read_ids_h.size());
    ASSERT_EQ(expected_output_representations_h.size(), expected_output_positions_in_reads_h.size());
    ASSERT_EQ(expected_output_representations_h.size(), expected_output_directions_of_representations_h.size());
    ASSERT_EQ(expected_output_unique_representations_h.size(), expected_output_first_occurrence_of_representations_h.size() - 1);
    ASSERT_EQ(expected_output_representations_h.size(), expected_output_first_occurrence_of_representations_h.back());

    DefaultDeviceAllocator allocator = create_default_device_allocator();

    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    device_buffer<representation_t> representations_d(input_representations_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_representations_h.data(), input_representations_h.size(), representations_d.data(), hip_stream); // H2D
    device_buffer<read_id_t> read_ids_d(input_read_ids_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_read_ids_h.data(), input_read_ids_h.size(), read_ids_d.data(), hip_stream); // H2D
    device_buffer<position_in_read_t> positions_in_reads_d(input_positions_in_reads_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_positions_in_reads_h.data(), input_positions_in_reads_h.size(), positions_in_reads_d.data(), hip_stream); // H2D
    device_buffer<DirectionOfRepresentation> directions_of_representations_d(input_directions_of_representations_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_directions_of_representations_h.data(), input_directions_of_representations_h.size(), directions_of_representations_d.data(), hip_stream); // H2D
    device_buffer<representation_t> unique_representations_d(input_unique_representations_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_unique_representations_h.data(), input_unique_representations_h.size(), unique_representations_d.data(), hip_stream); // H2D
    device_buffer<std::uint32_t> first_occurrence_of_representations_d(input_first_occurrence_of_representations_h.size(), allocator, hip_stream);
    cudautils::device_copy_n(input_first_occurrence_of_representations_h.data(), input_first_occurrence_of_representations_h.size(), first_occurrence_of_representations_d.data(), hip_stream); // H2D

    filter_out_most_common_representations(allocator,
                                           filtering_parameter,
                                           representations_d,
                                           read_ids_d,
                                           positions_in_reads_d,
                                           directions_of_representations_d,
                                           unique_representations_d,
                                           first_occurrence_of_representations_d,
                                           hip_stream);

    thrust::host_vector<representation_t> output_representations_h(representations_d.size());
    cudautils::device_copy_n(representations_d.data(), representations_d.size(), output_representations_h.data(), hip_stream); //D2H
    thrust::host_vector<read_id_t> output_read_ids_h(read_ids_d.size());
    cudautils::device_copy_n(read_ids_d.data(), read_ids_d.size(), output_read_ids_h.data(), hip_stream); //D2H
    thrust::host_vector<position_in_read_t> output_positions_in_reads_h(positions_in_reads_d.size());
    cudautils::device_copy_n(positions_in_reads_d.data(), positions_in_reads_d.size(), output_positions_in_reads_h.data(), hip_stream); //D2H
    thrust::host_vector<DirectionOfRepresentation> output_directions_of_representations_h(directions_of_representations_d.size());
    cudautils::device_copy_n(directions_of_representations_d.data(), directions_of_representations_d.size(), output_directions_of_representations_h.data(), hip_stream); // D2H
    thrust::host_vector<representation_t> output_unique_representations_h(unique_representations_d.size());
    cudautils::device_copy_n(unique_representations_d.data(), unique_representations_d.size(), output_unique_representations_h.data(), hip_stream); // D2H
    thrust::host_vector<std::uint32_t> output_first_occurrence_of_representations_h(first_occurrence_of_representations_d.size());
    cudautils::device_copy_n(first_occurrence_of_representations_d.data(), first_occurrence_of_representations_d.size(), output_first_occurrence_of_representations_h.data(), hip_stream); // D2H
    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

    ASSERT_EQ(expected_output_representations_h.size(), output_representations_h.size());
    ASSERT_EQ(expected_output_representations_h.size(), output_read_ids_h.size());
    ASSERT_EQ(expected_output_representations_h.size(), output_positions_in_reads_h.size());
    ASSERT_EQ(expected_output_representations_h.size(), output_directions_of_representations_h.size());

    for (std::int32_t i = 0; i < get_size(representations_d); ++i)
    {
        EXPECT_EQ(expected_output_representations_h[i], output_representations_h[i]) << "index: " << i;
        EXPECT_EQ(expected_output_read_ids_h[i], output_read_ids_h[i]) << "index: " << i;
        EXPECT_EQ(expected_output_positions_in_reads_h[i], output_positions_in_reads_h[i]) << "index: " << i;
        EXPECT_EQ(expected_output_directions_of_representations_h[i], output_directions_of_representations_h[i]) << "index: " << i;
    }

    ASSERT_EQ(expected_output_unique_representations_h, output_unique_representations_h);
    ASSERT_EQ(expected_output_first_occurrence_of_representations_h, output_first_occurrence_of_representations_h);
    ASSERT_EQ(output_unique_representations_h.size(), output_first_occurrence_of_representations_h.size() - 1);
    ASSERT_EQ(output_representations_h.size(), output_first_occurrence_of_representations_h.back());

    for (std::int32_t i = 0; i < get_size(output_unique_representations_h); ++i)
    {
        EXPECT_EQ(expected_output_unique_representations_h[i], output_unique_representations_h[i]) << "index: " << i;
        EXPECT_EQ(expected_output_first_occurrence_of_representations_h[i], output_first_occurrence_of_representations_h[i]) << "index: " << i;
    }
    ASSERT_EQ(expected_output_first_occurrence_of_representations_h.back(), output_first_occurrence_of_representations_h.back());

    representations_d.free();
    read_ids_d.free();
    positions_in_reads_d.free();
    directions_of_representations_d.free();
    unique_representations_d.free();
    first_occurrence_of_representations_d.free();

    CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, test_filter_out_most_common_representations_small_example)
{
    // For example this index initinally contains 20 sketch elements:
    // 0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19
    // 1  1  3  3  5  5  5  5  6  6  6  6  6  6  7  7  7  8  8  8 <- representations (before filtering)
    // 0  1  3  5  3  4  6  6  0  1  2  2  2  3  7  8  9  1  2  3 <- read_ids (before filtering)
    // 0  0  1  1  4  5  8  9  3  6  7  8  9  5  4  7  3  7  8  9 <- positions_in_reads (before filtering)
    // F  F  F  F  R  R  R  F  R  F  F  R  R  F  F  R  R  F  F  F <- directions_of_reads (before filtering)
    // 1  3  5  6  7  8    <- unique_representations (before filtering)
    // 0  2  4  8 14 17 20 <- first_occurrence_of_representations (before filtering)
    //
    // For filtering_parameter = 0.2:
    // sketch_elementes_with_that_representation/total_sketch_element >= filtering_parameter
    // sketch_elementes_with_that_representation/20 >= 0.2 <=>
    // sketch_elementes_with_that_representation >= 20 * 0.2 <=>
    // sketch_elementes_with_that_representation >= 4 <=>
    // sketch element with representations with 4 or more sketch elements will be removed
    //
    // In the example above that means that representations 5 and 6 will be removed and that the output would be:
    // 0  1  2  3  4  5  6  7  8  9
    // 1  1  3  3  7  7  7  8  8  8 <- representations (after filtering)
    // 0  1  3  5  7  8  9  1  2  3 <- read_ids (after filtering)
    // 0  0  1  1  4  7  3  7  8  9 <- positions_in_reads (after filtering)
    // F  F  F  F  F  R  R  F  F  F <- directions_of_reads (after filtering)
    // 1  3  7  8    <- unique_representations (after filtering)
    // 0  2  4  7 10 <- first_occurrence_of_representations (after filtering)

    const double filtering_parameter = 0.2;

    const std::vector<representation_t> input_representations_std({1, 1, 3, 3, 5, 5, 5, 5, 6, 6, 6, 6, 6, 6, 7, 7, 7, 8, 8, 8});
    const thrust::host_vector<representation_t> input_representations_h(std::begin(input_representations_std), std::end(input_representations_std));

    const std::vector<read_id_t> input_read_ids_std({0, 1, 3, 5, 3, 4, 6, 6, 0, 1, 2, 2, 2, 3, 7, 8, 9, 1, 2, 3});
    const thrust::host_vector<read_id_t> input_read_ids_h(std::begin(input_read_ids_std), std::end(input_read_ids_std));

    const std::vector<position_in_read_t> input_positions_in_reads_std({0, 0, 1, 1, 4, 5, 8, 9, 3, 6, 7, 8, 9, 5, 4, 7, 3, 7, 8, 9});
    const thrust::host_vector<position_in_read_t> input_positions_in_reads_h(std::begin(input_positions_in_reads_std), std::end(input_positions_in_reads_std));

    const std::vector<SketchElement::DirectionOfRepresentation> input_directions_of_representations_std({SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                         SketchElement::DirectionOfRepresentation::FORWARD});
    const thrust::host_vector<SketchElement::DirectionOfRepresentation> input_directions_of_representations_h(std::begin(input_directions_of_representations_std), std::end(input_directions_of_representations_std));

    const std::vector<representation_t> input_unique_representations_std({1, 3, 5, 6, 7, 8});
    const thrust::host_vector<representation_t> input_unique_representations_h(std::begin(input_unique_representations_std), std::end(input_unique_representations_std));

    const std::vector<std::uint32_t> input_first_occurrence_of_representations_std({0, 2, 4, 8, 14, 17, 20});
    const thrust::host_vector<std::uint32_t> input_first_occurrence_of_representations_h(std::begin(input_first_occurrence_of_representations_std), std::end(input_first_occurrence_of_representations_std));

    const std::vector<representation_t> expected_output_representations_std({1, 1, 3, 3, 7, 7, 7, 8, 8, 8});
    const thrust::host_vector<representation_t> expected_output_representations_h(std::begin(expected_output_representations_std), std::end(expected_output_representations_std));

    const std::vector<read_id_t> expected_output_read_ids_std({0, 1, 3, 5, 7, 8, 9, 1, 2, 3});
    const thrust::host_vector<read_id_t> expected_output_read_ids_h(std::begin(expected_output_read_ids_std), std::end(expected_output_read_ids_std));

    const std::vector<position_in_read_t> expected_output_positions_in_reads_std({0, 0, 1, 1, 4, 7, 3, 7, 8, 9});
    const thrust::host_vector<position_in_read_t> expected_output_positions_in_reads_h(std::begin(expected_output_positions_in_reads_std), std::end(expected_output_positions_in_reads_std));

    const std::vector<SketchElement::DirectionOfRepresentation> expected_output_directions_of_representations_std({SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                                   SketchElement::DirectionOfRepresentation::REVERSE,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD,
                                                                                                                   SketchElement::DirectionOfRepresentation::FORWARD});
    const thrust::host_vector<SketchElement::DirectionOfRepresentation> expected_output_directions_of_representations_h(std::begin(expected_output_directions_of_representations_std), std::end(expected_output_directions_of_representations_std));

    const std::vector<representation_t> expected_output_unique_representations_std({1, 3, 7, 8});
    const thrust::host_vector<representation_t> expected_output_unique_representations_h(std::begin(expected_output_unique_representations_std), std::end(expected_output_unique_representations_std));

    const std::vector<std::uint32_t> expected_output_first_occurrence_of_representations_std({0, 2, 4, 7, 10});
    const thrust::host_vector<std::uint32_t> expected_output_first_occurrence_of_representations_h(std::begin(expected_output_first_occurrence_of_representations_std), std::end(expected_output_first_occurrence_of_representations_std));

    test_filter_out_most_common_representations(filtering_parameter,
                                                input_representations_h,
                                                input_read_ids_h,
                                                input_positions_in_reads_h,
                                                input_directions_of_representations_h,
                                                input_unique_representations_h,
                                                input_first_occurrence_of_representations_h,
                                                expected_output_representations_h,
                                                expected_output_read_ids_h,
                                                expected_output_positions_in_reads_h,
                                                expected_output_directions_of_representations_h,
                                                expected_output_unique_representations_h,
                                                expected_output_first_occurrence_of_representations_h);
}

TEST(TestCudamapperIndexGPU, test_filter_out_most_common_representations_large_example)
{
    // Total sketch elements = 50'000'000
    //
    // Wanted filtering_threshold = 6
    // 6 = 50'000'000 * filtering_parameter <=> filtering_parameter = 6 / 50'000'000 * <=>
    // filtering_parameter = 0.00000012
    //
    //  6 <- filtering_threshold
    //  0  1  2  3  4  5  6  7  8  9 10 11 12 13 14 15|16 17 18 19 20 21 22 23 24 25 26
    //  0  0  0  0  0  0  1  1  1  1  1  2  2  2  2  2| 3  3  3  3  3  3  4  4  4  4  4... <- representations_before_compression
    //  0  1  2  3  4  5  1  2  3  4  5  2  3  4  5  6| 0  1  2  3  4  5  1  2  3  4  5... <- read_ids_before_compression
    // 10 11 12 13 14 15 11 12 13 14 15 12 13 14 15 16|10 11 12 13 14 15 11 12 13 14 15... <- positions_in_reads_before_compression
    //  F  R  F  R  F  R  F  R  F  R  F  R  F  R  F  R| F  R  F  R  F  R  F  R  F  R  F... <- directions_of_reads_before_compression
    //  0  1  2| 3  4  5| 6  7  8| 9 10 11..    <- unique_representations_before_filtering
    //  0  6 11|16 22 27|32 38 43|48 54 59... X <- first_occurrence_of_representation_before_filtering (with aditional element)
    //
    // after compression
    //  0  1  2  3  4  5  6  7  8  9|10 11 12 13 14 15 16 17 18 19|20 21
    //  1  1  1  1  1  2  2  2  2  2| 4  4  4  4  4  5  5  5  5  5| 7  7... <- expected_representations_after_compression
    //  1  2  3  4  5  2  3  4  5  6| 1  2  3  4  5  2  3  4  5  6| 1  2... <- expected_read_ids_after_compression
    // 11 12 13 14 15 12 13 14 15 16|11 12 13 14 15 12 13 14 15 16|11 12    <- expected_positions_in_reads_after_compression
    //  F  R  F  R  F  R  F  R  F  R| F  R  F  R  F  R  F  R  F  R| F  R    <- expected_directions_of_reads_before_compression
    //  1  2| 4  5| 7  8|...   <- expected_unique_representations_after_filtering
    //  0  5|10 15|20 25|... X <- first_occurrence_of_representation_after_filtering (with aditional element)

    const std::uint64_t total_sketch_elements = 50000000; // = 3125000 * 16
    const double filtering_parameter          = 0.00000012;

    thrust::host_vector<representation_t> input_representations_h;
    thrust::host_vector<read_id_t> input_read_ids_h;
    thrust::host_vector<position_in_read_t> input_positions_in_reads_h;
    thrust::host_vector<SketchElement::DirectionOfRepresentation> input_directions_of_representations_h;
    thrust::host_vector<representation_t> input_unique_representations_h;
    thrust::host_vector<std::uint32_t> input_first_occurrence_of_representations_h;
    thrust::host_vector<representation_t> expected_output_representations_h;
    thrust::host_vector<read_id_t> expected_output_read_ids_h;
    thrust::host_vector<position_in_read_t> expected_output_positions_in_reads_h;
    thrust::host_vector<SketchElement::DirectionOfRepresentation> expected_output_directions_of_representations_h;
    thrust::host_vector<representation_t> expected_output_unique_representations_h;
    thrust::host_vector<std::uint32_t> expected_output_first_occurrence_of_representations_h;

    representation_t current_representation                                   = 0;
    std::uint32_t current_first_occurrence_of_representation_before_filtering = 0;
    std::uint32_t current_first_occurrence_of_representation_after_filtering  = 0;

    for (std::uint64_t i = 0; i < total_sketch_elements;) // add 16 elements per iteration
    {
        // first 6 elements
        for (std::uint64_t j = 0; j < 6; ++j)
        {
            input_representations_h.push_back(current_representation);
            input_read_ids_h.push_back(j);
            input_positions_in_reads_h.push_back(j + 10);
            if (0 == j / 2)
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            else
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
        }
        input_unique_representations_h.push_back(current_representation);
        input_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_before_filtering);
        current_first_occurrence_of_representation_before_filtering += 6;
        ++current_representation;
        i += 6;

        // next 5 elements
        for (std::uint64_t j = 0; j < 5; ++j)
        {
            input_representations_h.push_back(current_representation);
            expected_output_representations_h.push_back(current_representation);
            input_read_ids_h.push_back(j + 1);
            expected_output_read_ids_h.push_back(j + 1);
            input_positions_in_reads_h.push_back(j + 11);
            expected_output_positions_in_reads_h.push_back(j + 11);
            if (0 == j / 2)
            {
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
                expected_output_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
            }
            else
            {
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
                expected_output_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            }
        }
        input_unique_representations_h.push_back(current_representation);
        expected_output_unique_representations_h.push_back(current_representation);
        input_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_before_filtering);
        expected_output_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_after_filtering);
        current_first_occurrence_of_representation_before_filtering += 5;
        current_first_occurrence_of_representation_after_filtering += 5;
        ++current_representation;
        i += 5;

        // next 5 elements
        for (std::uint64_t j = 0; j < 5; ++j)
        {
            input_representations_h.push_back(current_representation);
            expected_output_representations_h.push_back(current_representation);
            input_read_ids_h.push_back(j + 2);
            expected_output_read_ids_h.push_back(j + 2);
            input_positions_in_reads_h.push_back(j + 12);
            expected_output_positions_in_reads_h.push_back(j + 12);
            if (0 == j / 2)
            {
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
                expected_output_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
            }
            else
            {
                input_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
                expected_output_directions_of_representations_h.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
            }
        }
        input_unique_representations_h.push_back(current_representation);
        expected_output_unique_representations_h.push_back(current_representation);
        input_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_before_filtering);
        expected_output_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_after_filtering);
        current_first_occurrence_of_representation_before_filtering += 5;
        current_first_occurrence_of_representation_after_filtering += 5;
        ++current_representation;
        i += 5;
    }
    input_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_before_filtering);
    expected_output_first_occurrence_of_representations_h.push_back(current_first_occurrence_of_representation_after_filtering);

    test_filter_out_most_common_representations(filtering_parameter,
                                                input_representations_h,
                                                input_read_ids_h,
                                                input_positions_in_reads_h,
                                                input_directions_of_representations_h,
                                                input_unique_representations_h,
                                                input_first_occurrence_of_representations_h,
                                                expected_output_representations_h,
                                                expected_output_read_ids_h,
                                                expected_output_positions_in_reads_h,
                                                expected_output_directions_of_representations_h,
                                                expected_output_unique_representations_h,
                                                expected_output_first_occurrence_of_representations_h);
}

} // namespace index_gpu
} // namespace details

void test_function(const std::string& filename,
                   const read_id_t first_read_id,
                   const read_id_t past_the_last_read_id,
                   const read_id_t expected_smallest_read_id,
                   const read_id_t expected_largest_read_id,
                   const std::uint64_t kmer_size,
                   const std::uint64_t window_size,
                   const std::vector<representation_t>& expected_representations,
                   const std::vector<position_in_read_t>& expected_positions_in_reads,
                   const std::vector<read_id_t>& expected_read_ids,
                   const std::vector<SketchElement::DirectionOfRepresentation>& expected_directions_of_reads,
                   const std::vector<representation_t>& expected_unique_representations,
                   const std::vector<std::uint32_t>& expected_first_occurrence_of_representations,
                   const read_id_t expected_number_of_reads,
                   const position_in_read_t expected_number_of_basepairs_in_longest_read,
                   const double filtering_parameter = 1.0)
{
    std::unique_ptr<io::FastaParser> parser = io::create_kseq_fasta_parser(filename);
    DefaultDeviceAllocator allocator        = create_default_device_allocator();

    hipStream_t hip_stream;
    CGA_CU_CHECK_ERR(hipStreamCreate(&hip_stream));

    {
        IndexGPU<Minimizer> index(allocator,
                                  *parser,
                                  first_read_id,
                                  past_the_last_read_id,
                                  kmer_size,
                                  window_size,
                                  false,
                                  filtering_parameter,
                                  hip_stream);
        CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));

        ASSERT_EQ(index.number_of_reads(), expected_number_of_reads);
        if (0 == expected_number_of_reads)
        {
            return;
        }

        ASSERT_EQ(index.smallest_read_id(), expected_smallest_read_id);
        ASSERT_EQ(index.largest_read_id(), expected_largest_read_id);

        ASSERT_EQ(expected_number_of_basepairs_in_longest_read, index.number_of_basepairs_in_longest_read());

        // check arrays
        const device_buffer<representation_t>& representations_d                             = index.representations();
        const device_buffer<position_in_read_t>& positions_in_reads_d                        = index.positions_in_reads();
        const device_buffer<read_id_t>& read_ids_d                                           = index.read_ids();
        const device_buffer<SketchElement::DirectionOfRepresentation>& directions_of_reads_d = index.directions_of_reads();
        thrust::host_vector<representation_t> representations_h(representations_d.size());
        cudautils::device_copy_n(representations_d.data(), representations_d.size(), representations_h.data(), hip_stream); // D2H
        thrust::host_vector<position_in_read_t> positions_in_reads_h(positions_in_reads_d.size());
        cudautils::device_copy_n(positions_in_reads_d.data(), positions_in_reads_d.size(), positions_in_reads_h.data(), hip_stream); // D2H
        thrust::host_vector<read_id_t> read_ids_h(read_ids_d.size());
        cudautils::device_copy_n(read_ids_d.data(), read_ids_d.size(), read_ids_h.data(), hip_stream); // D2H
        thrust::host_vector<SketchElement::DirectionOfRepresentation> directions_of_reads_h(directions_of_reads_d.size());
        cudautils::device_copy_n(directions_of_reads_d.data(), directions_of_reads_d.size(), directions_of_reads_h.data(), hip_stream); // D2H
        CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
        ASSERT_EQ(representations_h.size(), expected_representations.size());
        ASSERT_EQ(positions_in_reads_h.size(), expected_positions_in_reads.size());
        ASSERT_EQ(read_ids_h.size(), expected_read_ids.size());
        ASSERT_EQ(directions_of_reads_h.size(), expected_directions_of_reads.size());
        ASSERT_EQ(representations_h.size(), positions_in_reads_h.size());
        ASSERT_EQ(positions_in_reads_h.size(), read_ids_h.size());
        ASSERT_EQ(read_ids_h.size(), directions_of_reads_h.size());
        for (std::size_t i = 0; i < expected_positions_in_reads.size(); ++i)
        {
            EXPECT_EQ(representations_h[i], expected_representations[i]) << "i: " << i;
            EXPECT_EQ(positions_in_reads_h[i], expected_positions_in_reads[i]) << "i: " << i;
            EXPECT_EQ(read_ids_h[i], expected_read_ids[i]) << "i: " << i;
            EXPECT_EQ(directions_of_reads_h[i], expected_directions_of_reads[i]) << "i: " << i;
        }

        const device_buffer<representation_t>& unique_representations_d           = index.unique_representations();
        const device_buffer<std::uint32_t>& first_occurrence_of_representations_d = index.first_occurrence_of_representations();
        thrust::host_vector<representation_t> unique_representations_h(unique_representations_d.size());
        cudautils::device_copy_n(unique_representations_d.data(), unique_representations_d.size(), unique_representations_h.data(), hip_stream); // D2H
        thrust::host_vector<std::uint32_t> first_occurrence_of_representations_h(first_occurrence_of_representations_d.size());
        cudautils::device_copy_n(first_occurrence_of_representations_d.data(), first_occurrence_of_representations_d.size(), first_occurrence_of_representations_h.data(), hip_stream); // D2H
        CGA_CU_CHECK_ERR(hipStreamSynchronize(hip_stream));
        ASSERT_EQ(expected_unique_representations.size() + 1, expected_first_occurrence_of_representations.size());
        ASSERT_EQ(unique_representations_h.size(), expected_unique_representations.size());
        ASSERT_EQ(first_occurrence_of_representations_h.size(), expected_first_occurrence_of_representations.size());
        for (std::size_t i = 0; i < expected_unique_representations.size(); ++i)
        {
            EXPECT_EQ(expected_unique_representations[i], unique_representations_h[i]) << "index: " << i;
            EXPECT_EQ(expected_first_occurrence_of_representations[i], first_occurrence_of_representations_h[i]) << "index: " << i;
        }
        EXPECT_EQ(expected_first_occurrence_of_representations.back(), expected_representations.size());
    }

    CGA_CU_CHECK_ERR(hipStreamDestroy(hip_stream));
}

TEST(TestCudamapperIndexGPU, GATT_4_1)
{
    // >read_0
    // GATT

    // GATT = 0b10001111
    // AATC = 0b00001101 <- minimizer

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/gatt.fasta";
    const std::uint64_t minimizer_size = 4;
    const std::uint64_t window_size    = 1;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b1101);
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b1101);
    expected_first_occurrence_of_representations.push_back(0);

    expected_first_occurrence_of_representations.push_back(1);

    const read_id_t expected_number_of_reads                              = 1;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 0;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 4;

    test_function(filename,
                  0,
                  1,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, GATT_2_3)
{
    // >read_0
    // GATT

    // kmer representation: forward, reverse
    // GA: <20> 31
    // AT: <03> 03
    // TT:  33 <00>

    // front end minimizers: representation, position_in_read, direction, read_id
    // GA : 20 0 F 0
    // GAT: 03 1 F 0

    // central minimizers
    // GATT: 00 2 R 0

    // back end minimizers
    // ATT: 00 2 R 0
    // TT : 00 2 R 0

    // All minimizers: GA(0f), AT(1f), AA(2r)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0        1        2
    // data arrays: GA(0f0), AT(1f0), AA(2r0)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/gatt.fasta";
    const std::uint64_t minimizer_size = 2;
    const std::uint64_t window_size    = 3;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b0000); // AA(2r0)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b0000);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b0011); // AT(1f0)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0011);
    expected_first_occurrence_of_representations.push_back(1);
    expected_representations.push_back(0b1000); // GA(0f0)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b1000);
    expected_first_occurrence_of_representations.push_back(2);

    expected_first_occurrence_of_representations.push_back(3);

    const read_id_t expected_number_of_reads                              = 1;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 0;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 4;

    test_function(filename,
                  0,
                  1,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, CCCATACC_2_8)
{
    // *** Read is shorter than one full window, the result should be empty ***

    // >read_0
    // CCCATACC

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/cccatacc.fasta";
    const std::uint64_t minimizer_size = 2;
    const std::uint64_t window_size    = 8;

    // all data arrays should be empty

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    const read_id_t expected_number_of_reads                              = 0;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 0;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 0;

    test_function(filename,
                  0,
                  1,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

// TODO: Cover this case as well
//TEST(TestCudamapperIndexGPU, CATCAAG_AAGCTA_3_5)
//{
//    // *** One Read is shorter than one full window, the other is not ***
//
//    // >read_0
//    // CATCAAG
//    // >read_1
//    // AAGCTA
//
//    // ** CATCAAG **
//
//    // kmer representation: forward, reverse
//    // CAT:  103 <032>
//    // ATC: <031> 203
//    // TCA: <310> 320
//    // CAA: <100> 332
//    // AAG: <002> 133
//
//    // front end minimizers: representation, position_in_read, direction, read_id
//    // CAT   : 032 0 R 0
//    // CATC  : 031 1 F 0
//    // CATCA : 031 1 F 0
//    // CATCAA: 031 1 F 0
//
//    // central minimizers
//    // CATCAAG: 002 4 F 0
//
//    // back end minimizers
//    // ATCAAG: 002 4 F 0
//    // TCAAG : 002 4 F 0
//    // CAAG  : 002 4 F 0
//    // AAG   : 002 4 F 0
//
//    // ** AAGCTA **
//    // ** read does not fit one array **
//
//    // All minimizers: ATG(0r0), ATC(1f0), AAG(4f0)
//
//    // (2r1) means position 2, reverse direction, read 1
//    // (1,2) means array block start at element 1 and has 2 elements
//
//    //              0         1         2
//    // data arrays: AAG(4f0), ATC(1f0), ATG(0r0)
//
//    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/catcaag_aagcta.fasta";
//    const std::uint64_t minimizer_size = 3;
//    const std::uint64_t window_size    = 5;
//
//    std::vector<representation_t> expected_representations;
//    std::vector<position_in_read_t> expected_positions_in_reads;
//    std::vector<read_id_t> expected_read_ids;
//    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
//    expected_representations.push_back(0b000010); // AAG(4f0)
//    expected_positions_in_reads.push_back(4);
//    expected_read_ids.push_back(0);
//    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
//    expected_representations.push_back(0b001101); // ATC(1f0)
//    expected_positions_in_reads.push_back(1);
//    expected_read_ids.push_back(0);
//    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
//    expected_representations.push_back(0b001110); // ATG(0r0)
//    expected_positions_in_reads.push_back(0);
//    expected_read_ids.push_back(0);
//    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
//
//    const read_id_t expected_number_of_reads                              = 1;
//    const position_in_read_t expected_number_of_basepairs_in_longest_read = 7;
//
//    test_function(filename,
//                  0,
//                  2,
//                  minimizer_size,
//                  window_size,
//                  expected_representations,
//                  expected_positions_in_reads,
//                  expected_read_ids,
//                  expected_directions_of_reads,
//                  expected_number_of_reads,
//                  expected_number_of_basepairs_in_longest_read); // <- only one read goes into index, the other is too short
//}

TEST(TestCudamapperIndexGPU, CCCATACC_3_5)
{
    // >read_0
    // CCCATACC

    // ** CCCATAC **

    // kmer representation: forward, reverse
    // CCC: <111> 222
    // CCA: <110> 322
    // CAT:  103 <032>
    // ATA: <030> 303
    // TAC:  301 <230>
    // ACC: <011> 223

    // front end minimizers: representation, position_in_read, direction
    // CCC   : 111 0 F
    // CCCA  : 110 1 F
    // CCCAT : 032 2 R
    // CCCATA: 030 3 F

    // central minimizers
    // CCCATAC: 030 3 F
    // CCATACC: 011 5 F

    // back end minimizers
    // CATACC: 011 5 F
    // ATACC : 011 5 F
    // TACC  : 011 5 F
    // ACC   : 011 5 F

    // All minimizers: CCC(0f), CCA(1f), ATG(2r), ATA(3f), ACC(5f)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0         1         2
    // data arrays: ACC(5f0), ATA(3f0), ATG(2r0), CCA(1f0), CCC(0f0)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/cccatacc.fasta";
    const std::uint64_t minimizer_size = 3;
    const std::uint64_t window_size    = 5;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b000101); // ACC(5f0)
    expected_positions_in_reads.push_back(5);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b000101);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b001100); // ATA(3f0)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b001100);
    expected_first_occurrence_of_representations.push_back(1);
    expected_representations.push_back(0b001110); // ATG(2r0)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b001110);
    expected_first_occurrence_of_representations.push_back(2);
    expected_representations.push_back(0b010100); // CCA(1f0)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b010100);
    expected_first_occurrence_of_representations.push_back(3);
    expected_representations.push_back(0b010101); // CCC(0f0)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b010101);
    expected_first_occurrence_of_representations.push_back(4);

    expected_first_occurrence_of_representations.push_back(5);

    const read_id_t expected_number_of_reads                              = 1;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 0;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 8;

    test_function(filename,
                  0,
                  1,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, CATCAAG_AAGCTA_3_2)
{
    // >read_0
    // CATCAAG
    // >read_1
    // AAGCTA

    // ** CATCAAG **

    // kmer representation: forward, reverse
    // CAT:  103 <032>
    // ATC: <031> 203
    // TCA: <310> 320
    // CAA: <100> 332
    // AAG: <002> 133

    // front end minimizers: representation, position_in_read, direction, read_id
    // CAT: 032 0 R 0

    // central minimizers
    // CATC: 031 1 F 0
    // ATCA: 031 1 F 0
    // TCAA: 100 3 F 0
    // CAAG: 002 4 F 0

    // back end minimizers
    // AAG: 002 4 F 0

    // All minimizers: ATC(1f), CAA(3f), AAG(4f), ATG(0r)

    // ** AAGCTA **

    // kmer representation: forward, reverse
    // AAG: <002> 133
    // AGC: <021> 213
    // GCT:  213 <021>
    // CTA: <130> 302

    // front end minimizers: representation, position_in_read, direction, read_id
    // AAG: 002 0 F 1

    // central minimizers
    // AAGC: 002 0 F 1
    // AGCT: 021 2 R 1 // only the last minimizer is saved
    // GCTA: 021 2 R 1

    // back end minimizers
    // CTA: 130 3 F 1

    // All minimizers: AAG(0f), AGC(2r), CTA(3f)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0         1         2         3         4         5         6
    // data arrays: AAG(4f0), AAG(0f1), AGC(2r1), ATC(1f0), ATG(0r0), CAA(3f0), CTA(3f1)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/catcaag_aagcta.fasta";
    const std::uint64_t minimizer_size = 3;
    const std::uint64_t window_size    = 2;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b000010); // AAG(4f0)
    expected_positions_in_reads.push_back(4);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0010);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b000010); // AAG(0f1)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b001001); // AGC(2r1)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b001001);
    expected_first_occurrence_of_representations.push_back(2);
    expected_representations.push_back(0b001101); // ATC(1f0)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b001101);
    expected_first_occurrence_of_representations.push_back(3);
    expected_representations.push_back(0b001110); // ATG(0r0)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b001110);
    expected_first_occurrence_of_representations.push_back(4);
    expected_representations.push_back(0b010000); // CAA(3f0)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(0);
    expected_unique_representations.push_back(0b010000);
    expected_first_occurrence_of_representations.push_back(5);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b011100); // CTA(3f1)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b011100);
    expected_first_occurrence_of_representations.push_back(6);

    expected_first_occurrence_of_representations.push_back(7);

    const read_id_t expected_number_of_reads                              = 2;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 1;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 7;

    test_function(filename,
                  0,
                  2,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, AAAACTGAA_GCCAAAG_2_3)
{
    // >read_0
    // AAAACTGAA
    // >read_1
    // GCCAAAG

    // ** AAAACTGAA **

    // kmer representation: forward, reverse
    // AA: <00> 33
    // AA: <00> 33
    // AA: <00> 33
    // AC: <01> 23
    // CT:  13 <02>
    // TG:  32 <10>
    // GA: <20> 31
    // AA: <00> 33

    // front end minimizers: representation, position_in_read, direction, read_id
    // AA : 00 0 F 0
    // AAA: 00 1 F 0

    // central minimizers
    // AAAA: 00 2 F 0
    // AAAC: 00 2 F 0
    // AACT: 00 2 F 0
    // ACTG: 01 3 F 0
    // CTGA: 02 4 R 0
    // TGAA: 00 7 F 0

    // back end minimizers
    // GAA: 00 7 F 0
    // AA : 00 7 F 0

    // All minimizers: AA(0f), AA(1f), AA(2f), AC(3f), AG(4r), AA (7f)

    // ** GCCAAAG **

    // kmer representation: forward, reverse
    // GC: <21> 21
    // CC: <11> 22
    // CA: <10> 32
    // AA: <00> 33
    // AA: <00> 33
    // AG: <03> 21

    // front end minimizers: representation, position_in_read, direction, read_id
    // GC : 21 0 F 0
    // GCC: 11 1 F 0

    // central minimizers
    // GCCA: 10 2 F 0
    // CCAA: 00 3 F 0
    // CAAA: 00 4 F 0
    // AAAG: 00 4 F 0

    // back end minimizers
    // AAG: 00 4 F 0
    // AG : 03 5 F 0

    // All minimizers: GC(0f), CC(1f), CA(2f), AA(3f), AA(4f), AG(5f)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0        1        2        3        4        5        6        7        8        9        10       11
    // data arrays: AA(0f0), AA(1f0), AA(2f0), AA(7f0), AA(3f1), AA(4f1), AC(3f0), AG(4r0), AG(5f1), CA(2f1), CC(1f1), GC(0f1)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/aaaactgaa_gccaaag.fasta";
    const std::uint64_t minimizer_size = 2;
    const std::uint64_t window_size    = 3;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b0000); // AA(0f0)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0000);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b0000); // AA(1f0)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0000); // AA(2f0)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0000); // AA(7f0)
    expected_positions_in_reads.push_back(7);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0000); // AA(3f1)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0000); // AA(4f1)
    expected_positions_in_reads.push_back(4);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0001); // AC(3f0)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0001);
    expected_first_occurrence_of_representations.push_back(6);
    expected_representations.push_back(0b0010); // AG(4r0)
    expected_positions_in_reads.push_back(4);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b0010);
    expected_first_occurrence_of_representations.push_back(7);
    expected_representations.push_back(0b0010); // AG(5f1)
    expected_positions_in_reads.push_back(5);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0100); // CA(2f1)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0100);
    expected_first_occurrence_of_representations.push_back(9);
    expected_representations.push_back(0b0101); // CC(1f1)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0101);
    expected_first_occurrence_of_representations.push_back(10);
    expected_representations.push_back(0b1001); // GC(0f1)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b1001);
    expected_first_occurrence_of_representations.push_back(11);

    expected_first_occurrence_of_representations.push_back(12);

    const read_id_t expected_number_of_reads                              = 2;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 1;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 9;

    test_function(filename,
                  0,
                  2,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, AAAACTGAA_GCCAAAG_2_3_only_second_read_in_index)
{
    // >read_0
    // AAAACTGAA
    // >read_1
    // GCCAAAG

    // ** AAAACTGAA **
    // only second read goes into index

    // ** GCCAAAG **

    // kmer representation: forward, reverse
    // GC: <21> 21
    // CC: <11> 22
    // CA: <10> 32
    // AA: <00> 33
    // AA: <00> 33
    // AG: <03> 21

    // front end minimizers: representation, position_in_read, direction, read_id
    // GC : 21 0 F 0
    // GCC: 11 1 F 0

    // central minimizers
    // GCCA: 10 2 F 0
    // CCAA: 00 3 F 0
    // CAAA: 00 4 F 0
    // AAAG: 00 4 F 0

    // back end minimizers
    // AAG: 00 4 F 0
    // AG : 03 5 F 0

    // All minimizers: GC(0f), CC(1f), CA(2f), AA(3f), AA(4f), AG(5f)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0        1        2        3        4        5
    // data arrays: AA(3f1), AA(4f1), AG(5f1), CA(2f1), CC(1f1), GC(0f1)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/aaaactgaa_gccaaag.fasta";
    const std::uint64_t minimizer_size = 2;
    const std::uint64_t window_size    = 3;

    // only take second read

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b0000); // AA(3f1)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b00);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b0000); // AA(4f1)
    expected_positions_in_reads.push_back(4);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0010); // AG(5f1)
    expected_positions_in_reads.push_back(5);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0010);
    expected_first_occurrence_of_representations.push_back(2);
    expected_representations.push_back(0b0100); // CA(2f1)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0100);
    expected_first_occurrence_of_representations.push_back(3);
    expected_representations.push_back(0b0101); // CC(1f1)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0101);
    expected_first_occurrence_of_representations.push_back(4);
    expected_representations.push_back(0b1001); // GC(0f1)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b1001);
    expected_first_occurrence_of_representations.push_back(5);

    expected_first_occurrence_of_representations.push_back(6);

    const read_id_t expected_number_of_reads                              = 1;
    const read_id_t expected_smallest_read_id                             = 1; // <- index is instructed to ignore first (0th) read
    const read_id_t expected_largest_read_id                              = 1;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 7;

    test_function(filename,
                  1, // <- only take second read
                  2,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read);
}

TEST(TestCudamapperIndexGPU, AAAACTGAA_GCCAAAG_2_3_filtering)
{
    // >read_0
    // AAAACTGAA
    // >read_1
    // GCCAAAG

    // ** AAAACTGAA **

    // kmer representation: forward, reverse
    // AA: <00> 33
    // AA: <00> 33
    // AA: <00> 33
    // AC: <01> 23
    // CT:  13 <02>
    // TG:  32 <10>
    // GA: <20> 31
    // AA: <00> 33

    // front end minimizers: representation, position_in_read, direction, read_id
    // AA : 00 0 F 0
    // AAA: 00 1 F 0

    // central minimizers
    // AAAA: 00 2 F 0
    // AAAC: 00 2 F 0
    // AACT: 00 2 F 0
    // ACTG: 01 3 F 0
    // CTGA: 02 4 R 0
    // TGAA: 00 7 F 0

    // back end minimizers
    // GAA: 00 7 F 0
    // AA : 00 7 F 0

    // All minimizers: AA(0f), AA(1f), AA(2f), AC(3f), AG(4r), AA (7f)

    // ** GCCAAAG **

    // kmer representation: forward, reverse
    // GC: <21> 21
    // CC: <11> 22
    // CA: <10> 32
    // AA: <00> 33
    // AA: <00> 33
    // AG: <03> 21

    // front end minimizers: representation, position_in_read, direction, read_id
    // GC : 21 0 F 0
    // GCC: 11 1 F 0

    // central minimizers
    // GCCA: 10 2 F 0
    // CCAA: 00 3 F 0
    // CAAA: 00 4 F 0
    // AAAG: 00 4 F 0

    // back end minimizers
    // AAG: 00 4 F 0
    // AG : 03 5 F 0

    // All minimizers: GC(0f), CC(1f), CA(2f), AA(3f), AA(4f), AG(5f)

    // (2r1) means position 2, reverse direction, read 1
    // (1,2) means array block start at element 1 and has 2 elements

    //              0        1        2        3        4        5        6        7        8        9        10       11
    // data arrays: AA(0f0), AA(1f0), AA(2f0), AA(7f0), AA(3f1), AA(4f1), AC(3f0), AG(4r0), AG(5f1), CA(2f1), CC(1f1), GC(0f1)

    // Total sketch elements: 12
    // Sketch elements per representation: AA = 6, AC = 1, AG = 2, CA = 1, CC = 1, GC = 1
    // filtering_parameter = 0.5 <=> filtering_threshold = 12 * 0.5 = 6
    // data arrays after filtering: AC(3f0), AG(4r0), AG(5f1), CA(2f1), CC(1f1), GC(0f1)

    const std::string filename         = std::string(CUDAMAPPER_BENCHMARK_DATA_DIR) + "/aaaactgaa_gccaaag.fasta";
    const std::uint64_t minimizer_size = 2;
    const std::uint64_t window_size    = 3;
    const double filtering_parameter   = 0.5;

    std::vector<representation_t> expected_representations;
    std::vector<position_in_read_t> expected_positions_in_reads;
    std::vector<read_id_t> expected_read_ids;
    std::vector<SketchElement::DirectionOfRepresentation> expected_directions_of_reads;
    std::vector<representation_t> expected_unique_representations;
    std::vector<std::uint32_t> expected_first_occurrence_of_representations;

    expected_representations.push_back(0b0001); // AC(3f0)
    expected_positions_in_reads.push_back(3);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0001);
    expected_first_occurrence_of_representations.push_back(0);
    expected_representations.push_back(0b0010); // AG(4r0)
    expected_positions_in_reads.push_back(4);
    expected_read_ids.push_back(0);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::REVERSE);
    expected_unique_representations.push_back(0b0010);
    expected_first_occurrence_of_representations.push_back(1);
    expected_representations.push_back(0b0010); // AG(5f1)
    expected_positions_in_reads.push_back(5);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_representations.push_back(0b0100); // CA(2f1)
    expected_positions_in_reads.push_back(2);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0100);
    expected_first_occurrence_of_representations.push_back(3);
    expected_representations.push_back(0b0101); // CC(1f1)
    expected_positions_in_reads.push_back(1);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b0101);
    expected_first_occurrence_of_representations.push_back(4);
    expected_representations.push_back(0b1001); // GC(0f1)
    expected_positions_in_reads.push_back(0);
    expected_read_ids.push_back(1);
    expected_directions_of_reads.push_back(SketchElement::DirectionOfRepresentation::FORWARD);
    expected_unique_representations.push_back(0b1001);
    expected_first_occurrence_of_representations.push_back(5);

    expected_first_occurrence_of_representations.push_back(6);

    const read_id_t expected_number_of_reads                              = 2;
    const read_id_t expected_smallest_read_id                             = 0;
    const read_id_t expected_largest_read_id                              = 1;
    const position_in_read_t expected_number_of_basepairs_in_longest_read = 9;

    test_function(filename,
                  0,
                  2,
                  expected_smallest_read_id,
                  expected_largest_read_id,
                  minimizer_size,
                  window_size,
                  expected_representations,
                  expected_positions_in_reads,
                  expected_read_ids,
                  expected_directions_of_reads,
                  expected_unique_representations,
                  expected_first_occurrence_of_representations,
                  expected_number_of_reads,
                  expected_number_of_basepairs_in_longest_read,
                  filtering_parameter);
}

} // namespace cudamapper
} // namespace claragenomics
