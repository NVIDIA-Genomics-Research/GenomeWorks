#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

// Implementation file for CUDA POA kernels.

#include "cudapoa_kernels.cuh"
#include "cudapoa_nw.cu"
#include "cudapoa_nw_banded.cu"
#include "cudapoa_topsort.cu"
#include "cudapoa_add_alignment.cu"
#include "cudapoa_generate_consensus.cu"
#include "cudapoa_generate_msa.cu"

#include <claragenomics/utils/cudautils.hpp>
#include <claragenomics/cudapoa/batch.hpp>

namespace claragenomics
{

namespace cudapoa
{

/**
 * @brief The main kernel that runs the partial order alignment
 *        algorithm.
 *
 * @param[out] consensus_d                  Device buffer for generated consensus
 * @param[in] sequences_d                   Device buffer with sequences for all windows
 * @param[in] base_weights_d                Device buffer with base weights for all windows
 * @param[in] sequence_lengths_d            Device buffer sequence lengths
 * @param[in] window_details_d              Device buffer with structs encapsulating sequence details per window
 * @param[in] total_windows                 Total number of windows to process
 * @param[in] scores_d                      Device scratch space that scores alignment matrix score
 * @param[in] alignment_graph_d             Device scratch space for backtrace alignment of graph
 * @param[in] alignment_read_d              Device scratch space for backtrace alignment of sequence
 * @param[in] nodes_d                       Device scratch space for storing unique nodes in graph
 * @param[in] incoming_edges_d              Device scratch space for storing incoming edges per node
 * @param[in] incoming_edges_count_d        Device scratch space for storing number of incoming edges per node
 * @param[in] outgoing_edges_d              Device scratch space for storing outgoing edges per node
 * @param[in] outgoing_edges_count_d        Device scratch space for storing number of outgoing edges per node
 * @param[in] incoming_edge_w_d             Device scratch space for storing weight of incoming edges
 * @param[in] outgoing_edge_w_d             Device scratch space for storing weight of outgoing edges
 * @param[in] sorted_poa_d                  Device scratch space for storing sorted graph
 * @param[in] node_id_to_pos_d              Device scratch space for mapping node ID to position in graph
 * @graph[in] node_alignments_d             Device scratch space for storing alignment nodes per node in graph
 * @param[in] node_alignment_count_d        Device scratch space for storing number of aligned nodes
 * @param[in] sorted_poa_local_edge_count_d Device scratch space for maintaining edge counts during topological sort
 * @param[in] node_marks_d_                 Device scratch space for storing node marks when running spoa accurate top sort
 * @param[in] check_aligned_nodes_d_        Device scratch space for storing check for aligned nodes
 * @param[in] nodes_to_visit_d_             Device scratch space for storing stack of nodes to be visited in topsort
 * @param[in] node_coverage_counts_d_       Device scratch space for storing coverage of each node in graph.
 * @param[in] gap_score                     Score for inserting gap into alignment
 * @param[in] mismatch_score                Score for finding a mismatch in alignment
 * @param[in] match_score                   Score for finding a match in alignment
 */
template <int32_t TPB = 64, bool cuda_banded_alignment = false, bool msa = false, typename ScoreT, typename SizeT>
__global__ void generatePOAKernel(uint8_t* consensus_d,
                                  uint8_t* sequences_d,
                                  int8_t* base_weights_d,
                                  SizeT* sequence_lengths_d,
                                  claragenomics::cudapoa::WindowDetails* window_details_d,
                                  int32_t total_windows,
                                  ScoreT* scores_d,
                                  SizeT* alignment_graph_d,
                                  SizeT* alignment_read_d,
                                  uint8_t* nodes_d,
                                  SizeT* incoming_edges_d,
                                  uint16_t* incoming_edge_count_d,
                                  SizeT* outgoing_edges_d,
                                  uint16_t* outgoing_edge_count_d,
                                  uint16_t* incoming_edge_w_d,
                                  uint16_t* outgoing_edge_w_d,
                                  SizeT* sorted_poa_d,
                                  SizeT* node_id_to_pos_d,
                                  SizeT* node_alignments_d,
                                  uint16_t* node_alignment_count_d,
                                  uint16_t* sorted_poa_local_edge_count_d,
                                  uint8_t* node_marks_d_,
                                  bool* check_aligned_nodes_d_,
                                  SizeT* nodes_to_visit_d_,
                                  uint16_t* node_coverage_counts_d_,
                                  ScoreT gap_score,
                                  ScoreT mismatch_score,
                                  ScoreT match_score,
                                  uint32_t max_sequences_per_poa,
                                  SizeT* sequence_begin_nodes_ids_d,
                                  uint16_t* outgoing_edges_coverage_d,
                                  uint16_t* outgoing_edges_coverage_count_d,
                                  uint32_t max_limit_nodes_per_window,
                                  uint32_t max_limit_nodes_per_window_banded,
                                  uint32_t max_limit_matrix_graph_dimension,
                                  uint32_t max_limit_matrix_graph_dimension_banded,
                                  uint32_t max_limit_consensus_size)
{
    // shared error indicator within a warp
    bool warp_error = false;

    int32_t nwindows_per_block = TPB / WARP_SIZE;
    int32_t warp_idx           = threadIdx.x / WARP_SIZE;
    int32_t lane_idx           = threadIdx.x % WARP_SIZE;
    int32_t window_idx         = blockIdx.x * nwindows_per_block + warp_idx;

    if (window_idx >= total_windows)
        return;

    // These are not being changed to int32_t to make use of larger range
    // without having to use 2 registers which would be needed for 64bit
    uint32_t max_nodes_per_window = cuda_banded_alignment ? max_limit_nodes_per_window_banded : max_limit_nodes_per_window;
    uint32_t max_graph_dimension  = cuda_banded_alignment ? max_limit_matrix_graph_dimension_banded : max_limit_matrix_graph_dimension;

    // Find the buffer offsets for each thread within the global memory buffers.
    uint8_t* nodes                        = &nodes_d[max_nodes_per_window * window_idx];
    SizeT* incoming_edges                 = &incoming_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* incoming_edge_count         = &incoming_edge_count_d[window_idx * max_nodes_per_window];
    SizeT* outoing_edges                  = &outgoing_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_count         = &outgoing_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* incoming_edge_weights       = &incoming_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_weights       = &outgoing_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    SizeT* sorted_poa                     = &sorted_poa_d[window_idx * max_nodes_per_window];
    SizeT* node_id_to_pos                 = &node_id_to_pos_d[window_idx * max_nodes_per_window];
    SizeT* node_alignments                = &node_alignments_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_ALIGNMENTS];
    uint16_t* node_alignment_count        = &node_alignment_count_d[window_idx * max_nodes_per_window];
    uint16_t* sorted_poa_local_edge_count = &sorted_poa_local_edge_count_d[window_idx * max_nodes_per_window];

    int32_t scores_width = window_details_d[window_idx].scores_width;
    size_t scores_offset = window_details_d[window_idx].scores_offset * max_graph_dimension;

    ScoreT* scores = 0;
    if (cuda_banded_alignment)
        scores = &scores_d[max_graph_dimension * CUDAPOA_BANDED_MAX_MATRIX_SEQUENCE_DIMENSION * window_idx];
    else
        scores = &scores_d[scores_offset];

    SizeT* alignment_graph         = &alignment_graph_d[max_graph_dimension * window_idx];
    SizeT* alignment_read          = &alignment_read_d[max_graph_dimension * window_idx];
    uint16_t* node_coverage_counts = &node_coverage_counts_d_[max_nodes_per_window * window_idx];

#ifdef SPOA_ACCURATE
    uint8_t* node_marks       = &node_marks_d_[max_nodes_per_window * window_idx];
    bool* check_aligned_nodes = &check_aligned_nodes_d_[max_nodes_per_window * window_idx];
    SizeT* nodes_to_visit     = &nodes_to_visit_d_[max_nodes_per_window * window_idx];
#endif

    SizeT* sequence_lengths = &sequence_lengths_d[window_details_d[window_idx].seq_len_buffer_offset];

    uint32_t num_sequences = window_details_d[window_idx].num_seqs;
    uint8_t* sequence      = &sequences_d[window_details_d[window_idx].seq_starts];
    int8_t* base_weights   = &base_weights_d[window_details_d[window_idx].seq_starts];

    uint8_t* consensus = &consensus_d[window_idx * max_limit_consensus_size];

    SizeT* sequence_begin_nodes_ids         = nullptr;
    uint16_t* outgoing_edges_coverage       = nullptr;
    uint16_t* outgoing_edges_coverage_count = nullptr;

    if (msa)
    {
        sequence_begin_nodes_ids      = &sequence_begin_nodes_ids_d[window_idx * max_sequences_per_poa];
        outgoing_edges_coverage       = &outgoing_edges_coverage_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES * max_sequences_per_poa];
        outgoing_edges_coverage_count = &outgoing_edges_coverage_count_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    }

    if (lane_idx == 0)
    {
        // Create backbone for window based on first sequence in window.
        nodes[0]                                     = sequence[0];
        sorted_poa[0]                                = 0;
        incoming_edge_count[0]                       = 0;
        node_alignment_count[0]                      = 0;
        node_id_to_pos[0]                            = 0;
        outgoing_edge_count[sequence_lengths[0] - 1] = 0;
        incoming_edge_weights[0]                     = base_weights[0];
        node_coverage_counts[0]                      = 1;
        if (msa)
        {
            sequence_begin_nodes_ids[0] = 0;
        }

        //Build the rest of the graphs
        for (SizeT nucleotide_idx = 1; nucleotide_idx < sequence_lengths[0]; nucleotide_idx++)
        {
            nodes[nucleotide_idx]                                          = sequence[nucleotide_idx];
            sorted_poa[nucleotide_idx]                                     = nucleotide_idx;
            outoing_edges[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES]   = nucleotide_idx;
            outgoing_edge_count[nucleotide_idx - 1]                        = 1;
            incoming_edges[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES]        = nucleotide_idx - SizeT(1);
            incoming_edge_weights[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES] = base_weights[nucleotide_idx - 1] + base_weights[nucleotide_idx];
            incoming_edge_count[nucleotide_idx]                            = 1;
            node_alignment_count[nucleotide_idx]                           = 0;
            node_id_to_pos[nucleotide_idx]                                 = nucleotide_idx;
            node_coverage_counts[nucleotide_idx]                           = 1;
            if (msa)
            {
                outgoing_edges_coverage[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES * max_sequences_per_poa] = 0;
                outgoing_edges_coverage_count[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES]                   = 1;
            }
        }

        // Clear error code for window.
        consensus[0] = CUDAPOA_KERNEL_NOERROR_ENCOUNTERED;
    }

    __syncwarp();

    // Align each subsequent read, add alignment to graph, run topoligical sort.
    for (SizeT s = 1; s < num_sequences; s++)
    {
        SizeT seq_len = sequence_lengths[s];
        sequence += sequence_lengths[s - 1];     // increment the pointer so it is pointing to correct sequence data
        base_weights += sequence_lengths[s - 1]; // increment the pointer so it is pointing to correct sequence data

        if (lane_idx == 0)
        {
            if (sequence_lengths[0] >= max_nodes_per_window)
            {
                consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
                consensus[1] = static_cast<uint8_t>(StatusType::node_count_exceeded_maximum_graph_size);
                warp_error   = true;
            }
        }

        warp_error = __shfl_sync(FULL_MASK, warp_error, 0);
        if (warp_error)
        {
            return;
        }

        // Run Needleman-Wunsch alignment between graph and new sequence.
        SizeT alignment_length;

        if (cuda_banded_alignment)
        {
            alignment_length = runNeedlemanWunschBanded<uint8_t, ScoreT, SizeT>(nodes,
                                                                                sorted_poa,
                                                                                node_id_to_pos,
                                                                                sequence_lengths[0],
                                                                                incoming_edge_count,
                                                                                incoming_edges,
                                                                                outgoing_edge_count,
                                                                                sequence,
                                                                                seq_len,
                                                                                scores,
                                                                                alignment_graph,
                                                                                alignment_read,
                                                                                gap_score,
                                                                                mismatch_score,
                                                                                match_score);
        }
        else
        {
            alignment_length = runNeedlemanWunsch<uint8_t, ScoreT, SizeT>(nodes,
                                                                          sorted_poa,
                                                                          node_id_to_pos,
                                                                          sequence_lengths[0],
                                                                          incoming_edge_count,
                                                                          incoming_edges,
                                                                          outgoing_edge_count,
                                                                          outoing_edges,
                                                                          sequence,
                                                                          seq_len,
                                                                          scores,
                                                                          scores_width,
                                                                          alignment_graph,
                                                                          alignment_read,
                                                                          gap_score,
                                                                          mismatch_score,
                                                                          match_score);
        }

        __syncwarp();

        if (alignment_length == UINT16_MAX)
        {
            if (lane_idx == 0)
            {
                consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
                consensus[1] = static_cast<uint8_t>(StatusType::loop_count_exceeded_upper_bound);
            }
            return;
        }

        if (lane_idx == 0)
        {

            // Add alignment to graph.
            SizeT new_node_count;
            uint8_t error_code = addAlignmentToGraph<msa>(new_node_count,
                                                          nodes, sequence_lengths[0],
                                                          node_alignments, node_alignment_count,
                                                          incoming_edges, incoming_edge_count,
                                                          outoing_edges, outgoing_edge_count,
                                                          incoming_edge_weights, outgoing_edge_weights,
                                                          alignment_length,
                                                          sorted_poa, alignment_graph,
                                                          sequence, alignment_read,
                                                          node_coverage_counts,
                                                          base_weights,
                                                          (sequence_begin_nodes_ids + s),
                                                          outgoing_edges_coverage,
                                                          outgoing_edges_coverage_count,
                                                          s,
                                                          max_sequences_per_poa,
                                                          max_nodes_per_window);

            if (error_code != 0)
            {
                consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
                consensus[1] = error_code;
                warp_error   = true;
            }
            else
            {
                sequence_lengths[0] = new_node_count;
                // Run a topsort on the graph.
#ifdef SPOA_ACCURATE
                // Exactly matches racon CPU results
                raconTopologicalSortDeviceUtil(sorted_poa,
                                               node_id_to_pos,
                                               new_node_count,
                                               incoming_edge_count,
                                               incoming_edges,
                                               node_alignment_count,
                                               node_alignments,
                                               node_marks,
                                               check_aligned_nodes,
                                               nodes_to_visit,
                                               cuda_banded_alignment,
                                               (uint16_t)max_nodes_per_window);
#else
                // Faster top sort
                topologicalSortDeviceUtil(sorted_poa,
                                          node_id_to_pos,
                                          new_node_count,
                                          incoming_edge_count,
                                          outoing_edges,
                                          outgoing_edge_count,
                                          sorted_poa_local_edge_count);
#endif
            }
        }

        __syncwarp();

        warp_error = __shfl_sync(FULL_MASK, warp_error, 0);
        if (warp_error)
        {
            return;
        }
    }
}

template <typename ScoreT, typename SizeT>
void generatePOAtemplated(claragenomics::cudapoa::OutputDetails* output_details_d,
                          claragenomics::cudapoa::InputDetails<SizeT>* input_details_d,
                          int32_t total_windows,
                          hipStream_t stream,
                          claragenomics::cudapoa::AlignmentDetails<ScoreT, SizeT>* alignment_details_d,
                          claragenomics::cudapoa::GraphDetails<SizeT>* graph_details_d,
                          ScoreT gap_score,
                          ScoreT mismatch_score,
                          ScoreT match_score,
                          bool cuda_banded_alignment,
                          uint32_t max_sequences_per_poa,
                          int8_t output_mask,
                          const BatchSize& batch_size)
{
    // unpack output details
    uint8_t* consensus_d                  = output_details_d->consensus;
    uint16_t* coverage_d                  = output_details_d->coverage;
    uint8_t* multiple_sequence_alignments = output_details_d->multiple_sequence_alignments;

    // unpack input details
    uint8_t* sequences_d            = input_details_d->sequences;
    int8_t* base_weights_d          = input_details_d->base_weights;
    SizeT* sequence_lengths_d       = input_details_d->sequence_lengths;
    WindowDetails* window_details_d = input_details_d->window_details;
    SizeT* sequence_begin_nodes_ids = input_details_d->sequence_begin_nodes_ids;

    // unpack alignment details
    ScoreT* scores         = alignment_details_d->scores;
    SizeT* alignment_graph = alignment_details_d->alignment_graph;
    SizeT* alignment_read  = alignment_details_d->alignment_read;
    // unpack graph details
    uint8_t* nodes                          = graph_details_d->nodes;
    SizeT* node_alignments                  = graph_details_d->node_alignments;
    uint16_t* node_alignment_count          = graph_details_d->node_alignment_count;
    SizeT* incoming_edges                   = graph_details_d->incoming_edges;
    uint16_t* incoming_edge_count           = graph_details_d->incoming_edge_count;
    SizeT* outgoing_edges                   = graph_details_d->outgoing_edges;
    uint16_t* outgoing_edge_count           = graph_details_d->outgoing_edge_count;
    uint16_t* incoming_edge_w               = graph_details_d->incoming_edge_weights;
    uint16_t* outgoing_edge_w               = graph_details_d->outgoing_edge_weights;
    SizeT* sorted_poa                       = graph_details_d->sorted_poa;
    SizeT* node_id_to_pos                   = graph_details_d->sorted_poa_node_map;
    uint16_t* sorted_poa_local_edge_count   = graph_details_d->sorted_poa_local_edge_count;
    int32_t* consensus_scores               = graph_details_d->consensus_scores;
    SizeT* consensus_predecessors           = graph_details_d->consensus_predecessors;
    uint8_t* node_marks                     = graph_details_d->node_marks;
    bool* check_aligned_nodes               = graph_details_d->check_aligned_nodes;
    SizeT* nodes_to_visit                   = graph_details_d->nodes_to_visit;
    uint16_t* node_coverage_counts          = graph_details_d->node_coverage_counts;
    uint16_t* outgoing_edges_coverage       = graph_details_d->outgoing_edges_coverage;
    uint16_t* outgoing_edges_coverage_count = graph_details_d->outgoing_edges_coverage_count;
    SizeT* node_id_to_msa_pos               = graph_details_d->node_id_to_msa_pos;

    int32_t nwindows_per_block = CUDAPOA_THREADS_PER_BLOCK / WARP_SIZE;
    int32_t nblocks            = (total_windows + nwindows_per_block - 1) / nwindows_per_block;

    CGA_CU_CHECK_ERR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

    int32_t consensus_num_blocks = (total_windows / CUDAPOA_MAX_CONSENSUS_PER_BLOCK) + 1;
    if (cuda_banded_alignment)
    {
        if (output_mask & OutputType::consensus)
        {
            generatePOAKernel<CUDAPOA_BANDED_THREADS_PER_BLOCK, true, false, ScoreT, SizeT>
                <<<total_windows, CUDAPOA_BANDED_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                 sequences_d,
                                                                                 base_weights_d,
                                                                                 sequence_lengths_d,
                                                                                 window_details_d,
                                                                                 total_windows,
                                                                                 scores,
                                                                                 alignment_graph,
                                                                                 alignment_read,
                                                                                 nodes,
                                                                                 incoming_edges,
                                                                                 incoming_edge_count,
                                                                                 outgoing_edges,
                                                                                 outgoing_edge_count,
                                                                                 incoming_edge_w,
                                                                                 outgoing_edge_w,
                                                                                 sorted_poa,
                                                                                 node_id_to_pos,
                                                                                 node_alignments,
                                                                                 node_alignment_count,
                                                                                 sorted_poa_local_edge_count,
                                                                                 node_marks,
                                                                                 check_aligned_nodes,
                                                                                 nodes_to_visit,
                                                                                 node_coverage_counts,
                                                                                 gap_score,
                                                                                 mismatch_score,
                                                                                 match_score,
                                                                                 max_sequences_per_poa,
                                                                                 sequence_begin_nodes_ids,
                                                                                 outgoing_edges_coverage,
                                                                                 outgoing_edges_coverage_count,
                                                                                 batch_size.max_nodes_per_window,
                                                                                 batch_size.max_nodes_per_window_banded,
                                                                                 batch_size.max_matrix_graph_dimension,
                                                                                 batch_size.max_matrix_graph_dimension_banded,
                                                                                 batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateConsensusKernel<true, SizeT>
                <<<consensus_num_blocks, CUDAPOA_MAX_CONSENSUS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                       coverage_d,
                                                                                       sequence_lengths_d,
                                                                                       window_details_d,
                                                                                       total_windows,
                                                                                       nodes,
                                                                                       incoming_edges,
                                                                                       incoming_edge_count,
                                                                                       outgoing_edges,
                                                                                       outgoing_edge_count,
                                                                                       incoming_edge_w,
                                                                                       sorted_poa,
                                                                                       node_id_to_pos,
                                                                                       node_alignments,
                                                                                       node_alignment_count,
                                                                                       consensus_scores,
                                                                                       consensus_predecessors,
                                                                                       node_coverage_counts,
                                                                                       batch_size.max_nodes_per_window,
                                                                                       batch_size.max_nodes_per_window_banded,
                                                                                       batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
        if (output_mask & OutputType::msa)
        {
            generatePOAKernel<CUDAPOA_BANDED_THREADS_PER_BLOCK, true, true, ScoreT, SizeT>
                <<<total_windows, CUDAPOA_BANDED_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                 sequences_d,
                                                                                 base_weights_d,
                                                                                 sequence_lengths_d,
                                                                                 window_details_d,
                                                                                 total_windows,
                                                                                 scores,
                                                                                 alignment_graph,
                                                                                 alignment_read,
                                                                                 nodes,
                                                                                 incoming_edges,
                                                                                 incoming_edge_count,
                                                                                 outgoing_edges,
                                                                                 outgoing_edge_count,
                                                                                 incoming_edge_w,
                                                                                 outgoing_edge_w,
                                                                                 sorted_poa,
                                                                                 node_id_to_pos,
                                                                                 node_alignments,
                                                                                 node_alignment_count,
                                                                                 sorted_poa_local_edge_count,
                                                                                 node_marks,
                                                                                 check_aligned_nodes,
                                                                                 nodes_to_visit,
                                                                                 node_coverage_counts,
                                                                                 gap_score,
                                                                                 mismatch_score,
                                                                                 match_score,
                                                                                 max_sequences_per_poa,
                                                                                 sequence_begin_nodes_ids,
                                                                                 outgoing_edges_coverage,
                                                                                 outgoing_edges_coverage_count,
                                                                                 batch_size.max_nodes_per_window,
                                                                                 batch_size.max_nodes_per_window_banded,
                                                                                 batch_size.max_matrix_graph_dimension,
                                                                                 batch_size.max_matrix_graph_dimension_banded,
                                                                                 batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateMSAKernel<true, SizeT>
                <<<total_windows, max_sequences_per_poa, 0, stream>>>(nodes,
                                                                      consensus_d,
                                                                      window_details_d,
                                                                      incoming_edge_count,
                                                                      incoming_edges,
                                                                      outgoing_edge_count,
                                                                      outgoing_edges,
                                                                      outgoing_edges_coverage,
                                                                      outgoing_edges_coverage_count,
                                                                      node_id_to_msa_pos,
                                                                      sequence_begin_nodes_ids,
                                                                      multiple_sequence_alignments,
                                                                      sequence_lengths_d,
                                                                      sorted_poa,
                                                                      node_alignments,
                                                                      node_alignment_count,
                                                                      max_sequences_per_poa,
                                                                      node_id_to_pos,
                                                                      node_marks,
                                                                      check_aligned_nodes,
                                                                      nodes_to_visit,
                                                                      batch_size.max_nodes_per_window,
                                                                      batch_size.max_nodes_per_window_banded,
                                                                      batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
    }
    else
    {
        if (output_mask & OutputType::consensus)
        {
            generatePOAKernel<CUDAPOA_THREADS_PER_BLOCK, false, false, ScoreT, SizeT>
                <<<nblocks, CUDAPOA_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                    sequences_d,
                                                                    base_weights_d,
                                                                    sequence_lengths_d,
                                                                    window_details_d,
                                                                    total_windows,
                                                                    scores,
                                                                    alignment_graph,
                                                                    alignment_read,
                                                                    nodes,
                                                                    incoming_edges,
                                                                    incoming_edge_count,
                                                                    outgoing_edges,
                                                                    outgoing_edge_count,
                                                                    incoming_edge_w,
                                                                    outgoing_edge_w,
                                                                    sorted_poa,
                                                                    node_id_to_pos,
                                                                    node_alignments,
                                                                    node_alignment_count,
                                                                    sorted_poa_local_edge_count,
                                                                    node_marks,
                                                                    check_aligned_nodes,
                                                                    nodes_to_visit,
                                                                    node_coverage_counts,
                                                                    gap_score,
                                                                    mismatch_score,
                                                                    match_score,
                                                                    max_sequences_per_poa,
                                                                    sequence_begin_nodes_ids,
                                                                    outgoing_edges_coverage,
                                                                    outgoing_edges_coverage_count,
                                                                    batch_size.max_nodes_per_window,
                                                                    batch_size.max_nodes_per_window_banded,
                                                                    batch_size.max_matrix_graph_dimension,
                                                                    batch_size.max_matrix_graph_dimension_banded,
                                                                    batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateConsensusKernel<false, SizeT>
                <<<consensus_num_blocks, CUDAPOA_MAX_CONSENSUS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                       coverage_d,
                                                                                       sequence_lengths_d,
                                                                                       window_details_d,
                                                                                       total_windows,
                                                                                       nodes,
                                                                                       incoming_edges,
                                                                                       incoming_edge_count,
                                                                                       outgoing_edges,
                                                                                       outgoing_edge_count,
                                                                                       incoming_edge_w,
                                                                                       sorted_poa,
                                                                                       node_id_to_pos,
                                                                                       node_alignments,
                                                                                       node_alignment_count,
                                                                                       consensus_scores,
                                                                                       consensus_predecessors,
                                                                                       node_coverage_counts,
                                                                                       batch_size.max_nodes_per_window,
                                                                                       batch_size.max_nodes_per_window_banded,
                                                                                       batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
        if (output_mask & OutputType::msa)
        {
            generatePOAKernel<CUDAPOA_THREADS_PER_BLOCK, false, true, ScoreT, SizeT>
                <<<nblocks, CUDAPOA_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                    sequences_d,
                                                                    base_weights_d,
                                                                    sequence_lengths_d,
                                                                    window_details_d,
                                                                    total_windows,
                                                                    scores,
                                                                    alignment_graph,
                                                                    alignment_read,
                                                                    nodes,
                                                                    incoming_edges,
                                                                    incoming_edge_count,
                                                                    outgoing_edges,
                                                                    outgoing_edge_count,
                                                                    incoming_edge_w,
                                                                    outgoing_edge_w,
                                                                    sorted_poa,
                                                                    node_id_to_pos,
                                                                    node_alignments,
                                                                    node_alignment_count,
                                                                    sorted_poa_local_edge_count,
                                                                    node_marks,
                                                                    check_aligned_nodes,
                                                                    nodes_to_visit,
                                                                    node_coverage_counts,
                                                                    gap_score,
                                                                    mismatch_score,
                                                                    match_score,
                                                                    max_sequences_per_poa,
                                                                    sequence_begin_nodes_ids,
                                                                    outgoing_edges_coverage,
                                                                    outgoing_edges_coverage_count,
                                                                    batch_size.max_nodes_per_window,
                                                                    batch_size.max_nodes_per_window_banded,
                                                                    batch_size.max_matrix_graph_dimension,
                                                                    batch_size.max_matrix_graph_dimension_banded,
                                                                    batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateMSAKernel<false, SizeT>
                <<<total_windows, max_sequences_per_poa, 0, stream>>>(nodes,
                                                                      consensus_d,
                                                                      window_details_d,
                                                                      incoming_edge_count,
                                                                      incoming_edges,
                                                                      outgoing_edge_count,
                                                                      outgoing_edges,
                                                                      outgoing_edges_coverage,
                                                                      outgoing_edges_coverage_count,
                                                                      node_id_to_msa_pos,
                                                                      sequence_begin_nodes_ids,
                                                                      multiple_sequence_alignments,
                                                                      sequence_lengths_d,
                                                                      sorted_poa,
                                                                      node_alignments,
                                                                      node_alignment_count,
                                                                      max_sequences_per_poa,
                                                                      node_id_to_pos,
                                                                      node_marks,
                                                                      check_aligned_nodes,
                                                                      nodes_to_visit,
                                                                      batch_size.max_nodes_per_window,
                                                                      batch_size.max_nodes_per_window_banded,
                                                                      batch_size.max_concensus_size);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
    }
}

// Host function call for POA kernel
void generatePOA(claragenomics::cudapoa::OutputDetails* output_details_d,
                 void* input_details_void,
                 int32_t total_windows,
                 hipStream_t stream,
                 void* alignment_details_void,
                 void* graph_details_void,
                 int16_t gap_score,
                 int16_t mismatch_score,
                 int16_t match_score,
                 bool cuda_banded_alignment,
                 uint32_t max_sequences_per_poa,
                 int8_t output_mask,
                 const BatchSize& batch_size)
{
    if (use32bitScore(batch_size, gap_score, mismatch_score, match_score))
    {
        if (use32bitSize(batch_size, cuda_banded_alignment))
        {
            auto* alignment_details_d = static_cast<claragenomics::cudapoa::AlignmentDetails<int32_t, int32_t>*>(alignment_details_void);
            auto* input_details_d     = static_cast<claragenomics::cudapoa::InputDetails<int32_t>*>(input_details_void);
            auto* graph_details_d     = static_cast<claragenomics::cudapoa::GraphDetails<int32_t>*>(graph_details_void);

            claragenomics::cudapoa::generatePOAtemplated<int32_t, int32_t>(output_details_d,
                                                                           input_details_d,
                                                                           total_windows,
                                                                           stream,
                                                                           alignment_details_d,
                                                                           graph_details_d,
                                                                           (int32_t)gap_score,
                                                                           (int32_t)mismatch_score,
                                                                           (int32_t)match_score,
                                                                           cuda_banded_alignment,
                                                                           max_sequences_per_poa,
                                                                           output_mask,
                                                                           batch_size);
        }
        else
        {
            auto* alignment_details_d = static_cast<claragenomics::cudapoa::AlignmentDetails<int32_t, int16_t>*>(alignment_details_void);
            auto* input_details_d     = static_cast<claragenomics::cudapoa::InputDetails<int16_t>*>(input_details_void);
            auto* graph_details_d     = static_cast<claragenomics::cudapoa::GraphDetails<int16_t>*>(graph_details_void);

            claragenomics::cudapoa::generatePOAtemplated<int32_t, int16_t>(output_details_d,
                                                                           input_details_d,
                                                                           total_windows,
                                                                           stream,
                                                                           alignment_details_d,
                                                                           graph_details_d,
                                                                           (int32_t)gap_score,
                                                                           (int32_t)mismatch_score,
                                                                           (int32_t)match_score,
                                                                           cuda_banded_alignment,
                                                                           max_sequences_per_poa,
                                                                           output_mask,
                                                                           batch_size);
        }
    }
    else
    {
        // if ScoreT is 16-bit, then it's safe to assume SizeT is 16-bit
        auto* alignment_details_d = static_cast<claragenomics::cudapoa::AlignmentDetails<int16_t, int16_t>*>(alignment_details_void);
        auto* input_details_d     = static_cast<claragenomics::cudapoa::InputDetails<int16_t>*>(input_details_void);
        auto* graph_details_d     = static_cast<claragenomics::cudapoa::GraphDetails<int16_t>*>(graph_details_void);

        claragenomics::cudapoa::generatePOAtemplated<int16_t, int16_t>(output_details_d,
                                                                       input_details_d,
                                                                       total_windows,
                                                                       stream,
                                                                       alignment_details_d,
                                                                       graph_details_d,
                                                                       gap_score,
                                                                       mismatch_score,
                                                                       match_score,
                                                                       cuda_banded_alignment,
                                                                       max_sequences_per_poa,
                                                                       output_mask,
                                                                       batch_size);
    }
}

void addAlignment(uint8_t* nodes,
                  void* node_count_void,
                  void* node_alignments_void, uint16_t* node_alignment_count,
                  void* incoming_edges_void, uint16_t* incoming_edge_count,
                  void* outgoing_edges_void, uint16_t* outgoing_edge_count,
                  uint16_t* incoming_edge_w, uint16_t* outgoing_edge_w,
                  void* alignment_length_void,
                  void* graph_void,
                  void* alignment_graph_void,
                  uint8_t* read,
                  void* alignment_read_void,
                  uint16_t* node_coverage_counts,
                  int8_t* base_weights,
                  void* sequence_begin_nodes_ids_void,
                  uint16_t* outgoing_edges_coverage,
                  uint16_t* outgoing_edges_coverage_count,
                  uint16_t s,
                  uint32_t max_sequences_per_poa,
                  uint32_t max_limit_nodes_per_window,
                  bool cuda_banded_alignment,
                  const BatchSize& batch_size)
{
    if (use32bitSize(batch_size, cuda_banded_alignment))
    {
        auto* node_count               = static_cast<int32_t*>(node_count_void);
        auto* node_alignments          = static_cast<int32_t*>(node_alignments_void);
        auto* incoming_edges           = static_cast<int32_t*>(incoming_edges_void);
        auto* outgoing_edges           = static_cast<int32_t*>(outgoing_edges_void);
        auto* alignment_length         = static_cast<int32_t*>(alignment_length_void);
        auto* graph                    = static_cast<int32_t*>(graph_void);
        auto* alignment_graph          = static_cast<int32_t*>(alignment_graph_void);
        auto* alignment_read           = static_cast<int32_t*>(alignment_read_void);
        auto* sequence_begin_nodes_ids = static_cast<int32_t*>(sequence_begin_nodes_ids_void);

        addAlignmentTemplated<int32_t>(nodes,
                                       node_count,
                                       node_alignments, node_alignment_count,
                                       incoming_edges, incoming_edge_count,
                                       outgoing_edges, outgoing_edge_count,
                                       incoming_edge_w, outgoing_edge_w,
                                       alignment_length,
                                       graph,
                                       alignment_graph,
                                       read,
                                       alignment_read,
                                       node_coverage_counts,
                                       base_weights,
                                       sequence_begin_nodes_ids,
                                       outgoing_edges_coverage,
                                       outgoing_edges_coverage_count,
                                       s,
                                       max_sequences_per_poa,
                                       max_limit_nodes_per_window);
    }
    else
    {
        auto* node_count               = static_cast<int16_t*>(node_count_void);
        auto* node_alignments          = static_cast<int16_t*>(node_alignments_void);
        auto* incoming_edges           = static_cast<int16_t*>(incoming_edges_void);
        auto* outgoing_edges           = static_cast<int16_t*>(outgoing_edges_void);
        auto* alignment_length         = static_cast<int16_t*>(alignment_length_void);
        auto* graph                    = static_cast<int16_t*>(graph_void);
        auto* alignment_graph          = static_cast<int16_t*>(alignment_graph_void);
        auto* alignment_read           = static_cast<int16_t*>(alignment_read_void);
        auto* sequence_begin_nodes_ids = static_cast<int16_t*>(sequence_begin_nodes_ids_void);

        addAlignmentTemplated<int16_t>(nodes,
                                       node_count,
                                       node_alignments, node_alignment_count,
                                       incoming_edges, incoming_edge_count,
                                       outgoing_edges, outgoing_edge_count,
                                       incoming_edge_w, outgoing_edge_w,
                                       alignment_length,
                                       graph,
                                       alignment_graph,
                                       read,
                                       alignment_read,
                                       node_coverage_counts,
                                       base_weights,
                                       sequence_begin_nodes_ids,
                                       outgoing_edges_coverage,
                                       outgoing_edges_coverage_count,
                                       s,
                                       max_sequences_per_poa,
                                       max_limit_nodes_per_window);
    }
}

void runNW(uint8_t* nodes,
           void* graph_void,
           void* node_id_to_pos_void,
           int32_t graph_count,
           uint16_t* incoming_edge_count,
           void* incoming_edges_void,
           uint16_t* outgoing_edge_count,
           void* outgoing_edges_void,
           uint8_t* read,
           uint16_t read_count,
           int16_t* scores,
           int32_t scores_width,
           void* alignment_graph_void,
           void* alignment_read_void,
           int16_t gap_score,
           int16_t mismatch_score,
           int16_t match_score,
           void* aligned_nodes_void,
           bool cuda_banded_alignment,
           const BatchSize& batch_size)
{
    if (use32bitSize(batch_size, cuda_banded_alignment))
    {
        auto* graph           = static_cast<int32_t*>(graph_void);
        auto* node_id_to_pos  = static_cast<int32_t*>(node_id_to_pos_void);
        auto* incoming_edges  = static_cast<int32_t*>(incoming_edges_void);
        auto* outgoing_edges  = static_cast<int32_t*>(outgoing_edges_void);
        auto* alignment_graph = static_cast<int32_t*>(alignment_graph_void);
        auto* alignment_read  = static_cast<int32_t*>(alignment_read_void);
        auto* aligned_nodes   = static_cast<int32_t*>(aligned_nodes_void);

        runNWtemplated<int32_t>(nodes,
                                graph,
                                node_id_to_pos,
                                static_cast<int32_t>(graph_count),
                                incoming_edge_count,
                                incoming_edges,
                                outgoing_edge_count,
                                outgoing_edges,
                                read,
                                static_cast<int32_t>(read_count),
                                scores,
                                scores_width,
                                alignment_graph,
                                alignment_read,
                                gap_score,
                                mismatch_score,
                                match_score,
                                aligned_nodes);
    }
    else
    {
        auto* graph           = static_cast<int16_t*>(graph_void);
        auto* node_id_to_pos  = static_cast<int16_t*>(node_id_to_pos_void);
        auto* incoming_edges  = static_cast<int16_t*>(incoming_edges_void);
        auto* outgoing_edges  = static_cast<int16_t*>(outgoing_edges_void);
        auto* alignment_graph = static_cast<int16_t*>(alignment_graph_void);
        auto* alignment_read  = static_cast<int16_t*>(alignment_read_void);
        auto* aligned_nodes   = static_cast<int16_t*>(aligned_nodes_void);

        runNWtemplated<int16_t>(nodes,
                                graph,
                                node_id_to_pos,
                                static_cast<int16_t>(graph_count),
                                incoming_edge_count,
                                incoming_edges,
                                outgoing_edge_count,
                                outgoing_edges,
                                read,
                                static_cast<int16_t>(read_count),
                                scores,
                                scores_width,
                                alignment_graph,
                                alignment_read,
                                gap_score,
                                mismatch_score,
                                match_score,
                                aligned_nodes);
    }
}

void generateConsensusTestHost(uint8_t* nodes,
                               int32_t node_count,
                               void* graph_void,
                               void* node_id_to_pos_void,
                               void* incoming_edges_void,
                               uint16_t* incoming_edge_count,
                               void* outgoing_edges_void,
                               uint16_t* outgoing_edge_count,
                               uint16_t* incoming_edge_w,
                               void* predecessors_void,
                               int32_t* scores,
                               uint8_t* consensus,
                               uint16_t* coverage,
                               uint16_t* node_coverage_counts,
                               void* node_alignments_void,
                               uint16_t* node_alignment_count,
                               uint32_t max_limit_consensus_size,
                               bool cuda_banded_alignment,
                               const BatchSize& batch_size)
{
    if (use32bitSize(batch_size, cuda_banded_alignment))
    {
        auto* graph           = static_cast<int32_t*>(graph_void);
        auto* node_id_to_pos  = static_cast<int32_t*>(node_id_to_pos_void);
        auto* incoming_edges  = static_cast<int32_t*>(incoming_edges_void);
        auto* outgoing_edges  = static_cast<int32_t*>(outgoing_edges_void);
        auto* predecessors    = static_cast<int32_t*>(predecessors_void);
        auto* node_alignments = static_cast<int32_t*>(node_alignments_void);

        generateConsensusTemplated(nodes,
                                   node_count,
                                   graph,
                                   node_id_to_pos,
                                   incoming_edges,
                                   incoming_edge_count,
                                   outgoing_edges,
                                   outgoing_edge_count,
                                   incoming_edge_w,
                                   predecessors,
                                   scores,
                                   consensus,
                                   coverage,
                                   node_coverage_counts,
                                   node_alignments,
                                   node_alignment_count,
                                   max_limit_consensus_size);
    }
    else
    {
        auto* graph           = static_cast<int16_t*>(graph_void);
        auto* node_id_to_pos  = static_cast<int16_t*>(node_id_to_pos_void);
        auto* incoming_edges  = static_cast<int16_t*>(incoming_edges_void);
        auto* outgoing_edges  = static_cast<int16_t*>(outgoing_edges_void);
        auto* predecessors    = static_cast<int16_t*>(predecessors_void);
        auto* node_alignments = static_cast<int16_t*>(node_alignments_void);

        generateConsensusTemplated(nodes,
                                   static_cast<int16_t>(node_count),
                                   graph,
                                   node_id_to_pos,
                                   incoming_edges,
                                   incoming_edge_count,
                                   outgoing_edges,
                                   outgoing_edge_count,
                                   incoming_edge_w,
                                   predecessors,
                                   scores,
                                   consensus,
                                   coverage,
                                   node_coverage_counts,
                                   node_alignments,
                                   node_alignment_count,
                                   max_limit_consensus_size);
    }
}

void runTopSort(void* sorted_poa_void,
                void* sorted_poa_node_map_void,
                int32_t node_count,
                uint16_t* incoming_edge_count,
                void* outgoing_edges_void,
                uint16_t* outgoing_edge_count,
                uint16_t* local_incoming_edge_count,
                bool cuda_banded_alignment,
                const BatchSize& batch_size)
{
    if (use32bitSize(batch_size, cuda_banded_alignment))
    {
        auto* sorted_poa          = static_cast<int32_t*>(sorted_poa_void);
        auto* sorted_poa_node_map = static_cast<int32_t*>(sorted_poa_node_map_void);
        auto* outgoing_edges      = static_cast<int32_t*>(outgoing_edges_void);

        runTopSortTemplated(sorted_poa,
                            sorted_poa_node_map,
                            node_count,
                            incoming_edge_count,
                            outgoing_edges,
                            outgoing_edge_count,
                            local_incoming_edge_count);
    }
    else
    {
        auto* sorted_poa          = static_cast<int16_t*>(sorted_poa_void);
        auto* sorted_poa_node_map = static_cast<int16_t*>(sorted_poa_node_map_void);
        auto* outgoing_edges      = static_cast<int16_t*>(outgoing_edges_void);

        runTopSortTemplated(sorted_poa,
                            sorted_poa_node_map,
                            static_cast<int16_t>(node_count),
                            incoming_edge_count,
                            outgoing_edges,
                            outgoing_edge_count,
                            local_incoming_edge_count);
    }
}

bool use32bitScore(const BatchSize& batch_size, const int16_t gap_score, const int16_t mismatch_score, const int16_t match_score)
{
    // theoretical max score takes place when sequence and graph completely match with each other
    int32_t upper_bound = batch_size.max_sequence_size * match_score;
    // theoretical min score takes place when sequence and graph do not include a single match
    // it is assumed max_sequence_size <= max_matrix_graph_dimension; gap_score and match_scores are negative, and match_score is positive
    int32_t lower_bound = batch_size.max_sequence_size * std::max(gap_score, mismatch_score) + (batch_size.max_matrix_graph_dimension - batch_size.max_sequence_size) * gap_score;
    // if theoretical upper or lower bound exceed the range represented by int16_t, then int32_t should be used
    return (upper_bound > INT16_MAX || (-lower_bound) > (INT16_MAX + 1));
}

bool use32bitSize(const BatchSize& batch_size, bool banded)
{
    int32_t max_length = batch_size.max_concensus_size;
    if (banded)
    {
        max_length = std::max(max_length, batch_size.max_matrix_graph_dimension_banded);
    }
    else
    {
        max_length = std::max(max_length, batch_size.max_matrix_graph_dimension);
    }
    max_length = std::max(max_length, batch_size.max_matrix_sequence_dimension);
    //if max array length in POA analysis exceeds the range represented by int16_t, then int32_t should be used
    return (max_length > INT16_MAX);
}

} // namespace cudapoa

} // namespace claragenomics
