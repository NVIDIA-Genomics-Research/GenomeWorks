#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

// Implementation file for CUDA POA kernels.

#include "cudapoa_kernels.cuh"
#include "cudapoa_nw.cu"
#include "cudapoa_nw_banded.cu"
#include "cudapoa_topsort.cu"
#include "cudapoa_add_alignment.cu"
#include "cudapoa_generate_consensus.cu"
#include "cudapoa_generate_msa.cu"
#include <cudautils/cudautils.hpp>

#include <stdio.h>

namespace claragenomics
{

namespace cudapoa
{

/**
 * @brief The main kernel that runs the partial order alignment
 *        algorithm.
 *
 * @param[out] consensus_d                Device buffer for generated consensus
 * @param[in] sequences_d                 Device buffer with sequences for all windows
 * @param[in] base_weights_d              Device buffer with base weights for all windows
 * @param[in] sequence_lengths_d          Device buffer sequence lengths
 * @param[in] window_details_d            Device buffer with structs 
 *                                        encapsulating sequence details per window
 * @param[in] total_window                Total number of windows to process
 * @param[in] scores                      Device scratch space that scores alignment matrix score
 * @param[in] alignment_graph_d           Device scratch space for backtrace alignment of graph
 * @param[in] alignment_read_d            Device scratch space for backtrace alignment of sequence
 * @param[in] nodes                       Device scratch space for storing unique nodes in graph
 * @param[in] incoming_edges              Device scratch space for storing incoming edges per node
 * @param[in] incoming_edges_count        Device scratch space for storing number of incoming edges per node
 * @param[in] outgoing_edges              Device scratch space for storing outgoing edges per node
 * @param[in] outgoing_edges_count        Device scratch space for storing number of outgoing edges per node
 * @param[in] incoming_edge_w             Device scratch space for storing weight of incoming edges
 * @param[in] outgoing_edge_w             Device scratch space for storing weight of outgoing edges
 * @param[in] sorted_poa                  Device scratch space for storing sorted graph
 * @param[in] node_id_to_pos              Device scratch space for mapping node ID to position in graph
 * @graph[in] node_alignments             Device scratch space for storing alignment nodes per node in graph
 * @param[in] node_alignment_count        Device scratch space for storing number of aligned nodes
 * @param[in] sorted_poa_local_edge_count Device scratch space for maintaining edge counts during topological sort
 * @param[in] node_marks_d_               Device scratch space for storing node marks when running spoa accurate top sort
 * @param[in] check_aligned_nodes_d_      Device scratch space for storing check for aligned nodes
 * @param[in] nodes_to_visit_d_           device scratch space for storing stack of nodes to be visited in topsort
 * @param[in] node_coverage_counts_d_     device scratch space for storing coverage of each node in graph.
 * @param[in] gap_score                   Score for inserting gap into alignment
 * @param[in] mismatch_score              Score for finding a mismatch in alignment
 * @param[in] match_score                 Score for finding a match in alignment
 */
template <int32_t TPB = 64, bool cuda_banded_alignment = false, bool msa = false>
__global__ void generatePOAKernel(uint8_t* consensus_d,
                                  uint8_t* sequences_d,
                                  int8_t* base_weights_d,
                                  uint16_t* sequence_lengths_d,
                                  claragenomics::cudapoa::WindowDetails* window_details_d,
                                  int32_t total_windows,
                                  int16_t* scores_d,
                                  int16_t* alignment_graph_d,
                                  int16_t* alignment_read_d,
                                  uint8_t* nodes_d,
                                  uint16_t* incoming_edges_d,
                                  uint16_t* incoming_edge_count_d,
                                  uint16_t* outgoing_edges_d,
                                  uint16_t* outgoing_edge_count_d,
                                  uint16_t* incoming_edge_w_d,
                                  uint16_t* outgoing_edge_w_d,
                                  uint16_t* sorted_poa_d,
                                  uint16_t* node_id_to_pos_d,
                                  uint16_t* node_alignments_d,
                                  uint16_t* node_alignment_count_d,
                                  uint16_t* sorted_poa_local_edge_count_d,
                                  uint8_t* node_marks_d_,
                                  bool* check_aligned_nodes_d_,
                                  uint16_t* nodes_to_visit_d_,
                                  uint16_t* node_coverage_counts_d_,
                                  int16_t gap_score,
                                  int16_t mismatch_score,
                                  int16_t match_score,
                                  uint32_t max_sequences_per_poa,
                                  uint16_t* sequence_begin_nodes_ids_d,
                                  uint16_t* outgoing_edges_coverage_d,
                                  uint16_t* outgoing_edges_coverage_count_d)
{

    // shared error indicator within a warp
    bool warp_error = false;

    int32_t nwindows_per_block = TPB / WARP_SIZE;
    int32_t warp_idx           = threadIdx.x / WARP_SIZE;
    int32_t lane_idx           = threadIdx.x % WARP_SIZE;
    int32_t window_idx         = blockIdx.x * nwindows_per_block + warp_idx;

    if (window_idx >= total_windows)
        return;

    // These are not being changed to int32_t to make use of larger range
    // without having to use 2 registers which would be needed for 64bit
    uint32_t matrix_sequence_dimension = cuda_banded_alignment ? CUDAPOA_BANDED_MAX_MATRIX_SEQUENCE_DIMENSION : CUDAPOA_MAX_MATRIX_SEQUENCE_DIMENSION;
    uint32_t max_nodes_per_window      = cuda_banded_alignment ? CUDAPOA_MAX_NODES_PER_WINDOW_BANDED : CUDAPOA_MAX_NODES_PER_WINDOW;
    uint32_t max_graph_dimension       = cuda_banded_alignment ? CUDAPOA_MAX_MATRIX_GRAPH_DIMENSION_BANDED : CUDAPOA_MAX_MATRIX_GRAPH_DIMENSION;

    // Find the buffer offsets for each thread within the global memory buffers.
    uint8_t* nodes                        = &nodes_d[max_nodes_per_window * window_idx];
    uint16_t* incoming_edges              = &incoming_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* incoming_edge_count         = &incoming_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* outoing_edges               = &outgoing_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_count         = &outgoing_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* incoming_edge_weights       = &incoming_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_weights       = &outgoing_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* sorted_poa                  = &sorted_poa_d[window_idx * max_nodes_per_window];
    uint16_t* node_id_to_pos              = &node_id_to_pos_d[window_idx * max_nodes_per_window];
    uint16_t* node_alignments             = &node_alignments_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_ALIGNMENTS];
    uint16_t* node_alignment_count        = &node_alignment_count_d[window_idx * max_nodes_per_window];
    uint16_t* sorted_poa_local_edge_count = &sorted_poa_local_edge_count_d[window_idx * max_nodes_per_window];

    int16_t* scores                = &scores_d[max_graph_dimension * matrix_sequence_dimension * window_idx];
    int16_t* alignment_graph       = &alignment_graph_d[max_graph_dimension * window_idx];
    int16_t* alignment_read        = &alignment_read_d[max_graph_dimension * window_idx];
    uint16_t* node_coverage_counts = &node_coverage_counts_d_[max_nodes_per_window * window_idx];

#ifdef SPOA_ACCURATE
    uint8_t* node_marks       = &node_marks_d_[max_nodes_per_window * window_idx];
    bool* check_aligned_nodes = &check_aligned_nodes_d_[max_nodes_per_window * window_idx];
    uint16_t* nodes_to_visit  = &nodes_to_visit_d_[max_nodes_per_window * window_idx];
#endif

    uint16_t* sequence_lengths = &sequence_lengths_d[window_details_d[window_idx].seq_len_buffer_offset];

    uint32_t num_sequences = window_details_d[window_idx].num_seqs;
    uint8_t* sequence      = &sequences_d[window_details_d[window_idx].seq_starts];
    int8_t* base_weights   = &base_weights_d[window_details_d[window_idx].seq_starts];

    uint8_t* consensus = &consensus_d[window_idx * CUDAPOA_MAX_CONSENSUS_SIZE];

    uint16_t* sequence_begin_nodes_ids      = nullptr;
    uint16_t* outgoing_edges_coverage       = nullptr;
    uint16_t* outgoing_edges_coverage_count = nullptr;

    if (msa)
    {
        sequence_begin_nodes_ids      = &sequence_begin_nodes_ids_d[window_idx * max_sequences_per_poa];
        outgoing_edges_coverage       = &outgoing_edges_coverage_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES * max_sequences_per_poa];
        outgoing_edges_coverage_count = &outgoing_edges_coverage_count_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    }

    if (lane_idx == 0)
    {
        // Create backbone for window based on first sequence in window.
        nodes[0]                                     = sequence[0];
        sorted_poa[0]                                = 0;
        incoming_edge_count[0]                       = 0;
        node_alignment_count[0]                      = 0;
        node_id_to_pos[0]                            = 0;
        outgoing_edge_count[sequence_lengths[0] - 1] = 0;
        incoming_edge_weights[0]                     = base_weights[0];
        node_coverage_counts[0]                      = 1;
        if (msa)
        {
            sequence_begin_nodes_ids[0] = 0;
        }

        //Build the rest of the graphs
        for (uint16_t nucleotide_idx = 1; nucleotide_idx < sequence_lengths[0]; nucleotide_idx++)
        {
            nodes[nucleotide_idx]                                          = sequence[nucleotide_idx];
            sorted_poa[nucleotide_idx]                                     = nucleotide_idx;
            outoing_edges[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES]   = nucleotide_idx;
            outgoing_edge_count[nucleotide_idx - 1]                        = 1;
            incoming_edges[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES]        = nucleotide_idx - uint16_t(1);
            incoming_edge_weights[nucleotide_idx * CUDAPOA_MAX_NODE_EDGES] = base_weights[nucleotide_idx - 1] + base_weights[nucleotide_idx];
            incoming_edge_count[nucleotide_idx]                            = 1;
            node_alignment_count[nucleotide_idx]                           = 0;
            node_id_to_pos[nucleotide_idx]                                 = nucleotide_idx;
            node_coverage_counts[nucleotide_idx]                           = 1;
            if (msa)
            {
                outgoing_edges_coverage[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES * max_sequences_per_poa] = 0;
                outgoing_edges_coverage_count[(nucleotide_idx - 1) * CUDAPOA_MAX_NODE_EDGES]                   = 1;
            }
        }

        // Clear error code for window.
        consensus[0] = CUDAPOA_KERNEL_NOERROR_ENCOUNTERED;
    }

    __syncwarp();

    //printf("window id %d, sequence %d\n", window_idx, num_sequences_in_window - 1);

    // Align each subsequent read, add alignment to graph, run topoligical sort.
    for (uint16_t s = 1; s < num_sequences; s++)
    {
        uint16_t seq_len = sequence_lengths[s];
        sequence += sequence_lengths[s - 1];     // increment the pointer so it is pointing to correct sequence data
        base_weights += sequence_lengths[s - 1]; // increment the pointer so it is pointing to correct sequence data

        if (lane_idx == 0)
        {
            if (sequence_lengths[0] >= CUDAPOA_MAX_NODES_PER_WINDOW)
            {
                consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
                consensus[1] = static_cast<uint8_t>(StatusType::node_count_exceeded_maximum_graph_size);
                warp_error   = true;
            }
        }

        warp_error = __shfl_sync(0xffffffff, warp_error, 0);
        if (warp_error)
        {
            return;
        }

        // Run Needleman-Wunsch alignment between graph and new sequence.
        uint16_t alignment_length;

        if (cuda_banded_alignment)
        {
            alignment_length = runNeedlemanWunschBanded(nodes,
                                                        sorted_poa,
                                                        node_id_to_pos,
                                                        sequence_lengths[0],
                                                        incoming_edge_count,
                                                        incoming_edges,
                                                        outgoing_edge_count,
                                                        outoing_edges,
                                                        sequence,
                                                        seq_len,
                                                        scores,
                                                        alignment_graph,
                                                        alignment_read,
                                                        gap_score,
                                                        mismatch_score,
                                                        match_score);
        }
        else
        {
            alignment_length = runNeedlemanWunsch<uint8_t, uint16_t, int16_t, TPB>(nodes,
                                                                                   sorted_poa,
                                                                                   node_id_to_pos,
                                                                                   sequence_lengths[0],
                                                                                   incoming_edge_count,
                                                                                   incoming_edges,
                                                                                   outgoing_edge_count,
                                                                                   outoing_edges,
                                                                                   sequence,
                                                                                   seq_len,
                                                                                   scores,
                                                                                   alignment_graph,
                                                                                   alignment_read,
                                                                                   gap_score,
                                                                                   mismatch_score,
                                                                                   match_score);
        }

        __syncwarp();
        //printf("%d %d %d\n", s, window_idx, alignment_length);
        if (alignment_length == UINT16_MAX)
        {
            if (lane_idx == 0)
            {
                consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
                consensus[1] = static_cast<uint8_t>(StatusType::loop_count_exceeded_upper_bound);
            }
            return;
        }

        if (lane_idx == 0)
        {

            // Add alignment to graph.
            //printf("running add\n");
            uint16_t new_node_count = addAlignmentToGraph<msa>(nodes, sequence_lengths[0],
                                                               node_alignments, node_alignment_count,
                                                               incoming_edges, incoming_edge_count,
                                                               outoing_edges, outgoing_edge_count,
                                                               incoming_edge_weights, outgoing_edge_weights,
                                                               alignment_length,
                                                               sorted_poa, alignment_graph,
                                                               sequence, alignment_read,
                                                               node_coverage_counts,
                                                               base_weights,
                                                               (sequence_begin_nodes_ids + s),
                                                               outgoing_edges_coverage,
                                                               outgoing_edges_coverage_count,
                                                               s,
                                                               max_sequences_per_poa);

            sequence_lengths[0] = new_node_count;
            // Run a topsort on the graph. Not strictly necessary at this point
            //printf("running topsort\n");
            if (new_node_count < CUDAPOA_MAX_NODES_PER_WINDOW)
            {
#ifdef SPOA_ACCURATE
                // Exactly matches racon CPU results
                raconTopologicalSortDeviceUtil(sorted_poa,
                                               node_id_to_pos,
                                               new_node_count,
                                               incoming_edge_count,
                                               incoming_edges,
                                               node_alignment_count,
                                               node_alignments,
                                               node_marks,
                                               check_aligned_nodes,
                                               nodes_to_visit,
                                               cuda_banded_alignment);
#else
                // Faster top sort
                topologicalSortDeviceUtil(sorted_poa,
                                          node_id_to_pos,
                                          new_node_count,
                                          incoming_edge_count,
                                          outoing_edges,
                                          outgoing_edge_count,
                                          sorted_poa_local_edge_count);
#endif
            }
        }

        __syncwarp();
    }
}

// Host function call for POA kernel.
void generatePOA(claragenomics::cudapoa::OutputDetails* output_details_d,
                 claragenomics::cudapoa::InputDetails* input_details_d,
                 int32_t total_windows,
                 hipStream_t stream,
                 claragenomics::cudapoa::AlignmentDetails* alignment_details_d,
                 claragenomics::cudapoa::GraphDetails* graph_details_d,
                 int16_t gap_score,
                 int16_t mismatch_score,
                 int16_t match_score,
                 bool cuda_banded_alignment,
                 uint32_t max_sequences_per_poa,
                 int8_t output_mask)
{
    // unpack output details
    uint8_t* consensus_d                  = output_details_d->consensus;
    uint16_t* coverage_d                  = output_details_d->coverage;
    uint8_t* multiple_sequence_alignments = output_details_d->multiple_sequence_alignments;

    // unpack input details
    uint8_t* sequences_d               = input_details_d->sequences;
    int8_t* base_weights_d             = input_details_d->base_weights;
    uint16_t* sequence_lengths_d       = input_details_d->sequence_lengths;
    WindowDetails* window_details_d    = input_details_d->window_details;
    uint16_t* sequence_begin_nodes_ids = input_details_d->sequence_begin_nodes_ids;

    // unpack alignment details
    int16_t* scores          = alignment_details_d->scores;
    int16_t* alignment_graph = alignment_details_d->alignment_graph;
    int16_t* alignment_read  = alignment_details_d->alignment_read;
    // unpack graph details
    uint8_t* nodes                          = graph_details_d->nodes;
    uint16_t* node_alignments               = graph_details_d->node_alignments;
    uint16_t* node_alignment_count          = graph_details_d->node_alignment_count;
    uint16_t* incoming_edges                = graph_details_d->incoming_edges;
    uint16_t* incoming_edge_count           = graph_details_d->incoming_edge_count;
    uint16_t* outgoing_edges                = graph_details_d->outgoing_edges;
    uint16_t* outgoing_edge_count           = graph_details_d->outgoing_edge_count;
    uint16_t* incoming_edge_w               = graph_details_d->incoming_edge_weights;
    uint16_t* outgoing_edge_w               = graph_details_d->outgoing_edge_weights;
    uint16_t* sorted_poa                    = graph_details_d->sorted_poa;
    uint16_t* node_id_to_pos                = graph_details_d->sorted_poa_node_map;
    uint16_t* sorted_poa_local_edge_count   = graph_details_d->sorted_poa_local_edge_count;
    int32_t* consensus_scores               = graph_details_d->consensus_scores;
    int16_t* consensus_predecessors         = graph_details_d->consensus_predecessors;
    uint8_t* node_marks                     = graph_details_d->node_marks;
    bool* check_aligned_nodes               = graph_details_d->check_aligned_nodes;
    uint16_t* nodes_to_visit                = graph_details_d->nodes_to_visit;
    uint16_t* node_coverage_counts          = graph_details_d->node_coverage_counts;
    uint16_t* outgoing_edges_coverage       = graph_details_d->outgoing_edges_coverage;
    uint16_t* outgoing_edges_coverage_count = graph_details_d->outgoing_edges_coverage_count;
    int16_t* node_id_to_msa_pos             = graph_details_d->node_id_to_msa_pos;

    int32_t nwindows_per_block = CUDAPOA_THREADS_PER_BLOCK / WARP_SIZE;
    int32_t nblocks            = (total_windows + nwindows_per_block - 1) / nwindows_per_block;

    CGA_CU_CHECK_ERR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

    int32_t consensus_num_blocks = (total_windows / CUDAPOA_MAX_CONSENSUS_PER_BLOCK) + 1;
    if (cuda_banded_alignment)
    {
        if (output_mask & OutputType::consensus)
        {
            generatePOAKernel<CUDAPOA_BANDED_THREADS_PER_BLOCK, true, false>
                <<<total_windows, CUDAPOA_BANDED_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                 sequences_d,
                                                                                 base_weights_d,
                                                                                 sequence_lengths_d,
                                                                                 window_details_d,
                                                                                 total_windows,
                                                                                 scores,
                                                                                 alignment_graph,
                                                                                 alignment_read,
                                                                                 nodes,
                                                                                 incoming_edges,
                                                                                 incoming_edge_count,
                                                                                 outgoing_edges,
                                                                                 outgoing_edge_count,
                                                                                 incoming_edge_w,
                                                                                 outgoing_edge_w,
                                                                                 sorted_poa,
                                                                                 node_id_to_pos,
                                                                                 node_alignments,
                                                                                 node_alignment_count,
                                                                                 sorted_poa_local_edge_count,
                                                                                 node_marks,
                                                                                 check_aligned_nodes,
                                                                                 nodes_to_visit,
                                                                                 node_coverage_counts,
                                                                                 gap_score,
                                                                                 mismatch_score,
                                                                                 match_score,
                                                                                 max_sequences_per_poa,
                                                                                 sequence_begin_nodes_ids,
                                                                                 outgoing_edges_coverage,
                                                                                 outgoing_edges_coverage_count);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateConsensusKernel<true>
                <<<consensus_num_blocks, CUDAPOA_MAX_CONSENSUS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                       coverage_d,
                                                                                       sequence_lengths_d,
                                                                                       window_details_d,
                                                                                       total_windows,
                                                                                       nodes,
                                                                                       incoming_edges,
                                                                                       incoming_edge_count,
                                                                                       outgoing_edges,
                                                                                       outgoing_edge_count,
                                                                                       incoming_edge_w,
                                                                                       sorted_poa,
                                                                                       node_id_to_pos,
                                                                                       node_alignments,
                                                                                       node_alignment_count,
                                                                                       consensus_scores,
                                                                                       consensus_predecessors,
                                                                                       node_coverage_counts);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
        if (output_mask & OutputType::msa)
        {
            generatePOAKernel<CUDAPOA_BANDED_THREADS_PER_BLOCK, true, true>
                <<<total_windows, CUDAPOA_BANDED_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                 sequences_d,
                                                                                 base_weights_d,
                                                                                 sequence_lengths_d,
                                                                                 window_details_d,
                                                                                 total_windows,
                                                                                 scores,
                                                                                 alignment_graph,
                                                                                 alignment_read,
                                                                                 nodes,
                                                                                 incoming_edges,
                                                                                 incoming_edge_count,
                                                                                 outgoing_edges,
                                                                                 outgoing_edge_count,
                                                                                 incoming_edge_w,
                                                                                 outgoing_edge_w,
                                                                                 sorted_poa,
                                                                                 node_id_to_pos,
                                                                                 node_alignments,
                                                                                 node_alignment_count,
                                                                                 sorted_poa_local_edge_count,
                                                                                 node_marks,
                                                                                 check_aligned_nodes,
                                                                                 nodes_to_visit,
                                                                                 node_coverage_counts,
                                                                                 gap_score,
                                                                                 mismatch_score,
                                                                                 match_score,
                                                                                 max_sequences_per_poa,
                                                                                 sequence_begin_nodes_ids,
                                                                                 outgoing_edges_coverage,
                                                                                 outgoing_edges_coverage_count);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateMSAKernel<true>
                <<<total_windows, max_sequences_per_poa, 0, stream>>>(nodes,
                                                                      consensus_d,
                                                                      window_details_d,
                                                                      incoming_edge_count,
                                                                      incoming_edges,
                                                                      outgoing_edge_count,
                                                                      outgoing_edges,
                                                                      outgoing_edges_coverage,
                                                                      outgoing_edges_coverage_count,
                                                                      node_id_to_msa_pos,
                                                                      sequence_begin_nodes_ids,
                                                                      multiple_sequence_alignments,
                                                                      sequence_lengths_d,
                                                                      sorted_poa,
                                                                      node_alignments,
                                                                      node_alignment_count,
                                                                      max_sequences_per_poa,
                                                                      node_id_to_pos,
                                                                      node_marks,
                                                                      check_aligned_nodes,
                                                                      nodes_to_visit);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
    }
    else
    {
        if (output_mask & OutputType::consensus)
        {
            generatePOAKernel<CUDAPOA_THREADS_PER_BLOCK, false, false>
                <<<nblocks, CUDAPOA_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                    sequences_d,
                                                                    base_weights_d,
                                                                    sequence_lengths_d,
                                                                    window_details_d,
                                                                    total_windows,
                                                                    scores,
                                                                    alignment_graph,
                                                                    alignment_read,
                                                                    nodes,
                                                                    incoming_edges,
                                                                    incoming_edge_count,
                                                                    outgoing_edges,
                                                                    outgoing_edge_count,
                                                                    incoming_edge_w,
                                                                    outgoing_edge_w,
                                                                    sorted_poa,
                                                                    node_id_to_pos,
                                                                    node_alignments,
                                                                    node_alignment_count,
                                                                    sorted_poa_local_edge_count,
                                                                    node_marks,
                                                                    check_aligned_nodes,
                                                                    nodes_to_visit,
                                                                    node_coverage_counts,
                                                                    gap_score,
                                                                    mismatch_score,
                                                                    match_score,
                                                                    max_sequences_per_poa,
                                                                    sequence_begin_nodes_ids,
                                                                    outgoing_edges_coverage,
                                                                    outgoing_edges_coverage_count);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateConsensusKernel<false>
                <<<consensus_num_blocks, CUDAPOA_MAX_CONSENSUS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                                       coverage_d,
                                                                                       sequence_lengths_d,
                                                                                       window_details_d,
                                                                                       total_windows,
                                                                                       nodes,
                                                                                       incoming_edges,
                                                                                       incoming_edge_count,
                                                                                       outgoing_edges,
                                                                                       outgoing_edge_count,
                                                                                       incoming_edge_w,
                                                                                       sorted_poa,
                                                                                       node_id_to_pos,
                                                                                       node_alignments,
                                                                                       node_alignment_count,
                                                                                       consensus_scores,
                                                                                       consensus_predecessors,
                                                                                       node_coverage_counts);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
        if (output_mask & OutputType::msa)
        {
            generatePOAKernel<CUDAPOA_THREADS_PER_BLOCK, false, true>
                <<<nblocks, CUDAPOA_THREADS_PER_BLOCK, 0, stream>>>(consensus_d,
                                                                    sequences_d,
                                                                    base_weights_d,
                                                                    sequence_lengths_d,
                                                                    window_details_d,
                                                                    total_windows,
                                                                    scores,
                                                                    alignment_graph,
                                                                    alignment_read,
                                                                    nodes,
                                                                    incoming_edges,
                                                                    incoming_edge_count,
                                                                    outgoing_edges,
                                                                    outgoing_edge_count,
                                                                    incoming_edge_w,
                                                                    outgoing_edge_w,
                                                                    sorted_poa,
                                                                    node_id_to_pos,
                                                                    node_alignments,
                                                                    node_alignment_count,
                                                                    sorted_poa_local_edge_count,
                                                                    node_marks,
                                                                    check_aligned_nodes,
                                                                    nodes_to_visit,
                                                                    node_coverage_counts,
                                                                    gap_score,
                                                                    mismatch_score,
                                                                    match_score,
                                                                    max_sequences_per_poa,
                                                                    sequence_begin_nodes_ids,
                                                                    outgoing_edges_coverage,
                                                                    outgoing_edges_coverage_count);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());

            generateMSAKernel<false>
                <<<total_windows, max_sequences_per_poa, 0, stream>>>(nodes,
                                                                      consensus_d,
                                                                      window_details_d,
                                                                      incoming_edge_count,
                                                                      incoming_edges,
                                                                      outgoing_edge_count,
                                                                      outgoing_edges,
                                                                      outgoing_edges_coverage,
                                                                      outgoing_edges_coverage_count,
                                                                      node_id_to_msa_pos,
                                                                      sequence_begin_nodes_ids,
                                                                      multiple_sequence_alignments,
                                                                      sequence_lengths_d,
                                                                      sorted_poa,
                                                                      node_alignments,
                                                                      node_alignment_count,
                                                                      max_sequences_per_poa,
                                                                      node_id_to_pos,
                                                                      node_marks,
                                                                      check_aligned_nodes,
                                                                      nodes_to_visit);
            CGA_CU_CHECK_ERR(hipPeekAtLastError());
        }
    }
}

} // namespace cudapoa

} // namespace claragenomics
