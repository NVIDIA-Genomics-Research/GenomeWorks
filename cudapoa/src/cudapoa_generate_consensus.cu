#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "cudapoa_kernels.cuh"

#include <claragenomics/utils/cudautils.hpp>

#include <stdio.h>

namespace claragenomics
{

namespace cudapoa
{

template <typename SizeT>
__device__
    SizeT
    branchCompletion(uint16_t max_score_id_pos,
                     uint8_t* nodes,
                     int32_t node_count,
                     SizeT* graph,
                     SizeT* incoming_edges,
                     uint16_t* incoming_edge_count,
                     SizeT* outgoing_edges,
                     uint16_t* outgoing_edge_count,
                     uint16_t* incoming_edge_w,
                     int32_t* scores,
                     SizeT* predecessors)
{
    SizeT node_id = graph[max_score_id_pos];

    // Go through all the outgoing edges of the node, and for
    // each of the end nodes of the edges clear the scores
    // for all the _other_ nodes that had edges to that end node.
    uint16_t out_edges = outgoing_edge_count[node_id];
    for (uint16_t oe = 0; oe < out_edges; oe++)
    {
        SizeT out_node_id          = outgoing_edges[node_id * CUDAPOA_MAX_NODE_EDGES + oe];
        uint16_t out_node_in_edges = incoming_edge_count[out_node_id];
        for (uint16_t ie = 0; ie < out_node_in_edges; ie++)
        {
            SizeT id = incoming_edges[out_node_id * CUDAPOA_MAX_NODE_EDGES + ie];
            if (id != node_id)
            {
                scores[id] = -1;
            }
        }
    }

    int32_t max_score  = 0;
    SizeT max_score_id = 0;
    // Run the same node weight traversal algorithm as always, to find the new
    // node with maximum weight.
    // We can start from the very next position in the graph rank because
    // the graph is topologically sorted and hence guarantees that successor of the current max
    // node will be processed again.
    for (SizeT graph_pos = max_score_id_pos + 1; graph_pos < node_count; graph_pos++)
    {
        node_id = graph[graph_pos];

        predecessors[node_id] = -1;
        int32_t score_node_id = -1;

        uint16_t in_edges = incoming_edge_count[node_id];
        for (uint16_t e = 0; e < in_edges; e++)
        {
            SizeT begin_node_id = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + e];
            if (scores[begin_node_id] == -1)
            {
                continue;
            }

            int32_t edge_w = static_cast<int32_t>(incoming_edge_w[node_id * CUDAPOA_MAX_NODE_EDGES + e]);
            if (score_node_id < edge_w ||
                (score_node_id == edge_w &&
                 scores[predecessors[node_id]] <= scores[begin_node_id]))
            {
                score_node_id         = edge_w;
                predecessors[node_id] = begin_node_id;
            }
        }

        if (predecessors[node_id] != -1)
        {
            score_node_id += scores[predecessors[node_id]];
        }

        if (max_score <= score_node_id)
        {
            max_score    = score_node_id;
            max_score_id = node_id;
        }
        //printf("max score %d, max score id %d, node id %d score %d\n", max_score, max_score_id, node_id, score_node_id);

        scores[node_id] = score_node_id;
    }

    return max_score_id;
}

/**
 * @brief Device function to generate consensus from a given graph.
 *        The input graph needs to be topologically sorted.
 *
 * @param[in] nodes                 Device buffer with unique nodes in graph
 * @param[in] node_count            Number of nodes in graph
 * @param[in] graph                 Device buffer with sorted graph
 * @param[in] node_id_to_pos        Device scratch space for mapping node ID to position in graph
 * @param[in] incoming_edges        Device buffer with incoming edges per node
 * @param[in] incoming_edges_count  Device buffer with number of incoming edges per node
 * @param[in] outgoing_edges        Device buffer with outgoing edges per node
 * @param[in] outgoing_edges_count  Device buffer with number of outgoing edges per node
 * @param[in] predecessors          Device buffer with predecessors of nodes while traversing graph during consensus
 * @param[in] scores                Device buffer with score of each node while traversing graph during consensus
 * @param[out] consensus            Device buffer for generated consensus
 * @param[out] coverate             Device buffer for coverage of each base in consensus
 * @param[out] node_coverage_counts Device buffer with coverage of each base in graph
 * @param[in] node_alignments       Device buffer with aligned nodes for each node in graph
 * @param[in] node_alignment)count  Device buffer with aligned nodes count for each node in graph
 */
template <typename SizeT>
__device__ void generateConsensus(uint8_t* nodes,
                                  int32_t node_count,
                                  SizeT* graph,
                                  SizeT* node_id_to_pos,
                                  SizeT* incoming_edges,
                                  uint16_t* incoming_edge_count,
                                  SizeT* outgoing_edges,
                                  uint16_t* outgoing_edge_count,
                                  uint16_t* incoming_edge_w,
                                  SizeT* predecessors,
                                  int32_t* scores,
                                  uint8_t* consensus,
                                  uint16_t* coverage,
                                  uint16_t* node_coverage_counts,
                                  SizeT* node_alignments,
                                  uint16_t* node_alignment_count,
                                  uint32_t max_limit_consensus_size)
{
    // Initialize scores and predecessors to default value.
    for (SizeT i = 0; i < node_count; i++)
    {
        predecessors[i] = -1;
        scores[i]       = -1;
    }

    SizeT max_score_id = 0;
    int32_t max_score  = -1;

    for (uint16_t graph_pos = 0; graph_pos < node_count; graph_pos++)
    {
        SizeT node_id     = graph[graph_pos];
        uint16_t in_edges = incoming_edge_count[node_id];

        int32_t score_node_id = scores[node_id];

        // For each node, go through it's incoming edges.
        // If the weight of any of the incoming edges is greater
        // than the score of the current node, or if the weight is equal
        // but the predecessors of the edge are heavier than the current node,
        // then update the score of the node to be the incoming edge weight.
        for (uint16_t e = 0; e < in_edges; e++)
        {
            int32_t edge_w      = static_cast<int32_t>(incoming_edge_w[node_id * CUDAPOA_MAX_NODE_EDGES + e]);
            SizeT begin_node_id = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + e];
            if (score_node_id < edge_w ||
                (score_node_id == edge_w &&
                 scores[predecessors[node_id]] <= scores[begin_node_id]))
            {
                score_node_id         = edge_w;
                predecessors[node_id] = begin_node_id;
            }
        }

        // Then update the score of the node to be the sum
        // of the score of the predecessor and itself.
        if (predecessors[node_id] != -1)
        {
            score_node_id += scores[predecessors[node_id]];
        }

        // Keep track of the highest weighted node.
        if (max_score <= score_node_id)
        {
            max_score_id = node_id;
            max_score    = score_node_id;
        }
        //printf("max score %d, max score id %d, node id %d score %d\n", max_score, max_score_id, node_id, score_node_id);

        scores[node_id] = score_node_id;
    }

    // If the node with maximum score isn't a leaf of the graph
    // then run a special branch completion function.
    uint16_t loop_count = 0;
    if (outgoing_edge_count[max_score_id] != 0)
    {
        while (outgoing_edge_count[max_score_id] != 0 && loop_count < node_count)
        {
            max_score_id = branchCompletion(node_id_to_pos[max_score_id],
                                            nodes,
                                            node_count,
                                            graph,
                                            incoming_edges,
                                            incoming_edge_count,
                                            outgoing_edges,
                                            outgoing_edge_count,
                                            incoming_edge_w,
                                            scores,
                                            predecessors);
            loop_count++;
        }
    }

    if (loop_count >= node_count)
    {
        consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
        consensus[1] = static_cast<uint8_t>(StatusType::loop_count_exceeded_upper_bound);
        return;
    }

    // Use consensus_pos to track which position to put new element in. Clip this to the maximum
    // size of consensus so as not to overwrite other good data.
    SizeT consensus_pos = 0;
    // Use consensus_count to track how many elements are in consensus. If more than the maximum
    // size, then consensus cannot be properly represented. So throw error.
    uint16_t consensus_count = 0;

    while (predecessors[max_score_id] != -1)
    {
        consensus[consensus_pos] = nodes[max_score_id];
        uint16_t cov             = node_coverage_counts[max_score_id];
        for (uint16_t a = 0; a < node_alignment_count[max_score_id]; a++)
        {
            cov += node_coverage_counts[node_alignments[max_score_id * CUDAPOA_MAX_NODE_ALIGNMENTS + a]];
        }
        coverage[consensus_pos] = cov;
        max_score_id            = predecessors[max_score_id];
        consensus_pos           = min(consensus_pos + 1, max_limit_consensus_size - 1);
        consensus_count++;
    }
    consensus[consensus_pos] = nodes[max_score_id];
    uint16_t cov             = node_coverage_counts[max_score_id];
    for (uint16_t a = 0; a < node_alignment_count[max_score_id]; a++)
    {
        cov += node_coverage_counts[node_alignments[max_score_id * CUDAPOA_MAX_NODE_ALIGNMENTS + a]];
    }
    coverage[consensus_pos] = cov;

    // Check consensus count against maximum size.
    if (consensus_count >= (max_limit_consensus_size - 1))
    {
        consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
        consensus[1] = static_cast<uint8_t>(StatusType::exceeded_maximum_sequence_size);
        return;
    }

    // Now we can increment consensus_pos without checking for upper bound because the max length
    // test above guarantees that consensus_pos <= (CUDAPOA_MAX_CONSENSUS_SIZE - 2).
    consensus_pos++;
    // Add EOL character at the end of the string.
    consensus[consensus_pos] = '\0';
}

template <bool cuda_banded_alignment = false, typename SizeT>
__global__ void generateConsensusKernel(uint8_t* consensus_d,
                                        uint16_t* coverage_d,
                                        SizeT* sequence_lengths_d,
                                        claragenomics::cudapoa::WindowDetails* window_details_d,
                                        int32_t total_windows,
                                        uint8_t* nodes_d,
                                        SizeT* incoming_edges_d,
                                        uint16_t* incoming_edge_count_d,
                                        SizeT* outgoing_edges_d,
                                        uint16_t* outgoing_edge_count_d,
                                        uint16_t* incoming_edge_w_d,
                                        SizeT* sorted_poa_d,
                                        SizeT* node_id_to_pos_d,
                                        SizeT* node_alignments_d,
                                        uint16_t* node_alignment_count_d,
                                        int32_t* consensus_scores_d,
                                        SizeT* consensus_predecessors_d,
                                        uint16_t* node_coverage_counts_d_,
                                        uint32_t max_limit_nodes_per_window,
                                        uint32_t max_limit_nodes_per_window_banded,
                                        uint32_t max_limit_consensus_size)
{
    //each thread will operate on a window
    int32_t window_idx = blockIdx.x * CUDAPOA_MAX_CONSENSUS_PER_BLOCK + threadIdx.x;

    if (window_idx >= total_windows)
        return;

    uint8_t* consensus = &consensus_d[window_idx * max_limit_consensus_size];

    if (consensus[0] == CUDAPOA_KERNEL_ERROR_ENCOUNTERED) //error during graph generation
        return;

    int32_t max_nodes_per_window = cuda_banded_alignment ? max_limit_nodes_per_window_banded : max_limit_nodes_per_window;

    // Find the buffer offsets for each thread within the global memory buffers.
    uint8_t* nodes                  = &nodes_d[max_nodes_per_window * window_idx];
    SizeT* incoming_edges           = &incoming_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* incoming_edge_count   = &incoming_edge_count_d[window_idx * max_nodes_per_window];
    SizeT* outgoing_edges           = &outgoing_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_count   = &outgoing_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* incoming_edge_weights = &incoming_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    SizeT* sorted_poa               = &sorted_poa_d[window_idx * max_nodes_per_window];
    SizeT* node_id_to_pos           = &node_id_to_pos_d[window_idx * max_nodes_per_window];
    SizeT* node_alignments          = &node_alignments_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_ALIGNMENTS];
    uint16_t* node_alignment_count  = &node_alignment_count_d[window_idx * max_nodes_per_window];
    uint16_t* node_coverage_counts  = &node_coverage_counts_d_[max_nodes_per_window * window_idx];
    SizeT* sequence_lengths         = &sequence_lengths_d[window_details_d[window_idx].seq_len_buffer_offset];

    //generate consensus
    uint16_t* coverage            = &coverage_d[window_idx * max_limit_consensus_size];
    int32_t* consensus_scores     = &consensus_scores_d[window_idx * max_nodes_per_window];
    SizeT* consensus_predecessors = &consensus_predecessors_d[window_idx * max_nodes_per_window];

    generateConsensus(nodes,
                      sequence_lengths[0],
                      sorted_poa,
                      node_id_to_pos,
                      incoming_edges,
                      incoming_edge_count,
                      outgoing_edges,
                      outgoing_edge_count,
                      incoming_edge_weights,
                      consensus_predecessors,
                      consensus_scores,
                      consensus,
                      coverage,
                      node_coverage_counts,
                      node_alignments,
                      node_alignment_count,
                      max_limit_consensus_size);
}

template <typename SizeT>
__global__ void generateConsensusTestKernel(uint8_t* nodes,
                                            int32_t node_count,
                                            SizeT* graph,
                                            SizeT* node_id_to_pos,
                                            SizeT* incoming_edges,
                                            uint16_t* incoming_edge_count,
                                            SizeT* outgoing_edges,
                                            uint16_t* outgoing_edge_count,
                                            uint16_t* incoming_edge_w,
                                            SizeT* predecessors,
                                            int32_t* scores,
                                            uint8_t* consensus,
                                            uint16_t* coverage,
                                            uint16_t* node_coverage_counts,
                                            SizeT* node_alignments,
                                            uint16_t* node_alignment_count,
                                            uint32_t max_limit_consensus_size)
{
    generateConsensus(nodes,
                      node_count,
                      graph,
                      node_id_to_pos,
                      incoming_edges,
                      incoming_edge_count,
                      outgoing_edges,
                      outgoing_edge_count,
                      incoming_edge_w,
                      predecessors,
                      scores,
                      consensus,
                      coverage,
                      node_coverage_counts,
                      node_alignments,
                      node_alignment_count,
                      max_limit_consensus_size);
}

template <typename SizeT>
void generateConsensusTemplated(uint8_t* nodes,
                                int32_t node_count,
                                SizeT* graph,
                                SizeT* node_id_to_pos,
                                SizeT* incoming_edges,
                                uint16_t* incoming_edge_count,
                                SizeT* outgoing_edges,
                                uint16_t* outgoing_edge_count,
                                uint16_t* incoming_edge_w,
                                SizeT* predecessors,
                                int32_t* scores,
                                uint8_t* consensus,
                                uint16_t* coverage,
                                uint16_t* node_coverage_counts,
                                SizeT* node_alignments,
                                uint16_t* node_alignment_count,
                                uint32_t max_limit_consensus_size)
{
    generateConsensusTestKernel<<<1, 1>>>(nodes,
                                          node_count,
                                          graph,
                                          node_id_to_pos,
                                          incoming_edges,
                                          incoming_edge_count,
                                          outgoing_edges,
                                          outgoing_edge_count,
                                          incoming_edge_w,
                                          predecessors,
                                          scores,
                                          consensus,
                                          coverage,
                                          node_coverage_counts,
                                          node_alignments,
                                          node_alignment_count,
                                          max_limit_consensus_size);
    CGA_CU_CHECK_ERR(hipPeekAtLastError());
};

} // namespace cudapoa

} // namespace claragenomics
