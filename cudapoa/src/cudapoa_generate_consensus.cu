#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "cudapoa_kernels.cuh"
#include <stdio.h>

namespace genomeworks
{

namespace cudapoa
{

__device__
    uint16_t
    branchCompletion(uint16_t max_score_id_pos,
                     uint8_t* nodes,
                     uint16_t node_count,
                     uint16_t* graph,
                     uint16_t* incoming_edges,
                     uint16_t* incoming_edge_count,
                     uint16_t* outgoing_edges,
                     uint16_t* outgoing_edge_count,
                     uint16_t* incoming_edge_w,
                     int32_t* scores,
                     int16_t* predecessors)
{
    uint16_t node_id = graph[max_score_id_pos];

    // Go through all the outgoing edges of the node, and for
    // each of the end nodes of the edges clear the scores
    // for all the _other_ nodes that had edges to that end node.
    uint16_t out_edges = outgoing_edge_count[node_id];
    for (uint16_t oe = 0; oe < out_edges; oe++)
    {
        uint16_t out_node_id       = outgoing_edges[node_id * CUDAPOA_MAX_NODE_EDGES + oe];
        uint16_t out_node_in_edges = incoming_edge_count[out_node_id];
        for (uint16_t ie = 0; ie < out_node_in_edges; ie++)
        {
            uint16_t id = incoming_edges[out_node_id * CUDAPOA_MAX_NODE_EDGES + ie];
            if (id != node_id)
            {
                scores[id] = -1;
            }
        }
    }

    int32_t max_score     = 0;
    uint16_t max_score_id = 0;
    // Run the same node weight traversal algorithm as always, to find the new
    // node with maximum weight.
    // We can start from the very next position in the graph rank because
    // the graph is topologically sorted and hence guarantees that successor of the current max
    // node will be processed again.
    for (uint16_t graph_pos = max_score_id_pos + 1; graph_pos < node_count; graph_pos++)
    {
        node_id = graph[graph_pos];

        predecessors[node_id] = -1;
        int32_t score_node_id = -1;

        uint16_t in_edges = incoming_edge_count[node_id];
        for (uint16_t e = 0; e < in_edges; e++)
        {
            uint16_t begin_node_id = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + e];
            if (scores[begin_node_id] == -1)
            {
                continue;
            }

            int32_t edge_w = static_cast<int32_t>(incoming_edge_w[node_id * CUDAPOA_MAX_NODE_EDGES + e]);
            if (score_node_id < edge_w ||
                (score_node_id == edge_w &&
                 scores[predecessors[node_id]] <= scores[begin_node_id]))
            {
                score_node_id         = edge_w;
                predecessors[node_id] = begin_node_id;
            }
        }

        if (predecessors[node_id] != -1)
        {
            score_node_id += scores[predecessors[node_id]];
        }

        if (max_score < score_node_id)
        {
            max_score    = score_node_id;
            max_score_id = node_id;
        }
        //printf("max score %d, max score id %d, node id %d score %d\n", max_score, max_score_id, node_id, score_node_id);

        scores[node_id] = score_node_id;
    }

    return max_score_id;
}

/**
 * @brief Device function to generate consensus from a given graph.
 *        The input graph needs to be topologically sorted.
 *
 * @param[in] nodes                 Device buffer with unique nodes in graph
 * @param[in] node_count            Number of nodes in graph
 * @param[in] graph                 Device buffer with sorted graph
 * @param[in] node_id_to_pos        Device scratch space for mapping node ID to position in graph
 * @param[in] incoming_edges        Device buffer with incoming edges per node
 * @param[in] incoming_edges_count  Device buffer with number of incoming edges per node
 * @param[in] outgoing_edges        Device buffer with outgoing edges per node
 * @param[in] outgoing_edges_count  Device buffer with number of outgoing edges per node
 * @param[in] predecessors          Device buffer with predecessors of nodes while traversing graph during consensus
 * @param[in] scores                Device buffer with score of each node while traversing graph during consensus
 * @param[out] consensus            Device buffer for generated consensus
 * @param[out] coverate             Device buffer for coverage of each base in consensus
 * @param[out] node_coverage_counts Device buffer with coverage of each base in graph
 * @param[in] node_alignments       Device buffer with aligned nodes for each node in graph
 * @param[in] node_alignment)count  Device buffer with aligned nodes count for each node in graph
 */
__device__ void generateConsensus(uint8_t* nodes,
                                  uint16_t node_count,
                                  uint16_t* graph,
                                  uint16_t* node_id_to_pos,
                                  uint16_t* incoming_edges,
                                  uint16_t* incoming_edge_count,
                                  uint16_t* outgoing_edges,
                                  uint16_t* outgoing_edge_count,
                                  uint16_t* incoming_edge_w,
                                  int16_t* predecessors,
                                  int32_t* scores,
                                  uint8_t* consensus,
                                  uint16_t* coverage,
                                  uint16_t* node_coverage_counts,
                                  uint16_t* node_alignments,
                                  uint16_t* node_alignment_count)
{
    // Initialize scores and predecessors to default value.
    for (uint16_t i = 0; i < node_count; i++)
    {
        predecessors[i] = -1;
        scores[i]       = -1;
    }

    uint16_t max_score_id = 0;
    int32_t max_score     = -1;

    for (uint16_t graph_pos = 0; graph_pos < node_count; graph_pos++)
    {
        uint16_t node_id  = graph[graph_pos];
        uint16_t in_edges = incoming_edge_count[node_id];

        int32_t score_node_id = scores[node_id];

        // For each node, go through it's incoming edges.
        // If the weight of any of the incoming edges is greater
        // than the score of the current node, or if the weight is equal
        // but the predecessors of the edge are heavier than the current node,
        // then update the score of the node to be the incoming edge weight.
        for (uint16_t e = 0; e < in_edges; e++)
        {
            int32_t edge_w         = static_cast<int32_t>(incoming_edge_w[node_id * CUDAPOA_MAX_NODE_EDGES + e]);
            uint16_t begin_node_id = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + e];
            if (score_node_id < edge_w ||
                (score_node_id == edge_w &&
                 scores[predecessors[node_id]] <= scores[begin_node_id]))
            {
                score_node_id         = edge_w;
                predecessors[node_id] = begin_node_id;
            }
        }

        // Then update the score of the node to be the sum
        // of the score of the predecessor and itself.
        if (predecessors[node_id] != -1)
        {
            score_node_id += scores[predecessors[node_id]];
        }

        // Keep track of the highest weighted node.
        if (max_score < score_node_id)
        {
            max_score_id = node_id;
            max_score    = score_node_id;
        }
        //printf("max score %d, max score id %d, node id %d score %d\n", max_score, max_score_id, node_id, score_node_id);

        scores[node_id] = score_node_id;
    }

    // If the node with maximum score isn't a leaf of the graph
    // then run a special branch completion function.
    uint16_t loop_count = 0;
    if (outgoing_edge_count[max_score_id] != 0)
    {
        while (outgoing_edge_count[max_score_id] != 0 && loop_count < node_count)
        {
            max_score_id = branchCompletion(node_id_to_pos[max_score_id],
                                            nodes,
                                            node_count,
                                            graph,
                                            incoming_edges,
                                            incoming_edge_count,
                                            outgoing_edges,
                                            outgoing_edge_count,
                                            incoming_edge_w,
                                            scores,
                                            predecessors);
            loop_count++;
        }
    }

    if (loop_count >= node_count)
    {
        consensus[0] = CUDAPOA_KERNEL_ERROR_ENCOUNTERED;
        consensus[1] = static_cast<uint8_t>(StatusType::loop_count_exceeded_upper_bound);
        return;
    }

    uint16_t consensus_pos = 0;
    while (predecessors[max_score_id] != -1)
    {
        consensus[consensus_pos] = nodes[max_score_id];
        uint16_t cov             = node_coverage_counts[max_score_id];
        for (uint16_t a = 0; a < node_alignment_count[max_score_id]; a++)
        {
            cov += node_coverage_counts[node_alignments[max_score_id * CUDAPOA_MAX_NODE_ALIGNMENTS + a]];
        }
        coverage[consensus_pos] = cov;
        max_score_id            = predecessors[max_score_id];
        consensus_pos++;
    }
    consensus[consensus_pos] = nodes[max_score_id];
    uint16_t cov             = node_coverage_counts[max_score_id];
    for (uint16_t a = 0; a < node_alignment_count[max_score_id]; a++)
    {
        cov += node_coverage_counts[node_alignments[max_score_id * CUDAPOA_MAX_NODE_ALIGNMENTS + a]];
    }
    coverage[consensus_pos] = cov;
    consensus_pos++;
    consensus[consensus_pos] = '\0';
}

template <bool cuda_banded_alignment = false>
__global__ void generateConsensusKernel(uint8_t* consensus_d,
                                        uint16_t* coverage_d,
                                        uint16_t* sequence_lengths_d,
                                        genomeworks::cudapoa::WindowDetails* window_details_d,
                                        uint32_t total_windows,
                                        uint8_t* nodes_d,
                                        uint16_t* incoming_edges_d,
                                        uint16_t* incoming_edge_count_d,
                                        uint16_t* outgoing_edges_d,
                                        uint16_t* outgoing_edge_count_d,
                                        uint16_t* incoming_edge_w_d,
                                        uint16_t* sorted_poa_d,
                                        uint16_t* node_id_to_pos_d,
                                        uint16_t* node_alignments_d,
                                        uint16_t* node_alignment_count_d,
                                        int32_t* consensus_scores_d,
                                        int16_t* consensus_predecessors_d,
                                        uint16_t* node_coverage_counts_d_)
{
    //each thread will operate on a window
    uint32_t window_idx = blockIdx.x * CUDAPOA_MAX_CONSENSUS_PER_BLOCK + threadIdx.x;

    if (window_idx >= total_windows)
        return;

    uint8_t* consensus = &consensus_d[window_idx * CUDAPOA_MAX_CONSENSUS_SIZE];

    if (consensus[0] == CUDAPOA_KERNEL_ERROR_ENCOUNTERED) //error during graph generation
        return;

    uint32_t max_nodes_per_window = cuda_banded_alignment ? CUDAPOA_MAX_NODES_PER_WINDOW_BANDED : CUDAPOA_MAX_NODES_PER_WINDOW;

    // Find the buffer offsets for each thread within the global memory buffers.
    uint8_t* nodes                  = &nodes_d[max_nodes_per_window * window_idx];
    uint16_t* incoming_edges        = &incoming_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* incoming_edge_count   = &incoming_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* outoing_edges         = &outgoing_edges_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* outgoing_edge_count   = &outgoing_edge_count_d[window_idx * max_nodes_per_window];
    uint16_t* incoming_edge_weights = &incoming_edge_w_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_EDGES];
    uint16_t* sorted_poa            = &sorted_poa_d[window_idx * max_nodes_per_window];
    uint16_t* node_id_to_pos        = &node_id_to_pos_d[window_idx * max_nodes_per_window];
    uint16_t* node_alignments       = &node_alignments_d[window_idx * max_nodes_per_window * CUDAPOA_MAX_NODE_ALIGNMENTS];
    uint16_t* node_alignment_count  = &node_alignment_count_d[window_idx * max_nodes_per_window];
    uint16_t* node_coverage_counts  = &node_coverage_counts_d_[max_nodes_per_window * window_idx];
    uint16_t* sequence_lengths      = &sequence_lengths_d[window_details_d[window_idx].seq_len_buffer_offset];

    //generate consensus
    uint16_t* coverage              = &coverage_d[window_idx * CUDAPOA_MAX_CONSENSUS_SIZE];
    int32_t* consensus_scores       = &consensus_scores_d[window_idx * max_nodes_per_window];
    int16_t* consensus_predecessors = &consensus_predecessors_d[window_idx * max_nodes_per_window];

    generateConsensus(nodes,
                      sequence_lengths[0],
                      sorted_poa,
                      node_id_to_pos,
                      incoming_edges,
                      incoming_edge_count,
                      outoing_edges,
                      outgoing_edge_count,
                      incoming_edge_weights,
                      consensus_predecessors,
                      consensus_scores,
                      consensus,
                      coverage,
                      node_coverage_counts,
                      node_alignments, node_alignment_count);
}

__global__ void generateConsensusTestKernel(uint8_t* nodes,
                                            uint16_t node_count,
                                            uint16_t* graph,
                                            uint16_t* node_id_to_pos,
                                            uint16_t* incoming_edges,
                                            uint16_t* incoming_edge_count,
                                            uint16_t* outgoing_edges,
                                            uint16_t* outgoing_edge_count,
                                            uint16_t* incoming_edge_w,
                                            int16_t* predecessors,
                                            int32_t* scores,
                                            uint8_t* consensus,
                                            uint16_t* coverage,
                                            uint16_t* node_coverage_counts,
                                            uint16_t* node_alignments,
                                            uint16_t* node_alignment_count)
{
    generateConsensus(nodes,
                      node_count,
                      graph,
                      node_id_to_pos,
                      incoming_edges,
                      incoming_edge_count,
                      outgoing_edges,
                      outgoing_edge_count,
                      incoming_edge_w,
                      predecessors,
                      scores,
                      consensus,
                      coverage,
                      node_coverage_counts,
                      node_alignments,
                      node_alignment_count);
}

void generateConsensusTestHost(uint8_t* nodes,
                               uint16_t node_count,
                               uint16_t* graph,
                               uint16_t* node_id_to_pos,
                               uint16_t* incoming_edges,
                               uint16_t* incoming_edge_count,
                               uint16_t* outgoing_edges,
                               uint16_t* outgoing_edge_count,
                               uint16_t* incoming_edge_w,
                               int16_t* predecessors,
                               int32_t* scores,
                               uint8_t* consensus,
                               uint16_t* coverage,
                               uint16_t* node_coverage_counts,
                               uint16_t* node_alignments,
                               uint16_t* node_alignment_count)
{
    generateConsensusTestKernel<<<1, 1>>>(nodes,
                                          node_count,
                                          graph,
                                          node_id_to_pos,
                                          incoming_edges,
                                          incoming_edge_count,
                                          outgoing_edges,
                                          outgoing_edge_count,
                                          incoming_edge_w,
                                          predecessors,
                                          scores,
                                          consensus,
                                          coverage,
                                          node_coverage_counts,
                                          node_alignments,
                                          node_alignment_count);
}

} // namespace cudapoa

} // namespace genomeworks
