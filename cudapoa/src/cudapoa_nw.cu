#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "cudapoa_kernels.cuh"
#include "cudastructs.cuh"

#include <claragenomics/utils/cudautils.hpp>

#include <stdio.h>

namespace claragenomics
{

namespace cudapoa
{

template <typename ScoreT>
__device__ __forceinline__
    ScoreT4<ScoreT>
    make_ScoreT4(ScoreT s0)
{
    ScoreT4<ScoreT> t;
    t.s0 = s0;
    t.s1 = s0;
    t.s2 = s0;
    t.s3 = s0;
    return t;
}

template <typename SeqT,
          typename IndexT,
          typename ScoreT,
          typename SizeT>
__device__ __forceinline__
    ScoreT4<ScoreT>
    computeScore(IndexT rIdx,
                 SeqT4<SeqT> read4,
                 IndexT gIdx,
                 SeqT graph_base,
                 uint16_t pred_count,
                 IndexT pred_idx,
                 SizeT* node_id_to_pos,
                 SizeT* incoming_edges,
                 ScoreT* scores,
                 int32_t scores_width,
                 ScoreT gap_score,
                 ScoreT match_score,
                 ScoreT mismatch_score)
{

    ScoreT4<ScoreT> char_profile;
    char_profile.s0 = (graph_base == read4.r0 ? match_score : mismatch_score);
    char_profile.s1 = (graph_base == read4.r1 ? match_score : mismatch_score);
    char_profile.s2 = (graph_base == read4.r2 ? match_score : mismatch_score);
    char_profile.s3 = (graph_base == read4.r3 ? match_score : mismatch_score);

    // The load instructions typically load data in 4B or 8B chunks.
    // If data is 16b (2B), then a 4B load chunk is loaded into register
    // and the necessary bits are extracted before returning. This wastes cycles
    // as each read of 16b issues a separate load command.
    // Instead it is better to load a 4B or 8B chunk into a register
    // using a single load inst, and then extracting necessary part of
    // of the data using bit arithmatic. Also reduces register count.

    ScoreT4<ScoreT>* pred_scores = (ScoreT4<ScoreT>*)&scores[pred_idx * scores_width];

    // loads 8 consecutive bytes (4 shorts)
    ScoreT4<ScoreT> score4 = pred_scores[rIdx];

    // need to load the next chunk of memory as well
    ScoreT4<ScoreT> score4_next = pred_scores[rIdx + 1];

    ScoreT4<ScoreT> score;

    score.s0 = max(score4.s0 + char_profile.s0,
                   score4.s1 + gap_score);
    score.s1 = max(score4.s1 + char_profile.s1,
                   score4.s2 + gap_score);
    score.s2 = max(score4.s2 + char_profile.s2,
                   score4.s3 + gap_score);
    score.s3 = max(score4.s3 + char_profile.s3,
                   score4_next.s0 + gap_score);

    // Perform same score updates as above, but for rest of predecessors.
    for (IndexT p = 1; p < pred_count; p++)
    {
        SizeT pred_idx = node_id_to_pos[incoming_edges[gIdx * CUDAPOA_MAX_NODE_EDGES + p]] + 1;

        ScoreT4<ScoreT>* pred_scores = (ScoreT4<ScoreT>*)&scores[pred_idx * scores_width];

        // Reasoning for 8B preload same as above.
        ScoreT4<ScoreT> score4      = pred_scores[rIdx];
        ScoreT4<ScoreT> score4_next = pred_scores[rIdx + 1];

        score.s0 = max(score4.s0 + char_profile.s0,
                       max(score.s0, score4.s1 + gap_score));

        score.s1 = max(score4.s1 + char_profile.s1,
                       max(score.s1, score4.s2 + gap_score));

        score.s2 = max(score4.s2 + char_profile.s2,
                       max(score.s2, score4.s3 + gap_score));

        score.s3 = max(score4.s3 + char_profile.s3,
                       max(score.s3, score4_next.s0 + gap_score));
    }

    return score;
}

/**
 * @brief Device function for running Needleman-Wunsch dynamic programming loop.
 *
 * @param[in] nodes                Device buffer with unique nodes in graph
 * @param[in] graph                Device buffer with sorted graph
 * @param[in] node_id_to_pos       Device scratch space for mapping node ID to position in graph
 * @param[in] incoming_edge_count  Device buffer with number of incoming edges per node
 * @param[in] incoming_edges       Device buffer with incoming edges per node
 * @param[in] outgoing_edge_count  Device buffer with number of outgoing edges per node
 * @param[in] outgoing_edges       Device buffer with outgoing edges per node
 * @param[in] read                 Device buffer with sequence (read) to align
 * @param[in] read_count           Number of bases in read
 * @param[out] scores              Device scratch space that scores alignment matrix score
 * @param[out] alignment_graph     Device scratch space for backtrace alignment of graph
 * @param[out] alignment_read      Device scratch space for backtrace alignment of sequence
 * @param[in] gap_score            Score for inserting gap into alignment
 * @param[in] mismatch_score       Score for finding a mismatch in alignment
 * @param[in] match_score          Score for finding a match in alignment
 *
 * @return Number of nodes in final alignment.
 */
template <typename SeqT,
          typename IndexT,
          typename ScoreT,
          typename SizeT,
          int32_t CPT = 4>
__device__
    uint16_t
    runNeedlemanWunsch(SeqT* nodes,
                       SizeT* graph,
                       SizeT* node_id_to_pos,
                       int32_t graph_count,
                       uint16_t* incoming_edge_count,
                       SizeT* incoming_edges,
                       uint16_t* outgoing_edge_count,
                       SizeT* outgoing_edges,
                       SeqT* read,
                       uint16_t read_count,
                       ScoreT* scores,
                       int32_t scores_width,
                       SizeT* alignment_graph,
                       SizeT* alignment_read,
                       ScoreT gap_score,
                       ScoreT mismatch_score,
                       ScoreT match_score)
{

    static_assert(CPT == 4,
                  "implementation currently supports only 4 cells per thread");

    int32_t lane_idx = threadIdx.x % WARP_SIZE;

    // Init horizonal boundary conditions (read).
    for (IndexT j = lane_idx; j < read_count + 1; j += WARP_SIZE)
    {
        scores[j] = j * gap_score;
    }

    if (lane_idx == 0)
    {
#ifdef NW_VERBOSE_PRINT
        printf("graph %d, read %d\n", graph_count, read_count);
#endif

        // Init vertical boundary (graph).
        for (IndexT graph_pos = 0; graph_pos < graph_count; graph_pos++)
        {
            uint16_t node_id    = graph[graph_pos];
            uint16_t i          = graph_pos + 1;
            uint16_t pred_count = incoming_edge_count[node_id];
            if (pred_count == 0)
            {
                scores[i * scores_width] = gap_score;
            }
            else
            {
                ScoreT penalty = SHRT_MIN;
                for (uint16_t p = 0; p < pred_count; p++)
                {
                    SizeT pred_node_id        = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p];
                    SizeT pred_node_graph_pos = node_id_to_pos[pred_node_id] + 1;
                    penalty                   = max(penalty, scores[pred_node_graph_pos * scores_width]);
                }
                scores[i * scores_width] = penalty + gap_score;
            }
        }
    }

    __syncwarp();

    // readpos_bound is the first multiple of (CPT * WARP_SIZE) that is larger than read_count.
    uint16_t readpos_bound = (((read_count - 1) / (WARP_SIZE * CPT)) + 1) * (WARP_SIZE * CPT);

    SeqT4<SeqT>* d_read4 = (SeqT4<SeqT>*)read;

    // Run DP loop for calculating scores. Process each row at a time, and
    // compute vertical and diagonal values in parallel.
    for (IndexT graph_pos = 0;
         graph_pos < graph_count;
         graph_pos++)
    {

        uint16_t node_id  = graph[graph_pos]; // node id for the graph node
        IndexT score_gIdx = graph_pos + 1;    // score matrix index for this graph node

        ScoreT first_element_prev_score = scores[score_gIdx * scores_width];

        uint16_t pred_count = incoming_edge_count[node_id];

        SizeT pred_idx = (pred_count == 0 ? 0 : node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1);

        SeqT graph_base = nodes[node_id];

        // readpos_bound is the first tb boundary multiple beyond read_count. This is done
        // so all threads in the block enter the loop. The loop has syncwarp, so if
        // any of the threads don't enter, then it'll cause a lock in the system.
        for (IndexT read_pos = lane_idx * CPT;
             read_pos < readpos_bound;
             read_pos += WARP_SIZE * CPT)
        {

            IndexT rIdx = read_pos / CPT;

            // To avoid doing extra work, we clip the extra warps that go beyond the read count.
            // Warp clipping hasn't shown to help too much yet, but might if we increase the tb
            // size in the future.

            SeqT4<SeqT> read4 = d_read4[rIdx];

            ScoreT4<ScoreT> score = make_ScoreT4((ScoreT)SHRT_MAX);

            if (read_pos < read_count)
            {
                score = computeScore<SeqT, IndexT, ScoreT>(rIdx, read4,
                                                           node_id, graph_base,
                                                           pred_count, pred_idx,
                                                           node_id_to_pos, incoming_edges,
                                                           scores, scores_width,
                                                           gap_score, match_score, mismatch_score);
            }
            // While there are changes to the horizontal score values, keep updating the matrix.
            // So loop will only run the number of time there are corrections in the matrix.
            // The any_sync warp primitive lets us easily check if any of the threads had an update.
            bool loop = true;
            while (__any_sync(FULL_MASK, loop))
            {

                // To increase instruction level parallelism, we compute the scores
                // in reverse order (score3 first, then score2, then score1, etc).
                // And then check if any of the scores had an update,
                // and if there's an update then we rerun the loop to capture the effects
                // of the change in the next loop.
                loop = false;

                // The shfl_up lets us grab a value from the lane below.
                ScoreT last_score = __shfl_up_sync(FULL_MASK, score.s3, 1);
                if (lane_idx == 0)
                {
                    last_score = first_element_prev_score;
                }

                ScoreT tscore = max(score.s2 + gap_score, score.s3);
                if (tscore > score.s3)
                {
                    score.s3 = tscore;
                    loop     = true;
                }

                tscore = max(score.s1 + gap_score, score.s2);
                if (tscore > score.s2)
                {
                    score.s2 = tscore;
                    loop     = true;
                }

                tscore = max(score.s0 + gap_score, score.s1);
                if (tscore > score.s1)
                {
                    score.s1 = tscore;
                    loop     = true;
                }

                tscore = max(last_score + gap_score, score.s0);
                if (tscore > score.s0)
                {
                    score.s0 = tscore;
                    loop     = true;
                }
            }

            // Copy over the last element score of the last lane into a register of first lane
            // which can be used to compute the first cell of the next warp.
            first_element_prev_score = __shfl_sync(FULL_MASK, score.s3, WARP_SIZE - 1);

            // Index into score matrix.
            if (read_pos < read_count)
            {
                scores[score_gIdx * scores_width + read_pos + 1] = score.s0;
                scores[score_gIdx * scores_width + read_pos + 2] = score.s1;
                scores[score_gIdx * scores_width + read_pos + 3] = score.s2;
                scores[score_gIdx * scores_width + read_pos + 4] = score.s3;
            }
            __syncwarp();
        }
    }

    uint16_t aligned_nodes = 0;
    if (lane_idx == 0)
    {
        // Find location of the maximum score in the matrix.
        IndexT i      = 0;
        IndexT j      = read_count;
        ScoreT mscore = SHRT_MIN;

        for (IndexT idx = 1; idx <= graph_count; idx++)
        {
            if (outgoing_edge_count[graph[idx - 1]] == 0)
            {
                ScoreT s = scores[idx * scores_width + j];
                if (mscore < s)
                {
                    mscore = s;
                    i      = idx;
                }
            }
        }

        // Fill in backtrace

        IndexT prev_i = 0;
        IndexT prev_j = 0;

        // Trace back from maximum score position to generate alignment.
        // Trace back is done by re-calculating the score at each cell
        // along the path to see which preceding cell the move could have
        // come from. This seems computaitonally more expensive, but doesn't
        // require storing any traceback buffer during alignment.
        int32_t loop_count = 0;
        while (!(i == 0 && j == 0) && loop_count < (read_count + graph_count + 2))
        {
            loop_count++;
            ScoreT scores_ij = scores[i * scores_width + j];
            bool pred_found  = false;

            // Check if move is diagonal.
            if (i != 0 && j != 0)
            {
                SizeT node_id       = graph[i - 1];
                ScoreT match_cost   = (nodes[node_id] == read[j - 1] ? match_score : mismatch_score);
                uint16_t pred_count = incoming_edge_count[node_id];
                SizeT pred_i        = (pred_count == 0 ? 0 : (node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1));

                if (scores_ij == (scores[pred_i * scores_width + (j - 1)] + match_cost))
                {
                    prev_i     = pred_i;
                    prev_j     = j - 1;
                    pred_found = true;
                }

                if (!pred_found)
                {
                    for (uint16_t p = 1; p < pred_count; p++)
                    {
                        pred_i = (node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p]] + 1);

                        if (scores_ij == (scores[pred_i * scores_width + (j - 1)] + match_cost))
                        {
                            prev_i     = pred_i;
                            prev_j     = j - 1;
                            pred_found = true;
                            break;
                        }
                    }
                }
            }

            // Check if move is vertical.
            if (!pred_found && i != 0)
            {
                SizeT node_id       = graph[i - 1];
                uint16_t pred_count = incoming_edge_count[node_id];
                SizeT pred_i        = (pred_count == 0 ? 0 : node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1);

                if (scores_ij == scores[pred_i * scores_width + j] + gap_score)
                {
                    prev_i     = pred_i;
                    prev_j     = j;
                    pred_found = true;
                }

                if (!pred_found)
                {
                    for (uint16_t p = 1; p < pred_count; p++)
                    {
                        pred_i = node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p]] + 1;

                        if (scores_ij == scores[pred_i * scores_width + j] + gap_score)
                        {
                            prev_i     = pred_i;
                            prev_j     = j;
                            pred_found = true;
                            break;
                        }
                    }
                }
            }

            // Check if move is horizontal.
            if (!pred_found && scores_ij == scores[i * scores_width + (j - 1)] + gap_score)
            {
                prev_i     = i;
                prev_j     = j - 1;
                pred_found = true;
            }

            alignment_graph[aligned_nodes] = (i == prev_i ? -1 : graph[i - 1]);
            alignment_read[aligned_nodes]  = (j == prev_j ? -1 : j - 1);
            aligned_nodes++;

            i = prev_i;
            j = prev_j;

        } // end of while
        if (loop_count >= (read_count + graph_count + 2))
        {
            aligned_nodes = UINT16_MAX;
        }

#ifdef NW_VERBOSE_PRINT
        printf("aligned nodes %d\n", aligned_nodes);
#endif
    }

    aligned_nodes = __shfl_sync(0xffffffff, aligned_nodes, 0);
    return aligned_nodes;
}

template <typename SizeT>
__global__ void runNeedlemanWunschKernel(uint8_t* nodes,
                                         SizeT* graph,
                                         SizeT* node_id_to_pos,
                                         int32_t graph_count,
                                         uint16_t* incoming_edge_count,
                                         SizeT* incoming_edges,
                                         uint16_t* outgoing_edge_count,
                                         SizeT* outgoing_edges,
                                         uint8_t* read,
                                         uint16_t read_count,
                                         int16_t* scores,
                                         int32_t scores_width,
                                         SizeT* alignment_graph,
                                         SizeT* alignment_read,
                                         int16_t gap_score,
                                         int16_t mismatch_score,
                                         int16_t match_score,
                                         SizeT* aligned_nodes)
{
    *aligned_nodes = runNeedlemanWunsch<uint8_t, uint16_t, int16_t, SizeT>(nodes,
                                                                           graph,
                                                                           node_id_to_pos,
                                                                           graph_count,
                                                                           incoming_edge_count,
                                                                           incoming_edges,
                                                                           outgoing_edge_count,
                                                                           outgoing_edges,
                                                                           read,
                                                                           read_count,
                                                                           scores,
                                                                           scores_width,
                                                                           alignment_graph,
                                                                           alignment_read,
                                                                           gap_score,
                                                                           mismatch_score,
                                                                           match_score);
}

// Host function that calls the kernel
template <typename SizeT>
void runNWtemplated(uint8_t* nodes,
                    SizeT* graph,
                    SizeT* node_id_to_pos,
                    int32_t graph_count,
                    uint16_t* incoming_edge_count,
                    SizeT* incoming_edges,
                    uint16_t* outgoing_edge_count,
                    SizeT* outgoing_edges,
                    uint8_t* read,
                    uint16_t read_count,
                    int16_t* scores,
                    int32_t scores_width,
                    SizeT* alignment_graph,
                    SizeT* alignment_read,
                    int16_t gap_score,
                    int16_t mismatch_score,
                    int16_t match_score,
                    SizeT* aligned_nodes)
{
    runNeedlemanWunschKernel<<<1, 64>>>(nodes,
                                        graph,
                                        node_id_to_pos,
                                        graph_count,
                                        incoming_edge_count,
                                        incoming_edges,
                                        outgoing_edge_count,
                                        outgoing_edges,
                                        read,
                                        read_count,
                                        scores,
                                        scores_width,
                                        alignment_graph,
                                        alignment_read,
                                        gap_score,
                                        mismatch_score,
                                        match_score,
                                        aligned_nodes);
    CGA_CU_CHECK_ERR(hipPeekAtLastError());
}

} // namespace cudapoa

} // namespace claragenomics
