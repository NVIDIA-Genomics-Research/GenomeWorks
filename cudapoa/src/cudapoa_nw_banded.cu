#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "cudapoa_kernels.cuh"
#include "cudastructs.cuh"

#include <claragenomics/utils/cudautils.hpp>

// Extract shorts from bit field.
#define EXTRACT_SHORT_FROM_BITFIELD(type, val, pos) (type)((val >> (16 * (pos))) & 0xffff)

#include <stdio.h>

namespace claragenomics
{

namespace cudapoa
{

__device__ uint16_t get_band_start_for_row(uint16_t row_idx, float gradient, uint16_t band_width, uint16_t max_column)
{

    int16_t start_pos = uint16_t(row_idx * gradient) - band_width / 2;

    start_pos = max(start_pos, 0);

    int16_t end_pos = start_pos + band_width;

    if (end_pos > max_column)
    {
        start_pos = max_column - band_width + CELLS_PER_THREAD;
    };

    start_pos = max(start_pos, 0);

    start_pos = start_pos - (start_pos % CELLS_PER_THREAD);

    return uint16_t(start_pos);
}

__device__ int16_t* get_score_ptr(int16_t* scores, uint16_t row, uint16_t column, float gradient, uint16_t band_width, uint16_t max_column)
{

    uint16_t band_start = get_band_start_for_row(row, gradient, band_width, max_column);

    uint16_t col_idx;

    if (column == 0)
    {
        col_idx = band_start;
    }
    else
    {
        col_idx = column - band_start;
    }

    return &scores[(col_idx) + row * CUDAPOA_BANDED_MAX_MATRIX_SEQUENCE_DIMENSION];
};

__device__ void set_score(int16_t* scores, uint16_t row, uint16_t column, int16_t value, float gradient, uint16_t band_width, uint16_t max_column)
{
    uint16_t band_start = get_band_start_for_row(row, gradient, band_width, max_column);

    uint16_t col_idx;
    if (column == 0)
    {
        col_idx = band_start;
    }
    else
    {
        col_idx = column - band_start;
    }

    scores[col_idx + row * CUDAPOA_BANDED_MAX_MATRIX_SEQUENCE_DIMENSION] = value;
}

__device__ void initialize_band(int16_t* scores, uint16_t row, int16_t value, float gradient, uint16_t band_width, uint16_t max_column)
{

    uint16_t band_start = get_band_start_for_row(row, gradient, band_width, max_column);
    uint16_t band_end   = band_start + band_width;

    uint16_t initialization_offset = (band_start == 0) ? 1 : band_start;

    set_score(scores, row, initialization_offset, value, gradient, band_width, max_column);

    for (uint16_t j = threadIdx.x + band_end; j < band_end + CUDAPOA_BANDED_MATRIX_RIGHT_PADDING; j += blockDim.x)
    {
        set_score(scores, row, j, value, gradient, band_width, max_column);
    }
};

__device__ int16_t get_score(int16_t* scores, uint16_t row, uint16_t column, float gradient, uint16_t bandwidth, uint16_t max_column, int16_t out_of_band_score_offset)
{
    uint16_t band_start = get_band_start_for_row(row, gradient, bandwidth, max_column);
    uint16_t band_end   = band_start + bandwidth;

    if (((column > band_end) || (column < band_start)) && column != 0)
    {
        return SHRT_MIN + out_of_band_score_offset;
    }
    else
    {
        return *get_score_ptr(scores, row, column, gradient, bandwidth, max_column);
    }
}

__device__ ScoreT4<int16_t> get_scores(uint16_t read_pos, int16_t* scores, uint16_t node,
                                       int16_t gap_score, int16_t char_profile0, int16_t char_profile1, int16_t char_profile2, int16_t char_profile3,
                                       float gradient, uint16_t bandwidth, int16_t default_value,
                                       uint16_t max_column)
{

    // The load instructions typically load data in 4B or 8B chunks.
    // If data is 16b (2B), then a 4B load chunk is loaded into register
    // and the necessary bits are extracted before returning. This wastes cycles
    // as each read of 16b issues a separate load command.
    // Instead it is better to load a 4B or 8B chunk into a register
    // using a single load inst, and then extracting necessary part of
    // of the data using bit arithmatic. Also reduces register count.

    uint16_t band_start = get_band_start_for_row(node, gradient, bandwidth, max_column);

    uint16_t band_end = band_start + bandwidth + CELLS_PER_THREAD;

    if (((read_pos + 1 > band_end) || (read_pos + 1 < band_start)) && read_pos + 1 != 0)
    {
        return ScoreT4<int16_t>{default_value, default_value, default_value, default_value};
    }
    else
    {
        ScoreT4<int16_t> scores_4;
        int16_t* score_ptr = get_score_ptr(scores, node, read_pos, gradient, bandwidth, max_column);

        // This loads 8 consecutive bytes (4 shorts).
        int64_t score_pred_i_1_64   = ((int64_t*)score_ptr)[0];
        int64_t score_pred_i_1_64_2 = ((int64_t*)score_ptr)[1];

        scores_4.s0 = max(EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 0) + char_profile0,
                          EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 1) + gap_score);
        scores_4.s1 = max(EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 1) + char_profile1,
                          EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 2) + gap_score);
        scores_4.s2 = max(EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 2) + char_profile2,
                          EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 3) + gap_score);
        scores_4.s3 = max(EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64, 3) + char_profile3,
                          EXTRACT_SHORT_FROM_BITFIELD(int16_t, score_pred_i_1_64_2, 0) + gap_score);

        return scores_4;
    }
}

__device__
    uint16_t
    runNeedlemanWunschBanded(uint8_t* nodes,
                             uint16_t* graph,
                             uint16_t* node_id_to_pos,
                             uint16_t graph_count,
                             uint16_t* incoming_edge_count,
                             uint16_t* incoming_edges,
                             uint16_t* outgoing_edge_count,
                             uint16_t* outgoing_edges,
                             uint8_t* read,
                             uint16_t read_length,
                             int16_t* scores,
                             int16_t* alignment_graph,
                             int16_t* alignment_read,
                             int16_t gap_score,
                             int16_t mismatch_score,
                             int16_t match_score)
{
    __shared__ int16_t first_element_prev_score;

    int16_t min_score_abs = abs(min(min(gap_score, mismatch_score), match_score) - 1);

    //Calculate gradient for the scores matrix
    float gradient = float(read_length + 1) / float(graph_count + 1);

    uint16_t band_width = blockDim.x * CELLS_PER_THREAD;
    int32_t thread_idx  = threadIdx.x;
    int32_t warp_idx    = thread_idx / WARP_SIZE;

    long long int start = clock64();

    uint16_t max_column = read_length + 1;

    // Initialise the horizontal boundary of the score matrix
    for (uint16_t j = thread_idx + 1; j < CUDAPOA_BANDED_MAX_MATRIX_SEQUENCE_DIMENSION; j += blockDim.x)
    {
        set_score(scores, 0, j, j * gap_score, gradient, band_width, max_column);
    }

    // Initialise the vertical boundary of the score matrix
    if (thread_idx == 0)
    {
#ifdef NW_VERBOSE_PRINT
        printf("graph %d, read %d\n", graph_count, read_length);
#endif

        for (uint16_t graph_pos = 0; graph_pos < graph_count; graph_pos++)
        {

            set_score(scores, 0, 0, 0, gradient, band_width, max_column);

            uint16_t node_id = graph[graph_pos];
            uint16_t i       = graph_pos + 1;

            uint16_t pred_count = incoming_edge_count[node_id];
            if (pred_count == 0)
            {
                set_score(scores, i, 0, gap_score, gradient, band_width, max_column);
            }
            else
            {
                int16_t penalty = SHRT_MIN;
                for (uint16_t p = 0; p < pred_count; p++)
                {
                    uint16_t pred_node_id        = incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p];
                    uint16_t pred_node_graph_pos = node_id_to_pos[pred_node_id] + 1;
                    penalty                      = max(penalty, get_score(scores, pred_node_graph_pos, 0, gradient, band_width, read_length + 1, min_score_abs));
                }
                set_score(scores, i, 0, penalty + gap_score, gradient, band_width, max_column);
            }
        }
    }

    __syncthreads();

    start = clock64();

    long long int serial = 0;

    // Maximum warps is total number of warps needed (based on fixed warp size and cells per thread)
    // to cover the full read. This number is <= max_cols.
    uint16_t max_warps = (((read_length - 1) / (WARP_SIZE * CELLS_PER_THREAD)) + 1);

    // compute vertical and diagonal values in parallel.
    for (uint16_t graph_pos = 0; graph_pos < graph_count; graph_pos++)
    {

        uint16_t node_id = graph[graph_pos];
        uint16_t i       = graph_pos + 1;

        uint16_t band_start = get_band_start_for_row(i, gradient, band_width, read_length + 1);

        initialize_band(scores, i, SHRT_MIN + min_score_abs, gradient, band_width, read_length + 1);

        if (thread_idx == 0)
        {
            first_element_prev_score = get_score(scores, i, 0, gradient, band_width, read_length + 1, min_score_abs);
        }

        uint16_t pred_count = incoming_edge_count[node_id];

        uint16_t pred_i_1 = (pred_count == 0 ? 0 : node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1);

        uint8_t n = nodes[node_id];

        uint16_t read_pos = thread_idx * CELLS_PER_THREAD + band_start;
        {

            int16_t score0, score1, score2, score3;
            uint16_t j0, j1, j2, j3;

            // To avoid doing extra work, we clip the extra warps that go beyond the read count.
            // Warp clipping hasn't shown to help too much yet, but might if we increase the tb
            // size in the future.

            if (warp_idx < max_warps)
            {
                int16_t char_profile0 = (n == read[read_pos + 0] ? match_score : mismatch_score);
                int16_t char_profile1 = (n == read[read_pos + 1] ? match_score : mismatch_score);
                int16_t char_profile2 = (n == read[read_pos + 2] ? match_score : mismatch_score);
                int16_t char_profile3 = (n == read[read_pos + 3] ? match_score : mismatch_score);
                // Index into score matrix.
                j0 = read_pos + 1;
                j1 = read_pos + 2;
                j2 = read_pos + 3;
                j3 = read_pos + 4;

                ScoreT4<int16_t> scores_4 = get_scores(read_pos, scores, pred_i_1, gap_score, char_profile0, char_profile1, char_profile2, char_profile3, gradient, band_width, SHRT_MIN + min_score_abs, read_length + 1);

                score0 = scores_4.s0;
                score1 = scores_4.s1;
                score2 = scores_4.s2;
                score3 = scores_4.s3;

                // Perform same score updates as above, but for rest of predecessors.
                for (uint16_t p = 1; p < pred_count; p++)
                {
                    int16_t pred_i_2          = node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p]] + 1;
                    ScoreT4<int16_t> scores_4 = get_scores(read_pos, scores, pred_i_2, gap_score, char_profile0, char_profile1, char_profile2, char_profile3, gradient, band_width, SHRT_MIN + min_score_abs, read_length + 1);

                    score0 = max(score0, scores_4.s0);
                    score1 = max(score1, scores_4.s1);
                    score2 = max(score2, scores_4.s2);
                    score3 = max(score3, scores_4.s3);
                }
            }

            long long int temp = clock64();

            for (int32_t tb_start = 0; tb_start < blockDim.x; tb_start += WARP_SIZE)
            {
                if (thread_idx >= tb_start && warp_idx < max_warps)
                {
                    int16_t last_score;
                    // While there are changes to the horizontal score values, keep updating the matrix.
                    // So loop will only run the number of time there are corrections in the matrix.
                    // The any_sync warp primitive lets us easily check if any of the threads had an update.
                    bool loop = true;

                    while (__any_sync(0xffffffff, loop))
                    {

                        // To increase instruction level parallelism, we compute the scores
                        // in reverse order (score3 first, then score2, then score1, etc).
                        // And then check if any of the scores had an update,
                        // and if there's an update then we rerun the loop to capture the effects
                        // of the change in the next loop.
                        loop = false;

                        // The shfl_up lets us grab a value from the lane below.
                        last_score = __shfl_up_sync(0xffffffff << 1, score3, 1);
                        if (thread_idx % 32 == 0)
                        {
                            last_score = score0;
                        }
                        if (thread_idx == tb_start)
                        {
                            last_score = first_element_prev_score;
                        }
                        __syncwarp();

                        bool check3    = false;
                        int16_t tscore = max(score2 + gap_score, score3);
                        if (tscore > score3)
                        {
                            score3 = tscore;
                            check3 = true;
                        }

                        bool check2 = false;
                        tscore      = max(score1 + gap_score, score2);
                        if (tscore > score2)
                        {
                            score2 = tscore;
                            check2 = true;
                        }

                        bool check1 = false;
                        tscore      = max(score0 + gap_score, score1);
                        if (tscore > score1)
                        {
                            score1 = tscore;
                            check1 = true;
                        }

                        bool check0 = false;
                        tscore      = max(last_score + gap_score, score0);
                        if (tscore > score0)
                        {
                            score0 = tscore;
                            check0 = true;
                        }
                        //TODO: See if using only one `check` variable affects performance.
                        loop = check0 || check1 || check2 || check3;
                    }

                    // Copy over the last element score of the last lane into a register of first lane
                    // which can be used to compute the first cell of the next warp.
                    if (thread_idx == tb_start + (WARP_SIZE - 1))
                    {
                        first_element_prev_score = score3;
                    }
                }

                __syncthreads();
            }

            // Index into score matrix.
            if (warp_idx < max_warps)
            {
                set_score(scores, i, j0, score0, gradient, band_width, max_column);
                set_score(scores, i, j1, score1, gradient, band_width, max_column);
                set_score(scores, i, j2, score2, gradient, band_width, max_column);
                set_score(scores, i, j3, score3, gradient, band_width, max_column);
            }

            serial += (clock64() - temp);

            __syncthreads();
        }
    }

    long long int nw = clock64() - start;
    //long long int tb = 0;

    start = clock64();

    uint16_t aligned_nodes = 0;
    if (thread_idx == 0)
    {
        // Find location of the maximum score in the matrix.
        int16_t i      = 0;
        int16_t j      = read_length;
        int16_t mscore = SHRT_MIN;

        for (int16_t idx = 1; idx <= graph_count; idx++)
        {
            if (outgoing_edge_count[graph[idx - 1]] == 0)
            {
                int16_t s = get_score(scores, idx, j, gradient, band_width, read_length + 1, min_score_abs);
                if (mscore < s)
                {
                    mscore = s;
                    i      = idx;
                }
            }
        }

        // Fill in backtrace
        int16_t prev_i = 0;
        int16_t prev_j = 0;

        int32_t loop_count = 0;
        while (!(i == 0 && j == 0) && loop_count < (read_length + graph_count + 2))
        {
            loop_count++;
            int16_t scores_ij = get_score(scores, i, j, gradient, band_width, read_length + 1, min_score_abs);
            bool pred_found   = false;
            // Check if move is diagonal.
            if (i != 0 && j != 0)
            {

                uint16_t node_id   = graph[i - 1];
                int16_t match_cost = (nodes[node_id] == read[j - 1] ? match_score : mismatch_score);

                uint16_t pred_count = incoming_edge_count[node_id];
                uint16_t pred_i     = (pred_count == 0 ? 0 : (node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1));

                if (scores_ij == (get_score(scores, pred_i, j - 1, gradient, band_width, read_length + 1, min_score_abs) + match_cost))
                {
                    prev_i     = pred_i;
                    prev_j     = j - 1;
                    pred_found = true;
                }

                if (!pred_found)
                {
                    for (uint16_t p = 1; p < pred_count; p++)
                    {
                        pred_i = (node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p]] + 1);

                        if (scores_ij == (get_score(scores, pred_i, j - 1, gradient, band_width, read_length + 1, min_score_abs) + match_cost))
                        {
                            prev_i     = pred_i;
                            prev_j     = j - 1;
                            pred_found = true;
                            break;
                        }
                    }
                }
            }

            // Check if move is vertical.
            if (!pred_found && i != 0)
            {
                uint16_t node_id    = graph[i - 1];
                uint16_t pred_count = incoming_edge_count[node_id];
                uint16_t pred_i     = (pred_count == 0 ? 0 : node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES]] + 1);

                if (scores_ij == get_score(scores, pred_i, j, gradient, band_width, read_length + 1, min_score_abs) + gap_score)
                {
                    prev_i     = pred_i;
                    prev_j     = j;
                    pred_found = true;
                }

                if (!pred_found)
                {
                    for (uint16_t p = 1; p < pred_count; p++)
                    {
                        pred_i = node_id_to_pos[incoming_edges[node_id * CUDAPOA_MAX_NODE_EDGES + p]] + 1;

                        if (scores_ij == get_score(scores, pred_i, j, gradient, band_width, read_length + 1, min_score_abs) + gap_score)
                        {
                            prev_i     = pred_i;
                            prev_j     = j;
                            pred_found = true;
                            break;
                        }
                    }
                }
            }

            // Check if move is horizontal.
            if (!pred_found && scores_ij == get_score(scores, i, j - 1, gradient, band_width, read_length + 1, min_score_abs) + gap_score)
            {
                prev_i     = i;
                prev_j     = j - 1;
                pred_found = true;
            }

            alignment_graph[aligned_nodes] = (i == prev_i ? -1 : graph[i - 1]);
            alignment_read[aligned_nodes]  = (j == prev_j ? -1 : j - 1);
            aligned_nodes++;

            i = prev_i;
            j = prev_j;
        }

        if (loop_count >= (read_length + graph_count + 2))
        {
            aligned_nodes = UINT16_MAX;
        }

#ifdef NW_VERBOSE_PRINT
        printf("aligned nodes %d\n", aligned_nodes);
#endif
    }
    aligned_nodes = __shfl_sync(0xffffffff, aligned_nodes, 0);
    return aligned_nodes;
}

} // namespace cudapoa

} // namespace claragenomics
