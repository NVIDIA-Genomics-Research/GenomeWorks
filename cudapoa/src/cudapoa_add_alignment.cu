#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "cudapoa_kernels.cuh"

#include <claragenomics/utils/cudautils.hpp>

#include <stdio.h>

namespace claragenomics
{

namespace cudapoa
{

/**
 * @brief Device function for adding a new alignment to the partial order alignment graph.
 *
 * @param[out] new_node_count             Number of nodes in graph after update
 * @param[in/out] nodes                   Device buffer with unique nodes in graph
 * @param[in] node_count                  Number of nodes in graph
 * @graph[in] node_alignments             Device buffer with alignment nodes per node in graph
 * @param[in] node_alignment_count        Device buffer with number of aligned nodes
 * @param[in] incoming_edges              Device buffer with incoming edges per node
 * @param[in] incoming_edges_count        Device buffer with number of incoming edges per node
 * @param[in] outgoing_edges              Device buffer with outgoing edges per node
 * @param[in] outgoing_edges_count        Device buffer with number of outgoing edges per node
 * @param[in] incoming_edge_w             Device buffer with weight of incoming edges
 * @param[in] outgoing_edge_w             Device buffer with weight of outgoing edges
 * @param[in] alignment_length            Total length of new alignment
 * @param[in] graph                       Device scratch space with sorted graph
 * @param[in] alignment_graph             Device buffer with nodes from graph in alignment
 * @param[in] read                        Device scratch space with sequence
 * @param[in] alignment_read              Device buffer with bases from read in alignment
 * @param[in] node_coverage_count         Device buffer with coverage of each node in graph
 * @param[in] base_weights                    Device buffer with weight of each node in read
 * @param[in] sequence_begin_nodes_ids        Device buffer with begining node of each sequence
 * @param[in] outgoing_edges_coverage         Device buffer with coverage of each edge in graph
 * @param[in] outgoing_edges_coverage_count   Device buffer with coverage count of each edge in graph
 * @param[in] s                               Current sequence id
 * @param[in] max_sequences_per_poa           Maximum sequences allowed in a graph
 * @param[in] max_limit_nodes_per_window      Upper limit of number of nodes per window
 *
 * @return Status code for any errors encountered.
 */

template <bool msa = false, typename SizeT>
__device__
    uint8_t
    addAlignmentToGraph(SizeT& new_node_count,
                        uint8_t* nodes,
                        SizeT node_count,
                        SizeT* node_alignments, uint16_t* node_alignment_count,
                        SizeT* incoming_edges, uint16_t* incoming_edge_count,
                        SizeT* outgoing_edges, uint16_t* outgoing_edge_count,
                        uint16_t* incoming_edge_w, uint16_t* /*outgoing_edge_w*/,
                        uint16_t alignment_length,
                        SizeT* /*graph*/,
                        SizeT* alignment_graph,
                        uint8_t* read,
                        SizeT* alignment_read,
                        uint16_t* node_coverage_counts,
                        int8_t* base_weights,
                        SizeT* sequence_begin_nodes_ids,
                        uint16_t* outgoing_edges_coverage,
                        uint16_t* outgoing_edges_coverage_count,
                        uint16_t s,
                        uint32_t max_sequences_per_poa,
                        uint32_t max_limit_nodes_per_window)
{
    //printf("Running addition for alignment %d\n", alignment_length);
    SizeT head_node_id   = -1;
    SizeT curr_node_id   = -1;
    uint16_t prev_weight = 0;

    // Basic algorithm is to iterate through the alignment of the read.
    // For each position in that alignment -
    //     if it's an insert in the read
    //         add a new node
    //     if it is aligned
    //         check if node base matches read base. if so, move on.
    //         if node base doesn't match, check other aligned nodes
    //             if none of the other aligned nodes match, add new node
    //             else use one of aligned nodes and move on.
    for (int16_t pos = alignment_length - 1; pos >= 0; pos--)
    {
        bool new_node  = false;
        SizeT read_pos = alignment_read[pos];
        // Case where base in read in an insert.
        if (read_pos != -1)
        {
            int8_t NODE_WEIGHT = base_weights[read_pos];

            //printf("%c ", read[read_pos]);
            uint8_t read_base   = read[read_pos];
            SizeT graph_node_id = alignment_graph[pos];
            if (graph_node_id == -1)
            {
                // No alignment node found in graph.
                // Create new node.
                curr_node_id = node_count++;
                if (node_count >= max_limit_nodes_per_window)
                {
                    return static_cast<uint8_t>(StatusType::node_count_exceeded_maximum_graph_size);
                }
                //printf("create new node %d\n", curr_node_id);
                new_node                           = true;
                nodes[curr_node_id]                = read_base;
                outgoing_edge_count[curr_node_id]  = 0;
                incoming_edge_count[curr_node_id]  = 0;
                node_alignment_count[curr_node_id] = 0;
                node_coverage_counts[curr_node_id] = 0;
            }
            else
            {
                // Get base information for aligned node in graph.
                uint8_t graph_base = nodes[graph_node_id];
                //printf("graph base %c\n", graph_base);

                // If bases match, then set current node id to graph node id.
                if (graph_base == read_base)
                {
                    //printf("graph and read base are same\n");
                    curr_node_id = graph_node_id;
                }
                else
                {
                    // Since bases don't match, iterate through all aligned nodes of
                    // graph node, and check against their bases. If a base matches,
                    // then set the current node as that aligned node.
                    uint16_t num_aligned_node = node_alignment_count[graph_node_id];
                    //printf("aligned nodes are %d\n", num_aligned_node);
                    SizeT aligned_node_id = -1;
                    //printf("looping through alignments\n");
                    for (uint16_t n = 0; n < num_aligned_node; n++)
                    {
                        SizeT aid = node_alignments[graph_node_id * CUDAPOA_MAX_NODE_ALIGNMENTS + n];
                        if (nodes[aid] == read_base)
                        {
                            aligned_node_id = aid;
                            break;
                        }
                    }

                    if (aligned_node_id != -1)
                    {
                        //printf("found aligned node %d\n", aligned_node_id);
                        curr_node_id = aligned_node_id;
                    }
                    else
                    {
                        // However, if none of the nodes in the aligned list match either,
                        // then create a new node and update the graph node (+ aligned nodes)
                        // with information about this new node since it also becomes an aligned
                        // node to the others.
                        new_node     = true;
                        curr_node_id = node_count++;
                        if (node_count >= max_limit_nodes_per_window)
                        {
                            return static_cast<uint8_t>(StatusType::node_count_exceeded_maximum_graph_size);
                        }
                        //printf("create new node %d\n", curr_node_id);
                        nodes[curr_node_id]                = read_base;
                        outgoing_edge_count[curr_node_id]  = 0;
                        incoming_edge_count[curr_node_id]  = 0;
                        node_alignment_count[curr_node_id] = 0;
                        node_coverage_counts[curr_node_id] = 0;
                        SizeT new_node_alignments          = 0;

                        for (uint16_t n = 0; n < num_aligned_node; n++)
                        {
                            SizeT aid                                                                         = node_alignments[graph_node_id * CUDAPOA_MAX_NODE_ALIGNMENTS + n];
                            uint16_t aid_count                                                                = node_alignment_count[aid];
                            node_alignments[aid * CUDAPOA_MAX_NODE_ALIGNMENTS + aid_count]                    = curr_node_id;
                            node_alignment_count[aid]                                                         = aid_count + 1;
                            node_alignments[curr_node_id * CUDAPOA_MAX_NODE_ALIGNMENTS + new_node_alignments] = aid;
                            new_node_alignments++;
                        }

                        node_alignments[graph_node_id * CUDAPOA_MAX_NODE_ALIGNMENTS + num_aligned_node] = curr_node_id;
                        node_alignment_count[graph_node_id]                                             = num_aligned_node + 1;

                        node_alignments[curr_node_id * CUDAPOA_MAX_NODE_ALIGNMENTS + new_node_alignments] = graph_node_id;
                        new_node_alignments++;

                        node_alignment_count[curr_node_id] = new_node_alignments;
                    }
                }
            }

            if (new_node)
            {
                //printf("new node %d\n", curr_node_id);
            }

            // for msa generation
            if (msa && (read_pos == 0))
            {
                //begin node of the sequence, add its node_id (curr_node_id) to sequence_begin_nodes_ids
                *sequence_begin_nodes_ids = curr_node_id;
                // printf("adding sequence_begin_nodes_ids = %d\n", curr_node_id);
            }

            // Create new edges if necessary.
            if (head_node_id != -1)
            {
                bool edge_exists  = false;
                uint16_t in_count = incoming_edge_count[curr_node_id];
                for (uint16_t e = 0; e < in_count; e++)
                {
                    if (incoming_edges[curr_node_id * CUDAPOA_MAX_NODE_EDGES + e] == head_node_id)
                    {
                        edge_exists = true;
                        incoming_edge_w[curr_node_id * CUDAPOA_MAX_NODE_EDGES + e] += (prev_weight + NODE_WEIGHT);
                        //printf("Update existing node from %d to %d with weight %d\n", head_node_id, curr_node_id, incoming_edge_w[curr_node_id * CUDAPOA_MAX_NODE_EDGES + e]);
                    }
                }
                if (!edge_exists)
                {
                    incoming_edges[curr_node_id * CUDAPOA_MAX_NODE_EDGES + in_count]  = head_node_id;
                    incoming_edge_w[curr_node_id * CUDAPOA_MAX_NODE_EDGES + in_count] = prev_weight + NODE_WEIGHT;
                    incoming_edge_count[curr_node_id]                                 = in_count + 1;
                    uint16_t out_count                                                = outgoing_edge_count[head_node_id];
                    outgoing_edges[head_node_id * CUDAPOA_MAX_NODE_EDGES + out_count] = curr_node_id;
                    if (msa)
                    {
                        outgoing_edges_coverage_count[head_node_id * CUDAPOA_MAX_NODE_EDGES + out_count]                     = 1;
                        outgoing_edges_coverage[(head_node_id * CUDAPOA_MAX_NODE_EDGES + out_count) * max_sequences_per_poa] = s;
                    }
                    outgoing_edge_count[head_node_id] = out_count + 1;
                    //printf("Created new edge %d to %d with weight %d\n", head_node_id, curr_node_id, prev_weight + NODE_WEIGHT);

                    if (out_count + 1 >= CUDAPOA_MAX_NODE_EDGES || in_count + 1 >= CUDAPOA_MAX_NODE_EDGES)
                    {
                        return static_cast<uint8_t>(StatusType::edge_count_exceeded_maximum_graph_size);
                        //printf("exceeded max edge count\n");
                    }
                }
                else if (msa) //if edge exists and for msa generation
                {
                    uint16_t out_count = outgoing_edge_count[head_node_id];
                    for (uint16_t e = 0; e < out_count; e++)
                    {
                        if (outgoing_edges[head_node_id * CUDAPOA_MAX_NODE_EDGES + e] == curr_node_id)
                        {
                            uint16_t out_edge_coverage_count                                                                                       = outgoing_edges_coverage_count[head_node_id * CUDAPOA_MAX_NODE_EDGES + e];
                            outgoing_edges_coverage[(head_node_id * CUDAPOA_MAX_NODE_EDGES + e) * max_sequences_per_poa + out_edge_coverage_count] = s;
                            outgoing_edges_coverage_count[head_node_id * CUDAPOA_MAX_NODE_EDGES + e]                                               = out_edge_coverage_count + 1;
                            break;
                        }
                    }
                }
            }

            head_node_id = curr_node_id;

            // If a node is seen within a graph, then it's part of some
            // read, hence its coverage is incremented by 1.
            node_coverage_counts[head_node_id]++;

            prev_weight = NODE_WEIGHT;
        }
    }
    //printf("final size %d\n", node_count);
    new_node_count = node_count;
    return static_cast<uint8_t>(StatusType::success);
}

// kernel that calls the addAlignmentToGraph device function
template <typename SizeT>
__global__ void addAlignmentKernel(uint8_t* nodes,
                                   SizeT* node_count,
                                   SizeT* node_alignments, uint16_t* node_alignment_count,
                                   SizeT* incoming_edges, uint16_t* incoming_edge_count,
                                   SizeT* outgoing_edges, uint16_t* outgoing_edge_count,
                                   uint16_t* incoming_edge_w, uint16_t* outgoing_edge_w,
                                   uint16_t* alignment_length,
                                   SizeT* graph,
                                   SizeT* alignment_graph,
                                   uint8_t* read,
                                   SizeT* alignment_read,
                                   uint16_t* node_coverage_counts,
                                   int8_t* base_weights,
                                   SizeT* sequence_begin_nodes_ids,
                                   uint16_t* outgoing_edges_coverage,
                                   uint16_t* outgoing_edges_coverage_count,
                                   uint16_t s,
                                   uint32_t max_sequences_per_poa,
                                   uint32_t max_limit_nodes_per_window)
{
    // all pointers will be allocated in unified memory visible to both host and device
    SizeT new_node_count;
    addAlignmentToGraph(new_node_count, nodes,
                        *node_count,
                        node_alignments, node_alignment_count,
                        incoming_edges, incoming_edge_count,
                        outgoing_edges, outgoing_edge_count,
                        incoming_edge_w, outgoing_edge_w,
                        *alignment_length,
                        graph,
                        alignment_graph,
                        read,
                        alignment_read,
                        node_coverage_counts,
                        base_weights,
                        sequence_begin_nodes_ids,
                        outgoing_edges_coverage,
                        outgoing_edges_coverage_count,
                        s,
                        max_sequences_per_poa,
                        max_limit_nodes_per_window);
    *node_count = new_node_count;
}

// Host function that calls the kernel
template <typename SizeT>
void addAlignmentTemplated(uint8_t* nodes,
                           SizeT* node_count,
                           SizeT* node_alignments, uint16_t* node_alignment_count,
                           SizeT* incoming_edges, uint16_t* incoming_edge_count,
                           SizeT* outgoing_edges, uint16_t* outgoing_edge_count,
                           uint16_t* incoming_edge_w, uint16_t* outgoing_edge_w,
                           uint16_t* alignment_length,
                           SizeT* graph,
                           SizeT* alignment_graph,
                           uint8_t* read,
                           SizeT* alignment_read,
                           uint16_t* node_coverage_counts,
                           int8_t* base_weights,
                           SizeT* sequence_begin_nodes_ids,
                           uint16_t* outgoing_edges_coverage,
                           uint16_t* outgoing_edges_coverage_count,
                           uint16_t s,
                           uint32_t max_sequences_per_poa,
                           uint32_t max_limit_nodes_per_window)
{
    addAlignmentKernel<SizeT><<<1, 1>>>(nodes,
                                        node_count,
                                        node_alignments, node_alignment_count,
                                        incoming_edges, incoming_edge_count,
                                        outgoing_edges, outgoing_edge_count,
                                        incoming_edge_w, outgoing_edge_w,
                                        alignment_length,
                                        graph,
                                        alignment_graph,
                                        read,
                                        alignment_read,
                                        node_coverage_counts,
                                        base_weights,
                                        sequence_begin_nodes_ids,
                                        outgoing_edges_coverage,
                                        outgoing_edges_coverage_count,
                                        s,
                                        max_sequences_per_poa,
                                        max_limit_nodes_per_window);
    CGA_CU_CHECK_ERR(hipPeekAtLastError());
}

} // namespace cudapoa

} // namespace claragenomics
