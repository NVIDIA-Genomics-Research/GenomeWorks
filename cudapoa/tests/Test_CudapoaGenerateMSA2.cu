/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "../src/cudapoa_kernels.cuh"

#include <claraparabricks/genomeworks/cudapoa/batch.hpp>
#include <claraparabricks/genomeworks/utils/genomeutils.hpp>

#include "gtest/gtest.h"
#include "spoa/spoa.hpp"
#include <assert.h>
#include <algorithm>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudapoa
{

using ::testing::TestWithParam;
using ::testing::ValuesIn;

class MSATest : public ::testing::Test
{
public:
    void SetUp() {}

    void initialize(const BatchConfig& batch_size,
                    uint32_t device_id     = 0,
                    hipStream_t stream    = 0,
                    int8_t output_mask     = OutputType::msa,
                    int16_t gap_score      = -8,
                    int16_t mismatch_score = -6,
                    int16_t match_score    = 8)
    {
        size_t total = 0, free = 0;
        hipSetDevice(device_id);
        hipMemGetInfo(&free, &total);
        size_t mem_per_batch = 0.9 * free;

        cudapoa_batch = genomeworks::cudapoa::create_batch(device_id, stream, mem_per_batch, output_mask, batch_size, gap_score, mismatch_score, match_score);
    }

    std::vector<std::string> spoa_generate_multiple_sequence_alignments(std::vector<std::string> sequences,
                                                                        spoa::AlignmentType atype = spoa::AlignmentType::kNW,
                                                                        int match_score           = 8,
                                                                        int mismatch_score        = -6,
                                                                        int gap_score             = -8)
    {
        auto alignment_engine = spoa::createAlignmentEngine(atype, match_score, mismatch_score, gap_score);
        auto graph            = spoa::createGraph();

        for (const auto& it : sequences)
        {
            auto alignment = alignment_engine->align(it, graph);
            graph->add_alignment(alignment, it);
        }

        std::vector<std::string> msa;
        graph->generate_multiple_sequence_alignment(msa);

        return msa;
    }

public:
    std::unique_ptr<genomeworks::cudapoa::Batch> cudapoa_batch;
};

TEST_F(MSATest, CudapoaMSA)
{
    std::minstd_rand rng(1);
    int num_sequences = 500;
    BatchConfig batch_size(1024, num_sequences);

    std::string backbone = genomeworks::genomeutils::generate_random_genome(50, rng);
    auto sequences       = genomeworks::genomeutils::generate_random_sequences(backbone, num_sequences, rng, 10, 5, 10);

    initialize(batch_size);
    Group poa_group;
    std::vector<StatusType> status;
    for (const auto& seq : sequences)
    {
        Entry e{};
        e.seq     = seq.c_str();
        e.weights = nullptr;
        e.length  = seq.length();
        poa_group.push_back(e);
    }
    ASSERT_EQ(cudapoa_batch->add_poa_group(status, poa_group), StatusType::success);

    std::vector<std::vector<std::string>> cudapoa_msa;
    std::vector<StatusType> output_status;

    cudapoa_batch->generate_poa();

    cudapoa_batch->get_msa(cudapoa_msa, output_status);

    ASSERT_EQ(output_status[0], StatusType::success);
    ASSERT_EQ(poa_group.size(), cudapoa_msa[0].size());

    auto spoa_msa = spoa_generate_multiple_sequence_alignments(sequences);

#ifndef SPOA_ACCURATE
    for (size_t i = 0; i < spoa_msa.size(); i++)
    {
        std::string msa = cudapoa_msa[0][i];
        msa.erase(std::remove(msa.begin(), msa.end(), '-'), msa.end());
        ASSERT_EQ(msa, sequences[i]);
    }
#else
    ASSERT_EQ(spoa_msa, cudapoa_msa[0]);
#endif
}

TEST_F(MSATest, CudapoaMSAFailure)
{
    std::minstd_rand rng(1);
    int num_sequences = 10;
    BatchConfig batch_size(1024, num_sequences);
    batch_size.max_consensus_size = batch_size.max_sequence_size;

    std::string backbone = genomeworks::genomeutils::generate_random_genome(batch_size.max_consensus_size - 1, rng);
    auto sequences       = genomeworks::genomeutils::generate_random_sequences(backbone, num_sequences, rng, 10, 5, 10);

    initialize(batch_size);
    Group poa_group;
    std::vector<StatusType> status;
    for (const auto& seq : sequences)
    {
        Entry e{};
        e.seq     = seq.c_str();
        e.weights = nullptr;
        e.length  = seq.length();
        poa_group.push_back(e);
    }
    ASSERT_EQ(cudapoa_batch->add_poa_group(status, poa_group), StatusType::success);

    std::vector<std::vector<std::string>> cudapoa_msa;
    std::vector<StatusType> output_status;

    cudapoa_batch->generate_poa();

    cudapoa_batch->get_msa(cudapoa_msa, output_status);

    ASSERT_EQ(output_status[0], StatusType::exceeded_maximum_sequence_size);
}

} // namespace cudapoa

} // namespace genomeworks

} // namespace claraparabricks
